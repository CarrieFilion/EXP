#include "hip/hip_runtime.h"
#include <SLGridMP2.h>

#include <iostream>
#include <iomanip>
#include <map>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

__constant__ float cuRscale, cuXmin, cuXmax, cuDxi;
__constant__ int   cuNumr, cuCmap;

void SLGridSph::initialize_cuda(std::vector<hipArray_t>& cuArray,
				thrust::host_vector<hipTextureObject_t>& tex)
{
  // Number of texture arrays
  //
  int ndim = (lmax+1)*nmax + 1;

  // Allocate CUDA array in device memory (a one-dimension 'channel')
  //
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  // hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

  // Interpolation data array
  //
  cuArray.resize(ndim);

  // Size of interpolation array
  //
  size_t tsize = numr*sizeof(float);

  // Create texture objects
  //
  tex.resize(ndim);
  thrust::fill(tex.begin(), tex.end(), 0);

  hipResourceDesc resDesc;

  // Specify texture object parameters
  //
  hipTextureDesc texDesc;

  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  thrust::host_vector<float> tt(numr);

  cuda_safe_call(hipMallocArray(&cuArray[0], &channelDesc, numr, 1),
		 __FILE__, __LINE__, "malloc cuArray");

  // Copy to device memory some data located at address h_data
  // in host memory
  for (int j=0; j<numr; j++) tt[j] = p0[j];

  cuda_safe_call(hipMemcpyToArray(cuArray[0], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

  // Specify texture
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray[0];

  cuda_safe_call(hipCreateTextureObject(&tex[0], &resDesc, &texDesc, NULL),
		 __FILE__, __LINE__, "create texture object");

  for (int l=0; l<=lmax; l++) {
    for (int n=0; n<nmax; n++) {
      int i = 1 + l*nmax + n;
      cuda_safe_call(hipMallocArray(&cuArray[i], &channelDesc, numr, 1),
		     __FILE__, __LINE__, "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      float fac = sqrt(table[l].ev[n+1]);
      for (int j=0; j<numr; j++) tt[j] = table[l].ef[n+1][j] / fac;

      cuda_safe_call(hipMemcpyToArray(cuArray[i], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

      // Specify texture
      resDesc.res.array.array = cuArray[i];

      cuda_safe_call(hipCreateTextureObject(&tex[i], &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");
    }
  }

  if (false) {
    printf("**HOST** Texture compare\n");
    {
      for (int l : {0, 1, 2}) {
	for (int j=0; j<10; j++) {
	  for (int i : {3980, 3990, 3995, 3999}) 
	    printf("%5d %5d %5d %13.7e\n", l, j, i, table[l].ef[j+1][i]);
	}
      }
    }
  }

  if (false) {
    std::cout << "cuInterpArray size = " << cuArray.size() << std::endl;
    unsigned cnt = 0;
    for (size_t i=0; i<cuArray.size(); i++) {
      std::ostringstream sout;
      sout << "trying to free cuArray [" << cnt++ << "]";
      cuda_safe_call(hipFreeArray(cuArray[i]), __FILE__, __LINE__, sout.str());
    }
    
    std::cout << "texture object array size = " << tex.size() << std::endl;
    for (size_t i=0; i<tex.size(); i++) {
      std::ostringstream sout;
      sout << "trying to free TextureObject [" << cnt++ << "]";
      cuda_safe_call(hipDestroyTextureObject(tex[i]), __FILE__, __LINE__, sout.str());
    }
    
    std::cout << "cuda memory freed" << std::endl;
    exit(-1);
  }
}

