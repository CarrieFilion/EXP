#include <SLGridMP2.h>

#include <iostream>
#include <iomanip>
#include <map>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

__constant__ float cuRscale, cuXmin, cuXmax, cuDxi;
__constant__ int   cuNumr, cuCmap;

void SLGridSph::initialize_cuda(hipChannelFormatDesc& channelDesc,
				std::vector<hipArray*>& cuArray,
				std::vector<hipResourceDesc>& resDesc,
				struct hipTextureDesc& texDesc,
				thrust::host_vector<hipTextureObject_t>& tex
				)
{
  // Number of texture arrays
  //
  int ndim = (lmax+1)*nmax;

  // Allocate CUDA array in device memory
  channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

  cuArray.resize(ndim);
  size_t tsize = BLOCK_SIZE*sizeof(float);

  // Create texture objects
  tex.resize(ndim);
  thrust::fill(tex.begin(), tex.end(), 0);

  resDesc.resize(ndim);

  // Specify texture object parameters
  //
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  thrust::host_vector<float> tt(numr);

  for (int l=0; l<=lmax; l++) {
    for (int n=0; n<nmax; n++) {
      int i = l*nmax + n;
      cuda_safe_call(hipMallocArray(&cuArray[i], &channelDesc, BLOCK_SIZE, 1), "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      for (int j=0; j<numr; j++) tt[j] = table[l].ef[n+1][j];

      cuda_safe_call(hipMemcpyToArray(cuArray[i], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), "copy texture to array");

      // Specify texture
      memset(&resDesc[i], 0, sizeof(resDesc));
      resDesc[i].resType = hipResourceTypeArray;
      resDesc[i].res.array.array = cuArray[i];

      cuda_safe_call(hipCreateTextureObject(&tex[i], &resDesc[i], &texDesc, NULL), "create texture object");
    }
  }
}
