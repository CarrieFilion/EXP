#include "hip/hip_runtime.h"
#include <SLGridMP2.h>

#include <iostream>
#include <iomanip>
#include <map>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

__constant__ double cuRscale, cuXmin, cuXmax, cuDxi;
__constant__ int   cuNumr, cuCmap;


__global__
void testFetch(hipTextureObject_t* T, double* f, int l, int j, int n, int nmax)
{
  int k = l*nmax + 1;
  *f = int2_as_double(tex1D<int2>(T[k+j], n));
}


double returnTestSph(thrust::host_vector<hipTextureObject_t>& tex, int l, int j, int n, int nmax)
{
  thrust::device_vector<hipTextureObject_t> t_d = tex;
  hipTextureObject_t *T = thrust::raw_pointer_cast(t_d.data());
  
  double* f;
  cuda_safe_call(hipMalloc(&f, sizeof(double)),  __FILE__, __LINE__, "hipMalloc");

  testFetch<<<1, 1>>>(T, f, l, j, n, nmax);

  hipDeviceSynchronize();

  double ret;
  cuda_safe_call(hipMemcpy(&ret, f, sizeof(double), hipMemcpyDeviceToHost), __FILE__, __LINE__, "hipMemcpy");

  cuda_safe_call(hipFree(f), __FILE__, __LINE__, "hipFree");

  return ret;
}

static std::vector<hipResourceDesc> resDesc;
static std::vector<hipTextureDesc>  texDesc;


void SLGridSph::initialize_cuda(std::vector<hipArray_t>& cuArray,
				thrust::host_vector<hipTextureObject_t>& tex)
{
  // Number of texture arrays
  //
  int ndim = (lmax+1)*nmax + 1;

  // Allocate CUDA array in device memory (a one-dimension 'channel')
  //
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();

  // Interpolation data array
  //
  cuArray.resize(ndim);

  // Size of interpolation array
  //
  size_t tsize = numr*sizeof(double);

  // Create texture objects
  //
  tex.resize(ndim);
  thrust::fill(tex.begin(), tex.end(), 0);

  // hipResourceDesc resDesc;
  resDesc.resize(ndim);

  // Specify texture object parameters
  //
  texDesc.resize(ndim);

  memset(&texDesc[0], 0, sizeof(hipTextureDesc));
  texDesc[0].addressMode[0] = hipAddressModeClamp;
  texDesc[0].filterMode = hipFilterModePoint;
  texDesc[0].readMode = hipReadModeElementType;
  texDesc[0].normalizedCoords = 0;

  thrust::host_vector<double> tt(numr);

  cuda_safe_call(hipMallocArray(&cuArray[0], &channelDesc, numr), __FILE__, __LINE__, "malloc cuArray");

  // Copy to device memory some data located at address h_data
  // in host memory
  for (int j=0; j<numr; j++) tt[j] = p0[j];

  cuda_safe_call(hipMemcpyToArray(cuArray[0], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

  // Specify texture
  memset(&resDesc[0], 0, sizeof(hipResourceDesc));
  resDesc[0].resType = hipResourceTypeArray;
  resDesc[0].res.array.array = cuArray[0];

  cuda_safe_call(hipCreateTextureObject(&tex[0], &resDesc[0], &texDesc[0], NULL), __FILE__, __LINE__, "create texture object");

  for (int l=0; l<=lmax; l++) {
    for (int n=0; n<nmax; n++) {
      int i = 1 + l*nmax + n;
      cuda_safe_call(hipMallocArray(&cuArray[i], &channelDesc, numr), __FILE__, __LINE__, "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      double fac = sqrt(table[l].ev[n+1]);
      for (int j=0; j<numr; j++) tt[j] = table[l].ef[n+1][j] / fac;

      cuda_safe_call(hipMemcpyToArray(cuArray[i], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");
      
      // Specify texture
      memset(&resDesc[i], 0, sizeof(hipResourceDesc));
      resDesc[i].resType = hipResourceTypeArray;
      resDesc[i].res.array.array = cuArray[i];

      memset(&texDesc[i], 0, sizeof(hipTextureDesc));
      texDesc[i].addressMode[0] = hipAddressModeClamp;
      texDesc[i].filterMode = hipFilterModePoint;
      texDesc[i].readMode = hipReadModeElementType;
      texDesc[i].normalizedCoords = 0;

      cuda_safe_call(hipCreateTextureObject(&tex[i], &resDesc[i], &texDesc[i], NULL), __FILE__, __LINE__, "create texture object");
    }
  }

  if (true) {
    std::cout << "**HOST** Texture compare" << std::endl;
    unsigned tot = 0, bad = 0;
    for (int l : {0, 1, 2}) {
      for (int j=0; j<nmax; j++) {
	for (int i : {1, 2, numr-2, numr-1}) {
	  double a = table[l].ef[j+1][i]/sqrt(table[l].ev[j+1]);
	  double b = returnTestSph(tex, l, j, i, nmax);
	  if (a>1.0e-18) {
	    if ( fabs((a - b)/a ) > 1.0e-7) {
	      std::cout << std::setw( 5) << l << std::setw( 5) << j
			<< std::setw( 5) << i << std::setw(15) << a
			<< std::setw(15) << (a-b)/a << std::endl;
	      bad++;
	    }
	  }
	  tot++;
	}
      }
    }
    std::cout << "**Found " << bad << "/" << tot << " bad values" << std::endl
	      << "**" << std::endl;
  }

  if (false) {
    std::cout << "cuInterpArray size = " << cuArray.size() << std::endl;
    unsigned cnt = 0;
    for (size_t i=0; i<cuArray.size(); i++) {
      std::ostringstream sout;
      sout << "trying to free cuArray [" << cnt++ << "]";
      cuda_safe_call(hipFreeArray(cuArray[i]), __FILE__, __LINE__, sout.str());
    }
    
    std::cout << "texture object array size = " << tex.size() << std::endl;
    for (size_t i=0; i<tex.size(); i++) {
      std::ostringstream sout;
      sout << "trying to free TextureObject [" << cnt++ << "]";
      cuda_safe_call(hipDestroyTextureObject(tex[i]), __FILE__, __LINE__, sout.str());
    }
    
    std::cout << "cuda memory freed" << std::endl;
    exit(-1);
  }
}

