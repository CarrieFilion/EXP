#include "hip/hip_runtime.h"
// -*- C++ -*-

#include <boost/make_shared.hpp>

/** CUDA strategy

(1) All cross sections with lookup tables to be copied to cuda texture objects

(2) Copy necessary constants to cuda symbol data

(3) Cross sections are then evaluated on the device so we need cuda
__device__ functions for each desired cross section

  Four grids
  ----------
  free free    2-dimensional
  rad recomb   1-dimensional
  col excite   1-dimensional (2 values)
  col ionize   1-dimensional

*/

// Define for debugging
//
// #define OFF_GRID_ALERT
// #define BOUNDS_CHECK
// #define VERBOSE

// Global symbols for coordinate transformation
//
__device__ __constant__
cuFP_t ionEminGrid, ionEmaxGrid, ionDeltaEGrid;

__device__ __constant__
cuFP_t ionCollideEmin, ionEmaxGrid, ionDeltaEGrid;

__global__
void testConstantsIon()
{
  printf("** Rscale = %f\n", cylRscale);
  printf("** Hscale = %f\n", cylHscale);
  printf("** Xmin   = %f\n", cylXmin);
  printf("** Xmax   = %f\n", cylXmax);
  printf("** Ymin   = %f\n", cylYmin);
  printf("** Ymax   = %f\n", cylYmax);
  printf("** Dxi    = %f\n", cylDxi);
  printf("** Dyi    = %f\n", cylDyi);
  printf("** Numx   = %d\n", cylNumx);
  printf("** Numy   = %d\n", cylNumy);
  printf("** Cmap   = %d\n", cylCmap);
}

void chdata::cuda_initialize()
{
  for (auto v : IonList) {
    // Indecies of ion state
    //
    cuZ.push_back(v.first.first );
    cuC.push_back(v.first.second);
    
    // Initialize Ion cross-section grids
    //
    if (not freeFreeGridComputed)
      v.second->freeFreeMakeEvGrid(0);

    if (v.first.second>1 and not radRecombGridComputed)
      radRecombMakeEvGrid(0);

    if (v.first.second[k]<=v.first.first[k]) {
      if (not exciteGridComputed) collExciteMakeGrid(id);
      if (not ionizeGridComputed) directIonMakeGrid(id);
    }
  }

}

void chdata:cuda_initialize_textures()
{
  size_t ionSize = cuZ.size();

  // Interpolation data array
  //
  cuFFarray.resize(ionSize);
  cuRCarray.resize(ionSize);
  cuCEarray.resize(ionSize);
  cuCIarray.resize(ionSize);

  std::vector<hipArray_t> cuFFarray, cuRCarray, cuCEarray, ciCIarray;
  std::vector<hipTextureObject_t> ff_d, rc_d, ce_d, ci_d;
  std::vector<int> cuZ, cuC;

  thrust::host_vector  <double> hEminArray, hEmaxArray;
  thrust::device_vector<double> dEminArray, dEmaxArray;


  // Texture array
  //
  ff_d.resize(ionSize);
  rc_d.resize(ionSize);
  ce_d.resize(ionSize);
  ci_d.resize(ionSize);

  thrust::fill(ff_d.begin(), ff_d.end(), 0);
  thrust::fill(rc_d.begin(), rc_d.end(), 0);
  thrust::fill(ce_d.begin(), ce_d.end(), 0);
  thrust::fill(ci_d.begin(), ci_d.end(), 0);

  size_t k = 0;

  for (auto v : IonList) {

    IonPtr I = v.second;

    // The free-free array
    if (true) {
      hipTextureDesc texDesc;

      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.readMode = hipReadModeElementType;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
  
      // Temporary storage
      //
      cuFP_t *d_Interp;
      cuda_safe_call(hipMalloc((void **)&d_Interp, I->NfreeFreeGrid*I->kffsteps*sizeof(cuFP_t)),
		   __FILE__, __LINE__,
		     "Error allocating d_Interp for texture construction");
  
      std::vector<cuFP_t> h_buffer(NfreeFreeGrid*kffsteps, 0.0);

      // Copy vectors to buffer
    //
      for (int i = 0; i < I->NfreeFreeGrid; i++) {
	for (int j = 0; j < kffsteps; j++) {
	  h_buffer[i + j*I->NfreeFreeGrid] = I->freeFreeGrid[i][j];
	}
      }
    
    // Copy data to device
    cuda_safe_call(hipMemcpy(d_Interp, &h_buffer[0], I->NfreeFreeGrid*sizeof(cuFP_t), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");
    
    // hipArray Descriptor
    //
#if cuREAL == 4
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    // cuda Array
    //
    cuda_safe_call(hipMalloc3DArray(&cuFFarray[k], &channelDesc, make_hipExtent(I->NfreeFreeGrid, I->kffsteps, 1), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");
    
    // Array creation
    //
    hipMemcpy3DParms copyParams = {0};
  
    copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, NfreeFreeGrid*sizeof(cuFP_t), I->NfreeFreeGrid, I->kffsteps);
    copyParams.dstArray = cuFFarray[k];
    copyParams.extent   = make_hipExtent(I->NfreeFreeGrid, I->kffsteps, 1);
    copyParams.kind     = hipMemcpyDeviceToDevice;
    
    cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");

    hipResourceDesc resDesc;

    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array  = cuFFarray[k];
    
    cuda_safe_call
      (hipCreateTextureObject(&ff_d[k], &resDesc, &texDesc, NULL),
       __FILE__, __LINE__, "Failure in 2d texture creation");
    
    cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
  }

  // Radiative recombination texture (1-d)
  //
  if (cuC[k]>1) {
    // Allocate CUDA array in device memory (a one-dimension 'channel')
    //
#if cuREAL == 4
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    
    // Size of interpolation array
    //
    size_t tsize = I->NradRecombGrid*sizeof(cuFP_t);

    hipTextureDesc texDesc;

    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    
    thrust::host_vector<cuFP_t> tt(I->NradRecombGrid);

    cuda_safe_call(hipMallocArray(&cuRCarray[k], &channelDesc, I->NradRecombGrid), __FILE__, __LINE__, "malloc cuArray");

    // Copy to device memory some data located at address h_data
    // in host memory
    for (size_t n = 0; n < I->NradRecombGrid; n++) tt[n] = I->radRecombGrid[n];
    
    cuda_safe_call(hipMemcpyToArray(cuRCarray[k], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

    // Specify texture

    hipResourceDesc resDesc;

    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuRCarray[k];

    cuda_safe_call(hipCreateTextureObject(&rc_d[k], &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");
  }

  // The collisional excitation array

  if (cuC[k] <= cuZ[k]) {

    hipTextureDesc texDesc;

    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
  
    // Temporary storage
    //
    cuFP_t *d_Interp;
    cuda_safe_call(hipMalloc((void **)&d_Interp, I->NcollideGrid*2*sizeof(cuFP_t)),
		   __FILE__, __LINE__,
		   "Error allocating d_Interp for texture construction");
  
    std::vector<cuFP_t> h_buffer(I->NcollideGrid*2, 0.0);

    // Copy vectors to buffer
    //
    for (int i = 0; i < I->NcollideGrid; i++) {
      h_buffer[i                ] = I->collideDataGrid[i].first;
      h_buffer[i + NfreeFreeGrid] = I->collideDataGrid[i].second;
    }
    
    // Copy data to device
    cuda_safe_call(hipMemcpy(d_Interp, &h_buffer[0], I->NcollideGrid*sizeof(cuFP_t), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");
    
    // hipArray Descriptor
    //
#if cuREAL == 4
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    // cuda Array
    //
    cuda_safe_call(hipMalloc3DArray(&cuCEarray[k], &channelDesc, make_hipExtent(I->NcollideGrid, 2, 1), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");
    
    // Array creation
    //
    hipMemcpy3DParms copyParams = {0};
  
    copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, NcollideGrid*sizeof(cuFP_t), NcollideGrid, 2);
    copyParams.dstArray = cuInterpArray[2];
    copyParams.extent   = make_hipExtent(I->NcollideGrid, 2, 1);
    copyParams.kind     = hipMemcpyDeviceToDevice;
    
    cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");

    hipResourceDesc resDesc;

    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array  = cuCEarray[k];
    
    cuda_safe_call
      (hipCreateTextureObject(&ce_d[k], &resDesc, &texDesc, NULL),
       __FILE__, __LINE__, "Failure in 2d texture creation");
    
    cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
  }

  if (cuZ[k] <= cuC[k]) {

    // Allocate CUDA array in device memory (a one-dimension 'channel')
    //
#if cuREAL == 4
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    
    // Size of interpolation array
    //
    size_t tsize = I->NionizeGrid*sizeof(cuFP_t);

    hipTextureDesc texDesc;

    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    
    thrust::host_vector<cuFP_t> tt(NionizeGrid);

    cuda_safe_call(hipMallocArray(&cuCIarray[k], &channelDesc, I->NionizeGrid), __FILE__, __LINE__, "malloc cuArray");

    // Copy to device memory some data located at address h_data
    // in host memory
    for (size_t n = 0; n < I->NionizeGrid; n++) tt[n] = I->ionizeDataGrid[n];
    
    cuda_safe_call(hipMemcpyToArray(cuCIarray[k], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

    // Specify texture

    hipResourceDesc resDesc;

    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuCIarray[k];

    cuda_safe_call(hipCreateTextureObject(&ci_d[k], &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");
  }

  // Grid values for the various cross sections
  /*
  std::vector< double > kgrid;
  std::vector< double > kgr10;
  std::vector< double > nugrid;

  int kffsteps;
  int nusteps;
  */
}

void Ion::initialize_grid_constants()
{
  // Copy constants to device
  //
  
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylRscale), &f.rscale, sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylRscale");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylHscale), &f.hscale, sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylHscale");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylXmin),   &f.xmin,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylXmin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylXmax),   &f.xmax,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylXmax");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylDxi),    &f.dxi,    sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylDxi");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylNumx),   &f.numx,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylNumx");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylYmin),   &f.ymin,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylYmin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylYmax),   &f.ymax,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylYmax");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylDyi),    &f.dyi,    sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylDxi");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylNumy),   &f.numy,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylNumy");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCmap),   &f.cmap,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCmap");

}


__global__ void coordKernelCyl
(dArray<cudaParticle> in, dArray<cuFP_t> mass, dArray<cuFP_t> phi,
 dArray<cuFP_t> Xfac, dArray<cuFP_t> Yfac,
 dArray<int> IndX, dArray<int> IndY,
 unsigned int stride, PII lohi, cuFP_t rmax)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i = tid*stride + n;	// Particle counter
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {

#ifdef BOUNDS_CHECK
      if (npart>=in._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle p = in._v[npart];
    
      cuFP_t xx=0.0, yy=0.0, zz=0.0;

      if (cylOrient) {
	for (int k=0; k<3; k++) xx += cylBody[0+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) yy += cylBody[3+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) zz += cylBody[6+k]*(p.pos[k] - cylCen[k]);
      } else {
	xx = p.pos[0] - cylCen[0];
	yy = p.pos[1] - cylCen[1];
	zz = p.pos[2] - cylCen[2];
      }
      
      cuFP_t R2 = xx*xx + yy*yy;
      cuFP_t r2 = R2 + zz*zz;
      cuFP_t R  = sqrt(R2);
      cuFP_t r  = sqrt(r2);
#ifdef BOUNDS_CHECK
      if (i>=mass._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      mass._v[i] = -1.0;
      
      if (r<=rmax) {
	
	mass._v[i] = p.mass;
	
	phi._v[i] = atan2(yy, xx);

#ifdef BOUNDS_CHECK
	if (i>=phi._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	// Interpolation indices
	//
	cuFP_t X  = (cu_r_to_xi_cyl(R) - cylXmin)/cylDxi;
	cuFP_t Y  = (cu_z_to_y_cyl(zz) - cylYmin)/cylDyi;

	int indX = floor(X);
	int indY = floor(Y);
	
	if (indX<0) indX = 0;
	if (indX>cylNumx-2) indX = cylNumx - 2;
	
	if (indY<0) indY = 0;
	if (indY>cylNumy-2) indY = cylNumy - 2;
	
	Xfac._v[i] = cuFP_t(indX+1) - X;
	IndX._v[i] = indX;

	Yfac._v[i] = cuFP_t(indY+1) - Y;
	IndY._v[i] = indY;

#ifdef OFF_GRID_ALERT
	if (Xfac._v[i]<-0.5 or Xfac._v[i]>1.5) printf("X off grid: x=%f\n", X);
	if (Yfac._v[i]<-0.5 or Yfac._v[i]>1.5) printf("Y off grid: y=%f\n", Y);
#endif
#ifdef BOUNDS_CHECK
	if (i>=Xfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=IndX._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=Yfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=IndY._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      }
    }
  }
}


__global__ void coefKernelCyl
(dArray<cuFP_t> coef, dArray<hipTextureObject_t> tex,
 dArray<cuFP_t> Mass, dArray<cuFP_t> Phi,
 dArray<cuFP_t> Xfac, dArray<cuFP_t> Yfac,
 dArray<int> indX, dArray<int> indY,
 int stride, int m, unsigned int nmax, PII lohi)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of particles to be evaluated
  //
  const unsigned int N = lohi.second - lohi.first;

  const cuFP_t norm = -4.0*M_PI;	// Biorthogonality factor

  for (int istr=0; istr<stride; istr++) {

    int i = tid*stride + istr;	// Particle counter

    if (i<N) {			// Allow for grid padding

      cuFP_t mass = Mass._v[i];
      
#ifdef BOUNDS_CHECK
      if (i>=Mass._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif      
      if (mass>0.0) {
#ifdef BOUNDS_CHECK
	if (i>=Phi._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	cuFP_t phi  = Phi._v[i];
	cuFP_t cosp = cos(phi*m);
	cuFP_t sinp = sin(phi*m);
	
	// Do the interpolation
	//
	cuFP_t delx0 = Xfac._v[i];
	cuFP_t dely0 = Yfac._v[i];
	cuFP_t delx1 = 1.0 - delx0;
	cuFP_t dely1 = 1.0 - dely0;

#ifdef BOUNDS_CHECK
	if (i>=Xfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=Yfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	cuFP_t c00 = delx0*dely0;
	cuFP_t c10 = delx1*dely0;
	cuFP_t c01 = delx0*dely1;
	cuFP_t c11 = delx1*dely1;

	int   indx = indX._v[i];
	int   indy = indY._v[i];

#ifdef BOUNDS_CHECK
	if (i>=indX._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=indY._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	for (int n=0; n<nmax; n++) {

	  // Texture maps are packed in slices
	  // ---------------------------------
	  // potC, rforceC, zforceC, potS, rforceS, zforceS
	  // 0     1        2        3     4        5

	  int k = m*nmax + n;

#if cuREAL == 4
	  cuFP_t d00  = tex3D<float>(tex._v[k], indx,   indy  , 0);
	  cuFP_t d10  = tex3D<float>(tex._v[k], indx+1, indy  , 0);
	  cuFP_t d01  = tex3D<float>(tex._v[k], indx,   indy+1, 0);
	  cuFP_t d11  = tex3D<float>(tex._v[k], indx+1, indy+1, 0);

#else
	  cuFP_t d00  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy  , 0));
	  cuFP_t d10  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy  , 0));
	  cuFP_t d01  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy+1, 0));
	  cuFP_t d11  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy+1, 0));
#endif

#ifdef BOUNDS_CHECK
	  if (k>=tex._s)            printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	  if ((2*n+0)*N+i>=coef._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	  coef._v[(2*n+0)*N + i] = (c00*d00 + c10*d10 + c01*d01 + c11*d11) * cosp * norm * mass;

	  if (m>0) {
	    // potS tables are offset from potC tables by +3
	    //
#if cuREAL == 4
	    d00  = tex3D<float>(tex._v[k], indx,   indy  , 3);
	    d10  = tex3D<float>(tex._v[k], indx+1, indy  , 3);
	    d01  = tex3D<float>(tex._v[k], indx,   indy+1, 3);
	    d11  = tex3D<float>(tex._v[k], indx+1, indy+1, 3);
#else
	    d00  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy  , 3));
	    d10  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy  , 3));
	    d01  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy+1, 3));
	    d11  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy+1, 3));
#endif

	    coef._v[(2*n+1)*N + i] = (c00*d00 + c10*d10 + c01*d01 + c11*d11) * sinp * norm * mass;

#ifdef BOUNDS_CHECK
	    if ((2*n+1)*N+i>=coef._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	  } // m>0
	  else {
	    coef._v[(2*n+1)*N + i] = 0.0;
	  }

	} // norder loop

      } else {
	// No contribution from off-grid particles
	for (int n=0; n<nmax; n++) {
	  coef._v[(2*n+0)*N + i] = 0.0;
	  if (m) coef._v[(2*n+1)*N + i] = 0.0;
	}

      } // mass value check

    } // particle index check

  } // stride loop
}

__global__ void
forceKernelCyl(dArray<cudaParticle> in, dArray<cuFP_t> coef,
	       dArray<hipTextureObject_t> tex,
	       int stride, unsigned int mmax, unsigned int nmax, PII lohi,
	       cuFP_t rmax, cuFP_t cylmass, bool external)
{
  // Thread ID
  //
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;

  // Maximum radius squared
  //
  const cuFP_t rmax2 = rmax*rmax;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {
      
#ifdef BOUNDS_CHECK
      if (npart>=in._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle p = in._v[npart];
      
      cuFP_t acc[3] = {0.0, 0.0, 0.0};
      cuFP_t xx=0.0, yy=0.0, zz=0.0;

      if (cylOrient) {
	for (int k=0; k<3; k++) xx += cylBody[0+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) yy += cylBody[3+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) zz += cylBody[6+k]*(p.pos[k] - cylCen[k]);
      } else {
	xx = p.pos[0] - cylCen[0];
	yy = p.pos[1] - cylCen[1];
	zz = p.pos[2] - cylCen[2];
      }

      cuFP_t phi = atan2(yy, xx);
      cuFP_t R2  = xx*xx + yy*yy;
      cuFP_t  R  = sqrt(R2) + FSMALL;
      
      const cuFP_t ratmin = 0.75;
      const cuFP_t maxerf = 3.0;
      const cuFP_t midpt  = ratmin + 0.5*(1.0 - ratmin);
      const cuFP_t rsmth  = 0.5*(1.0 - ratmin)/maxerf;

      cuFP_t ratio = sqrt( (R2 + zz*zz)/rmax2 );
      cuFP_t mfactor = 1.0, frac = 1.0, cfrac = 0.0;

      if (ratio >= 1.0) {
	// cfrac      = 1.0 - mfactor;
	cfrac = 1.0;
      } else if (ratio > ratmin) {
	frac  = 0.5*(1.0 - erf( (ratio - midpt)/rsmth )) * mfactor;
	cfrac = 1.0 - frac;
      } else {
	frac  = mfactor;
      }

      cuFP_t fr = 0.0;
      cuFP_t fz = 0.0;
      cuFP_t fp = 0.0;
      cuFP_t pp = 0.0;
      
      if (ratio < 1.0) {

	cuFP_t X  = (cu_r_to_xi_cyl(R) - cylXmin)/cylDxi;
	cuFP_t Y  = (cu_z_to_y_cyl(zz) - cylYmin)/cylDyi;

	int indX = floor(X);
	int indY = floor(Y);
	
	if (indX < 0) indX = 0;
	if (indY < 0) indY = 0;
	if (indX >= cylNumx) indX = cylNumx - 1;
	if (indY >= cylNumy) indY = cylNumy - 1;

	cuFP_t delx0 = cuFP_t(indX+1) - X;
	cuFP_t dely0 = cuFP_t(indY+1) - Y;

#ifdef OFF_GRID_ALERT
	if (delx0<0.0 or delx0>1.0) printf("X off grid: x=%f\n", delx0);
	if (dely0<0.0 or dely0>1.0) printf("Y off grid: y=%f\n", dely0);
#endif

	cuFP_t delx1 = 1.0 - delx0;
	cuFP_t dely1 = 1.0 - dely0;
      
	cuFP_t c00 = delx0*dely0;
	cuFP_t c10 = delx1*dely0;
	cuFP_t c01 = delx0*dely1;
	cuFP_t c11 = delx1*dely1;

	cuFP_t cos1 = cos(phi);
	cuFP_t sin1 = sin(phi);

	cuFP_t ccos = 1.0;
	cuFP_t ssin = 0.0;

	for (int mm=0; mm<=mmax; mm++) {

	  for (int n=0; n<nmax; n++) {
      
	    cuFP_t fac0 = coef._v[Imn(mm, 'c', n, nmax)];
	    cuFP_t fac1 = fac0 * ccos;
	    cuFP_t fac2 = fac0 * ssin;
      
	    // Texture table index
	    //
	    int k = mm*nmax + n;

	    pp += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 0) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 0) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 0) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 0) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 0)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 0)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 0)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 0)) * c11 
#endif
	       );
	    
	    fr += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 1) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 1) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 1) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 1) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 1)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 1)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 1)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 1)) * c11 
#endif
	       );
      
	    fz += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 2) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 2) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 2) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 2) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 2)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 2)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 2)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 2)) * c11 
#endif
	       );
	    
	    fp += fac2 * mm *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 0) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 0) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 0) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 0) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 0)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 0)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 0)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 0)) * c11 
#endif
	       );
      
      
	    if (mm) {
	
	      cuFP_t fac0 =  coef._v[Imn(mm, 's', n, nmax)];
	      cuFP_t fac1 =  fac0 * ssin;
	      cuFP_t fac2 = -fac0 * ccos;

	      pp += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 3) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 3) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 3) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 3) * c11
#else		 
 		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 3)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 3)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 3)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 3)) * c11 
#endif
		 );
	      
	      fr += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 4) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 4) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 4) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 4) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 4)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 4)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 4)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 4)) * c11 
#endif
		 );
	      
	      fz += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 5) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 5) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 5) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 5) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 5)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 5)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 5)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 5)) * c11 
#endif
		 );
	      
	      fp += fac2 * mm *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 3) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 3) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 3) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 3) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 3)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 3)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 3)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 3)) * c11 
#endif
		 );
	      
	    }
	  }
	  
	  // Trig recursion to squeeze avoid internal FP fct call
	  //
	  cuFP_t cosM = ccos;
	  cuFP_t sinM = ssin;

	  ccos = cosM * cos1 - sinM * sin1;
	  ssin = sinM * cos1 + cosM * sin1;
	}

	acc[0] += ( fr*xx/R - fp*yy/R2 ) * frac;
	acc[1] += ( fr*yy/R + fp*xx/R2 ) * frac;
	acc[2] += fz * frac;
      }

      if (ratio > ratmin) {

	cuFP_t r3 = R2 + zz*zz;
	pp = -cylmass/sqrt(r3);	// -M/r
	fr = pp/r3;		// -M/r^3

	acc[0] += xx*fr * cfrac;
	acc[1] += yy*fr * cfrac;
	acc[2] += zz*fr * cfrac;
      }

      if (cylOrient) {
	for (int j=0; j<3; j++) {
	  for (int k=0; k<3; k++) in._v[npart].acc[j] += cylOrig[3*j+k]*acc[k];
	}
      } else {
	for (int j=0; j<3; j++) in._v[npart].acc[j] += acc[j];
      }

      if (external)
	in._v[npart].potext += pp;
      else
	in._v[npart].pot    += pp;

    } // Particle index block

  } // END: stride loop

}



template<typename T>
class LessAbs : public std::binary_function<bool, T, T>
{
public:
  T operator()( const T &a, const T &b ) const
  {
    return (fabs(a) < fabs(b));
  }
};


void Cylinder::cudaStorage::resize_coefs
(int ncylorder, int mmax, int N, int gridSize)
{
  // Reserve space for coefficient reduction
  //
  if (dN_coef.capacity() < 2*ncylorder*N)
    dN_coef.reserve(2*ncylorder*N);
  
  if (dc_coef.capacity() < 2*ncylorder*gridSize)
    dc_coef.reserve(2*ncylorder*gridSize);
  
  if (m_d .capacity() < N) m_d .reserve(N);
  if (X_d .capacity() < N) X_d .reserve(N);
  if (Y_d .capacity() < N) Y_d .reserve(N);
  if (p_d .capacity() < N) p_d .reserve(N);
  if (iX_d.capacity() < N) iX_d.reserve(N);
  if (iY_d.capacity() < N) iY_d.reserve(N);
  
  
  // Set space for current step
  //
  dN_coef.resize(2*ncylorder*N);
  dc_coef.resize(2*ncylorder*gridSize);
  dw_coef.resize(2*ncylorder);	// This will stay fixed

  // Space for coordinate arrays on the current step
  //
  m_d .resize(N);
  X_d .resize(N);
  Y_d .resize(N);
  p_d .resize(N);
  iX_d.resize(N);
  iY_d.resize(N);
}

void Cylinder::zero_coefs()
{
  Component::cuRingType cr = *cC->cuRing.get();
  Cylinder ::cuRingType ar = *cuRing.get();
  
  for (int n=0; n<cuStreams; n++) {
    // Resize output array
    //
    ar->df_coef.resize((mmax+1)*2*ncylorder);
    
    // Zero output array
    //
    thrust::fill(thrust::cuda::par.on(cr->stream),
		 ar->df_coef.begin(), ar->df_coef.end(), 0.0);

    // Advance iterators
    //
    ++cr;			// Component stream
    ++ar;			// Force method storage
  }
}

static bool initialize_cuda_cyl = true;
static unsigned dbg_id = 0;

void Cylinder::determine_coefficients_cuda()
{
  if (initialize_cuda_cyl) {
    initialize_cuda();
    initialize_mapping_constants();
    initialize_cuda_cyl = false;
    // Only copy texture memory once
    t_d = tex;
  }

  std::cout << std::scientific;

  int deviceCount = 0;
  cuda_safe_call(hipGetDeviceCount(&deviceCount),
		 __FILE__, __LINE__, "could not get device count");

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceCount-1);

  // This will stay fixed for the entire run
  //
  host_coefs.resize((2*mmax+1)*ncylorder);

  // Set component center and orientation
  //
  std::vector<cuFP_t> ctr;
  for (auto v : cC->getCenter(Component::Local | Component::Centered)) ctr.push_back(v);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCen), &ctr[0], sizeof(cuFP_t)*3,
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCen");

  bool orient = (cC->EJ & Orient::AXIS) && !cC->EJdryrun;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrient), &orient,   sizeof(bool),  size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylOrient");

  if (orient) {
    std::vector<cuFP_t> trans(9);
    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++) trans[i*3+j] = cC->orient->transformBody()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylBody), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylBody");
  }

  Component::cuRingType cr = *cC->cuRing.get();
  Cylinder::cuRingType  ar = *cuRing.get();

  // For debugging (set to false to disable)
  //
  static bool firstime = false;

  if (firstime) {
    testConstantsCyl<<<1, 1, 0, cr->stream>>>();
    hipDeviceSynchronize();
    firstime = false;
  }
  
  // Zero counter and coefficients
  //
  unsigned Ntot = 0;
  use[0]      = 0.0;
  cylmass0[0] = 0.0;
  thrust::fill(host_coefs.begin(), host_coefs.end(), 0.0);

  // Zero out coefficient storage
  //
  zero_coefs();

  // Maximum radius on grid
  //
  cuFP_t rmax = rcylmax * acyl * M_SQRT1_2;

  // Copy particles to host vector
  //
  cC->ParticlesToCuda();

  // Loop over bunches
  //
  size_t psize  = cC->host_particles.size();

  Component::hostPartItr begin = cC->host_particles.begin();
  Component::hostPartItr first = begin;
  Component::hostPartItr last  = begin;
  Component::hostPartItr end   = cC->host_particles.end();

  if (psize <= cC->bunchSize) last = end;
  else std::advance(last, cC->bunchSize);

  // Set up stream and data arrays for asynchronous evaluation
  //
  std::vector<hipStream_t> f_s;
  thrust::device_vector<unsigned int> f_use;
  thrust::device_vector<cuFP_t>       f_mass;

  while (std::distance(first, last)) {
    
    cr->first = first;
    cr->last  = last;
    cr->id    = ++dbg_id;

    // Copy bunch to device
    //
    cC->HostToDev(cr);

    // Sort particles and get coefficient size
    //
    PII lohi = cC->CudaSortByLevel(cr, mlevel, mlevel);

    // Compute grid
    //
    unsigned int N         = lohi.second - lohi.first;
    unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

#ifdef VERBOSE
    static debug_max_count = 10;
    static debug_cur_count = 0;
    if (debug_cur_count++ < debug_max_count) {
      std::cout << std::endl << "**" << std::endl
		<< "** N      = " << N           << std::endl
		<< "** I low  = " << lohi.first  << std::endl
		<< "** I high = " << lohi.second << std::endl
		<< "** Stride = " << stride      << std::endl
		<< "** Block  = " << BLOCK_SIZE  << std::endl
		<< "** Grid   = " << gridSize    << std::endl
		<< "** Xcen   = " << ctr[0]     << std::endl
		<< "** Ycen   = " << ctr[1]     << std::endl
		<< "** Zcen   = " << ctr[2]     << std::endl
		<< "**" << std::endl;
    }
#endif
  
    if (N) {

      // Adjust cached storage, if necessary
      //
      ar->resize_coefs(ncylorder, mmax, N, gridSize);

      // Shared memory size for the reduction
      //
      int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);
    

      thrust::counting_iterator<int> index_begin(0);
      thrust::counting_iterator<int> index_end(gridSize*2*ncylorder);

      // Do the work
      //
				// Compute the coordinate
				// transformation
				// 
      coordKernelCyl<<<gridSize, BLOCK_SIZE, 0, cr->stream>>>
	(toKernel(cr->cuda_particles), toKernel(ar->m_d), toKernel(ar->p_d),
	 toKernel(ar->X_d), toKernel(ar->Y_d), toKernel(ar->iX_d),
	 toKernel(ar->iY_d), stride, lohi, rmax);
      
				// Compute the coefficient
				// contribution for each order
      int osize = ncylorder*2;	// 
      auto beg = ar->df_coef.begin();

      for (int m=0; m<=mmax; m++) {

	coefKernelCyl<<<gridSize, BLOCK_SIZE, 0, cr->stream>>>
	  (toKernel(ar->dN_coef), toKernel(t_d), toKernel(ar->m_d), toKernel(ar->p_d),
	   toKernel(ar->X_d), toKernel(ar->Y_d), toKernel(ar->iX_d), toKernel(ar->iY_d),
	   stride, m, ncylorder, lohi);
      
				// Begin the reduction per grid block
				//
	reduceSum<cuFP_t, BLOCK_SIZE><<<gridSize, BLOCK_SIZE, sMemSize, cr->stream>>>
	  (toKernel(ar->dc_coef), toKernel(ar->dN_coef), osize, N);
      
				// Finish the reduction for this order
				// in parallel
	thrust::reduce_by_key
	  (
	   thrust::cuda::par.on(cr->stream),
	   thrust::make_transform_iterator(index_begin, key_functor(gridSize)),
	   thrust::make_transform_iterator(index_end,   key_functor(gridSize)),
	   ar->dc_coef.begin(), thrust::make_discard_iterator(), ar->dw_coef.begin()
	   );

	thrust::transform(thrust::cuda::par.on(cr->stream),
			  ar->dw_coef.begin(), ar->dw_coef.end(),
			  beg, beg, thrust::plus<cuFP_t>());

	thrust::advance(beg, 2*ncylorder);
      }
    
      // Compute number and total mass of particles used in coefficient
      // determination
      //
      thrust::sort(thrust::cuda::par.on(cr->stream), ar->m_d.begin(), ar->m_d.end());

      // Asynchronously cache result for host side to prevent stream block
      //
      hipStream_t s1;		// Create a new non blocking stream
      hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
      f_s.push_back(s1);

      size_t fsz = f_s.size();	// Augment the data vectors
      f_use. resize(fsz);
      f_mass.resize(fsz);
				// Call the kernel on a single thread
				// 
      reduceUseCyl<<<1, 1, 0, s1>>>(ar->m_d.begin(), ar->m_d.end(),
				    &f_use[fsz-1], &f_mass[fsz-1]);

      Ntot += N;
    }

    // Advance iterators
    //
    first = last;
    size_t nadv = std::distance(first, end);
    if (nadv < cC->bunchSize) last = end;
    else std::advance(last, cC->bunchSize);

    // Advance stream iterators
    //
    ++cr;			// Coefficient stream
    ++ar;			// Force method storage
  }

  if (Ntot == 0) {
    return;
  }

  // Accumulate the coefficients from the device to the host
  //
  for (auto r : cuRingData) {
    thrust::host_vector<cuFP_t> ret = r.df_coef;
    int offst = 0;
    for (int m=0; m<=mmax; m++) {
      for (size_t j=0; j<ncylorder; j++) {
	host_coefs[Imn(m, 'c', j, ncylorder)] += ret[2*j+offst];
	if (m>0) host_coefs[Imn(m, 's', j, ncylorder)] += ret[2*j+1+offst];
      }
      offst += 2*ncylorder;
    }
  }

  // Get the on-grid count and mass from the threads
  //
  for (auto & s : f_s) {	// Synchronize and dallocate streams
    hipStreamSynchronize(s);
    hipStreamDestroy(s);
  }
				// Copy the data from the device
  thrust::host_vector<unsigned int> f_ret1(f_use);
  thrust::host_vector<cuFP_t>       f_ret2(f_mass);

				// Sum counts and mass
  for (auto v : f_ret1) use[0]      += v;
  for (auto v : f_ret2) cylmass0[0] += v;

  // DEBUG, only useful for CUDAtest branch
  //
  if (false) {
    constexpr bool compareC = false;

    if (compareC) {
      std::cout << std::string(2*4+4*20, '-') << std::endl
		<< "---- Cylindrical "      << std::endl
		<< std::string(2*4+4*20, '-') << std::endl;
      std::cout << "M=0 coefficients" << std::endl
		<< std::setprecision(10);

      std::cout << std::setw(4)  << "n"
		<< std::setw(4)  << "i"
		<< std::setw(20) << "GPU"
		<< std::setw(20) << "CPU"
		<< std::setw(20) << "diff"
		<< std::setw(20) << "rel diff"
		<< std::endl;
    } else {
      std::cout << std::string(2*4+20, '-') << std::endl
		<< "---- Cylindrical "      << std::endl
		<< std::string(2*4+20, '-') << std::endl;
      std::cout << "M=0 coefficients" << std::endl
		<< std::setprecision(10);

      std::cout << std::setw(4)  << "n"
		<< std::setw(4)  << "i"
		<< std::setw(20) << "GPU"
		<< std::endl;
    }

    int i = Imn(0, 'c', 0, ncylorder);
    auto cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(0, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(0, n, 'c');
      if (compareC) {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::setw(20) << b
		  << std::setw(20) << a - b
		  << std::setw(20) << (a - b)/fabs(*cmax)
		  << std::endl;
      } else {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::endl;
      }
    }

    std::cout << "M=1c coefficients" << std::endl;

    i = Imn(1, 'c', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(1, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(1, n, 'c');
      if (compareC) {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::setw(20) << b
		  << std::setw(20) << a - b
		  << std::setw(20) << (a - b)/fabs(*cmax)
		  << std::endl;
      } else {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::endl;
      }
    }

    std::cout << "M=1s coefficients" << std::endl;

    i = Imn(1, 's', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(1, 's', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(1, n, 's');
      if (compareC) {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::setw(20) << b
		  << std::setw(20) << a - b
		  << std::setw(20) << (a - b)/fabs(*cmax)
		  << std::endl;
      } else {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::endl;
      }
    }

    std::cout << "M=2c coefficients" << std::endl;

    i = Imn(2, 'c', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(2, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(2, n, 'c');
      if (compareC) {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::setw(20) << b
		  << std::setw(20) << a - b
		  << std::setw(20) << (a - b)/fabs(*cmax)
		  << std::endl;
      } else {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::endl;
      }
    }
    
    std::cout << "M=2s coefficients" << std::endl;

    i = Imn(2, 's', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(2, 's', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(2, n, 's');
      if (compareC) {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::setw(20) << b
		  << std::setw(20) << a - b
		  << std::setw(20) << (a - b)/fabs(*cmax)
		  << std::endl;
      } else {
	std::cout << std::setw(4)  << n
		  << std::setw(4)  << i
		  << std::setw(20) << a
		  << std::endl;
      }
    }

    std::cout << std::string(2*4+4*20, '-') << std::endl;
  }


  //
  // TEST comparison of coefficients for debugging
  //
  if (false) {

    struct Element
    {
      double d;
      double f;
      
      int  m;
      int  n;
      
      char cs;
    }
    elem;

    std::multimap<double, Element> compare;

    std::ofstream out("test_cyl.dat");

    // m loop
    for (int m=0; m<=mmax; m++) {
	
      if (m==0) {
	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 'c';
	  elem.d = ortho->get_coef(m, n, 'c');
	  elem.f = host_coefs[Imn(m, 'c', n, ncylorder)];
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);
	  
	  compare.insert(std::make_pair(test, elem));;
	    
	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 'c'
	      << std::setw( 5) << Imn(m, 'c', n, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	}

      } else {
	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 'c';
	  elem.d = ortho->get_coef(m, n, 'c');
	  elem.f = host_coefs[Imn(m, 'c', n, ncylorder)];

	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 'c'
	      << std::setw( 5) << Imn(m, 'c', n, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);

	  compare.insert(std::make_pair(test, elem));;
	}

	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 's';
	  elem.d = ortho->get_coef(m, n, 's');
	  elem.f = host_coefs[Imn(m, 's', n, ncylorder)];

	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 's'
	      << std::setw( 5) << Imn(m, 's', n-1, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);
	  
	  compare.insert(std::make_pair(test, elem));;
	}
      }
    }
    
    std::map<double, Element>::iterator best = compare.begin();
    std::map<double, Element>::iterator midl = best;
    std::advance(midl, compare.size()/2);
    std::map<double, Element>::reverse_iterator last = compare.rbegin();
    
    std::cout << std::string(3*2 + 3*20 + 20, '-') << std::endl
	      << "---- Cylinder coefficients" << std::endl
	      << std::string(3*2 + 3*20 + 20, '-') << std::endl;

    std::cout << "Best case: ["
	      << std::setw( 2) << best->second.m << ", "
	      << std::setw( 2) << best->second.n << ", "
	      << std::setw( 2) << best->second.cs << "] = "
	      << std::setw(20) << best->second.d
	      << std::setw(20) << best->second.f
	      << std::setw(20) << fabs(best->second.d - best->second.f)
	      << std::endl;
  
    std::cout << "Mid case:  ["
	      << std::setw( 2) << midl->second.m << ", "
	      << std::setw( 2) << midl->second.n << ", "
	      << std::setw( 2) << midl->second.cs << "] = "
	      << std::setw(20) << midl->second.d
	      << std::setw(20) << midl->second.f
	      << std::setw(20) << fabs(midl->second.d - midl->second.f)
	      << std::endl;
    
    std::cout << "Last case: ["
	      << std::setw( 2) << last->second.m << ", "
	      << std::setw( 2) << last->second.n << ", "
	      << std::setw( 2) << last->second.cs << "] = "
	      << std::setw(20) << last->second.d
	      << std::setw(20) << last->second.f
	      << std::setw(20) << fabs(last->second.d - last->second.f)
	      << std::endl;
  }

}


void Cylinder::determine_acceleration_cuda()
{
  if (initialize_cuda_cyl) {
    initialize_cuda();
    initialize_mapping_constants();
    initialize_cuda_cyl = false;
    // Only copy texture memory once
    t_d = tex;
  }

  std::cout << std::scientific;

  int deviceCount = 0;
  cuda_safe_call(hipGetDeviceCount(&deviceCount),
		 __FILE__, __LINE__, "could not get device count");

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceCount-1);

  Component::cuRingType cr = *cC->cuRing.get();

  // Assign component center and orientation
  //
  std::vector<cuFP_t> ctr;
  for (auto v : cC->getCenter(Component::Local | Component::Centered)) ctr.push_back(v);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCen), &ctr[0], sizeof(cuFP_t)*3,
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCen");

  bool orient = (cC->EJ & Orient::AXIS) && !cC->EJdryrun;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrient), &orient,   sizeof(bool),  size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylOrient");

  if (orient) {
    std::vector<cuFP_t> trans(9);
    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++)
	trans[i*3+j] = cC->orient->transformBody()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylBody), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylBody");

    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++)
	trans[i*3+j] = cC->orient->transformOrig()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrig), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylOrig");
  }

  // Copy particles to host vector
  //
  cC->ParticlesToCuda();

  // Loop over bunches
  //
  size_t psize  = cC->host_particles.size();

  Component::hostPartItr begin = cC->host_particles.begin();
  Component::hostPartItr first = begin;
  Component::hostPartItr last  = begin;
  Component::hostPartItr end   = cC->host_particles.end();

  if (psize <= cC->bunchSize) last = end;
  else std::advance(last, cC->bunchSize);

  unsigned Ntot = 0;

  while (std::distance(first, last)) {

    cr->first = first;
    cr->last  = last;
    cr->id    = ++dbg_id;

    // Copy bunch to device
    //
    cC->HostToDev(cr);

    // Sort particles and get coefficient size
    //
    PII lohi = cC->CudaSortByLevel(cr, mlevel, multistep);

    // Compute grid
    //
    unsigned int N         = lohi.second - lohi.first;
    unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
    if (N>0) {

      Ntot += N;

      if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

#ifdef VERBOSE
      static debug_max_count = 10;
      static debug_cur_count = 0;
      if (debug_cur_count++ < debug_max_count) {
	std::cout << std::endl << "**" << std::endl
		  << "** N      = " << N          << std::endl
		  << "** Stride = " << stride     << std::endl
		  << "** Block  = " << BLOCK_SIZE << std::endl
		  << "** Grid   = " << gridSize   << std::endl
		  << "** Xcen   = " << ctr[0]     << std::endl
		  << "** Ycen   = " << ctr[1]     << std::endl
		  << "** Zcen   = " << ctr[2]     << std::endl
		  << "**" << std::endl;
      }
#endif
    
      // Shared memory size for the reduction
      //
      int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);
      
      // Maximum radius on grid
      //
      cuFP_t rmax = rcylmax * acyl;
      
      // Do the work
      //
      forceKernelCyl<<<gridSize, BLOCK_SIZE, sMemSize, cr->stream>>>
	(toKernel(cr->cuda_particles), toKernel(dev_coefs), toKernel(t_d),
	 stride, mmax, ncylorder, lohi, rmax, cylmass, use_external);
      
      // Copy particles back to host
      //
      cC->DevToHost(cr);
    }
      
    // Advance iterators
    //
    first = last;
    size_t nadv = std::distance(first, end);
    if (nadv < cC->bunchSize) last = end;
    else std::advance(last, cC->bunchSize);

    // Advance stream iterators
    //
    ++cr;			// Component
				// Force method ring not needed here
  }

  cC->CudaToParticles();

  // DEBUGGING TEST
  if (false) {
    std::cout << std::string(10+7*16, '-') << std::endl;
    std::cout << "---- Acceleration in Cylinder [T=" << tnow
	      << ", N=" << Ntot << ", level=" << mlevel
	      << ", mass=" << cylmass
	      << ", name=" << cC->name << "]" << std::endl;
    std::cout << std::string(10+7*16, '-') << std::endl;
    first = last = begin;
    std::advance(last, 5);
    std::copy(first, last, std::ostream_iterator<cudaParticle>(std::cout, "\n"));
    first = begin;
    last  = end;
    std::advance(first, psize-5);
    std::copy(first, last, std::ostream_iterator<cudaParticle>(std::cout, "\n"));
    std::cout << std::string(10+7*16, '-') << std::endl;
  }
}

void Cylinder::HtoD_coefs()
{
  // Check size
  host_coefs.resize((2*mmax+1)*ncylorder); // Should stay fixed, no reserve

  // Copy from EmpCylSL
  
  // m loop
  //
  for (int m=0; m<=mmax; m++) {
    
    // n loop
    //
    for (int n=0; n<ncylorder; n++) {
      host_coefs[Imn(m, 'c', n, ncylorder)] = ortho->get_coef(m, n, 'c');
      if (m>0) host_coefs[Imn(m, 's', n, ncylorder)] = ortho->get_coef(m, n, 's');
    }
  }

  // Copy to device
  dev_coefs = host_coefs;
}


void Cylinder::DtoH_coefs(int M)
{
  // Copy from host device to EmpCylSL

  // m loop
  //
  for (int m=0; m<=mmax; m++) {
    
    // n loop
    //
    for (int n=0; n<ncylorder; n++) {
      ortho->set_coef(M, m, n, 'c') = host_coefs[Imn(m, 'c', n, ncylorder)];
      if (m>0) ortho->set_coef(M, m, n, 's') = host_coefs[Imn(m, 's', n, ncylorder)];
    }
  }
}

void chdata::destroy_cuda()
{
  for (size_t i=0; i<tex.size(); i++) {
    std::ostringstream sout;
    sout << "trying to free TextureObject [" << i << "]";
    cuda_safe_call(hipDestroyTextureObject(tex[i]),
		   __FILE__, __LINE__, sout.str());
  }

  for (size_t i=0; i<cuInterpArray.size(); i++) {
    std::ostringstream sout;
    sout << "trying to free cuPitch [" << i << "]";
    cuda_safe_call(hipFree(cuInterpArray[i]),
		     __FILE__, __LINE__, sout.str());
  }
}
