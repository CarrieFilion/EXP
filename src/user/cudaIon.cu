#include "hip/hip_runtime.h"
// -*- C++ -*-

#include <iostream>
#include <iomanip>
#include <boost/make_shared.hpp>

#include <Ion.H>

// Global symbols for coordinate transformation
//
__device__ __constant__
cuFP_t ionEminGrid, ionEmaxGrid, ionDeltaEGrid;

__device__ __constant__
int ionEgridNumber, ionRadRecombNumber;

__global__
void testConstantsIon()
{
  printf("** Egrid(min) = %f\n", ionEminGrid);
  printf("** Egrid(max) = %f\n", ionEmaxGrid);
  printf("** Egrid(del) = %f\n", ionDeltaEGrid);
  printf("** Egrid(num) = %d\n", ionEgridNumber);
  printf("** Rgrid(num) = %d\n", ionRadRecombNumber);
}

void chdata::cuda_initialize_textures()
{
  size_t ionSize = cuZ.size();

  // Interpolation data array
  //
  cuF0array.resize(ionSize);
  cuFFarray.resize(ionSize);
  cuRCarray.resize(ionSize);
  cuCEarray.resize(ionSize);
  cuCIarray.resize(ionSize);

  NColl.    resize(ionSize);
  NIonz.    resize(ionSize);

  // Texture object array
  //
  ff_0.resize(ionSize);
  ff_d.resize(ionSize);
  rc_d.resize(ionSize);
  ce_d.resize(ionSize);
  ci_d.resize(ionSize);

  thrust::fill(ff_0.begin(), ff_0.end(), 0);
  thrust::fill(ff_d.begin(), ff_d.end(), 0);
  thrust::fill(rc_d.begin(), rc_d.end(), 0);
  thrust::fill(ce_d.begin(), ce_d.end(), 0);
  thrust::fill(ci_d.begin(), ci_d.end(), 0);

  size_t k = 0;

  for (auto v : IonList) {

    IonPtr I = v.second;

    // The free-free array
    if (cuC[k]>1) {
      hipTextureDesc texDesc;

      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.readMode = hipReadModeElementType;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
  
      // Temporary storage
      //
      cuFP_t *d_Interp0, *d_Interp1;

      cuda_safe_call(hipMalloc((void **)&d_Interp0, I->NfreeFreeGrid*sizeof(cuFP_t)),
		     __FILE__, __LINE__,
		     "Error allocating d_Interp0 for texture construction");
  
      std::vector<cuFP_t> h_buffer0(I->NfreeFreeGrid, 0.0);

      cuda_safe_call(hipMalloc((void **)&d_Interp1, I->NfreeFreeGrid*CHCUMK*sizeof(cuFP_t)),
		     __FILE__, __LINE__,
		     "Error allocating d_Interp1 for texture construction");
  
      std::vector<cuFP_t> h_buffer1(I->NfreeFreeGrid*CHCUMK, 0.0);

      // Temporary storage
      //
      cuFP_t *d_Interp;
      cuda_safe_call(hipMalloc((void **)&d_Interp, I->NfreeFreeGrid*CHCUMK*sizeof(cuFP_t)),
		     __FILE__, __LINE__,
		     "Error allocating d_Interp for texture construction");

      double delC = 1.0/(CHCUMK-1);

      // Copy cross section values to buffer
      //
      for (int i = 0; i < I->NfreeFreeGrid; i++) {

	h_buffer0[i] = I->freeFreeGrid[i].back();
	
	// Unit normalized cumulative distribution
	//
	std::vector<double> temp(I->kffsteps);
	for (int j = 0; j < I->kffsteps; j++) {	
	  temp[j] = I->freeFreeGrid[i][j]/h_buffer0[i];
	}

	// Remap to even grid
	//
	int j = 1;
	for (int k = 0; k < I->kffsteps; k++) {	
	  double C = delC*j;	// Interpolate
	  if (temp[k] >= C and temp[k-1]< C) {
	    double D = temp[k] - temp[k-1];
	    double A = (C - temp[k-1])/D;
	    double B = (temp[k  ] - C)/D;
	    h_buffer1[i + j*I->NfreeFreeGrid] = I->kgrid[k-1]*A + I->kgrid[k]*B;
	    j++;
	  }
	}

	// End points
	//
	h_buffer1[i                              ] = I->kgrid[0];
	h_buffer1[i + (CHCUMK-1)*I->NfreeFreeGrid] = I->kgrid[I->kffsteps-1];
      }

      // Copy 1-dim data to device
      //
      size_t tsize = I->NfreeFreeGrid*sizeof(cuFP_t);

      cuda_safe_call(hipMemcpyToArray(cuF0array[k], 0, 0, &d_Interp0, tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

      // Specify 1-d texture

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuF0array[k];

      cuda_safe_call(hipCreateTextureObject(&ff_0[k], &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");

      // Copy data to device
      cuda_safe_call(hipMemcpy(d_Interp, &h_buffer1[0], I->NfreeFreeGrid*CHCUMK*sizeof(cuFP_t), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");
    
      // cuda 2d Array Descriptor
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
      // cuda 2d Array
      //
      cuda_safe_call(hipMalloc3DArray(&cuFFarray[k], &channelDesc, make_hipExtent(I->NfreeFreeGrid, I->kffsteps, 1), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");
      
      // Array creation
      //
      hipMemcpy3DParms copyParams = {0};
  
      copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, I->NfreeFreeGrid*sizeof(cuFP_t), I->NfreeFreeGrid, I->kffsteps);
      copyParams.dstArray = cuFFarray[k];
      copyParams.extent   = make_hipExtent(I->NfreeFreeGrid, I->kffsteps, 1);
      copyParams.kind     = hipMemcpyDeviceToDevice;
      
      cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array  = cuFFarray[k];
    
      cuda_safe_call
	(hipCreateTextureObject(&ff_d[k], &resDesc, &texDesc, NULL),
	 __FILE__, __LINE__, "Failure in 2d texture creation");
      
      cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
    }

    // Radiative recombination texture (1-d)
    //
    if (cuC[k]>1) {
      // Allocate CUDA array in device memory (a one-dimension 'channel')
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    
      // Size of interpolation array
      //
      size_t tsize = I->NradRecombGrid*sizeof(cuFP_t);

      hipTextureDesc texDesc;
      
      memset(&texDesc, 0, sizeof(hipTextureDesc));
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.readMode = hipReadModeElementType;
      texDesc.normalizedCoords = 0;
      
      thrust::host_vector<cuFP_t> tt(I->NradRecombGrid);
      
      cuda_safe_call(hipMallocArray(&cuRCarray[k], &channelDesc, I->NradRecombGrid), __FILE__, __LINE__, "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      for (size_t n = 0; n < I->NradRecombGrid; n++) tt[n] = I->radRecombGrid[n];
    
      cuda_safe_call(hipMemcpyToArray(cuRCarray[k], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

      // Specify texture

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuRCarray[k];
      
      cuda_safe_call(hipCreateTextureObject(&rc_d[k], &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");
    }

    // The collisional excitation array

    if (cuC[k] <= cuZ[k]) {

      NColl[k] = I->NcollideGrid;

      hipTextureDesc texDesc;

      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.readMode = hipReadModeElementType;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
  
      // Temporary storage
      //
      cuFP_t *d_Interp;
      cuda_safe_call(hipMalloc((void **)&d_Interp, I->NcollideGrid*2*sizeof(cuFP_t)),
		     __FILE__, __LINE__,
		     "Error allocating d_Interp for texture construction");
  
      std::vector<cuFP_t> h_buffer(I->NcollideGrid*2, 0.0);

      // Copy vectors to buffer
      //
      for (int i = 0; i < I->NcollideGrid; i++) {
	h_buffer[i                  ] = I->collideDataGrid[i].back().first;
	h_buffer[i + I->NcollideGrid] = I->collideDataGrid[i].back().second;
      }
      
      // Copy data to device
      cuda_safe_call(hipMemcpy(d_Interp, &h_buffer[0], I->NcollideGrid*sizeof(cuFP_t), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");
    
      // hipArray Descriptor
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
      // cuda Array
      //
      cuda_safe_call(hipMalloc3DArray(&cuCEarray[k], &channelDesc, make_hipExtent(I->NcollideGrid, 2, 1), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");
    
      // Array creation
      //
      hipMemcpy3DParms copyParams = {0};
      
      copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, I->NcollideGrid*sizeof(cuFP_t), I->NcollideGrid, 2);
      copyParams.dstArray = cuCEarray[k];
      copyParams.extent   = make_hipExtent(I->NcollideGrid, 2, 1);
      copyParams.kind     = hipMemcpyDeviceToDevice;
      
      cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");
      
      hipResourceDesc resDesc;
      
      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array  = cuCEarray[k];
      
      cuda_safe_call
	(hipCreateTextureObject(&ce_d[k], &resDesc, &texDesc, NULL),
	 __FILE__, __LINE__, "Failure in 2d texture creation");
      
      cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
    }

    if (cuZ[k] <= cuC[k]) {

      NIonz[k] = I->NionizeGrid;

      // Allocate CUDA array in device memory (a one-dimension 'channel')
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    
      // Size of interpolation array
      //
      size_t tsize = I->NionizeGrid*sizeof(cuFP_t);
      
      hipTextureDesc texDesc;

      memset(&texDesc, 0, sizeof(hipTextureDesc));
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.readMode = hipReadModeElementType;
      texDesc.normalizedCoords = 0;
      
      thrust::host_vector<cuFP_t> tt(I->NionizeGrid);
      
      cuda_safe_call(hipMallocArray(&cuCIarray[k], &channelDesc, I->NionizeGrid), __FILE__, __LINE__, "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      for (size_t n = 0; n < I->NionizeGrid; n++) tt[n] = I->ionizeDataGrid[n];
      
      cuda_safe_call(hipMemcpyToArray(cuCIarray[k], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");
      
      // Specify texture

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuCIarray[k];
      
      cuda_safe_call(hipCreateTextureObject(&ci_d[k], &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");
    }
    
  } // END: IonList

}

void chdata::cuda_initialize_grid_constants()
{
  double Emin, Emax, delE;
  int NE, NR;

  for (auto v : IonList) {
    Emin = v.second->EminGrid;
    Emax = v.second->EmaxGrid;
    delE = v.second->DeltaEGrid;

    NE   = v.second->NfreeFreeGrid;

    if (v.first.second>1) {
      NR = v.second->NradRecombGrid;
      break;
    }
  }

  cuFP_t f;

  // Copy constants to device
  //
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEminGrid), &(f=Emin),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEminGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEmaxGrid), &(f=Emax),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEmaxGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionDeltaEGrid), &(f=delE),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionDeltaEGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEgridNumber), &NE,
				    sizeof(int), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEgridNumber");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionRadRecombNumber), &NR,
				    sizeof(int), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionRadRecombNumber");

}


__global__ void testFreeFree
(dArray<cuFP_t> energy,
 dArray<cuFP_t> randsl,
 dArray<cuFP_t> ph, dArray<cuFP_t> xc,
 hipTextureObject_t tex1,
 hipTextureObject_t tex2)
{
  // value of h-bar * c in eV*nm
  //
  constexpr double hbc = 197.327;

  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {

    cuFP_t E = energy._v[tid];

    // Enforce minimum and maximum energies
    //
    if (E<ionEminGrid) E = ionEminGrid;
    if (E>ionEmaxGrid) E = ionEmaxGrid;

    size_t indx = std::floor( (E - ionEminGrid)/ionDeltaEGrid );
    
    if (indx >= ionEgridNumber - 1) indx = ionEgridNumber-2;

    double eA = ionEminGrid + ionDeltaEGrid*indx;
    double eB = ionEminGrid + ionDeltaEGrid*(indx+1);

    double A = (eB - E)/ionDeltaEGrid;
    double B = (E - eA)/ionDeltaEGrid;

    // Location in cumulative cross section grid
    //
    double rn = randsl._v[tid];
    double dC = 1.0/(CHCUMK-1);
    int lb    = rn/dC;
    cuFP_t k[4];

    // Interpolate the cross section array
    //
#if cuREAL == 4
    k[0]  = tex3D<float>(tex2, indx,   lb  , 0);
    k[1]  = tex3D<float>(tex2, indx+1, lb  , 0);
    k[2]  = tex3D<float>(tex2, indx,   lb+1, 0);
    k[3]  = tex3D<float>(tex2, indx+1, lb+1, 0);
#else
    k[0] = int2_as_double(tex3D<int2>(tex2, indx,   lb  , 0));
    k[1] = int2_as_double(tex3D<int2>(tex2, indx+1, lb  , 0));
    k[2] = int2_as_double(tex3D<int2>(tex2, indx,   lb+1, 0));
    k[3] = int2_as_double(tex3D<int2>(tex2, indx+1, lb+1, 0));
#endif

    // Linear interpolation
    //
    double a = (rn - dC*(lb+0)) / dC;
    double b = (dC*(lb+1) - rn) / dC;

    double K = A*(a*k[0] + b*k[2]) + B*(a*k[1] + b*k[3]);

    // Assign the photon energy
    //
    ph._v[tid] = pow(10, K) * hbc;

    // Use the integrated cross section from the differential grid
    //

    xc._v[tid] = 
#if cuREAL == 4
      A*tex1D<float>(tex1, indx  ) +
      B*tex1D<float>(tex1, indx+1) ;
#else
    A*int2_as_double(tex1D<int2>(tex1, indx  )) +
      B*int2_as_double(tex1D<int2>(tex1, indx+1)) ;
#endif
  }

  __syncthreads();
}

void chdata::testCross(int Nenergy)
{
  // Loop over ions and tabulate statistics
  //
  size_t k = 0;

  thrust::host_vector<cuFP_t> energy_h(Nenergy), randsl_h(Nenergy);

  for (auto v : IonList) {

    IonPtr I = v.second;

    // Make an energy grid
    //
    double dE = (I->EmaxGrid - I->EminGrid)/(Nenergy-1);
    for (int i=0; i<Nenergy; i++) {
      energy_h[i] = I->EminGrid + dE*i;
      randsl_h[i] = static_cast<cuFP_t>(rand())/RAND_MAX;
    }

    thrust::device_vector<cuFP_t> energy_d = energy_h;
    thrust::device_vector<cuFP_t> randsl_d = randsl_h;

    // Only free-free for non-neutral species
    if (cuC[k]>1) {

      thrust::device_vector<cuFP_t> ph_d(Nenergy), xc_d(Nenergy);

      unsigned int gridSize  = Nenergy/BLOCK_SIZE;
      if (Nenergy > gridSize*BLOCK_SIZE) gridSize++;

      testFreeFree<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), toKernel(randsl_d),
					     toKernel(ph_d), toKernel(xc_d),
					     ff_0[k], ff_d[k]);
      
      thrust::host_vector<cuFP_t> ph_h = ph_d;
      thrust::host_vector<cuFP_t> xc_h = xc_d;

      std::vector<double> ph_0(Nenergy), xc_0(Nenergy);

      for (int i=0; i<Nenergy; i++) {
	auto ret = I->freeFreeCrossTest(energy_h[i], randsl_h[i], 0);
	xc_0[i] = (xc_h[i] - ret.first )/ret.first;
	ph_0[i] = (ph_h[i] - ret.second)/ret.second;
      }

      std::sort(xc_0.begin(), xc_0.end());
      std::sort(ph_0.begin(), ph_0.end());

      std::vector<double> quantiles = {0.01, 0.05, 0.1, 0.2, 0.5, 0.8, 0.9, 0.95, 0.99};

      std::cout << "Ion (" << I->Z << ", " << I->C << ")" << std::endl;
      for (auto v : quantiles) {
	int indx = std::min<int>(std::floor(v*Nenergy+0.5), Nenergy-1);
	std::cout << std::setw(10) << v
		  << " | " << std::setw(14) << xc_0[indx]
		  << " | " << std::setw(14) << ph_0[indx]
		  << std::endl;
      }

    }

  } // END: Ion list

}
