#include "hip/hip_runtime.h"
// -*- C++ -*-

#include <cudaUtil.cuH>
#include <cudaReduce.cuH>

#include <Component.H>
#include "UserSat.H"

// Global symbols
//
__device__ __constant__
cuFP_t userSatMass, userSatCore2, userSatCen[3], userSatPos[3];

__device__ __constant__
bool userSatShadow;

__global__ void
userSatForceKernel(dArray<cudaParticle> P, dArray<int> I,
		   int stride, PII lohi)
{
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {
      
#ifdef BOUNDS_CHECK
      if (npart>=P._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle & p = P._v[I._v[npart]];
      
      cuFP_t rr = userSatCore2;
      for (int k=0; k<3; k++) {
	cuFP_t f = p.pos[k] - userSatCen[k] - userSatPos[k];
	rr += f*f;
      }

      rr = pow(rr, -0.5);
    
      cuFP_t ffac = -userSatMass*rr*rr*rr;

      // Add acceration
      for (int k=0; k<3; k++)
	p.acc[k] += ffac*(p.pos[k] - userSatCen[k] - userSatPos[k]);

      p.potext += -userSatMass*rr;

      // Add the shadow satellite
      if (userSatShadow) {
	rr = userSatCore2;
	for (int k=0; k<3; k++) {
	  cuFP_t f = p.pos[k] - userSatCen[k] + userSatPos[k];
	  rr += f*f;
	}

	rr = pow(rr, -0.5);
	
	ffac = -userSatMass*rr*rr*rr;
	
	// Add acceration
	for (int k=0; k<3; k++)
	  p.acc[k] += ffac*(p.pos[k] - userSatCen[k] + userSatPos[k]);

	p.potext += -userSatMass*rr;
      }

    } // Particle index block

  } // END: stride loop

}


void UserSat::determine_acceration_and_potential_cuda()
{
  // Sanity check
  //
  int nbodies = cC->Number();
  if (nbodies != static_cast<int>(cC->Particles().size())) {
    std::cerr << "UserSat: ooops! number=" << nbodies
	      << " but particle size=" << cC->Particles().size() << endl;
    nbodies = static_cast<int>(cC->Particles().size());
  }
  
  if (nbodies==0) {		// Return if there are no particles
    if (verbose and zbflag) {
      cout << "Process " << myid << ": in UserSat, nbodies=0" 
	   << " for Component <" << cC->name << "> at T=" << tnow
	   << endl;
      zbflag = false;
    }
    return;
  }

  zbflag = true;

  double rs[3];

  if (traj_type==circ) {
    double phi = phase + omega*tnow;
    rs[0] = r0*cos(phi);
    rs[1] = r0*sin(phi);
    rs[2] = 0.0;
  }
  else
    traj->get_satellite_orbit(tnow - toffset, &rs[0]);

  double satmass = mass * 
    0.5*(1.0 + erf( (tnow - ton) /delta )) *
    0.5*(1.0 + erf( (toff - tnow)/delta )) ;
    
  if (shadow) satmass *= 0.5;

  if (orbit && myid==0 && mlevel==0 && tnow>tlast) {
    std::ofstream out (orbfile.c_str(), ios::app);
    if (out) {
      out << setw(15) << tnow;
      for (int k=0; k<3; k++) out << setw(15) << rs[k];
      out << endl;
      tlast = tnow;
    } else {
      std::cout << "Error opening trajectory file: " << orbfile << endl;
    }
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cC->cudaDevice);
  cuda_check_last_error_mpi("hipGetDeviceProperties", __FILE__, __LINE__, myid);

  // Stream structure iterators
  //
  auto cr = cC->cuStream;

  // Assign expansion center
  //
  std::vector<cuFP_t> ctr, sps;

  for (auto v : component->getCenter(Component::Inertial))
    ctr.push_back(v);

  for (int k=0; k<3; k++) {
    sps[k] = rs[k];
    if (pinning) ctr[k] += c0->com[k];
  }

  cuFP_t cuSatCore2 = core * core, cuSatMass = satmass;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userSatCore2), &cuSatCore2, sizeof(cuFP_t),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying userSatCore");
  
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userSatMass), &cuSatMass, sizeof(cuFP_t),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying userSatMass");
  
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userSatCen), &ctr[0], sizeof(cuFP_t)*3,
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying userSatCen");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userSatPos), &sps[0], sizeof(cuFP_t)*3,
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying userSatPos");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userSatShadow), &shadow,  sizeof(bool),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying userSatShadow");

  // Get particle index range for levels [mlevel, multistep]
  //
  PII lohi = cC->CudaGetLevelRange(mlevel, multistep);

  // Compute grid
  //
  unsigned int N         = lohi.second - lohi.first;
  unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
  unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
  if (N>0) {

    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

    unsigned int Nthread = gridSize*BLOCK_SIZE;

    // Shared memory size for the reduction
    //
    int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);
    
    // Do the work
    //
    userSatForceKernel<<<gridSize, BLOCK_SIZE, sMemSize, cr->stream>>>
      (toKernel(cr->cuda_particles), toKernel(cr->indx1), stride, lohi);
  }
}
