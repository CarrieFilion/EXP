#include "hip/hip_runtime.h"
#include <cudaElastic.cuH>

// Atomic radii in picometers from Clementi, E.; Raimond, D. L.;
// Reinhardt, W. P. (1967). "Atomic Screening Constants from SCF
// Functions. II. Atoms with 37 to 86 Electrons". Journal of Chemical
// Physics 47 (4): 1300-1307.  See also Paper 1, ref. therein.
//
const int numRadii = 87;
__constant__ int cudaRadii[numRadii];

// Cross section interpolation arrays for electron-neutral and
// proton-neutral interactions
//
thrust::device_vector<cuFP_t> xsc_H, xsc_He, xsc_pH, xsc_pHe;
__constant__ cuFP_t cuH_H, cuHe_H, cuPH_H, cuPHe_H;
__constant__ cuFP_t cuH_Emin, cuHe_Emin, cuPH_Emin, cuPHe_Emin;

// For construction of evenly spaced interpolation arrays
//
thrust::host_vector<cuFP_t>
resampleArray(const std::vector<cuFP_t>& x, const std::vector<cuFP_t>& y,
	      cuFP_t& dx)
{
  // Get minimum grid spacing
  cuFP_t minH = std::numeric_limits<cuFP_t>::max();
  for (int i=0; i<x.size()-1; i++)
    minH = std::min<cuFP_t>(minH, x[i+1]- x[i]);

  // Resample based on minimum spacing
  int numH = std:floor( (x.back() - x.front())/minH ) + 1;

  thrust::host_vector<cuFP_t> Y(numH);
  
  dx = (x.back() - x.front())/(numH - 1);

  for (int i=0; i<numH; i++) {
    cuFP_t xx = x.back() + dx*i, yy;
    if (xx <= x.front()) {
      yy = y.front();
    } else if (xx >= x.back()) {
      yy = y.back();
    } else {
      auto ub = std::lower_bound(x.begin(), x.end(), xx);
      auto lb = ub++;
      if (ub == x.end()) ub = lb--;
      cuFP_t a = (x - *lb)(*ub - *lb);
      cuFP_t b = (*ub - x)(*ub - *lb);
      yy = a*y[lb - ev_H.begin()] + b*y[ub - ev_H.begin()];
    }
    Y.push_back(yy);
  }

  return Y;
}

// Initialize cross-section look up and interpolation arrays.  Data
// input could be generalized here . . . for later.
//
void cudaElasticInit()
{
  std::vector<int> radii(numRadii, 0);

  radii[1]  =  53;
  radii[2]  =  31;
  radii[3]  =  167;
  radii[4]  =  112;
  radii[5]  =  87;
  radii[6]  =  67;
  radii[7]  =  56;
  radii[8]  =  48;
  radii[9]  =  42;
  radii[10] =  38;
  radii[11] =  190;
  radii[12] =  145;
  radii[13] =  118;
  radii[14] =  111;
  radii[15] =  98;
  radii[16] =  180;
  radii[17] =  79;
  radii[18] =  188;
  radii[19] =  243;
  radii[20] =  194;
  radii[21] =  184;
  radii[22] =  176;
  radii[23] =  171;
  radii[24] =  166;
  radii[25] =  161;
  radii[26] =  156;
  radii[27] =  152;
  radii[28] =  149;
  radii[29] =  145;
  radii[30] =  152;
  radii[31] =  136;
  radii[32] =  125;
  radii[33] =  114;
  radii[34] =  103;
  radii[35] =  94;
  radii[36] =  88;
  radii[37] =  265;
  radii[38] =  219;
  radii[39] =  212;
  radii[40] =  206;
  radii[41] =  198;
  radii[42] =  190;
  radii[43] =  183;
  radii[44] =  178;
  radii[45] =  173;
  radii[46] =  169;
  radii[47] =  172;
  radii[48] =  161;
  radii[49] =  193;
  radii[50] =  217;
  radii[51] =  133;
  radii[52] =  123;
  radii[53] =  198;
  radii[54] =  108;
  radii[55] =  298;
  radii[56] =  268;
  radii[59] =  247;
  radii[60] =  206;
  radii[61] =  205;
  radii[62] =  238;
  radii[63] =  231;
  radii[64] =  233;
  radii[65] =  225;
  radii[66] =  228;
  radii[68] =  226;
  radii[69] =  222;
  radii[70] =  222;
  radii[71] =  217;
  radii[72] =  208;
  radii[73] =  200;
  radii[74] =  193;
  radii[75] =  188;
  radii[76] =  185;
  radii[77] =  180;
  radii[78] =  177;
  radii[79] =  166;
  radii[80] =  171;
  radii[81] =  156;
  radii[82] =  202;
  radii[83] =  143;
  radii[84] =  135;
  radii[86] =  120;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cudaRadii), &radii[0], sizeof(cuFP_t)*numRadii), 
		 __FILE__, __LINE__, "Error copying cudaRadii");

  // Total cross section from Malik & Trefftz, 1960, Zeitschrift fur Astrophysik, 50, 96-109

  // Column 1 in eV, Column2 in Bohr cross section (pi*a_0^2) units

  std::vector<cuFP_t> eV_H, xs_H;

  eV_H.push_back(0.66356077923727);	xs_H.push_back(28.864);
  eV_H.push_back(0.66576762346358);	xs_H.push_back(29.5088);
  eV_H.push_back(0.71282701193426);	xs_H.push_back(28.2574);
  eV_H.push_back(0.73590227585419);	xs_H.push_back(27.4989);
  eV_H.push_back(0.75936666273882);	xs_H.push_back(26.8542);
  eV_H.push_back(0.80889140369906);	xs_H.push_back(26.3234);
  eV_H.push_back(0.83209592175062);	xs_H.push_back(25.6028);
  eV_H.push_back(0.90677351672548);	xs_H.push_back(24.9204);
  eV_H.push_back(0.95590913472103);	xs_H.push_back(24.2757);
  eV_H.push_back(1.031106467362);	xs_H.push_back(23.745);
  eV_H.push_back(1.05457085424663);	xs_H.push_back(23.1003);
  eV_H.push_back(1.10409559520687);	xs_H.push_back(22.5694);
  eV_H.push_back(1.17903305901478);	xs_H.push_back(21.9628);
  eV_H.push_back(1.22881766880808);	xs_H.push_back(21.5079);
  eV_H.push_back(1.2782131556367);	xs_H.push_back(20.9391);
  eV_H.push_back(1.35379961124236);	xs_H.push_back(20.5221);
  eV_H.push_back(1.45506137966837);	xs_H.push_back(20.1052);
  eV_H.push_back(1.58185287994542);	xs_H.push_back(19.6504);
  eV_H.push_back(1.75999228472356);	xs_H.push_back(19.1958);
  eV_H.push_back(1.91233528596856);	xs_H.push_back(18.7032);
  eV_H.push_back(2.06519530374007);	xs_H.push_back(18.3623);
  eV_H.push_back(2.24360682247953);	xs_H.push_back(17.9835);
  eV_H.push_back(2.42186867854026);	xs_H.push_back(17.5668);
  eV_H.push_back(2.60026659158166);	xs_H.push_back(17.188);
  eV_H.push_back(2.77893661858437);	xs_H.push_back(16.8851);
  eV_H.push_back(2.9830084838763);	xs_H.push_back(16.5064);
  eV_H.push_back(3.21287675270137);	xs_H.push_back(16.1657);
  eV_H.push_back(3.39141072272342);	xs_H.push_back(15.8249);
  eV_H.push_back(3.64683049251644);	xs_H.push_back(15.4463);
  eV_H.push_back(3.87695726960477);	xs_H.push_back(15.1815);
  eV_H.push_back(4.0810291348967);	xs_H.push_back(14.8028);
  eV_H.push_back(4.31091100941984);	xs_H.push_back(14.4621);
  eV_H.push_back(4.54091533522557);	xs_H.push_back(14.1593);
  eV_H.push_back(4.71957175653021);	xs_H.push_back(13.8564);
  eV_H.push_back(4.97525003458648);	xs_H.push_back(13.5537);
  eV_H.push_back(5.28200410318252);	xs_H.push_back(13.1753);
  eV_H.push_back(5.53768238123879);	xs_H.push_back(12.8726);
  eV_H.push_back(5.74227126305723);	xs_H.push_back(12.6456);
  eV_H.push_back(5.97267015410688);	xs_H.push_back(12.4566);
  eV_H.push_back(6.15132657541152);	xs_H.push_back(12.1537);
  eV_H.push_back(6.40726336173105);	xs_H.push_back(11.9268);
  eV_H.push_back(6.61198830053015);	xs_H.push_back(11.7378);
  eV_H.push_back(6.81683569061185);	xs_H.push_back(11.5866);
  eV_H.push_back(6.99562816889715);	xs_H.push_back(11.3216);
  eV_H.push_back(7.20035310769626);	xs_H.push_back(11.1326);
  eV_H.push_back(7.43061594176524);	xs_H.push_back(10.9057);
  eV_H.push_back(7.71209062335465);	xs_H.push_back(10.641);
  eV_H.push_back(7.96789135269351);	xs_H.push_back(10.3762);
  eV_H.push_back(8.27517604351412);	xs_H.push_back(10.1495);
  eV_H.push_back(8.50530282060245);	xs_H.push_back(9.88464);
  eV_H.push_back(8.76123960692198);	xs_H.push_back(9.6578);
  eV_H.push_back(9.06852429774258);	xs_H.push_back(9.43109);
  eV_H.push_back(9.35012143061459);	xs_H.push_back(9.20432);
  eV_H.push_back(9.55484636941369);	xs_H.push_back(9.01526);
  eV_H.push_back(9.78536771174593);	xs_H.push_back(8.86419);
  eV_H.push_back(10.0157666027956);	xs_H.push_back(8.6752);
  eV_H.push_back(10.2717033891151);	xs_H.push_back(8.44835);
  eV_H.push_back(10.5533005219871);	xs_H.push_back(8.22158);
  eV_H.push_back(10.8349112605572);	xs_H.push_back(7.9948);
  eV_H.push_back(11.1421823456797);	xs_H.push_back(7.7681);
  eV_H.push_back(11.4237930842498);	xs_H.push_back(7.54133);
  eV_H.push_back(11.6798523218519);	xs_H.push_back(7.35241);
  eV_H.push_back(11.9615991174026);	xs_H.push_back(7.16356);
  eV_H.push_back(12.2176583550047);	xs_H.push_back(6.97464);
  eV_H.push_back(12.4223832938038);	xs_H.push_back(6.78558);
  eV_H.push_back(12.7041164836565);	xs_H.push_back(6.59673);
  eV_H.push_back(12.9858496735092);	xs_H.push_back(6.40788);
  eV_H.push_back(13.2163710158414);	xs_H.push_back(6.25682);
  eV_H.push_back(13.4212320116212);	xs_H.push_back(6.10568);
  eV_H.push_back(13.600541506433);	xs_H.push_back(5.9924);;

  cuFP_t dx;

  xsc_H = resampleArray(eV_H, xs_H, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_Emin), &eV_H[0], sizeof(cuFP_t)),
		 __FILE__, __LINE__, "Error copying cuH_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_H), &dx, sizeof(cuFP_t)),
		 __FILE__, __LINE__, "Error copying cuH_H");

  // Total cross section from LaBahn & Callaway, 1966, Phys. Rev., 147, 50, 28-40
  //
  std::vector<cuFP_t> eV_He, xs_He;

  eV_He.push_back(0.0972135);	xs_He.push_back(62.2773619684373);
  eV_He.push_back(0.212908);	xs_He.push_back(65.3193661349083);
  eV_He.push_back(0.251768);	xs_He.push_back(66.6419766420696);
  eV_He.push_back(0.440878);	xs_He.push_back(67.8332685763108);
  eV_He.push_back(0.704798);	xs_He.push_back(68.6287198361997);
  eV_He.push_back(1.11846);	xs_He.push_back(68.7641224795695);
  eV_He.push_back(1.5694);	xs_He.push_back(68.5696578943123);
  eV_He.push_back(1.86971);	xs_He.push_back(68.109100411296 );
  eV_He.push_back(2.20762);	xs_He.push_back(67.6491712468105);
  eV_He.push_back(2.50774);	xs_He.push_back(66.9903792673527);
  eV_He.push_back(2.77027);	xs_He.push_back(66.3312731286295);
  eV_He.push_back(3.07045);	xs_He.push_back(65.7387687541625);
  eV_He.push_back(3.25779);	xs_He.push_back(65.0790342969086);
  eV_He.push_back(3.44532);	xs_He.push_back(64.6178484953617);
  eV_He.push_back(3.78297);	xs_He.push_back(63.8930830701785);
  eV_He.push_back(4.0079);	xs_He.push_back(63.2339769314554);
  eV_He.push_back(4.2329);	xs_He.push_back(62.6405300791922);
  eV_He.push_back(4.57055);	xs_He.push_back(61.9160788132744);
  eV_He.push_back(4.79555);	xs_He.push_back(61.3229461202767);
  eV_He.push_back(5.02048);	xs_He.push_back(60.6635258222882);
  eV_He.push_back(5.20782);	xs_He.push_back(60.0041055242997);
  eV_He.push_back(5.50788);	xs_He.push_back(59.2790259398512);
  eV_He.push_back(5.883);	xs_He.push_back(58.4226277824826);
  eV_He.push_back(6.14552);	xs_He.push_back(57.7635216437595);
  eV_He.push_back(6.48318);	xs_He.push_back(57.0390703778416);
  eV_He.push_back(6.89576);	xs_He.push_back(56.0507253290223);
  eV_He.push_back(7.27088);	xs_He.push_back(55.1943271716537);
  eV_He.push_back(7.68347);	xs_He.push_back(54.2059821228344);
  eV_He.push_back(7.98353);	xs_He.push_back(53.4809025383858);
  eV_He.push_back(8.32118);	xs_He.push_back(52.756451272468);
  eV_He.push_back(8.6963);	xs_He.push_back(51.9000531150995);
  eV_He.push_back(8.99617);	xs_He.push_back(50.9767390342094);
  eV_He.push_back(9.33408);	xs_He.push_back(50.5168098697239);
  eV_He.push_back(9.67173);	xs_He.push_back(49.7920444445407);
  eV_He.push_back(9.97191);	xs_He.push_back(49.1995400700737);
  eV_He.push_back(10.2346);	xs_He.push_back(48.6726949820667);
  eV_He.push_back(10.4596);	xs_He.push_back(48.0795622890689);
  eV_He.push_back(10.7222);	xs_He.push_back(47.5527172010619);
  eV_He.push_back(10.9849);	xs_He.push_back(47.0921597180456);
  eV_He.push_back(11.2852);	xs_He.push_back(46.565628789304 );
  eV_He.push_back(11.5478);	xs_He.push_back(46.038783701297 );
  eV_He.push_back(11.773);	xs_He.push_back(45.57759789975  );
  eV_He.push_back(12.0731);	xs_He.push_back(44.9191200795576);
  eV_He.push_back(12.3734);	xs_He.push_back(44.4585625965413);
  eV_He.push_back(12.7488);	xs_He.push_back(43.866686540605 );
  eV_He.push_back(13.0489);	xs_He.push_back(43.2738680068726);
  eV_He.push_back(13.3867);	xs_He.push_back(42.6153901866802);
  eV_He.push_back(13.7996);	xs_He.push_back(41.9578548442838);
  eV_He.push_back(14.1375);	xs_He.push_back(41.4976115205329);
  eV_He.push_back(14.4752);	xs_He.push_back(40.9054213053313);
  eV_He.push_back(14.8132);	xs_He.push_back(40.5114655865711);
  eV_He.push_back(15.1509);	xs_He.push_back(39.8529877663787);
  eV_He.push_back(15.4889);	xs_He.push_back(39.4590320476185);
  eV_He.push_back(15.7893);	xs_He.push_back(39.064762169593 );
  eV_He.push_back(16.1271);	xs_He.push_back(38.5385454001167);
  eV_He.push_back(16.465);	xs_He.push_back(38.0123286306404);
  eV_He.push_back(16.8404);	xs_He.push_back(37.4864260204295);
  eV_He.push_back(17.1407);	xs_He.push_back(37.0258685374132);
  eV_He.push_back(17.5162);	xs_He.push_back(36.566253532193 );
  eV_He.push_back(17.8917);	xs_He.push_back(36.1063243677075);
  eV_He.push_back(18.2672);	xs_He.push_back(35.6463952032219);
  eV_He.push_back(18.6426);	xs_He.push_back(35.120492593011 );
  eV_He.push_back(19.0181);	xs_He.push_back(34.6608775877908);
  eV_He.push_back(19.3561);	xs_He.push_back(34.2669218690307);
  eV_He.push_back(19.6941);	xs_He.push_back(33.8729661502705);
  eV_He.push_back(20.0321);	xs_He.push_back(33.478696272245);
  eV_He.push_back(20.3324);	xs_He.push_back(33.0844263942195);
  eV_He.push_back(20.708);	xs_He.push_back(32.6907848347247);
  eV_He.push_back(21.046);	xs_He.push_back(32.2968291159645);
  eV_He.push_back(21.3839);	xs_He.push_back(31.836899951479 );
  eV_He.push_back(21.7594);	xs_He.push_back(31.37700220292  );
  eV_He.push_back(22.1725);	xs_He.push_back(30.9174814454794);
  eV_He.push_back(22.548);	xs_He.push_back(30.523808470058 );
  eV_He.push_back(22.9612);	xs_He.push_back(30.1304182379755);
  eV_He.push_back(23.3369);	xs_He.push_back(29.8689434814172);
  eV_He.push_back(23.5995);	xs_He.push_back(29.3421612252633);
  eV_He.push_back(23.9752);	xs_He.push_back(29.080686468705 );
  eV_He.push_back(24.3506);	xs_He.push_back(28.4886847490626);
  eV_He.push_back(24.8389);	xs_He.push_back(28.0958914195842);
  eV_He.push_back(25.2144);	xs_He.push_back(27.6360879188048);
  eV_He.push_back(25.8906);	xs_He.push_back(27.1125729190106);
  eV_He.push_back(26.3788);	xs_He.push_back(26.5875499547427);
  eV_He.push_back(26.7543);	xs_He.push_back(26.1277464539633);
  eV_He.push_back(27.2427);	xs_He.push_back(25.734953124485 );
  eV_He.push_back(27.7686);	xs_He.push_back(25.342473954272 );
  eV_He.push_back(28.2944);	xs_He.push_back(24.8177337333429);
  eV_He.push_back(28.8205);	xs_He.push_back(24.5574841979195);
  eV_He.push_back(29.2337);	xs_He.push_back(24.164093965837 );
  eV_He.push_back(29.8723);	xs_He.push_back(23.706363916209 );
  eV_He.push_back(30.3607);	xs_He.push_back(23.3135705867306);
  eV_He.push_back(30.8868);	xs_He.push_back(23.1194201607388);
  eV_He.push_back(31.4127);	xs_He.push_back(22.7269409905258);
  eV_He.push_back(31.9388);	xs_He.push_back(22.4666600391759);
  eV_He.push_back(32.615);	xs_He.push_back(21.9431450393817);
  eV_He.push_back(33.3665);	xs_He.push_back(21.5524251610547);
  eV_He.push_back(34.0429);	xs_He.push_back(21.2272389054817);
  eV_He.push_back(34.6064);	xs_He.push_back(20.8350424786075);
  eV_He.push_back(35.0948);	xs_He.push_back(20.5083796744872);
  eV_He.push_back(35.5457);	xs_He.push_back(20.2475018205331);
  eV_He.push_back(36.0342);	xs_He.push_back(20.0530372352759);
  eV_He.push_back(36.4851);	xs_He.push_back(19.7921907972484);
  eV_He.push_back(37.0112);	xs_He.push_back(19.59800895533  );
  eV_He.push_back(37.5372);	xs_He.push_back(19.2716288945485);
  eV_He.push_back(38.0258);	xs_He.push_back(19.0771957252179);
  eV_He.push_back(38.5519);	xs_He.push_back(18.8830138832995);
  eV_He.push_back(38.9277);	xs_He.push_back(18.6876696520993);
  eV_He.push_back(39.4537);	xs_He.push_back(18.4273887007494);
  eV_He.push_back(39.9423);	xs_He.push_back(18.2329555314187);
  eV_He.push_back(40.506);	xs_He.push_back(18.0390878487657);
  eV_He.push_back(40.9569);	xs_He.push_back(17.7782099948116);
  eV_He.push_back(41.483);	xs_He.push_back(17.5840595688197);
  eV_He.push_back(41.9715);	xs_He.push_back(17.3895949835625);
  eV_He.push_back(42.3472);	xs_He.push_back(17.1281516429308);
  eV_He.push_back(42.7606);	xs_He.push_back(16.9991892645009);
  eV_He.push_back(43.174);	xs_He.push_back(16.8702583019976);
  eV_He.push_back(43.5498);	xs_He.push_back(16.6749140707974);
  eV_He.push_back(43.9631);	xs_He.push_back(16.479852582936);
  eV_He.push_back(44.339);	xs_He.push_back(16.3506074611673);
  eV_He.push_back(44.6772);	xs_He.push_back(16.2210795960598);
  eV_He.push_back(45.2033);	xs_He.push_back(16.0268977541414);
  eV_He.push_back(45.504);	xs_He.push_back(15.9631862551266);
  eV_He.push_back(45.8798);	xs_He.push_back(15.8339411333579);
  eV_He.push_back(46.2556);	xs_He.push_back(15.7046960115892);
  eV_He.push_back(46.5939);	xs_He.push_back(15.5751681464817);
  eV_He.push_back(46.857);	xs_He.push_back(15.5111424882016);
  eV_He.push_back(47.1952);	xs_He.push_back(15.3816146230941);
  eV_He.push_back(47.571);	xs_He.push_back(15.2523695013254);
  eV_He.push_back(47.8717);	xs_He.push_back(15.188626586384 );
  eV_He.push_back(48.1348);	xs_He.push_back(15.1246009281039);
  eV_He.push_back(48.4354);	xs_He.push_back(14.9947903196575);
  eV_He.push_back(48.7736);	xs_He.push_back(14.8652310386235);
  eV_He.push_back(49.0368);	xs_He.push_back(14.8673359057014);
  eV_He.push_back(49.2622);	xs_He.push_back(14.7368969787244);
  eV_He.push_back(49.5254);	xs_He.push_back(14.7389704298757);
  eV_He.push_back(49.7885);	xs_He.push_back(14.6749447715956);
  eV_He.push_back(49.9763);	xs_He.push_back(14.4781239918482);

  xsc_He = resampleArray(eV_He, xs_He, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_Emin), &eV_He[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuHe_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuHe_H");

  // Interpolated from Figure 1 of "Elastic scattering and charge
  // transfer in slow collisions: isotopes of H and H + colliding with
  // isotopes of H and with He" by Predrag S Krstić and David R Schultz,
  // 1999 J. Phys. B: At. Mol. Opt. Phys. 32 3485
  //

  std::vector<cuFP_t> eV_pH, xs_pH;

  eV_pH.push_back(-0.994302);	xs_pH.push_back(2.86205);
  eV_pH.push_back(-0.897482);	xs_pH.push_back(2.90929);
  eV_pH.push_back(-0.801179);	xs_pH.push_back(2.86016);
  eV_pH.push_back(-0.691555);	xs_pH.push_back(2.89417);
  eV_pH.push_back(-0.588753);	xs_pH.push_back(2.85638);
  eV_pH.push_back(-0.49242);	xs_pH.push_back(2.81291);
  eV_pH.push_back(-0.395965);	xs_pH.push_back(2.79213);
  eV_pH.push_back(-0.292839);	xs_pH.push_back(2.8148);
  eV_pH.push_back(-0.19019);	xs_pH.push_back(2.74866);
  eV_pH.push_back(-0.0872765);	xs_pH.push_back(2.73165);
  eV_pH.push_back(0.00935082);	xs_pH.push_back(2.74299);
  eV_pH.push_back(0.112152);	xs_pH.push_back(2.7052);
  eV_pH.push_back(0.208688);	xs_pH.push_back(2.69953);
  eV_pH.push_back(0.311612);	xs_pH.push_back(2.68441);
  eV_pH.push_back(0.401578);	xs_pH.push_back(2.65417);
  eV_pH.push_back(0.517468);	xs_pH.push_back(2.65606);
  eV_pH.push_back(0.613862);	xs_pH.push_back(2.62394);
  eV_pH.push_back(0.716846);	xs_pH.push_back(2.62016);
  eV_pH.push_back(0.819688);	xs_pH.push_back(2.58992);
  eV_pH.push_back(0.909797);	xs_pH.push_back(2.58614);
  eV_pH.push_back(1.01906);	xs_pH.push_back(2.55213);
  eV_pH.push_back(1.1092);	xs_pH.push_back(2.55402);
  eV_pH.push_back(1.21203);	xs_pH.push_back(2.52189);
  eV_pH.push_back(1.3085);	xs_pH.push_back(2.50488);
  eV_pH.push_back(1.41149);	xs_pH.push_back(2.5011);
  eV_pH.push_back(1.52077);	xs_pH.push_back(2.47087);
  eV_pH.push_back(1.61715);	xs_pH.push_back(2.43685);
  eV_pH.push_back(1.71368);	xs_pH.push_back(2.42929);
  eV_pH.push_back(1.81666);	xs_pH.push_back(2.42551);
  eV_pH.push_back(1.9131);	xs_pH.push_back(2.40094);
  eV_pH.push_back(2.0159);	xs_pH.push_back(2.36315);

  xsc_pH = resampleArray(eV_pH, xs_pH, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPH_Emin), &eV_pH[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPH_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPH_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPH_H");

  // Interpolated from the top panel of Figure 4, op. cit.
  //
  std::vector<cuFP_t> eV_pHe, xs_pHe;

  eV_pHe.push_back(-0.984127);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.904762);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.825397);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.753968);	xs_pHe.push_back(2.64444);
  eV_pHe.push_back(-0.674603);	xs_pHe.push_back(2.6);
  eV_pHe.push_back(-0.595238);	xs_pHe.push_back(2.57778);
  eV_pHe.push_back(-0.515873);	xs_pHe.push_back(2.57778);
  eV_pHe.push_back(-0.444444);	xs_pHe.push_back(2.55556);
  eV_pHe.push_back(-0.373016);	xs_pHe.push_back(2.48889);
  eV_pHe.push_back(-0.293651);	xs_pHe.push_back(2.44444);
  eV_pHe.push_back(-0.214286);	xs_pHe.push_back(2.46667);
  eV_pHe.push_back(-0.142857);	xs_pHe.push_back(2.44444);
  eV_pHe.push_back(-0.0634921);	xs_pHe.push_back(2.4);
  eV_pHe.push_back(0.015873);	xs_pHe.push_back(2.37778);
  eV_pHe.push_back(0.0952381);	xs_pHe.push_back(2.37778);
  eV_pHe.push_back(0.166667);	xs_pHe.push_back(2.33333);
  eV_pHe.push_back(0.246032);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.325397);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.404762);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.47619);	xs_pHe.push_back(2.24444);
  eV_pHe.push_back(0.555556);	xs_pHe.push_back(2.2);
  eV_pHe.push_back(0.634921);	xs_pHe.push_back(2.17778);
  eV_pHe.push_back(0.706349);	xs_pHe.push_back(2.2);
  eV_pHe.push_back(0.785714);	xs_pHe.push_back(2.17778);
  eV_pHe.push_back(0.865079);	xs_pHe.push_back(2.13333);
  eV_pHe.push_back(0.936508);	xs_pHe.push_back(2.08889);
  eV_pHe.push_back(1.01587);	xs_pHe.push_back(2.06667);
  eV_pHe.push_back(1.09524);	xs_pHe.push_back(2.08889);
  eV_pHe.push_back(1.16667);	xs_pHe.push_back(2.06667);
  eV_pHe.push_back(1.24603);	xs_pHe.push_back(2.04444);
  eV_pHe.push_back(1.3254);	xs_pHe.push_back(2.02222);
  eV_pHe.push_back(1.40476);	xs_pHe.push_back(1.97778);
  eV_pHe.push_back(1.47619);	xs_pHe.push_back(1.93333);
  eV_pHe.push_back(1.55556);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.63492);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.71429);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.79365);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.87302);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.95238);	xs_pHe.push_back(1.91111);

  xsc_pHe = resampleArray(eV_pHe, xs_pHe, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPHe_Emin), &eV_pHe[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPHe_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPHe_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPHe_H");
}

__device__
cuFP_t cudaGeometric(int Z)
{
  if (Z>0 and Z< numRadii) {
    return cudaRadii[Z] * 1.0e-3;
  } else {
    return 0.0;
  }
}
		 
__device__
cuFP_t cudaElasticInterp(cuFP_t E, cuFP_t Emin, cuFP_t H, dArray<cuFP_t> xsc)
{
  int indx = 0;
  if (E >= Emin+H*xsc._s) indx = xsc._s - 2;
  else                    indx = floor( (E - Emin)/H );

  cuFP_t a = E - Emin - H*indx;
  cuFP_t b = Emin + H*(indx+1) - E;

  return a*xsc._v[indx] + b*xsc._v[indx+1];
}

