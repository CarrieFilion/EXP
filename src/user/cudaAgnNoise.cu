#include "hip/hip_runtime.h"
// -*- C++ -*-

#include <cudaUtil.cuH>
#include <cudaReduce.cuH>

#include <Component.H>
#include "UserAgnNoise.H"

// Global device symbols for CUDA kernel
//
__device__ __constant__
cuFP_t userAgnR0, userAgnTau1, userAgnTau2, userAgnEps;

__device__ __constant__
int userAgnLoc;

// Cuda implementation of AGN mass setup
//
__global__ void
userAgnSetupKernel(dArray<cudaParticle> P, int stride, cuFP_t tau1)
{
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {

    int npart = tid*stride + n; // Index in the stride
    
    if (npart < P._s) {
      
      cudaParticle & p = P._v[npart];
      
      p.datr[userAgnLoc+0] = p.mass;    // Initial mass
      p.datr[userAgnLoc+1] = -32.0*tau1; // Large negative value
      
    } // Particle index block

  } // END: stride loop
  
}

// Cuda implementation of AGN mass update with no level control
//
__global__ void
userAgnNoiseKernel(dArray<cudaParticle> P, int stride,
		   cuFP_t tnow, cuFP_t tev)
{
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  
  for (int n=0; n<stride; n++) {
    int npart = tid*stride + n;	// Index in the stride

    if (npart < P._s) {
      
      cudaParticle & p = P._v[npart];
      
      if (tnow > tev) {
	cuFP_t rr = 0.0;
	for (int k=0; k<3; k++) rr += p.pos[k]*p.pos[k];
	if (rr < userAgnR0*userAgnR0) {
	  p.datr[userAgnLoc+1] = tnow;
	}
      }

      p.mass = p.datr[userAgnLoc] *
	(1.0 - userAgnEps*exp(-(tnow - p.datr[userAgnLoc+1])/userAgnTau2));
      
    } // Particle index block

  } // END: stride loop

}

__global__
void testConstantsAgnNoise(cuFP_t tnow)
{
  printf("------------------------------\n");
  printf("---cudaAgnNoise constants-----\n");
  printf("------------------------------\n");
  printf("   Time   = %e\n", tnow          );
  printf("   R0     = %e\n", userAgnR0     );
  printf("   Tau2   = %e\n", userAgnTau2   );
  printf("   eps    = %e\n", userAgnEps    );
  printf("   loc    = %d\n", userAgnLoc    );
  printf("------------------------------\n");
}

// Cuda implementation of AGN mass update with level control
//
__global__ void
userAgnNoiseKernel(dArray<cudaParticle> P, dArray<int> I, int stride,
		   PII lohi, cuFP_t tnow, cuFP_t tev)
{
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n; // Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < P._s) {
      
      cudaParticle & p = P._v[I._v[npart]];
      
      if (tnow > tev) {
	cuFP_t rr = 0.0;
	for (int k=0; k<3; k++) rr += p.pos[k]*p.pos[k];
	if (rr < userAgnR0*userAgnR0) {
	  p.datr[userAgnLoc+1] = tnow;
	}
      }

      p.mass = p.datr[userAgnLoc]*(1.0 - userAgnEps*exp(-(tnow - p.datr[userAgnLoc+1])/userAgnTau2));
      
    } // Particle index block

  } // END: stride loop

  // DONE
}


void UserAgnNoise::determine_acceleration_and_potential_cuda()
{
  // Sanity check
  //
  int nbodies = cC->Number();
  if (nbodies != static_cast<int>(cC->Particles().size())) {
    std::cerr << "UserAgnNoise: ooops! number=" << nbodies
	      << " but particle size=" << cC->Particles().size() << endl;
    nbodies = static_cast<int>(cC->Particles().size());
  }
  
  if (nbodies==0) {		// Return if there are no particles
    if (VERBOSE>4) {
      std::cout << "Process " << myid << ": in UserAgnNoise, nbodies=0" 
		<< " for Component <" << cC->name << "> at T=" << tnow
		<< std::endl;
    }
    return;
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cC->cudaDevice);
  cuda_check_last_error_mpi("hipGetDeviceProperties", __FILE__, __LINE__, myid);

  // Stream structure iterators
  //
  auto cr = cC->cuStream;

  // VERBOSE diagnostic output on first ncheck calls
  //
  static bool first_time = true;
  
  if (first_time) {

    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userAgnR0), &R0, sizeof(cuFP_t),
				      size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying userAgnR0");
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userAgnTau2), &tau2, sizeof(cuFP_t),
				      size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying userAgnTau2");
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userAgnEps), &eps, sizeof(cuFP_t),
				      size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying userSatCen");
    
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(userAgnLoc), &loc, sizeof(int),
				      size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying userSatPos");

    // Verbose check of Cuda constants
    if (myid==0) {
      testConstantsAgnNoise<<<1, 1, 0, cr->stream>>>(tnow);
      hipDeviceSynchronize();
      cuda_check_last_error_mpi("hipDeviceSynchronize", __FILE__, __LINE__, myid);
    }

    first_time = false;
  }

  // Compute grid
  //
  unsigned int N         = cr->cuda_particles.size();
  unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
  unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
  if (N>0) {

    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

    // Do the work
    //
    userAgnNoiseKernel<<<gridSize, BLOCK_SIZE, 0, cr->stream>>>
      (toKernel(cr->cuda_particles), stride, tnow, tev);
  }

}


 void UserAgnNoise::setup_decay_cuda(void)
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cC->cudaDevice);
  cuda_check_last_error_mpi("hipGetDeviceProperties", __FILE__, __LINE__, myid);

  // Stream structure iterators
  //
  auto cr = cC->cuStream;

  // Compute grid
  //
  unsigned int N         = cr->cuda_particles.size();
  unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
  unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
  if (N>0) {

    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

    // Do the work
    //
    userAgnSetupKernel<<<gridSize, BLOCK_SIZE, 0, cr->stream>>>
      (toKernel(cr->cuda_particles), stride, tau1);
  }
}
