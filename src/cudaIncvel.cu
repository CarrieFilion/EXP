#include "hip/hip_runtime.h"
// -*- C++ -*-

#include <Component.H>
#include <expand.h>
#include <cudaUtil.cuH>
#include <cudaReduce.cuH>
#include <cudaParticle.cuH>

#include <boost/make_shared.hpp>

__global__ void velocityKick
(dArray<cudaParticle> P, dArray<int> I, cuFP_t dt, int dim, int stride, PII lohi)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Particle counter
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {

#ifdef BOUNDS_CHECK
      if (npart>=P._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle * p = &P._v[I._v[npart]];
    
      for (int k=0; k<dim; k++) p->vel[k] += p->acc[k]*dt;
    }
  }
}

__global__ void velocityDebug
(dArray<cudaParticle> P, dArray<int> I, int stride, PII lohi)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Particle counter
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second and npart < I._s) {

      cudaParticle & p = P._v[I._v[npart]];
    
      printf("%d vel a=(%13.6e %13.6e %13.6e) p=%13.6e\n", i, p.acc[0], p.acc[1], p.acc[2], p.pot);
    }
  }
}


void incr_velocity_cuda(cuFP_t dt, int mlevel)
{
  for (auto c : comp->components) {

    auto cr = c->cuStream;

    PII lohi = {0, cr->cuda_particles.size()};

    if (multistep) {		// Get particle range
      lohi = c->CudaGetLevelRange(mlevel, multistep);
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, c->cudaDevice);

    // Compute grid
    //
    unsigned int N         = lohi.second - lohi.first;
    unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
    if (N>0) {
      
      if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

      // Do the work
      //
      velocityKick<<<gridSize, BLOCK_SIZE>>>
	(toKernel(c->cuStream->cuda_particles),
	 toKernel(c->cuStream->indx1), dt, c->dim, stride, lohi);
    }

    // DEBUGGING output
    //
    if (false) {
      PII lohi(0, std::min<int>(3, cr->cuda_particles.size()));

      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, c->cudaDevice);

      // Compute grid
      //
      unsigned int N         = lohi.second - lohi.first;
      unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
      unsigned int gridSize  = N/BLOCK_SIZE/stride;
      
      if (N>0) {
	
	if (N > gridSize*BLOCK_SIZE*stride) gridSize++;
	
	// Do the work
	//
	velocityDebug<<<gridSize, BLOCK_SIZE>>>
	  (toKernel(cr->cuda_particles), toKernel(cr->indx1), stride, lohi);
      }
    }
    // END: DEBUG
  }
  // END: component loop
}
