#include "hip/hip_runtime.h"
// -*- C++ -*-

#include <thrust/tuple.h>

#include <cudaReduce.cuH>
#include <Component.H>
#include <Cube.H>

#include "expand.H"

// Define for debugging
//
// #define BOUNDS_CHECK
// #define VERBOSE_RPT
// #define VERBOSE_DBG

// Global symbols for cube construction
//
__device__ __constant__
int cubeNumX, cubeNumY, cubeNumZ, cubeNX, cubeNY, cubeNZ, cubeNdim;

__device__ __constant__
cuFP_t cubeDfac;

__device__ __constant__
std::complex<cuFP_t> cubeKfac;

// Index functions for coefficients based on Eigen Tensor packing order
//
__host__ __device__
int Index(int i, int j, int k, int numx, int numy)
{
  return k*numx*numy + j*numx + i;
}

// Index function for modulus coefficients
//
__host__ __device__
thrust::tuple<int, int, int> Indices(int indx, int numx, int numy)
{
  int k = indx/(numx*numy);
  int j = indx/numx - k*numy;
  int i = indx - (j + k*numy)*numx;

  return {i, j, k};
}


__global__
void testConstantsCube()
{
  printf("-------------------------\n");
  printf("---CubeBasis constants---\n");
  printf("-------------------------\n");
  printf("   Numx   = %d\n", cubeNumX );
  printf("   Numy   = %d\n", cubeNumY );
  printf("   Numy   = %d\n", cubeNumZ );
  printf("   Nx     = %d\n", cubeNX   );
  printf("   Ny     = %d\n", cubeNY   );
  printf("   Nz     = %d\n", cubeNZ   );
  printf("   Dfac   = %e\n", cubeDfac );
  // printf("   Kfac   = (%e, %e)\n",
  // cubeKfac.real, cubeKfac.imag);
  printf("-------------------------\n");
}

// Initialize for streams
//
void Cube::cuda_initialize()
{
  // Nothing so far
}

// Copy constants to device
//
void Cube::initialize_constants()
{
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeNX), &nmaxx, sizeof(int),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeNX");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeNY), &nmaxy, sizeof(int),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeNY");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeNZ), &nmaxz, sizeof(int),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeNZ");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeNumX), &imx, sizeof(int),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeNumX");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeNumY), &imy, sizeof(int),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeNumY");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeNumZ), &imz, sizeof(int),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeNumZ");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeNdim), &osize, sizeof(int),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeNdim");

  cuFP_t dfac = 2.0*M_PI;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeDfac), &dfac, sizeof(cuFP_t),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeDfac");

  std::complex<cuFP_t> kfac = std::complex<cuFP_t>(0.0, 2.0*M_PI);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cubeKfac), &kfac,
				    sizeof(std::complex<cuFP_t>),
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cubeKfac");
}

__global__ void coefKernelCube
(dArray<cudaParticle> P, dArray<int> I, dArray<thrust::complex<cuFP_t>> coef,
 int stride, PII lohi)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int N   = lohi.second - lohi.first;

  for (int n=0; n<stride; n++) {

    // Particle counter
    //
    int i     = tid*stride + n;
    int npart = i + lohi.first;

    if (npart < lohi.second) {	// Check that particle index is in
				// range for consistency with other
				// kernels

#ifdef BOUNDS_CHECK
      if (npart>=P._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle & p = P._v[I._v[npart]];
      
      cuFP_t pos[3] = {p.pos[0], p.pos[1], p.pos[2]};
      cuFP_t mm     = p.mass;

      for (int k=0; k<3; k++) {
	if (pos[k]<0.0)	pos[k] += std::floor(-pos[k]) + 1.0;
	if (pos[k]>1.0) pos[k] -= std::floor( pos[k]);
      }

      // Index loop
      //
      for (int s=0; s<cubeNdim; s++) {

	// Get the tensor indices
	int ii, jj, kk;
	thrust::tie(ii, jj, kk) = Indices(s, cubeNumX, cubeNumY);

	// Wave numbers
	ii -= cubeNX;
	jj -= cubeNY;
	kk -= cubeNZ;

	// Make the the basis function
	double expon = pos[0]*ii + pos[1]*jj + pos[2]*kk;
	double norm  = sqrt(M_PI*(ii*ii + jj*jj + kk*kk));
			    
	coef._v[s*N + i] += -mm*thrust::exp(thrust::complex<cuFP_t>(0.0, -expon))/norm;
      }
      // index loop

    } // particle index check

  } // stride loop

}


__global__ void
forceKernelCube(dArray<cudaParticle> P, dArray<int> I,
		dArray<thrust::complex<cuFP_t>> coef, int stride, PII lohi)
{
  // Thread ID
  //
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {	// Check that particle index is in
				// range
      
#ifdef BOUNDS_CHECK
      if (npart>=P._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle & p = P._v[I._v[npart]];
      
      thrust::complex<cuFP_t> acc[3] = {0.0, 0.0, 0.0};
      cuFP_t xx=0.0, yy=0.0, zz=0.0, mm = p.mass;
      cuFP_t pos[3] = {p.pos[0], p.pos[1], p.pos[2]};
      cuFP_t ind[3];

      // Index loop
      //
      for (int s=0; s<cubeNdim; s++) {

	thrust::tie(ind[0], ind[1], ind[2]) = Indices(s, cubeNumX, cubeNumY);

	ind[0] -= cubeNX;
	ind[1] -= cubeNY;
	ind[2] -= cubeNZ;

	// Recursion multipliers
	cuFP_t expon = 0.0, norm = 0.0;
	for (int k=0; k<3; k++) {
	  expon += pos[k]*ind[k];
	  norm  += ind[k]*ind[k];
	}
	norm = sqrt(M_PI*norm);

	thrust::complex<cuFP_t> pfac  =
	  thrust::exp(thrust::complex<cuFP_t>(0.0, cubeDfac*expon))/norm * coef._v[s];

	for (int k=0; k<3; k++) {
	  acc[k] += -thrust::complex<cuFP_t>(0.0, cubeDfac*ind[k]) * pfac;
	}
      }
      // index loop

      for (int k=0; k<3; k++) p.acc[k] = acc[k].real();

    } // particle index check

  } // stride loop
}

template<typename T>
class LessAbs : public std::binary_function<bool, T, T>
{
public:
  bool operator()( const T &a, const T &b ) const
  {
    return (thrust::abs(a) < thrust::abs(b));
  }
};

void Cube::cudaStorage::resize_coefs(int N, int osize, int gridSize, int stride)
{
  // Reserve space for coefficient reduction
  //
  if (dN_coef.capacity() < osize*N)
    dN_coef.reserve(osize*N);
  
  if (dc_coef.capacity() < osize*gridSize)
    dc_coef.reserve(osize*gridSize);
  
  // Set space for current step
  //
  dN_coef.resize(osize*N);
  dc_coef.resize(osize*gridSize);
  dw_coef.resize(osize);	// This will stay fixed
}


void Cube::cuda_zero_coefs()
{
  auto cr = component->cuStream;
  
  // Resize output array
  //
  cuS.df_coef.resize(osize);
    
  // Zero output array
  //
  thrust::fill(thrust::cuda::par.on(cr->stream),
	       cuS.df_coef.begin(), cuS.df_coef.end(), 0.0);
}

void Cube::determine_coefficients_cuda()
{
  // Only do this once but copying mapping coefficients and textures
  // must be done every time
  //
  if (initialize_cuda_cube) {
    initialize_cuda();
    initialize_cuda_cube = false;
  }

  // Copy coordinate mapping
  //
  initialize_constants();


  std::cout << std::scientific;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, component->cudaDevice);
  cuda_check_last_error_mpi("hipGetDeviceProperties", __FILE__, __LINE__, myid);

  // This will stay fixed for the entire run
  //
  host_coefs.resize(osize);

  // Get the stream for this component
  //
  auto cs = component->cuStream;

  // VERBOSE diagnostic output on first call
  //
  static bool firstime = true;

  if (firstime and myid==0 and VERBOSE>4) {
    testConstantsCube<<<1, 1, 0, cs->stream>>>();
    hipDeviceSynchronize();
    cuda_check_last_error_mpi("hipDeviceSynchronize", __FILE__, __LINE__, myid);
    firstime = false;
  }
  
  // Zero counter and coefficients
  //
  thrust::fill(host_coefs.begin(), host_coefs.end(), 0.0);

  // Zero out coefficient storage
  //
  cuda_zero_coefs();

  // Get sorted particle range for mlevel
  //
  PII lohi = component->CudaGetLevelRange(mlevel, mlevel), cur;

  if (false) {
    for (int n=0; n<numprocs; n++) {
      if (myid==n) std::cout << "[" << myid << "] mlevel=" << mlevel
			     << " coef check (lo, hi) = (" << lohi.first << ", "
			     << lohi.second << ")" << std::endl
			     << std::string(60, '-') << std::endl;
      MPI_Barrier(MPI_COMM_WORLD);
    }
  }
  
  unsigned int Ntotal = lohi.second - lohi.first;
  unsigned int Npacks = Ntotal/component->bunchSize + 1;

  // Loop over bunches
  //
  for (int n=0; n<Npacks; n++) {

    // Current bunch
    //
    cur. first = lohi.first + component->bunchSize*n;
    cur.second = lohi.first + component->bunchSize*(n+1);
    cur.second = std::min<unsigned int>(cur.second, lohi.second);

    if (cur.second <= cur.first) break;
    
    // Compute grid
    //
    unsigned int N         = cur.second - cur.first;
    unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    unsigned int gridSize  = N/BLOCK_SIZE/stride;
  
    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

#ifdef VERBOSE_RPT
    static unsigned debug_max_count = 100;
    static unsigned debug_cur_count = 0;
    if (debug_cur_count++ < debug_max_count) {
      std::cout << std::endl
		<< "** -------------------------" << std::endl
		<< "** cudaCube coefficients" << std::endl
		<< "** -------------------------" << std::endl
		<< "** N      = " << N            << std::endl
		<< "** Npacks = " << Npacks       << std::endl
		<< "** I low  = " << cur.first    << std::endl
		<< "** I high = " << cur.second   << std::endl
		<< "** Stride = " << stride       << std::endl
		<< "** Block  = " << BLOCK_SIZE   << std::endl
		<< "** Grid   = " << gridSize     << std::endl
		<< "** Level  = " << mlevel       << std::endl
		<< "** lo     = " << lohi.first   << std::endl
		<< "** hi     = " << lohi.second  << std::endl
		<< "**" << std::endl;
  }
#endif
  
    // Adjust cached storage, if necessary
    //
    cuS.resize_coefs(N, osize, gridSize, stride);
    
    // Shared memory size for the reduction
    //
    int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);
    
    // Compute the coefficient contribution for each order
    //
    auto beg  = cuS.df_coef.begin();
    std::vector<thrust::device_vector<cuFP_t>::iterator> bg, bh;

    coefKernelCube<<<gridSize, BLOCK_SIZE, 0, cs->stream>>>
      (toKernel(cs->cuda_particles), toKernel(cs->indx1),
       toKernel(cuS.dN_coef), stride, lohi);
      
    // Begin the reduction by blocks [perhaps this should use a
    // stride?]
    //
    unsigned int gridSize1 = N/BLOCK_SIZE;
    if (N > gridSize1*BLOCK_SIZE) gridSize1++;

    reduceSum<thrust::complex<cuFP_t>, BLOCK_SIZE>
      <<<gridSize1, BLOCK_SIZE, sMemSize, cs->stream>>>
      (toKernel(cuS.dc_coef), toKernel(cuS.dN_coef), osize, N);
      
    // Finish the reduction for this order in parallel
    //
    thrust::counting_iterator<int> index_begin(0);
    thrust::counting_iterator<int> index_end(gridSize1*osize);

    // The key_functor indexes the sum reduced series by array index
    //
    thrust::reduce_by_key
      (
       thrust::cuda::par.on(cs->stream),
       thrust::make_transform_iterator(index_begin, key_functor(gridSize1)),
       thrust::make_transform_iterator(index_end,   key_functor(gridSize1)),
       cuS.dc_coef.begin(), thrust::make_discard_iterator(), cuS.dw_coef.begin()
       );
    
    thrust::transform(thrust::cuda::par.on(cs->stream),
		      cuS.dw_coef.begin(), cuS.dw_coef.end(),
		      beg, beg, thrust::plus<thrust::complex<cuFP_t>>());
    
    thrust::advance(beg, osize);

    use1 += N;			// Increment particle count
  }

  // Accumulate the coefficients from the device to the host
  //
  host_coefs = cuS.df_coef;

  // DEBUG
  //
  if (false) {
    std::cout << std::string(3*4+4*20, '-') << std::endl
	      << "---- Polar "      << std::endl
	      << std::string(3*4+4*20, '-') << std::endl
	      << std::setprecision(10);

    std::cout << std::setw(4)  << "i"
	      << std::setw(4)  << "j"
	      << std::setw(4)  << "k"
	      << std::setw(20) << "GPU"
	      << std::setw(20) << "CPU"
	      << std::setw(20) << "diff"
	      << std::setw(20) << "rel diff"
	      << std::endl;
    
    auto cmax = std::max_element(host_coefs.begin(), host_coefs.begin()+osize,
				 LessAbs<thrust::complex<cuFP_t>>());

    for (int n=0; n<osize; n++) {
      int i, j, k;
      thrust::tie(i, j, k) = Indices(n, nmaxx, nmaxy);
      auto a = static_cast<std::complex<double>>(host_coefs[n]);
      auto b = expcoef[0](i, j, k);
      auto c = std::abs(a - b);
      std::cout << std::setw(4)  << i
		<< std::setw(4)  << j
		<< std::setw(4)  << k
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << c
		<< std::setw(20) << c/thrust::abs(*cmax)
		<< std::endl;
    }

    std::cout << std::string(3*4+4*20, '-') << std::endl;
  }


  //
  // TEST comparison of coefficients for debugging
  //
  if (false) {

    struct Element
    {
      std::complex<double> d;
      std::complex<double> f;
      
      int    i;
      int    j;
      int    k;
    }
    elem;

    std::multimap<double, Element> compare;

    std::ofstream out("test_cube.dat");

    // m loop
    for (int n=0; n<osize; n++) {
	
      thrust::tie(elem.i, elem.j, elem.k) = Indices(n, nmaxx, nmaxy);

      elem.d = expcoef[0](elem.i, elem.j, elem.k);
      elem.f = static_cast<std::complex<double>>(host_coefs[n]);
	  
      double test = std::abs(elem.d - elem.f);
      if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);
	  
      compare.insert(std::make_pair(test, elem));
	    
      out << std::setw( 5) << elem.i
	  << std::setw( 5) << elem.j
	  << std::setw( 5) << elem.k
	  << std::setw( 5) << n
	  << std::setw(20) << elem.d
	  << std::setw(20) << elem.f
	  << std::endl;
    }
    
    std::map<double, Element>::iterator best = compare.begin();
    std::map<double, Element>::iterator midl = best;
    std::advance(midl, compare.size()/2);
    std::map<double, Element>::reverse_iterator last = compare.rbegin();
    
    std::cout << std::string(3*3 + 3*20 + 20, '-') << std::endl
	      << "---- Cube coefficients" << std::endl
	      << std::string(3*3 + 3*20 + 20, '-') << std::endl;

    std::cout << "Best case: ["
	      << std::setw( 3) << best->second.i << ", "
	      << std::setw( 3) << best->second.j << ", "
	      << std::setw( 3) << best->second.k << "] = "
	      << std::setw(20) << best->second.d
	      << std::setw(20) << best->second.f
	      << std::setw(20) << fabs(best->second.d - best->second.f)
	      << std::endl;
  
    std::cout << "Mid case:  ["
	      << std::setw( 3) << midl->second.i << ", "
	      << std::setw( 3) << midl->second.j << ", "
	      << std::setw( 3) << midl->second.k << "] = "
	      << std::setw(20) << midl->second.d
	      << std::setw(20) << midl->second.f
	      << std::setw(20) << fabs(midl->second.d - midl->second.f)
	      << std::endl;
    
    std::cout << "Last case: ["
	      << std::setw( 3) << last->second.i << ", "
	      << std::setw( 3) << last->second.j << ", "
	      << std::setw( 3) << last->second.k << "] = "
	      << std::setw(20) << last->second.d
	      << std::setw(20) << last->second.f
	      << std::setw(20) << fabs(last->second.d - last->second.f)
	      << std::endl;
  }

}


void Cube::determine_acceleration_cuda()
{
  // Only do this once but copying mapping coefficients and textures
  // must be done every time
  //
  if (initialize_cuda_cube) {
    initialize_cuda();
    initialize_cuda_cube = false;
  }

  // Copy coordinate mapping
  //
  initialize_constants();

  std::cout << std::scientific;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cC->cudaDevice);
  cuda_check_last_error_mpi("hipGetDeviceProperties", __FILE__, __LINE__, myid);

  auto cs = cC->cuStream;

  // Get particle index range for levels [mlevel, multistep]
  //
  PII lohi = cC->CudaGetLevelRange(mlevel, multistep);

  // Compute grid
  //
  unsigned int N         = lohi.second - lohi.first;
  unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
  unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
  if (N>0) {

    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

#ifdef VERBOSE_RPT
    static unsigned debug_max_count = 100;
    static unsigned debug_cur_count = 0;
    if (debug_cur_count++ < debug_max_count) {
      std::cout << std::endl
		<< "** -------------------------" << std::endl
		<< "** cudaCube acceleration" << std::endl
		<< "** -------------------------" << std::endl
		<< "** N      = " << N            << std::endl
		<< "** Stride = " << stride       << std::endl
		<< "** Block  = " << BLOCK_SIZE   << std::endl
		<< "** Grid   = " << gridSize     << std::endl
		<< "** Level  = " << mlevel       << std::endl
		<< "** lo     = " << lohi.first   << std::endl
		<< "** hi     = " << lohi.second  << std::endl
		<< "**" << std::endl;
    }
#endif
    
    // Shared memory size for the reduction
    //
    int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);
      
    forceKernelCube<<<gridSize, BLOCK_SIZE, sMemSize, cs->stream>>>
      (toKernel(cs->cuda_particles), toKernel(cs->indx1),
       toKernel(dev_coefs), stride, lohi);
  }
}

void Cube::HtoD_coefs()
{
  // Check size
  host_coefs.resize(osize);

  // Copy from Cube
  for (int i=0; i<host_coefs.size(); i++)
    host_coefs[i] = expcoef[0].data()[i];

  // Copy to device
  dev_coefs = host_coefs;
}


void Cube::DtoH_coefs(unsigned M)
{
  // Copy from host device to Cube
  for (int i=0; i<expcoef[0].size(); i++)
    expcoef[0].data()[i] = host_coefs[i];
}

void Cube::multistep_update_cuda()
{
  // The plan: for the current active level search above and below for
  // particles for correction to coefficient matrix
  //

  //! Sort the device vector by level changes
  auto chg = component->CudaSortLevelChanges();

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, component->cudaDevice);
  cuda_check_last_error_mpi("hipGetDeviceProperties", __FILE__, __LINE__, myid);
  auto cs = component->cuStream;
  
  // Step through all levels
  //
  for (int olev=mfirst[mstep]; olev<=multistep; olev++) {

    for (int nlev=0; nlev<=multistep; nlev++) {

      if (olev == nlev) continue;

      // Get range of update block in particle index
      //
      unsigned int Ntotal = chg[olev][nlev].second - chg[olev][nlev].first;

      if (Ntotal==0) continue; // No particles [from, to]=[olev, nlev]

      unsigned int Npacks = Ntotal/component->bunchSize + 1;

      // Zero out coefficient storage
      //
      cuda_zero_coefs();

#ifdef VERBOSE_DBG
      std::cout << "[" << myid << ", " << tnow
		<< "] Adjust cube: Ntotal=" << Ntotal << " Npacks=" << Npacks
		<< " for (m, d)=(" << olev << ", " << nlev << ")" << std::endl;
#endif

      // Loop over bunches
      //
      for (int n=0; n<Npacks; n++) {

	PII cur;
	
	// Current bunch
	//
	cur. first = chg[olev][nlev].first + component->bunchSize*n;
	cur.second = chg[olev][nlev].first + component->bunchSize*(n+1);
	cur.second = std::min<unsigned int>(cur.second, chg[olev][nlev].second);

	if (cur.second <= cur.first) break;
    
	// Compute grid
	//
	unsigned int N         = cur.second - cur.first;
	unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
	unsigned int gridSize  = N/BLOCK_SIZE/stride;
	
	if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

    
	// Adjust cached storage, if necessary
	//
	cuS.resize_coefs(N, osize, gridSize, stride);
	
	// Shared memory size for the reduction
	//
	int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);
    
	// Compute the coefficient contribution for each order
	//
	auto beg  = cuS.df_coef.begin();

	// Do the work!
	//
	coefKernelCube<<<gridSize, BLOCK_SIZE, 0, cs->stream>>>
	  (toKernel(cs->cuda_particles), toKernel(cs->indx1),
	   toKernel(cuS.dN_coef), stride, cur);
      
	unsigned int gridSize1 = N/BLOCK_SIZE;
	if (N > gridSize1*BLOCK_SIZE) gridSize1++;

	reduceSum<thrust::complex<cuFP_t>, BLOCK_SIZE>
	  <<<gridSize1, BLOCK_SIZE, sMemSize, cs->stream>>>
	  (toKernel(cuS.dc_coef), toKernel(cuS.dN_coef), osize, N);
	  
	// Finish the reduction for this order in parallel
	//
	thrust::counting_iterator<int> index_begin(0);
	thrust::counting_iterator<int> index_end(gridSize1*osize);

	// The key_functor indexes the sum reduced series by array index
	//
	thrust::reduce_by_key
	  (
	   thrust::cuda::par.on(cs->stream),
	   thrust::make_transform_iterator(index_begin, key_functor(gridSize1)),
	   thrust::make_transform_iterator(index_end,   key_functor(gridSize1)),
	   cuS.dc_coef.begin(), thrust::make_discard_iterator(), cuS.dw_coef.begin()
	   );

	thrust::transform(thrust::cuda::par.on(cs->stream),
			  cuS.dw_coef.begin(), cuS.dw_coef.end(),
			  beg, beg, thrust::plus<thrust::complex<cuFP_t>>());
	
	thrust::advance(beg, osize);
      }
      // END: bunches

      // Accumulate the coefficients from the device to the host
      //
      thrust::host_vector<thrust::complex<cuFP_t>> ret = cuS.df_coef;

      // Decrement current level and increment new level using the
      // Cube update matricies
      //
      for (int i=0; i<osize; i++) {
	std::complex<double> val = ret[i];
	differ1[0][olev].data()[i] -= val;
	differ1[0][nlev].data()[i] += val;
      }
    }
    // DONE: Inner loop
  }
  // DONE: Outer loop
}


void Cube::destroy_cuda()
{
  // Nothing
}


