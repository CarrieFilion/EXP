#include "hip/hip_runtime.h"
// -*- C++ -*-

#include <iostream>
#include <iomanip>

#include <Ion.H>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <Timer.h>
#include <cudaUtil.cuH>

thrust::device_vector<cuFP_t> xsc_H, xsc_He, xsc_pH, xsc_pHe;
__constant__ cuFP_t cuH_H, cuHe_H, cuPH_H, cuPHe_H;
__constant__ cuFP_t cuH_Emin, cuHe_Emin, cuPH_Emin, cuPHe_Emin;

// Atomic radii in picometers from Clementi, E.; Raimond, D. L.;
// Reinhardt, W. P. (1967). "Atomic Screening Constants from SCF
// Functions. II. Atoms with 37 to 86 Electrons". Journal of Chemical
// Physics 47 (4): 1300-1307.  See also Paper 1, ref. therein.
//
const int numRadii = 87;
__constant__ int cudaRadii[numRadii];

// For construction of evenly spaced interpolation arrays
//
thrust::host_vector<cuFP_t>
resampleArray(const std::vector<cuFP_t>& x, const std::vector<cuFP_t>& y,
	      cuFP_t& dx)
{
  // Get minimum grid spacing
  cuFP_t minH = std::numeric_limits<cuFP_t>::max();
  for (int i=0; i<x.size()-1; i++)
    minH = std::min<cuFP_t>(minH, x[i+1]- x[i]);

  // Resample based on minimum spacing
  int numH = int( (x.back() - x.front())/minH ) + 1;

  thrust::host_vector<cuFP_t> Y(numH);
  
  dx = (x.back() - x.front())/(numH - 1);

  for (int i=0; i<numH; i++) {
    cuFP_t xx = x.front() + dx*i, yy;
    if (xx <= x.front()) {
      yy = y.front();
    } else if (xx >= x.back()) {
      yy = y.back();
    } else {
      auto lb = std::lower_bound(x.begin(), x.end(), xx);
      auto ub = lb;
      if (lb!=x.begin()) lb--;
      if (ub == x.end()) ub = lb--;
      auto a = (*ub - xx)/(*ub - *lb);
      auto b = (xx - *lb)/(*ub - *lb);
      yy = a*y[lb - x.begin()] + b*y[ub - x.begin()];
    }
    Y[i] = yy;
  }

  return Y;
}

// Initialize cross-section look up and interpolation arrays.  Data
// input could be generalized here . . . for later.
//
void cudaElasticInit()
{
  std::vector<int> radii(numRadii, 0);

  radii[1]  =  53;
  radii[2]  =  31;
  radii[3]  =  167;
  radii[4]  =  112;
  radii[5]  =  87;
  radii[6]  =  67;
  radii[7]  =  56;
  radii[8]  =  48;
  radii[9]  =  42;
  radii[10] =  38;
  radii[11] =  190;
  radii[12] =  145;
  radii[13] =  118;
  radii[14] =  111;
  radii[15] =  98;
  radii[16] =  180;
  radii[17] =  79;
  radii[18] =  188;
  radii[19] =  243;
  radii[20] =  194;
  radii[21] =  184;
  radii[22] =  176;
  radii[23] =  171;
  radii[24] =  166;
  radii[25] =  161;
  radii[26] =  156;
  radii[27] =  152;
  radii[28] =  149;
  radii[29] =  145;
  radii[30] =  152;
  radii[31] =  136;
  radii[32] =  125;
  radii[33] =  114;
  radii[34] =  103;
  radii[35] =  94;
  radii[36] =  88;
  radii[37] =  265;
  radii[38] =  219;
  radii[39] =  212;
  radii[40] =  206;
  radii[41] =  198;
  radii[42] =  190;
  radii[43] =  183;
  radii[44] =  178;
  radii[45] =  173;
  radii[46] =  169;
  radii[47] =  172;
  radii[48] =  161;
  radii[49] =  193;
  radii[50] =  217;
  radii[51] =  133;
  radii[52] =  123;
  radii[53] =  198;
  radii[54] =  108;
  radii[55] =  298;
  radii[56] =  268;
  radii[59] =  247;
  radii[60] =  206;
  radii[61] =  205;
  radii[62] =  238;
  radii[63] =  231;
  radii[64] =  233;
  radii[65] =  225;
  radii[66] =  228;
  radii[68] =  226;
  radii[69] =  222;
  radii[70] =  222;
  radii[71] =  217;
  radii[72] =  208;
  radii[73] =  200;
  radii[74] =  193;
  radii[75] =  188;
  radii[76] =  185;
  radii[77] =  180;
  radii[78] =  177;
  radii[79] =  166;
  radii[80] =  171;
  radii[81] =  156;
  radii[82] =  202;
  radii[83] =  143;
  radii[84] =  135;
  radii[86] =  120;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cudaRadii), &radii[0], sizeof(int)*numRadii), 
		 __FILE__, __LINE__, "Error copying cudaRadii");

  // Total cross section from Malik & Trefftz, 1960, Zeitschrift fur Astrophysik, 50, 96-109

  // Column 1 in eV, Column2 in Bohr cross section (pi*a_0^2) units

  std::vector<cuFP_t> eV_H, xs_H;

  eV_H.push_back(0.66356077923727);	xs_H.push_back(28.864);
  eV_H.push_back(0.66576762346358);	xs_H.push_back(29.5088);
  eV_H.push_back(0.71282701193426);	xs_H.push_back(28.2574);
  eV_H.push_back(0.73590227585419);	xs_H.push_back(27.4989);
  eV_H.push_back(0.75936666273882);	xs_H.push_back(26.8542);
  eV_H.push_back(0.80889140369906);	xs_H.push_back(26.3234);
  eV_H.push_back(0.83209592175062);	xs_H.push_back(25.6028);
  eV_H.push_back(0.90677351672548);	xs_H.push_back(24.9204);
  eV_H.push_back(0.95590913472103);	xs_H.push_back(24.2757);
  eV_H.push_back(1.031106467362);	xs_H.push_back(23.745);
  eV_H.push_back(1.05457085424663);	xs_H.push_back(23.1003);
  eV_H.push_back(1.10409559520687);	xs_H.push_back(22.5694);
  eV_H.push_back(1.17903305901478);	xs_H.push_back(21.9628);
  eV_H.push_back(1.22881766880808);	xs_H.push_back(21.5079);
  eV_H.push_back(1.2782131556367);	xs_H.push_back(20.9391);
  eV_H.push_back(1.35379961124236);	xs_H.push_back(20.5221);
  eV_H.push_back(1.45506137966837);	xs_H.push_back(20.1052);
  eV_H.push_back(1.58185287994542);	xs_H.push_back(19.6504);
  eV_H.push_back(1.75999228472356);	xs_H.push_back(19.1958);
  eV_H.push_back(1.91233528596856);	xs_H.push_back(18.7032);
  eV_H.push_back(2.06519530374007);	xs_H.push_back(18.3623);
  eV_H.push_back(2.24360682247953);	xs_H.push_back(17.9835);
  eV_H.push_back(2.42186867854026);	xs_H.push_back(17.5668);
  eV_H.push_back(2.60026659158166);	xs_H.push_back(17.188);
  eV_H.push_back(2.77893661858437);	xs_H.push_back(16.8851);
  eV_H.push_back(2.9830084838763);	xs_H.push_back(16.5064);
  eV_H.push_back(3.21287675270137);	xs_H.push_back(16.1657);
  eV_H.push_back(3.39141072272342);	xs_H.push_back(15.8249);
  eV_H.push_back(3.64683049251644);	xs_H.push_back(15.4463);
  eV_H.push_back(3.87695726960477);	xs_H.push_back(15.1815);
  eV_H.push_back(4.0810291348967);	xs_H.push_back(14.8028);
  eV_H.push_back(4.31091100941984);	xs_H.push_back(14.4621);
  eV_H.push_back(4.54091533522557);	xs_H.push_back(14.1593);
  eV_H.push_back(4.71957175653021);	xs_H.push_back(13.8564);
  eV_H.push_back(4.97525003458648);	xs_H.push_back(13.5537);
  eV_H.push_back(5.28200410318252);	xs_H.push_back(13.1753);
  eV_H.push_back(5.53768238123879);	xs_H.push_back(12.8726);
  eV_H.push_back(5.74227126305723);	xs_H.push_back(12.6456);
  eV_H.push_back(5.97267015410688);	xs_H.push_back(12.4566);
  eV_H.push_back(6.15132657541152);	xs_H.push_back(12.1537);
  eV_H.push_back(6.40726336173105);	xs_H.push_back(11.9268);
  eV_H.push_back(6.61198830053015);	xs_H.push_back(11.7378);
  eV_H.push_back(6.81683569061185);	xs_H.push_back(11.5866);
  eV_H.push_back(6.99562816889715);	xs_H.push_back(11.3216);
  eV_H.push_back(7.20035310769626);	xs_H.push_back(11.1326);
  eV_H.push_back(7.43061594176524);	xs_H.push_back(10.9057);
  eV_H.push_back(7.71209062335465);	xs_H.push_back(10.641);
  eV_H.push_back(7.96789135269351);	xs_H.push_back(10.3762);
  eV_H.push_back(8.27517604351412);	xs_H.push_back(10.1495);
  eV_H.push_back(8.50530282060245);	xs_H.push_back(9.88464);
  eV_H.push_back(8.76123960692198);	xs_H.push_back(9.6578);
  eV_H.push_back(9.06852429774258);	xs_H.push_back(9.43109);
  eV_H.push_back(9.35012143061459);	xs_H.push_back(9.20432);
  eV_H.push_back(9.55484636941369);	xs_H.push_back(9.01526);
  eV_H.push_back(9.78536771174593);	xs_H.push_back(8.86419);
  eV_H.push_back(10.0157666027956);	xs_H.push_back(8.6752);
  eV_H.push_back(10.2717033891151);	xs_H.push_back(8.44835);
  eV_H.push_back(10.5533005219871);	xs_H.push_back(8.22158);
  eV_H.push_back(10.8349112605572);	xs_H.push_back(7.9948);
  eV_H.push_back(11.1421823456797);	xs_H.push_back(7.7681);
  eV_H.push_back(11.4237930842498);	xs_H.push_back(7.54133);
  eV_H.push_back(11.6798523218519);	xs_H.push_back(7.35241);
  eV_H.push_back(11.9615991174026);	xs_H.push_back(7.16356);
  eV_H.push_back(12.2176583550047);	xs_H.push_back(6.97464);
  eV_H.push_back(12.4223832938038);	xs_H.push_back(6.78558);
  eV_H.push_back(12.7041164836565);	xs_H.push_back(6.59673);
  eV_H.push_back(12.9858496735092);	xs_H.push_back(6.40788);
  eV_H.push_back(13.2163710158414);	xs_H.push_back(6.25682);
  eV_H.push_back(13.4212320116212);	xs_H.push_back(6.10568);
  eV_H.push_back(13.600541506433);	xs_H.push_back(5.9924);;

  cuFP_t dx;

  xsc_H = resampleArray(eV_H, xs_H, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_Emin), &eV_H[0], sizeof(cuFP_t)),
		 __FILE__, __LINE__, "Error copying cuH_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_H), &dx, sizeof(cuFP_t)),
		 __FILE__, __LINE__, "Error copying cuH_H");

  // Total cross section from LaBahn & Callaway, 1966, Phys. Rev., 147, 50, 28-40
  //
  std::vector<cuFP_t> eV_He, xs_He;

  eV_He.push_back(0.0972135);	xs_He.push_back(62.2773619684373);
  eV_He.push_back(0.212908);	xs_He.push_back(65.3193661349083);
  eV_He.push_back(0.251768);	xs_He.push_back(66.6419766420696);
  eV_He.push_back(0.440878);	xs_He.push_back(67.8332685763108);
  eV_He.push_back(0.704798);	xs_He.push_back(68.6287198361997);
  eV_He.push_back(1.11846);	xs_He.push_back(68.7641224795695);
  eV_He.push_back(1.5694);	xs_He.push_back(68.5696578943123);
  eV_He.push_back(1.86971);	xs_He.push_back(68.109100411296 );
  eV_He.push_back(2.20762);	xs_He.push_back(67.6491712468105);
  eV_He.push_back(2.50774);	xs_He.push_back(66.9903792673527);
  eV_He.push_back(2.77027);	xs_He.push_back(66.3312731286295);
  eV_He.push_back(3.07045);	xs_He.push_back(65.7387687541625);
  eV_He.push_back(3.25779);	xs_He.push_back(65.0790342969086);
  eV_He.push_back(3.44532);	xs_He.push_back(64.6178484953617);
  eV_He.push_back(3.78297);	xs_He.push_back(63.8930830701785);
  eV_He.push_back(4.0079);	xs_He.push_back(63.2339769314554);
  eV_He.push_back(4.2329);	xs_He.push_back(62.6405300791922);
  eV_He.push_back(4.57055);	xs_He.push_back(61.9160788132744);
  eV_He.push_back(4.79555);	xs_He.push_back(61.3229461202767);
  eV_He.push_back(5.02048);	xs_He.push_back(60.6635258222882);
  eV_He.push_back(5.20782);	xs_He.push_back(60.0041055242997);
  eV_He.push_back(5.50788);	xs_He.push_back(59.2790259398512);
  eV_He.push_back(5.883);	xs_He.push_back(58.4226277824826);
  eV_He.push_back(6.14552);	xs_He.push_back(57.7635216437595);
  eV_He.push_back(6.48318);	xs_He.push_back(57.0390703778416);
  eV_He.push_back(6.89576);	xs_He.push_back(56.0507253290223);
  eV_He.push_back(7.27088);	xs_He.push_back(55.1943271716537);
  eV_He.push_back(7.68347);	xs_He.push_back(54.2059821228344);
  eV_He.push_back(7.98353);	xs_He.push_back(53.4809025383858);
  eV_He.push_back(8.32118);	xs_He.push_back(52.756451272468);
  eV_He.push_back(8.6963);	xs_He.push_back(51.9000531150995);
  eV_He.push_back(8.99617);	xs_He.push_back(50.9767390342094);
  eV_He.push_back(9.33408);	xs_He.push_back(50.5168098697239);
  eV_He.push_back(9.67173);	xs_He.push_back(49.7920444445407);
  eV_He.push_back(9.97191);	xs_He.push_back(49.1995400700737);
  eV_He.push_back(10.2346);	xs_He.push_back(48.6726949820667);
  eV_He.push_back(10.4596);	xs_He.push_back(48.0795622890689);
  eV_He.push_back(10.7222);	xs_He.push_back(47.5527172010619);
  eV_He.push_back(10.9849);	xs_He.push_back(47.0921597180456);
  eV_He.push_back(11.2852);	xs_He.push_back(46.565628789304 );
  eV_He.push_back(11.5478);	xs_He.push_back(46.038783701297 );
  eV_He.push_back(11.773);	xs_He.push_back(45.57759789975  );
  eV_He.push_back(12.0731);	xs_He.push_back(44.9191200795576);
  eV_He.push_back(12.3734);	xs_He.push_back(44.4585625965413);
  eV_He.push_back(12.7488);	xs_He.push_back(43.866686540605 );
  eV_He.push_back(13.0489);	xs_He.push_back(43.2738680068726);
  eV_He.push_back(13.3867);	xs_He.push_back(42.6153901866802);
  eV_He.push_back(13.7996);	xs_He.push_back(41.9578548442838);
  eV_He.push_back(14.1375);	xs_He.push_back(41.4976115205329);
  eV_He.push_back(14.4752);	xs_He.push_back(40.9054213053313);
  eV_He.push_back(14.8132);	xs_He.push_back(40.5114655865711);
  eV_He.push_back(15.1509);	xs_He.push_back(39.8529877663787);
  eV_He.push_back(15.4889);	xs_He.push_back(39.4590320476185);
  eV_He.push_back(15.7893);	xs_He.push_back(39.064762169593 );
  eV_He.push_back(16.1271);	xs_He.push_back(38.5385454001167);
  eV_He.push_back(16.465);	xs_He.push_back(38.0123286306404);
  eV_He.push_back(16.8404);	xs_He.push_back(37.4864260204295);
  eV_He.push_back(17.1407);	xs_He.push_back(37.0258685374132);
  eV_He.push_back(17.5162);	xs_He.push_back(36.566253532193 );
  eV_He.push_back(17.8917);	xs_He.push_back(36.1063243677075);
  eV_He.push_back(18.2672);	xs_He.push_back(35.6463952032219);
  eV_He.push_back(18.6426);	xs_He.push_back(35.120492593011 );
  eV_He.push_back(19.0181);	xs_He.push_back(34.6608775877908);
  eV_He.push_back(19.3561);	xs_He.push_back(34.2669218690307);
  eV_He.push_back(19.6941);	xs_He.push_back(33.8729661502705);
  eV_He.push_back(20.0321);	xs_He.push_back(33.478696272245);
  eV_He.push_back(20.3324);	xs_He.push_back(33.0844263942195);
  eV_He.push_back(20.708);	xs_He.push_back(32.6907848347247);
  eV_He.push_back(21.046);	xs_He.push_back(32.2968291159645);
  eV_He.push_back(21.3839);	xs_He.push_back(31.836899951479 );
  eV_He.push_back(21.7594);	xs_He.push_back(31.37700220292  );
  eV_He.push_back(22.1725);	xs_He.push_back(30.9174814454794);
  eV_He.push_back(22.548);	xs_He.push_back(30.523808470058 );
  eV_He.push_back(22.9612);	xs_He.push_back(30.1304182379755);
  eV_He.push_back(23.3369);	xs_He.push_back(29.8689434814172);
  eV_He.push_back(23.5995);	xs_He.push_back(29.3421612252633);
  eV_He.push_back(23.9752);	xs_He.push_back(29.080686468705 );
  eV_He.push_back(24.3506);	xs_He.push_back(28.4886847490626);
  eV_He.push_back(24.8389);	xs_He.push_back(28.0958914195842);
  eV_He.push_back(25.2144);	xs_He.push_back(27.6360879188048);
  eV_He.push_back(25.8906);	xs_He.push_back(27.1125729190106);
  eV_He.push_back(26.3788);	xs_He.push_back(26.5875499547427);
  eV_He.push_back(26.7543);	xs_He.push_back(26.1277464539633);
  eV_He.push_back(27.2427);	xs_He.push_back(25.734953124485 );
  eV_He.push_back(27.7686);	xs_He.push_back(25.342473954272 );
  eV_He.push_back(28.2944);	xs_He.push_back(24.8177337333429);
  eV_He.push_back(28.8205);	xs_He.push_back(24.5574841979195);
  eV_He.push_back(29.2337);	xs_He.push_back(24.164093965837 );
  eV_He.push_back(29.8723);	xs_He.push_back(23.706363916209 );
  eV_He.push_back(30.3607);	xs_He.push_back(23.3135705867306);
  eV_He.push_back(30.8868);	xs_He.push_back(23.1194201607388);
  eV_He.push_back(31.4127);	xs_He.push_back(22.7269409905258);
  eV_He.push_back(31.9388);	xs_He.push_back(22.4666600391759);
  eV_He.push_back(32.615);	xs_He.push_back(21.9431450393817);
  eV_He.push_back(33.3665);	xs_He.push_back(21.5524251610547);
  eV_He.push_back(34.0429);	xs_He.push_back(21.2272389054817);
  eV_He.push_back(34.6064);	xs_He.push_back(20.8350424786075);
  eV_He.push_back(35.0948);	xs_He.push_back(20.5083796744872);
  eV_He.push_back(35.5457);	xs_He.push_back(20.2475018205331);
  eV_He.push_back(36.0342);	xs_He.push_back(20.0530372352759);
  eV_He.push_back(36.4851);	xs_He.push_back(19.7921907972484);
  eV_He.push_back(37.0112);	xs_He.push_back(19.59800895533  );
  eV_He.push_back(37.5372);	xs_He.push_back(19.2716288945485);
  eV_He.push_back(38.0258);	xs_He.push_back(19.0771957252179);
  eV_He.push_back(38.5519);	xs_He.push_back(18.8830138832995);
  eV_He.push_back(38.9277);	xs_He.push_back(18.6876696520993);
  eV_He.push_back(39.4537);	xs_He.push_back(18.4273887007494);
  eV_He.push_back(39.9423);	xs_He.push_back(18.2329555314187);
  eV_He.push_back(40.506);	xs_He.push_back(18.0390878487657);
  eV_He.push_back(40.9569);	xs_He.push_back(17.7782099948116);
  eV_He.push_back(41.483);	xs_He.push_back(17.5840595688197);
  eV_He.push_back(41.9715);	xs_He.push_back(17.3895949835625);
  eV_He.push_back(42.3472);	xs_He.push_back(17.1281516429308);
  eV_He.push_back(42.7606);	xs_He.push_back(16.9991892645009);
  eV_He.push_back(43.174);	xs_He.push_back(16.8702583019976);
  eV_He.push_back(43.5498);	xs_He.push_back(16.6749140707974);
  eV_He.push_back(43.9631);	xs_He.push_back(16.479852582936);
  eV_He.push_back(44.339);	xs_He.push_back(16.3506074611673);
  eV_He.push_back(44.6772);	xs_He.push_back(16.2210795960598);
  eV_He.push_back(45.2033);	xs_He.push_back(16.0268977541414);
  eV_He.push_back(45.504);	xs_He.push_back(15.9631862551266);
  eV_He.push_back(45.8798);	xs_He.push_back(15.8339411333579);
  eV_He.push_back(46.2556);	xs_He.push_back(15.7046960115892);
  eV_He.push_back(46.5939);	xs_He.push_back(15.5751681464817);
  eV_He.push_back(46.857);	xs_He.push_back(15.5111424882016);
  eV_He.push_back(47.1952);	xs_He.push_back(15.3816146230941);
  eV_He.push_back(47.571);	xs_He.push_back(15.2523695013254);
  eV_He.push_back(47.8717);	xs_He.push_back(15.188626586384 );
  eV_He.push_back(48.1348);	xs_He.push_back(15.1246009281039);
  eV_He.push_back(48.4354);	xs_He.push_back(14.9947903196575);
  eV_He.push_back(48.7736);	xs_He.push_back(14.8652310386235);
  eV_He.push_back(49.0368);	xs_He.push_back(14.8673359057014);
  eV_He.push_back(49.2622);	xs_He.push_back(14.7368969787244);
  eV_He.push_back(49.5254);	xs_He.push_back(14.7389704298757);
  eV_He.push_back(49.7885);	xs_He.push_back(14.6749447715956);
  eV_He.push_back(49.9763);	xs_He.push_back(14.4781239918482);

  xsc_He = resampleArray(eV_He, xs_He, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_Emin), &eV_He[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuHe_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuHe_H");

  // Interpolated from Figure 1 of "Elastic scattering and charge
  // transfer in slow collisions: isotopes of H and H + colliding with
  // isotopes of H and with He" by Predrag S Krstić and David R Schultz,
  // 1999 J. Phys. B: At. Mol. Opt. Phys. 32 3485
  //

  std::vector<cuFP_t> eV_pH, xs_pH;

  eV_pH.push_back(-0.994302);	xs_pH.push_back(2.86205);
  eV_pH.push_back(-0.897482);	xs_pH.push_back(2.90929);
  eV_pH.push_back(-0.801179);	xs_pH.push_back(2.86016);
  eV_pH.push_back(-0.691555);	xs_pH.push_back(2.89417);
  eV_pH.push_back(-0.588753);	xs_pH.push_back(2.85638);
  eV_pH.push_back(-0.49242);	xs_pH.push_back(2.81291);
  eV_pH.push_back(-0.395965);	xs_pH.push_back(2.79213);
  eV_pH.push_back(-0.292839);	xs_pH.push_back(2.8148);
  eV_pH.push_back(-0.19019);	xs_pH.push_back(2.74866);
  eV_pH.push_back(-0.0872765);	xs_pH.push_back(2.73165);
  eV_pH.push_back(0.00935082);	xs_pH.push_back(2.74299);
  eV_pH.push_back(0.112152);	xs_pH.push_back(2.7052);
  eV_pH.push_back(0.208688);	xs_pH.push_back(2.69953);
  eV_pH.push_back(0.311612);	xs_pH.push_back(2.68441);
  eV_pH.push_back(0.401578);	xs_pH.push_back(2.65417);
  eV_pH.push_back(0.517468);	xs_pH.push_back(2.65606);
  eV_pH.push_back(0.613862);	xs_pH.push_back(2.62394);
  eV_pH.push_back(0.716846);	xs_pH.push_back(2.62016);
  eV_pH.push_back(0.819688);	xs_pH.push_back(2.58992);
  eV_pH.push_back(0.909797);	xs_pH.push_back(2.58614);
  eV_pH.push_back(1.01906);	xs_pH.push_back(2.55213);
  eV_pH.push_back(1.1092);	xs_pH.push_back(2.55402);
  eV_pH.push_back(1.21203);	xs_pH.push_back(2.52189);
  eV_pH.push_back(1.3085);	xs_pH.push_back(2.50488);
  eV_pH.push_back(1.41149);	xs_pH.push_back(2.5011);
  eV_pH.push_back(1.52077);	xs_pH.push_back(2.47087);
  eV_pH.push_back(1.61715);	xs_pH.push_back(2.43685);
  eV_pH.push_back(1.71368);	xs_pH.push_back(2.42929);
  eV_pH.push_back(1.81666);	xs_pH.push_back(2.42551);
  eV_pH.push_back(1.9131);	xs_pH.push_back(2.40094);
  eV_pH.push_back(2.0159);	xs_pH.push_back(2.36315);

  xsc_pH = resampleArray(eV_pH, xs_pH, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPH_Emin), &eV_pH[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPH_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPH_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPH_H");

  // Interpolated from the top panel of Figure 4, op. cit.
  //
  std::vector<cuFP_t> eV_pHe, xs_pHe;

  eV_pHe.push_back(-0.984127);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.904762);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.825397);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.753968);	xs_pHe.push_back(2.64444);
  eV_pHe.push_back(-0.674603);	xs_pHe.push_back(2.6);
  eV_pHe.push_back(-0.595238);	xs_pHe.push_back(2.57778);
  eV_pHe.push_back(-0.515873);	xs_pHe.push_back(2.57778);
  eV_pHe.push_back(-0.444444);	xs_pHe.push_back(2.55556);
  eV_pHe.push_back(-0.373016);	xs_pHe.push_back(2.48889);
  eV_pHe.push_back(-0.293651);	xs_pHe.push_back(2.44444);
  eV_pHe.push_back(-0.214286);	xs_pHe.push_back(2.46667);
  eV_pHe.push_back(-0.142857);	xs_pHe.push_back(2.44444);
  eV_pHe.push_back(-0.0634921);	xs_pHe.push_back(2.4);
  eV_pHe.push_back(0.015873);	xs_pHe.push_back(2.37778);
  eV_pHe.push_back(0.0952381);	xs_pHe.push_back(2.37778);
  eV_pHe.push_back(0.166667);	xs_pHe.push_back(2.33333);
  eV_pHe.push_back(0.246032);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.325397);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.404762);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.47619);	xs_pHe.push_back(2.24444);
  eV_pHe.push_back(0.555556);	xs_pHe.push_back(2.2);
  eV_pHe.push_back(0.634921);	xs_pHe.push_back(2.17778);
  eV_pHe.push_back(0.706349);	xs_pHe.push_back(2.2);
  eV_pHe.push_back(0.785714);	xs_pHe.push_back(2.17778);
  eV_pHe.push_back(0.865079);	xs_pHe.push_back(2.13333);
  eV_pHe.push_back(0.936508);	xs_pHe.push_back(2.08889);
  eV_pHe.push_back(1.01587);	xs_pHe.push_back(2.06667);
  eV_pHe.push_back(1.09524);	xs_pHe.push_back(2.08889);
  eV_pHe.push_back(1.16667);	xs_pHe.push_back(2.06667);
  eV_pHe.push_back(1.24603);	xs_pHe.push_back(2.04444);
  eV_pHe.push_back(1.3254);	xs_pHe.push_back(2.02222);
  eV_pHe.push_back(1.40476);	xs_pHe.push_back(1.97778);
  eV_pHe.push_back(1.47619);	xs_pHe.push_back(1.93333);
  eV_pHe.push_back(1.55556);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.63492);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.71429);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.79365);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.87302);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.95238);	xs_pHe.push_back(1.91111);

  xsc_pHe = resampleArray(eV_pHe, xs_pHe, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPHe_Emin), &eV_pHe[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPHe_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPHe_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPHe_H");
}

__device__
cuFP_t cudaGeometric(int Z)
{
  if (Z>0 and Z< numRadii) {
    return cudaRadii[Z] * 1.0e-3;
  } else {
    return 0.0;
  }
}
		 
__device__
cuFP_t cudaElasticInterp(cuFP_t E, cuFP_t Emin, cuFP_t H, dArray<cuFP_t> xsc,
			 bool pin = true)
{
  int indx = 0;
  if (E >= Emin+H*xsc._s) indx = xsc._s - 2;
  if (E <  Emin)          indx = 0;
  else                    indx = floor( (E - Emin)/H );

  cuFP_t a = (E - Emin - H*(indx+0))/H;
  cuFP_t b = (Emin + H*(indx+1) - E)/H;

  // Enforce return value to grid boundaries for off-grid ordinates.
  // Otherwise, values will be extrapolated.
  if (pin) {
    if (a < 0.0) return xsc._v[0];
    if (b < 0.0) return xsc._v[xsc._s-1];
  }

  return a*xsc._v[indx] + b*xsc._v[indx+1];
}


// Global symbols for coordinate transformation
//
__device__ __constant__
cuFP_t ionEminGrid, ionEmaxGrid, ionDeltaEGrid;

__device__ __constant__
int ionEgridNumber, ionRadRecombNumber;

thrust::host_vector<cuIonElement> cuIonElem;

__global__
void testConstantsIon()
{
  printf("** Egrid(min) = %f\n", ionEminGrid);
  printf("** Egrid(max) = %f\n", ionEmaxGrid);
  printf("** Egrid(del) = %f\n", ionDeltaEGrid);
  printf("** Egrid(num) = %d\n", ionEgridNumber);
  printf("** Rgrid(num) = %d\n", ionRadRecombNumber);
}

void chdata::cuda_initialize_textures()
{
  size_t ionSize = IonList.size();

  // Interpolation data array
  //
  cuF0array.resize(ionSize, 0);
  cuFFarray.resize(ionSize, 0);
  cuRCarray.resize(ionSize, 0);
  cuCEarray.resize(ionSize, 0);
  cuCIarray.resize(ionSize, 0);
  cuPIarray.resize(ionSize   );

  // Texture object array
  //
  cuIonElem.resize(ionSize);

  // Total photo-ionization rate
  //
  std::vector<cuFP_t> phRate(ionSize, 0.0);

  size_t k = 0;

  for (auto v : IonList) {

    IonPtr I = v.second;
    cuIonElement& E = cuIonElem[k];

    // The free-free array
    if (E.C>1) {
      hipTextureDesc texDesc;

      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.readMode = hipReadModeElementType;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
  
      // Temporary storage
      //
      std::vector<cuFP_t> h_buffer0(I->NfreeFreeGrid, 0.0);

      cuFP_t *d_Interp;

      cuda_safe_call(hipMalloc((void **)&d_Interp, I->NfreeFreeGrid*CHCUMK*sizeof(cuFP_t)),
		     __FILE__, __LINE__,
		     "Error allocating d_Interp1 for texture construction");
  
      std::vector<cuFP_t> h_buffer1(I->NfreeFreeGrid*CHCUMK, 0.0);

      double delC = 1.0/(CHCUMK-1);

      // Copy cross section values to buffer
      //
      for (int i = 0; i < I->NfreeFreeGrid; i++) {

	h_buffer0[i] = I->freeFreeGrid[i].back();
	
	// Unit normalized cumulative distribution
	//
	size_t tsize = I->freeFreeGrid[i].size();
	std::vector<double> temp(tsize);
	for (int j = 0; j < tsize; j++) {	
	  temp[j] = I->freeFreeGrid[i][j]/h_buffer0[i];
	}

	// End points
	//
	h_buffer1[i                              ] = I->kgrid[0];
	h_buffer1[i + (CHCUMK-1)*I->NfreeFreeGrid] = I->kgrid[tsize-1];

	// Remap to even grid
	//
	for (int j=1; j<CHCUMK-1; j++) {

	  double C = delC*j;

	  // Points to first element that is not < C
	  // but may be equal
	  std::vector<double>::iterator lb = 
	    std::lower_bound(temp.begin(), temp.end(), C);
    
	  // Assign upper end of range to the
	  // found element
	  //
	  std::vector<double>::iterator ub = lb;
	  //
	  // If is the first element, increment
	  // the upper boundary
	  //
	  if (lb == temp.begin()) { if (temp.size()>1) ub++; }
	  //
	  // Otherwise, decrement the lower boundary
	  //
	  else { lb--; }
    
	  // Compute the associated indices
	  //
	  size_t ii = lb - temp.begin();
	  size_t jj = ub - temp.begin();
	  double kk = I->kgrid[ii];
	  
	  // Linear interpolation
	  //
	  if (*ub > *lb) {
	    double d = *ub - *lb;
	    double a = (C - *lb) / d;
	    double b = (*ub - C) / d;
	    /*
	    std::cout << "[a, b] = [" << a << ", " << b << "]"
		      << ", c = " << C << std::endl;
	    */
	    kk  = a * I->kgrid[ii] + b * I->kgrid[jj];
	  }

	  h_buffer1[i + j*I->NfreeFreeGrid] = kk;

	} // END: cumululative array loop

      } // END: energy loop

      // Copy 1-dim data to device
      //
      size_t tsize = I->NfreeFreeGrid*sizeof(cuFP_t);

      // Allocate CUDA array in device memory (a one-dimension 'channel')
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc<int2>();
#endif
      
      std::cout << "Allocating cuF0array[" << k << "]" << std::endl;
      cuda_safe_call(hipMallocArray(&cuF0array[k], &channelDesc1, I->NfreeFreeGrid), __FILE__, __LINE__, "malloc cuArray");

      cuda_safe_call(hipMemcpyToArray(cuF0array[k], 0, 0, &h_buffer0[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

      // Specify 1-d texture

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuF0array[k];

      cuda_safe_call(hipCreateTextureObject(&E.ff_0, &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");

      // Copy data to device
      tsize = I->NfreeFreeGrid*CHCUMK*sizeof(cuFP_t);
      cuda_safe_call(hipMemcpy(d_Interp, &h_buffer1[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");
    
      // cuda 2d Array Descriptor
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<int2>();
#endif
      // cuda 2d Array
      //
      cuda_safe_call(hipMalloc3DArray(&cuFFarray[k], &channelDesc2, make_hipExtent(I->NfreeFreeGrid, CHCUMK, 1), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");
      
      // Array creation
      //
      hipMemcpy3DParms copyParams = {0};
  
      copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, I->NfreeFreeGrid*sizeof(cuFP_t), I->NfreeFreeGrid, CHCUMK);
      copyParams.dstArray = cuFFarray[k];
      copyParams.extent   = make_hipExtent(I->NfreeFreeGrid, CHCUMK, 1);
      copyParams.kind     = hipMemcpyDeviceToDevice;
      
      cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array  = cuFFarray[k];
    
      cuda_safe_call
	(hipCreateTextureObject(&E.ff_d, &resDesc, &texDesc, NULL),
	 __FILE__, __LINE__, "Failure in 2d texture creation");
      
      cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
    }

    // Radiative recombination texture (1-d)
    //
    if (E.C>1) {
      // Allocate CUDA array in device memory (a one-dimension 'channel')
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    
      // Size of interpolation array
      //
      size_t tsize = I->NradRecombGrid*sizeof(cuFP_t);

      hipTextureDesc texDesc;
      
      memset(&texDesc, 0, sizeof(hipTextureDesc));
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.readMode = hipReadModeElementType;
      texDesc.normalizedCoords = 0;
      
      thrust::host_vector<cuFP_t> tt(I->NradRecombGrid);
      
      cuda_safe_call(hipMallocArray(&cuRCarray[k], &channelDesc, I->NradRecombGrid), __FILE__, __LINE__, "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      for (size_t n = 0; n < I->NradRecombGrid; n++) tt[n] = I->radRecombGrid[n];
    
      cuda_safe_call(hipMemcpyToArray(cuRCarray[k], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

      // Specify texture

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuRCarray[k];
      
      cuda_safe_call(hipCreateTextureObject(&E.rc_d, &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");
    }

    // The collisional excitation array

    if (E.C <= E.Z) {

      E.ceEmin = I->collideEmin;
      E.ceEmax = I->collideEmax;
      E.ceDelE = I->delCollideE;
      E.NColl  = I->NcollideGrid;

      std::cout << " k=" << k
		<< " Emin=" << E.ceEmin
		<< " Emax=" << E.ceEmax
		<< " delE=" << E.ceDelE
		<< std::endl;

      hipTextureDesc texDesc;
      
      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.readMode = hipReadModeElementType;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
  
      // Temporary storage
      //
      cuFP_t *d_Interp;
      std::cout << "Size(" << I->Z << ", " << I->C << ")="
		<< I->NcollideGrid << std::endl;
      cuda_safe_call(hipMalloc((void **)&d_Interp, I->NcollideGrid*2*sizeof(cuFP_t)),
		     __FILE__, __LINE__,
		     "Error allocating d_Interp for texture construction");
  
      std::vector<cuFP_t> h_buffer(I->NcollideGrid*2, 0.0);

      // Copy vectors to buffer
      //
      for (int i = 0; i < I->NcollideGrid; i++) {
	h_buffer[i                  ] = I->collideDataGrid[i].back().first;
	h_buffer[i + I->NcollideGrid] = I->collideDataGrid[i].back().second;
      }
      
      // Copy data to device
      cuda_safe_call(hipMemcpy(d_Interp, &h_buffer[0], I->NcollideGrid*2*sizeof(cuFP_t), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");
    
      // hipArray Descriptor
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
      // cuda Array
      //
      cuda_safe_call(hipMalloc3DArray(&cuCEarray[k], &channelDesc, make_hipExtent(I->NcollideGrid, 2, 1), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");
    
      // Array creation
      //
      hipMemcpy3DParms copyParams = {0};
      
      copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, I->NcollideGrid*sizeof(cuFP_t), I->NcollideGrid, 2);
      copyParams.dstArray = cuCEarray[k];
      copyParams.extent   = make_hipExtent(I->NcollideGrid, 2, 1);
      copyParams.kind     = hipMemcpyDeviceToDevice;
      
      cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");
      
      hipResourceDesc resDesc;
      
      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array  = cuCEarray[k];
      
      cuda_safe_call
	(hipCreateTextureObject(&E.ce_d, &resDesc, &texDesc, NULL),
	 __FILE__, __LINE__, "Failure in 2d texture creation");
      
      cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
    }

    if (E.C <= E.Z) {

      E.ciEmin = I->ionizeEmin;
      E.ciEmax = I->ionizeEmax;
      E.ciDelE = I->DeltaEGrid;
      E.NIonz  = I->NionizeGrid;

      // Allocate CUDA array in device memory (a one-dimension 'channel')
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    
      // Size of interpolation array
      //
      size_t tsize = I->NionizeGrid*sizeof(cuFP_t);
      
      hipTextureDesc texDesc;

      memset(&texDesc, 0, sizeof(hipTextureDesc));
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.readMode = hipReadModeElementType;
      texDesc.normalizedCoords = 0;
      
      thrust::host_vector<cuFP_t> tt(I->NionizeGrid);
      
      std::cout << "Size(" << I->Z << ", " << I->C << ")="
		<< I->NionizeGrid << std::endl;

      cuda_safe_call(hipMallocArray(&cuCIarray[k], &channelDesc, I->NionizeGrid), __FILE__, __LINE__, "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      for (size_t n = 0; n < I->NionizeGrid; n++) tt[n] = I->ionizeDataGrid[n];
      
      cuda_safe_call(hipMemcpyToArray(cuCIarray[k], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");
      
      // Specify texture

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuCIarray[k];
      
      cuda_safe_call(hipCreateTextureObject(&E.ci_d, &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");

      // Photoionization array
      //
      if (I->ib_type != Ion::none) {

	thrust::host_vector<cuFP_t> piCum(CHCUMK, 0.0);
	piCum[CHCUMK-1] = 1.0;
      
	double delC = 1.0/(CHCUMK-1);
      
	if (not I->IBinit) I->IBcreate();
      
	E.piTotl = I->IBtotl;

	// Copy cross section values to buffer
	//
	for (int j=1; j<CHCUMK-1; j++) {

	  // Location in cumulative cross section grid
	  //
	  double C = delC*j;

	  // Interpolate the cross section array
	  //
	
	  // Points to first element that is not < rn
	  // but may be equal
	  std::vector<double>::iterator lb = 
	    std::lower_bound(I->IBcum.begin(), I->IBcum.end(), C);
	  
	  // Assign upper end of range to the
	  // found element
	  //
	  std::vector<double>::iterator ub = lb;
	  //
	  // If is the first element, increment
	  // the upper boundary
	  //
	  if (lb == I->IBcum.begin()) { if (I->IBcum.size()>1) ub++; }
	  //
	  // Otherwise, decrement the lower boundary
	  //
	  else { lb--; }
	  
	  // Compute the associated indices
	  //
	  size_t ii = lb - I->IBcum.begin();
	  size_t jj = ub - I->IBcum.begin();
	  double nu = I->nugrid[ii];
	  
	  // Linear interpolation
	  //
	  if (*ub > *lb) {
	    double d = *ub - *lb;
	    double a = (C - *lb) / d;
	    double b = (*ub - C) / d;
	    nu  = a * I->nugrid[ii] + b * I->nugrid[jj];
	  }
	  
	  piCum[j] = (nu - 1.0)*I->ip;
	}
	
	std::cout << "Allocating pi_0[" << k << "]" << std::endl;

	// Create storage on device
	cuPIarray[k] = piCum;

	// Assign pointer
	E.pi_0 = thrust::raw_pointer_cast(&cuPIarray[k][0]);

      } // END: cumululative array loop

    } // END: ions with electrons
    
    // Increment counter
    k++;	
    
  } // END: IonList

}

void chdata::cuda_initialize_grid_constants()
{
  double Emin, Emax, delE;
  int NE, NR;

  for (auto v : IonList) {
    Emin = v.second->EminGrid;
    Emax = v.second->EmaxGrid;
    delE = v.second->DeltaEGrid;

    NE   = v.second->NfreeFreeGrid;

    if (v.first.second>1) {
      NR = v.second->NradRecombGrid;
      break;
    }
  }

  cuFP_t f;

  // Copy constants to device
  //
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEminGrid), &(f=Emin),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEminGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEmaxGrid), &(f=Emax),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEmaxGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionDeltaEGrid), &(f=delE),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionDeltaEGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEgridNumber), &NE,
				    sizeof(int), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEgridNumber");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionRadRecombNumber), &NR,
				    sizeof(int), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionRadRecombNumber");
}


__device__
void computeFreeFree
(cuFP_t E, cuFP_t rr, cuFP_t& ph, cuFP_t& xc, cuIonElement& elem)
{
  // value of h-bar * c in eV*nm
  //
  constexpr double hbc = 197.327;

  // Enforce minimum and maximum energies
  //
  if (E<ionEminGrid) E = ionEminGrid;
  if (E>ionEmaxGrid) E = ionEmaxGrid;

  size_t indx = std::floor( (E - ionEminGrid)/ionDeltaEGrid );
    
  if (indx >= ionEgridNumber - 1) indx = ionEgridNumber-2;

  double eA = ionEminGrid + ionDeltaEGrid*indx;
  double eB = ionEminGrid + ionDeltaEGrid*(indx+1);
  
  double A = (eB - E)/ionDeltaEGrid;
  double B = (E - eA)/ionDeltaEGrid;
  
  // Location in cumulative cross section grid
  //
  double rn = rr;
  double dC = 1.0/(CHCUMK-1);
  int lb    = rn/dC;
  cuFP_t k[4];

  // Interpolate the cross section array
  //
#if cuREAL == 4
  k[0]  = tex3D<float>(elem.ff_d, indx,   lb  , 0);
  k[1]  = tex3D<float>(elem.ff_d, indx+1, lb  , 0);
  k[2]  = tex3D<float>(elem.ff_d, indx,   lb+1, 0);
  k[3]  = tex3D<float>(elem.ff_d, indx+1, lb+1, 0);
#else
  k[0] = int2_as_double(tex3D<int2>(elem.ff_d, indx,   lb  , 0));
  k[1] = int2_as_double(tex3D<int2>(elem.ff_d, indx+1, lb  , 0));
  k[2] = int2_as_double(tex3D<int2>(elem.ff_d, indx,   lb+1, 0));
  k[3] = int2_as_double(tex3D<int2>(elem.ff_d, indx+1, lb+1, 0));
#endif
  
  // Linear interpolation
  //
  double a = (rn - dC*(lb+0)) / dC;
  double b = (dC*(lb+1) - rn) / dC;

  double K = A*(a*k[0] + b*k[2]) + B*(a*k[1] + b*k[3]);

  // Assign the photon energy
  //
  ph = pow(10, K) * hbc;

  // Use the integrated cross section from the differential grid
  //

  xc = 
#if cuREAL == 4
    A*tex1D<float>(elem.ff_0, indx  ) +
    B*tex1D<float>(elem.ff_0, indx+1) ;
#else
    A*int2_as_double(tex1D<int2>(elem.ff_0, indx  )) +
    B*int2_as_double(tex1D<int2>(elem.ff_0, indx+1)) ;
#endif
}


__global__
void testFreeFree
(dArray<cuFP_t> energy,
 dArray<cuFP_t> randsl,
 dArray<cuFP_t> ph, dArray<cuFP_t> xc,
 cuIonElement elem)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {
    computeFreeFree(energy._v[tid], randsl._v[tid], 
		    ph._v[tid], xc._v[tid], elem);
  }

  __syncthreads();
}


__device__
void computeColExcite
(cuFP_t E, cuFP_t& ph, cuFP_t& xc, cuIonElement& elem)
{
  if (E < elem.ceEmin or E > elem.ceEmax) {

    xc = 0.0;
    ph= 0.0;

  } else {

    // Interpolate the values
    //
    int indx = std::floor( (E - elem.ceEmin)/elem.ceDelE );
    
    // Sanity check
    //
    if (indx > elem.NColl-2) indx = elem.NColl - 2;
    if (indx < 0)            indx = 0;
    
    double eA   = elem.ceEmin + elem.ceDelE*indx;
    double eB   = elem.ceEmin + elem.ceDelE*(indx+1);
    
    double A = (eB - E)/elem.ceDelE;
    double B = (E - eA)/elem.ceDelE;
    
#if cuREAL == 4
    xc = 
      A*tex3D<float>(elem.ce_d, indx,   0, 0) +
      B*tex3D<float>(elem.ce_d, indx+1, 0, 0) ;
    ph = 
      A*tex3D<float>(elem.ce_d, indx,   1, 0) +
      B*tex3D<float>(elem.ce_d, indx+1, 1, 0) ;
#else
    xc = 
      A*int2_as_double(tex3D<int2>(elem.ce_d, indx  , 0, 0)) +
      B*int2_as_double(tex3D<int2>(elem.ce_d, indx+1, 0, 0)) ;
    ph= 
      A*int2_as_double(tex3D<int2>(elem.ce_d, indx  , 1, 0)) +
      B*int2_as_double(tex3D<int2>(elem.ce_d, indx+1, 1, 0)) ;
#endif
  }
  // DONE
}

__global__ void testColExcite
(dArray<cuFP_t> energy,
 dArray<cuFP_t> ph, dArray<cuFP_t> xc, cuIonElement elem)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {
    computeColExcite(energy._v[tid], ph._v[tid], xc._v[tid], elem);
  }

  __syncthreads();
}

__device__
void computeColIonize
(cuFP_t E, cuFP_t& xc, cuIonElement& elem)
{
  if (E < elem.ciEmin or E > elem.ciEmax) {

    xc = 0.0;

  } else {

    // Interpolate the values
    //
    int indx = std::floor( (E - elem.ciEmin)/elem.ciDelE );

    // Sanity check
    //
    if (indx > elem.NIonz-2) indx = elem.NIonz - 2;
    if (indx < 0)            indx = 0;
    
    double eA   = elem.ciEmin + elem.ciDelE*indx;
    double eB   = elem.ciEmin + elem.ciDelE*(indx+1);
    
    double A = (eB - E)/elem.ciDelE;
    double B = (E - eA)/elem.ciDelE;
    
#if cuREAL == 4
    xc = 
      A*tex1D<float>(elem.ci_d, indx  ) +
      B*tex1D<float>(elem.ci_d, indx+1) ;
#else
    xc = 
      A*int2_as_double(tex1D<int2>(elem.ci_d, indx  )) +
      B*int2_as_double(tex1D<int2>(elem.ci_d, indx+1)) ;
#endif
  }
}


__device__
void computePhotoIonize
(cuFP_t rr, cuFP_t& ph, cuFP_t& xc, cuIonElement& elem)
{
  constexpr cuFP_t dC = 1.0/CHCUMK;
  int indx  = rr/dC;
  if (indx > CHCUMK-2) indx = CHCUMK - 2;

  // Linear interpolation
  //
  double a = (rr - dC*(indx+0)) / dC;
  double b = (dC*(indx+1) - rr) / dC;

  ph = a*elem.pi_0[indx+0] + b*elem.pi_0[indx+1];
  xc = elem.piTotl;
}


__global__ void testColIonize
(dArray<cuFP_t> energy, dArray<cuFP_t> xc, cuIonElement elem)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {
    computeColIonize(energy._v[tid], xc._v[tid], elem);
  }

  __syncthreads();
}

__device__
void computeRadRecomb
(cuFP_t E, cuFP_t& xc, cuIonElement& elem)
{
  if (E < ionEminGrid or E > ionEmaxGrid) {

    xc = 0.0;

  } else {

    // Interpolate the values
    //
    int indx = std::floor( (E - ionEminGrid)/ionDeltaEGrid );

    // Sanity check
    //
    if (indx > ionRadRecombNumber-2) indx = ionRadRecombNumber - 2;
    if (indx < 0)                    indx = 0;
    
    double eA   = ionEminGrid + ionDeltaEGrid*indx;
    double eB   = ionEminGrid + ionDeltaEGrid*(indx+1);
    
    double A = (eB - E)/ionDeltaEGrid;
    double B = (E - eA)/ionDeltaEGrid;
    
#if cuREAL == 4
    xc = 
      A*tex1D<float>(elem.rc_d, indx  ) +
      B*tex1D<float>(elem.rc_d, indx+1) ;
#else
    xc = 
      A*int2_as_double(tex1D<int2>(elem.rc_d, indx  )) +
      B*int2_as_double(tex1D<int2>(elem.rc_d, indx+1)) ;
#endif
  }
  // DONE
}

__global__
void testRadRecomb
(dArray<cuFP_t> energy, dArray<cuFP_t> xc, cuIonElement elem)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {
    computeRadRecomb(energy._v[tid], xc._v[tid], elem);
  }

  __syncthreads();
}


void chdata::testCross(int Nenergy)
{
  // Timers
  //
  Timer serial, cuda;

  // Loop over ions and tabulate statistics
  //
  size_t k = 0;

  thrust::host_vector<cuFP_t> energy_h(Nenergy), randsl_h(Nenergy);

  for (auto v : IonList) {

    IonPtr I = v.second;
    cuIonElement& E = cuIonElem[k];

    // Make an energy grid
    //
    double dE = (I->EmaxGrid - I->EminGrid)/(Nenergy-1) * 0.999;
    for (int i=0; i<Nenergy; i++) {
      energy_h[i] = I->EminGrid + dE*i;
      randsl_h[i] = static_cast<cuFP_t>(rand())/RAND_MAX;
    }

    thrust::device_vector<cuFP_t> energy_d = energy_h;
    thrust::device_vector<cuFP_t> randsl_d = randsl_h;

    // Only free-free for non-neutral species

    thrust::device_vector<cuFP_t> eFF_d(Nenergy), xFF_d(Nenergy);
    thrust::device_vector<cuFP_t> eCE_d(Nenergy), xCE_d(Nenergy);
    thrust::device_vector<cuFP_t> xCI_d(Nenergy), xRC_d(Nenergy);

    unsigned int gridSize  = Nenergy/BLOCK_SIZE;
    if (Nenergy > gridSize*BLOCK_SIZE) gridSize++;

    cuda.start();

    if (E.C>1)
      testFreeFree<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), toKernel(randsl_d),
					     toKernel(eFF_d), toKernel(xFF_d),
					     cuIonElem[k]);

    if (E.C<=E.Z)
      testColExcite<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), 
					      toKernel(eCE_d), toKernel(xCE_d), cuIonElem[k]);
      
    if (E.C<=E.Z)
      testColIonize<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), 
					      toKernel(xCI_d), cuIonElem[k]);
      
    if (E.C>1)
      testRadRecomb<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), 
					      toKernel(xRC_d), cuIonElem[k]);
      
    std::cout << "k=" << k << " delE=" << E.ceDelE << std::endl;

    thrust::host_vector<cuFP_t> eFF_h = eFF_d;
    thrust::host_vector<cuFP_t> xFF_h = xFF_d;
    thrust::host_vector<cuFP_t> eCE_h = eCE_d;
    thrust::host_vector<cuFP_t> xCE_h = xCE_d;
    thrust::host_vector<cuFP_t> xCI_h = xCI_d;
    thrust::host_vector<cuFP_t> xRC_h = xRC_d;
    
    cuda.stop();
    
    std::vector<double> eFF_0(Nenergy, 0), xFF_0(Nenergy, 0);
    std::vector<double> eCE_0(Nenergy, 0), xCE_0(Nenergy, 0);
    std::vector<double> xCI_0(Nenergy, 0), xRC_0(Nenergy, 0);
    
    serial.start();
    
    for (int i=0; i<Nenergy; i++) {
				// Free-free
      auto retFF = I->freeFreeCrossTest(energy_h[i], randsl_h[i], 0);
      if (retFF.first>0.0)
	xFF_0[i]   = (xFF_h[i] - retFF.first )/retFF.first;
      if (retFF.second>0.0)
	eFF_0[i]   = (eFF_h[i] - retFF.second)/retFF.second;

				// Collisional excitation
      auto retCE = I->collExciteCross(energy_h[i], 0).back();
      if (retCE.first>0.0) {
	xCE_0[i]   = (xCE_h[i] - retCE.first )/retCE.first;
	/*
	std::cout << std::setw( 4) << cuZ[k]
		  << std::setw( 4) << cuC[k]
		  << std::setw(14) << energy_h[i]
		  << std::setw(14) << xCE_h[i]
		  << std::setw(14) << retCE.first
		  << std::endl;
	*/
      }
				// Collisional ionization

      auto retCI = I->directIonCross(energy_h[i], 0);
      if (retCI>0.0) {
	xCI_0[i]   = (xCI_h[i] - retCI)/retCI;
	/*
	std::cout << std::setw( 4) << cuZ[k]
		  << std::setw( 4) << cuC[k]
		  << std::setw(14) << energy_h[i]
		  << std::setw(14) << xCI_h[i]
		  << std::setw(14) << retCI
		  << std::endl;
	*/
      }

				// Radiative recombination

      auto retRC = I->radRecombCross(energy_h[i], 0).back();
      if (retRC>0.0) {
	xRC_0[i]   = (xRC_h[i] - retRC)/retRC;
	/*
	std::cout << std::setw( 4) << cuZ[k]
		  << std::setw( 4) << cuC[k]
		  << std::setw(14) << energy_h[i]
		  << std::setw(14) << xRC_h[i]
		  << std::setw(14) << retRC
		  << std::endl;
	*/
      }

      /*
      if (retCE.second>0.0)
	eCE_0[i]   = (eCE_h[i] - retCE.second)/retCE.second;

      if (cuC[k]<=cuZ[k])
	std::cout << std::setw(14) << xCE_h[i]
		  << std::setw(14) << eCE_h[i]
		  << std::endl;
      */
    }

    serial.stop();

    std::sort(xFF_0.begin(), xFF_0.end());
    std::sort(eFF_0.begin(), eFF_0.end());
    std::sort(xCE_0.begin(), xCE_0.end());
    std::sort(eCE_0.begin(), eCE_0.end());
    std::sort(xCI_0.begin(), xCI_0.end());
    std::sort(xRC_0.begin(), xRC_0.end());
    
    std::vector<double> quantiles = {0.01, 0.05, 0.1, 0.2, 0.5, 0.8, 0.9, 0.95, 0.99};

    std::cout << "Ion (" << I->Z << ", " << I->C << ")" << std::endl;
    for (auto v : quantiles) {
      int indx = std::min<int>(std::floor(v*Nenergy+0.5), Nenergy-1);
      double FF_xc = 0.0, FF_ph = 0.0, CE_xc = 0.0, CE_ph = 0.0;
      double CI_xc = 0.0, RC_xc = 0.0;
      
      if (E.C>1) {
	FF_xc = xFF_0[indx];
	FF_ph = eFF_0[indx];
	RC_xc = xRC_0[indx];
      }

      if (E.C<=E.Z) {
	CE_xc = xCE_0[indx];
	CE_ph = eCE_0[indx];
	CI_xc = xCI_0[indx];
      }

      std::cout << std::setw(10) << v
		<< " | " << std::setw(14) << FF_xc
		<< " | " << std::setw(14) << FF_ph
		<< " | " << std::setw(14) << CE_xc
		<< " | " << std::setw(14) << CE_ph
		<< " | " << std::setw(14) << CI_xc
		<< " | " << std::setw(14) << RC_xc
		<< std::endl;
    }

    k++;

  } // END: Ion list

  std::cout << std::endl
	    << "Serial time: " << serial() << std::endl
	    << "Cuda time  : " << cuda()   << std::endl;
}
