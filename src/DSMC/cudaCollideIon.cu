#include "hip/hip_runtime.h"
#include <algorithm>
#include <numeric>

#include <Component.H>
#include <Ion.H>
#include <CollideIon.H>
#include <TreeDSMC.H>
#include <EXPException.H>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cudaUtil.cuH>

// Sanity debug PP flag
// #define SANITY_DEBUG

// Use explicit energy conservation in COM frame
//
#define EXPLICIT_ECOM

// Number of pairs to store in partInteraction
//
constexpr int MAX_PAIRS = 64;
constexpr int PLIST_LEN = MAX_PAIRS*MAX_PAIRS;

// Relative error for energy conservation check
//
constexpr cuFP_t EDEL_TOL = 1.0e-09;

// Some thrust definitions for species handling
//@{
typedef thrust::pair<unsigned short, unsigned short> cuSpeciesKey;

struct cuSpeciesDef
{
  cuSpeciesKey sp;
  int k, I;
};

typedef thrust::tuple<int, cuSpeciesDef, cuSpeciesDef, cuFP_t> cuInteract;

//@}

// Swap value in device code
template <class T>
__device__
void cuSwap(T & x, T & y)
{
  T t = x;
  x   = y;
  y   = t;
}

// Constants for device defintions
__constant__ cuFP_t cuH_H, cuHe_H, cuPH_H, cuPHe_H, cuEsu;
__constant__ cuFP_t cuH_Emin, cuHe_Emin, cuPH_Emin, cuPHe_Emin;

#ifdef XC_DEEP9
__device__ unsigned long long w_countr[11];
__device__ cuFP_t w_weight[11];
#endif

// Charged particle type
enum cudaElasticType { electron, proton };

// Atomic radii in picometers from Clementi, E.; Raimond, D. L.;
// Reinhardt, W. P. (1967). "Atomic Screening Constants from SCF
// Functions. II. Atoms with 37 to 86 Electrons". Journal of Chemical
// Physics 47 (4): 1300-1307.  See also Paper 1, ref. therein.
//
const int numRadii = 87;
__constant__ int cudaRadii[numRadii];

// For construction of evenly spaced interpolation arrays
//
thrust::host_vector<cuFP_t>
resampleArray(const std::vector<cuFP_t>& x, const std::vector<cuFP_t>& y,
	      cuFP_t& dx)
{
  // Get minimum grid spacing
  cuFP_t minH = std::numeric_limits<cuFP_t>::max();
  for (int i=0; i<x.size()-1; i++)
    minH = std::min<cuFP_t>(minH, x[i+1]- x[i]);

  // Resample based on minimum spacing
  int numH = int( (x.back() - x.front())/minH ) + 1;

  thrust::host_vector<cuFP_t> Y(numH);
  
  dx = (x.back() - x.front())/(numH - 1);

  for (int i=0; i<numH; i++) {
    cuFP_t xx = x.front() + dx*i, yy;
    if (xx <= x.front()) {
      yy = y.front();
    } else if (xx >= x.back()) {
      yy = y.back();
    } else {
      auto lb = std::lower_bound(x.begin(), x.end(), xx);
      auto ub = lb;
      if (lb!=x.begin()) lb--;
      if (ub == x.end()) ub = lb--;
      auto a = (*ub - xx)/(*ub - *lb);
      auto b = (xx - *lb)/(*ub - *lb);
      yy = a*y[lb - x.begin()] + b*y[ub - x.begin()];
    }
    Y[i] = yy;
  }

  return Y;
}

// Initialize cross-section look up and interpolation arrays.  Data
// input could be generalized here . . . for later.
//
void CollideIon::cudaElasticInit()
{
  std::vector<int> radii(numRadii, 0);

  radii[1]  =  53;
  radii[2]  =  31;
  radii[3]  =  167;
  radii[4]  =  112;
  radii[5]  =  87;
  radii[6]  =  67;
  radii[7]  =  56;
  radii[8]  =  48;
  radii[9]  =  42;
  radii[10] =  38;
  radii[11] =  190;
  radii[12] =  145;
  radii[13] =  118;
  radii[14] =  111;
  radii[15] =  98;
  radii[16] =  180;
  radii[17] =  79;
  radii[18] =  188;
  radii[19] =  243;
  radii[20] =  194;
  radii[21] =  184;
  radii[22] =  176;
  radii[23] =  171;
  radii[24] =  166;
  radii[25] =  161;
  radii[26] =  156;
  radii[27] =  152;
  radii[28] =  149;
  radii[29] =  145;
  radii[30] =  152;
  radii[31] =  136;
  radii[32] =  125;
  radii[33] =  114;
  radii[34] =  103;
  radii[35] =  94;
  radii[36] =  88;
  radii[37] =  265;
  radii[38] =  219;
  radii[39] =  212;
  radii[40] =  206;
  radii[41] =  198;
  radii[42] =  190;
  radii[43] =  183;
  radii[44] =  178;
  radii[45] =  173;
  radii[46] =  169;
  radii[47] =  172;
  radii[48] =  161;
  radii[49] =  193;
  radii[50] =  217;
  radii[51] =  133;
  radii[52] =  123;
  radii[53] =  198;
  radii[54] =  108;
  radii[55] =  298;
  radii[56] =  268;
  radii[59] =  247;
  radii[60] =  206;
  radii[61] =  205;
  radii[62] =  238;
  radii[63] =  231;
  radii[64] =  233;
  radii[65] =  225;
  radii[66] =  228;
  radii[68] =  226;
  radii[69] =  222;
  radii[70] =  222;
  radii[71] =  217;
  radii[72] =  208;
  radii[73] =  200;
  radii[74] =  193;
  radii[75] =  188;
  radii[76] =  185;
  radii[77] =  180;
  radii[78] =  177;
  radii[79] =  166;
  radii[80] =  171;
  radii[81] =  156;
  radii[82] =  202;
  radii[83] =  143;
  radii[84] =  135;
  radii[86] =  120;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cudaRadii), &radii[0], sizeof(int)*numRadii), 
		 __FILE__, __LINE__, "Error copying cudaRadii");

  // Total cross section from Malik & Trefftz, 1960, Zeitschrift fur Astrophysik, 50, 96-109

  // Column 1 in eV, Column2 in Bohr cross section (pi*a_0^2) units

  std::vector<cuFP_t> eV_H, xs_H;

  eV_H.push_back(0.66356077923727);	xs_H.push_back(28.864);
  eV_H.push_back(0.66576762346358);	xs_H.push_back(29.5088);
  eV_H.push_back(0.71282701193426);	xs_H.push_back(28.2574);
  eV_H.push_back(0.73590227585419);	xs_H.push_back(27.4989);
  eV_H.push_back(0.75936666273882);	xs_H.push_back(26.8542);
  eV_H.push_back(0.80889140369906);	xs_H.push_back(26.3234);
  eV_H.push_back(0.83209592175062);	xs_H.push_back(25.6028);
  eV_H.push_back(0.90677351672548);	xs_H.push_back(24.9204);
  eV_H.push_back(0.95590913472103);	xs_H.push_back(24.2757);
  eV_H.push_back(1.031106467362);	xs_H.push_back(23.745);
  eV_H.push_back(1.05457085424663);	xs_H.push_back(23.1003);
  eV_H.push_back(1.10409559520687);	xs_H.push_back(22.5694);
  eV_H.push_back(1.17903305901478);	xs_H.push_back(21.9628);
  eV_H.push_back(1.22881766880808);	xs_H.push_back(21.5079);
  eV_H.push_back(1.2782131556367);	xs_H.push_back(20.9391);
  eV_H.push_back(1.35379961124236);	xs_H.push_back(20.5221);
  eV_H.push_back(1.45506137966837);	xs_H.push_back(20.1052);
  eV_H.push_back(1.58185287994542);	xs_H.push_back(19.6504);
  eV_H.push_back(1.75999228472356);	xs_H.push_back(19.1958);
  eV_H.push_back(1.91233528596856);	xs_H.push_back(18.7032);
  eV_H.push_back(2.06519530374007);	xs_H.push_back(18.3623);
  eV_H.push_back(2.24360682247953);	xs_H.push_back(17.9835);
  eV_H.push_back(2.42186867854026);	xs_H.push_back(17.5668);
  eV_H.push_back(2.60026659158166);	xs_H.push_back(17.188);
  eV_H.push_back(2.77893661858437);	xs_H.push_back(16.8851);
  eV_H.push_back(2.9830084838763);	xs_H.push_back(16.5064);
  eV_H.push_back(3.21287675270137);	xs_H.push_back(16.1657);
  eV_H.push_back(3.39141072272342);	xs_H.push_back(15.8249);
  eV_H.push_back(3.64683049251644);	xs_H.push_back(15.4463);
  eV_H.push_back(3.87695726960477);	xs_H.push_back(15.1815);
  eV_H.push_back(4.0810291348967);	xs_H.push_back(14.8028);
  eV_H.push_back(4.31091100941984);	xs_H.push_back(14.4621);
  eV_H.push_back(4.54091533522557);	xs_H.push_back(14.1593);
  eV_H.push_back(4.71957175653021);	xs_H.push_back(13.8564);
  eV_H.push_back(4.97525003458648);	xs_H.push_back(13.5537);
  eV_H.push_back(5.28200410318252);	xs_H.push_back(13.1753);
  eV_H.push_back(5.53768238123879);	xs_H.push_back(12.8726);
  eV_H.push_back(5.74227126305723);	xs_H.push_back(12.6456);
  eV_H.push_back(5.97267015410688);	xs_H.push_back(12.4566);
  eV_H.push_back(6.15132657541152);	xs_H.push_back(12.1537);
  eV_H.push_back(6.40726336173105);	xs_H.push_back(11.9268);
  eV_H.push_back(6.61198830053015);	xs_H.push_back(11.7378);
  eV_H.push_back(6.81683569061185);	xs_H.push_back(11.5866);
  eV_H.push_back(6.99562816889715);	xs_H.push_back(11.3216);
  eV_H.push_back(7.20035310769626);	xs_H.push_back(11.1326);
  eV_H.push_back(7.43061594176524);	xs_H.push_back(10.9057);
  eV_H.push_back(7.71209062335465);	xs_H.push_back(10.641);
  eV_H.push_back(7.96789135269351);	xs_H.push_back(10.3762);
  eV_H.push_back(8.27517604351412);	xs_H.push_back(10.1495);
  eV_H.push_back(8.50530282060245);	xs_H.push_back(9.88464);
  eV_H.push_back(8.76123960692198);	xs_H.push_back(9.6578);
  eV_H.push_back(9.06852429774258);	xs_H.push_back(9.43109);
  eV_H.push_back(9.35012143061459);	xs_H.push_back(9.20432);
  eV_H.push_back(9.55484636941369);	xs_H.push_back(9.01526);
  eV_H.push_back(9.78536771174593);	xs_H.push_back(8.86419);
  eV_H.push_back(10.0157666027956);	xs_H.push_back(8.6752);
  eV_H.push_back(10.2717033891151);	xs_H.push_back(8.44835);
  eV_H.push_back(10.5533005219871);	xs_H.push_back(8.22158);
  eV_H.push_back(10.8349112605572);	xs_H.push_back(7.9948);
  eV_H.push_back(11.1421823456797);	xs_H.push_back(7.7681);
  eV_H.push_back(11.4237930842498);	xs_H.push_back(7.54133);
  eV_H.push_back(11.6798523218519);	xs_H.push_back(7.35241);
  eV_H.push_back(11.9615991174026);	xs_H.push_back(7.16356);
  eV_H.push_back(12.2176583550047);	xs_H.push_back(6.97464);
  eV_H.push_back(12.4223832938038);	xs_H.push_back(6.78558);
  eV_H.push_back(12.7041164836565);	xs_H.push_back(6.59673);
  eV_H.push_back(12.9858496735092);	xs_H.push_back(6.40788);
  eV_H.push_back(13.2163710158414);	xs_H.push_back(6.25682);
  eV_H.push_back(13.4212320116212);	xs_H.push_back(6.10568);
  eV_H.push_back(13.600541506433);	xs_H.push_back(5.9924);

  cuFP_t dx;

  xsc_H = resampleArray(eV_H, xs_H, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_Emin), &eV_H[0], sizeof(cuFP_t)),
		 __FILE__, __LINE__, "Error copying cuH_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuH_H), &dx, sizeof(cuFP_t)),
		 __FILE__, __LINE__, "Error copying cuH_H");

  // Total cross section from LaBahn & Callaway, 1966, Phys. Rev., 147, 50, 28-40
  //
  std::vector<cuFP_t> eV_He, xs_He;

  eV_He.push_back(0.0972135);	xs_He.push_back(62.2773619684373);
  eV_He.push_back(0.212908);	xs_He.push_back(65.3193661349083);
  eV_He.push_back(0.251768);	xs_He.push_back(66.6419766420696);
  eV_He.push_back(0.440878);	xs_He.push_back(67.8332685763108);
  eV_He.push_back(0.704798);	xs_He.push_back(68.6287198361997);
  eV_He.push_back(1.11846);	xs_He.push_back(68.7641224795695);
  eV_He.push_back(1.5694);	xs_He.push_back(68.5696578943123);
  eV_He.push_back(1.86971);	xs_He.push_back(68.109100411296 );
  eV_He.push_back(2.20762);	xs_He.push_back(67.6491712468105);
  eV_He.push_back(2.50774);	xs_He.push_back(66.9903792673527);
  eV_He.push_back(2.77027);	xs_He.push_back(66.3312731286295);
  eV_He.push_back(3.07045);	xs_He.push_back(65.7387687541625);
  eV_He.push_back(3.25779);	xs_He.push_back(65.0790342969086);
  eV_He.push_back(3.44532);	xs_He.push_back(64.6178484953617);
  eV_He.push_back(3.78297);	xs_He.push_back(63.8930830701785);
  eV_He.push_back(4.0079);	xs_He.push_back(63.2339769314554);
  eV_He.push_back(4.2329);	xs_He.push_back(62.6405300791922);
  eV_He.push_back(4.57055);	xs_He.push_back(61.9160788132744);
  eV_He.push_back(4.79555);	xs_He.push_back(61.3229461202767);
  eV_He.push_back(5.02048);	xs_He.push_back(60.6635258222882);
  eV_He.push_back(5.20782);	xs_He.push_back(60.0041055242997);
  eV_He.push_back(5.50788);	xs_He.push_back(59.2790259398512);
  eV_He.push_back(5.883);	xs_He.push_back(58.4226277824826);
  eV_He.push_back(6.14552);	xs_He.push_back(57.7635216437595);
  eV_He.push_back(6.48318);	xs_He.push_back(57.0390703778416);
  eV_He.push_back(6.89576);	xs_He.push_back(56.0507253290223);
  eV_He.push_back(7.27088);	xs_He.push_back(55.1943271716537);
  eV_He.push_back(7.68347);	xs_He.push_back(54.2059821228344);
  eV_He.push_back(7.98353);	xs_He.push_back(53.4809025383858);
  eV_He.push_back(8.32118);	xs_He.push_back(52.756451272468);
  eV_He.push_back(8.6963);	xs_He.push_back(51.9000531150995);
  eV_He.push_back(8.99617);	xs_He.push_back(50.9767390342094);
  eV_He.push_back(9.33408);	xs_He.push_back(50.5168098697239);
  eV_He.push_back(9.67173);	xs_He.push_back(49.7920444445407);
  eV_He.push_back(9.97191);	xs_He.push_back(49.1995400700737);
  eV_He.push_back(10.2346);	xs_He.push_back(48.6726949820667);
  eV_He.push_back(10.4596);	xs_He.push_back(48.0795622890689);
  eV_He.push_back(10.7222);	xs_He.push_back(47.5527172010619);
  eV_He.push_back(10.9849);	xs_He.push_back(47.0921597180456);
  eV_He.push_back(11.2852);	xs_He.push_back(46.565628789304 );
  eV_He.push_back(11.5478);	xs_He.push_back(46.038783701297 );
  eV_He.push_back(11.773);	xs_He.push_back(45.57759789975  );
  eV_He.push_back(12.0731);	xs_He.push_back(44.9191200795576);
  eV_He.push_back(12.3734);	xs_He.push_back(44.4585625965413);
  eV_He.push_back(12.7488);	xs_He.push_back(43.866686540605 );
  eV_He.push_back(13.0489);	xs_He.push_back(43.2738680068726);
  eV_He.push_back(13.3867);	xs_He.push_back(42.6153901866802);
  eV_He.push_back(13.7996);	xs_He.push_back(41.9578548442838);
  eV_He.push_back(14.1375);	xs_He.push_back(41.4976115205329);
  eV_He.push_back(14.4752);	xs_He.push_back(40.9054213053313);
  eV_He.push_back(14.8132);	xs_He.push_back(40.5114655865711);
  eV_He.push_back(15.1509);	xs_He.push_back(39.8529877663787);
  eV_He.push_back(15.4889);	xs_He.push_back(39.4590320476185);
  eV_He.push_back(15.7893);	xs_He.push_back(39.064762169593 );
  eV_He.push_back(16.1271);	xs_He.push_back(38.5385454001167);
  eV_He.push_back(16.465);	xs_He.push_back(38.0123286306404);
  eV_He.push_back(16.8404);	xs_He.push_back(37.4864260204295);
  eV_He.push_back(17.1407);	xs_He.push_back(37.0258685374132);
  eV_He.push_back(17.5162);	xs_He.push_back(36.566253532193 );
  eV_He.push_back(17.8917);	xs_He.push_back(36.1063243677075);
  eV_He.push_back(18.2672);	xs_He.push_back(35.6463952032219);
  eV_He.push_back(18.6426);	xs_He.push_back(35.120492593011 );
  eV_He.push_back(19.0181);	xs_He.push_back(34.6608775877908);
  eV_He.push_back(19.3561);	xs_He.push_back(34.2669218690307);
  eV_He.push_back(19.6941);	xs_He.push_back(33.8729661502705);
  eV_He.push_back(20.0321);	xs_He.push_back(33.478696272245);
  eV_He.push_back(20.3324);	xs_He.push_back(33.0844263942195);
  eV_He.push_back(20.708);	xs_He.push_back(32.6907848347247);
  eV_He.push_back(21.046);	xs_He.push_back(32.2968291159645);
  eV_He.push_back(21.3839);	xs_He.push_back(31.836899951479 );
  eV_He.push_back(21.7594);	xs_He.push_back(31.37700220292  );
  eV_He.push_back(22.1725);	xs_He.push_back(30.9174814454794);
  eV_He.push_back(22.548);	xs_He.push_back(30.523808470058 );
  eV_He.push_back(22.9612);	xs_He.push_back(30.1304182379755);
  eV_He.push_back(23.3369);	xs_He.push_back(29.8689434814172);
  eV_He.push_back(23.5995);	xs_He.push_back(29.3421612252633);
  eV_He.push_back(23.9752);	xs_He.push_back(29.080686468705 );
  eV_He.push_back(24.3506);	xs_He.push_back(28.4886847490626);
  eV_He.push_back(24.8389);	xs_He.push_back(28.0958914195842);
  eV_He.push_back(25.2144);	xs_He.push_back(27.6360879188048);
  eV_He.push_back(25.8906);	xs_He.push_back(27.1125729190106);
  eV_He.push_back(26.3788);	xs_He.push_back(26.5875499547427);
  eV_He.push_back(26.7543);	xs_He.push_back(26.1277464539633);
  eV_He.push_back(27.2427);	xs_He.push_back(25.734953124485 );
  eV_He.push_back(27.7686);	xs_He.push_back(25.342473954272 );
  eV_He.push_back(28.2944);	xs_He.push_back(24.8177337333429);
  eV_He.push_back(28.8205);	xs_He.push_back(24.5574841979195);
  eV_He.push_back(29.2337);	xs_He.push_back(24.164093965837 );
  eV_He.push_back(29.8723);	xs_He.push_back(23.706363916209 );
  eV_He.push_back(30.3607);	xs_He.push_back(23.3135705867306);
  eV_He.push_back(30.8868);	xs_He.push_back(23.1194201607388);
  eV_He.push_back(31.4127);	xs_He.push_back(22.7269409905258);
  eV_He.push_back(31.9388);	xs_He.push_back(22.4666600391759);
  eV_He.push_back(32.615);	xs_He.push_back(21.9431450393817);
  eV_He.push_back(33.3665);	xs_He.push_back(21.5524251610547);
  eV_He.push_back(34.0429);	xs_He.push_back(21.2272389054817);
  eV_He.push_back(34.6064);	xs_He.push_back(20.8350424786075);
  eV_He.push_back(35.0948);	xs_He.push_back(20.5083796744872);
  eV_He.push_back(35.5457);	xs_He.push_back(20.2475018205331);
  eV_He.push_back(36.0342);	xs_He.push_back(20.0530372352759);
  eV_He.push_back(36.4851);	xs_He.push_back(19.7921907972484);
  eV_He.push_back(37.0112);	xs_He.push_back(19.59800895533  );
  eV_He.push_back(37.5372);	xs_He.push_back(19.2716288945485);
  eV_He.push_back(38.0258);	xs_He.push_back(19.0771957252179);
  eV_He.push_back(38.5519);	xs_He.push_back(18.8830138832995);
  eV_He.push_back(38.9277);	xs_He.push_back(18.6876696520993);
  eV_He.push_back(39.4537);	xs_He.push_back(18.4273887007494);
  eV_He.push_back(39.9423);	xs_He.push_back(18.2329555314187);
  eV_He.push_back(40.506);	xs_He.push_back(18.0390878487657);
  eV_He.push_back(40.9569);	xs_He.push_back(17.7782099948116);
  eV_He.push_back(41.483);	xs_He.push_back(17.5840595688197);
  eV_He.push_back(41.9715);	xs_He.push_back(17.3895949835625);
  eV_He.push_back(42.3472);	xs_He.push_back(17.1281516429308);
  eV_He.push_back(42.7606);	xs_He.push_back(16.9991892645009);
  eV_He.push_back(43.174);	xs_He.push_back(16.8702583019976);
  eV_He.push_back(43.5498);	xs_He.push_back(16.6749140707974);
  eV_He.push_back(43.9631);	xs_He.push_back(16.479852582936);
  eV_He.push_back(44.339);	xs_He.push_back(16.3506074611673);
  eV_He.push_back(44.6772);	xs_He.push_back(16.2210795960598);
  eV_He.push_back(45.2033);	xs_He.push_back(16.0268977541414);
  eV_He.push_back(45.504);	xs_He.push_back(15.9631862551266);
  eV_He.push_back(45.8798);	xs_He.push_back(15.8339411333579);
  eV_He.push_back(46.2556);	xs_He.push_back(15.7046960115892);
  eV_He.push_back(46.5939);	xs_He.push_back(15.5751681464817);
  eV_He.push_back(46.857);	xs_He.push_back(15.5111424882016);
  eV_He.push_back(47.1952);	xs_He.push_back(15.3816146230941);
  eV_He.push_back(47.571);	xs_He.push_back(15.2523695013254);
  eV_He.push_back(47.8717);	xs_He.push_back(15.188626586384 );
  eV_He.push_back(48.1348);	xs_He.push_back(15.1246009281039);
  eV_He.push_back(48.4354);	xs_He.push_back(14.9947903196575);
  eV_He.push_back(48.7736);	xs_He.push_back(14.8652310386235);
  eV_He.push_back(49.0368);	xs_He.push_back(14.8673359057014);
  eV_He.push_back(49.2622);	xs_He.push_back(14.7368969787244);
  eV_He.push_back(49.5254);	xs_He.push_back(14.7389704298757);
  eV_He.push_back(49.7885);	xs_He.push_back(14.6749447715956);
  eV_He.push_back(49.9763);	xs_He.push_back(14.4781239918482);

  xsc_He = resampleArray(eV_He, xs_He, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuHe_Emin), &eV_He[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuHe_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuHe_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuHe_H");

  // Interpolated from Figure 1 of "Elastic scattering and charge
  // transfer in slow collisions: isotopes of H and H + colliding with
  // isotopes of H and with He" by Predrag S Krstić and David R Schultz,
  // 1999 J. Phys. B: At. Mol. Opt. Phys. 32 3485
  //

  std::vector<cuFP_t> eV_pH, xs_pH;

  eV_pH.push_back(-0.994302);	xs_pH.push_back(2.86205);
  eV_pH.push_back(-0.897482);	xs_pH.push_back(2.90929);
  eV_pH.push_back(-0.801179);	xs_pH.push_back(2.86016);
  eV_pH.push_back(-0.691555);	xs_pH.push_back(2.89417);
  eV_pH.push_back(-0.588753);	xs_pH.push_back(2.85638);
  eV_pH.push_back(-0.49242);	xs_pH.push_back(2.81291);
  eV_pH.push_back(-0.395965);	xs_pH.push_back(2.79213);
  eV_pH.push_back(-0.292839);	xs_pH.push_back(2.8148);
  eV_pH.push_back(-0.19019);	xs_pH.push_back(2.74866);
  eV_pH.push_back(-0.0872765);	xs_pH.push_back(2.73165);
  eV_pH.push_back(0.00935082);	xs_pH.push_back(2.74299);
  eV_pH.push_back(0.112152);	xs_pH.push_back(2.7052);
  eV_pH.push_back(0.208688);	xs_pH.push_back(2.69953);
  eV_pH.push_back(0.311612);	xs_pH.push_back(2.68441);
  eV_pH.push_back(0.401578);	xs_pH.push_back(2.65417);
  eV_pH.push_back(0.517468);	xs_pH.push_back(2.65606);
  eV_pH.push_back(0.613862);	xs_pH.push_back(2.62394);
  eV_pH.push_back(0.716846);	xs_pH.push_back(2.62016);
  eV_pH.push_back(0.819688);	xs_pH.push_back(2.58992);
  eV_pH.push_back(0.909797);	xs_pH.push_back(2.58614);
  eV_pH.push_back(1.01906);	xs_pH.push_back(2.55213);
  eV_pH.push_back(1.1092);	xs_pH.push_back(2.55402);
  eV_pH.push_back(1.21203);	xs_pH.push_back(2.52189);
  eV_pH.push_back(1.3085);	xs_pH.push_back(2.50488);
  eV_pH.push_back(1.41149);	xs_pH.push_back(2.5011);
  eV_pH.push_back(1.52077);	xs_pH.push_back(2.47087);
  eV_pH.push_back(1.61715);	xs_pH.push_back(2.43685);
  eV_pH.push_back(1.71368);	xs_pH.push_back(2.42929);
  eV_pH.push_back(1.81666);	xs_pH.push_back(2.42551);
  eV_pH.push_back(1.9131);	xs_pH.push_back(2.40094);
  eV_pH.push_back(2.0159);	xs_pH.push_back(2.36315);

  xsc_pH = resampleArray(eV_pH, xs_pH, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPH_Emin), &eV_pH[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPH_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPH_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPH_H");

  // Interpolated from the top panel of Figure 4, op. cit.
  //
  std::vector<cuFP_t> eV_pHe, xs_pHe;

  eV_pHe.push_back(-0.984127);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.904762);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.825397);	xs_pHe.push_back(2.68889);
  eV_pHe.push_back(-0.753968);	xs_pHe.push_back(2.64444);
  eV_pHe.push_back(-0.674603);	xs_pHe.push_back(2.6);
  eV_pHe.push_back(-0.595238);	xs_pHe.push_back(2.57778);
  eV_pHe.push_back(-0.515873);	xs_pHe.push_back(2.57778);
  eV_pHe.push_back(-0.444444);	xs_pHe.push_back(2.55556);
  eV_pHe.push_back(-0.373016);	xs_pHe.push_back(2.48889);
  eV_pHe.push_back(-0.293651);	xs_pHe.push_back(2.44444);
  eV_pHe.push_back(-0.214286);	xs_pHe.push_back(2.46667);
  eV_pHe.push_back(-0.142857);	xs_pHe.push_back(2.44444);
  eV_pHe.push_back(-0.0634921);	xs_pHe.push_back(2.4);
  eV_pHe.push_back(0.015873);	xs_pHe.push_back(2.37778);
  eV_pHe.push_back(0.0952381);	xs_pHe.push_back(2.37778);
  eV_pHe.push_back(0.166667);	xs_pHe.push_back(2.33333);
  eV_pHe.push_back(0.246032);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.325397);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.404762);	xs_pHe.push_back(2.28889);
  eV_pHe.push_back(0.47619);	xs_pHe.push_back(2.24444);
  eV_pHe.push_back(0.555556);	xs_pHe.push_back(2.2);
  eV_pHe.push_back(0.634921);	xs_pHe.push_back(2.17778);
  eV_pHe.push_back(0.706349);	xs_pHe.push_back(2.2);
  eV_pHe.push_back(0.785714);	xs_pHe.push_back(2.17778);
  eV_pHe.push_back(0.865079);	xs_pHe.push_back(2.13333);
  eV_pHe.push_back(0.936508);	xs_pHe.push_back(2.08889);
  eV_pHe.push_back(1.01587);	xs_pHe.push_back(2.06667);
  eV_pHe.push_back(1.09524);	xs_pHe.push_back(2.08889);
  eV_pHe.push_back(1.16667);	xs_pHe.push_back(2.06667);
  eV_pHe.push_back(1.24603);	xs_pHe.push_back(2.04444);
  eV_pHe.push_back(1.3254);	xs_pHe.push_back(2.02222);
  eV_pHe.push_back(1.40476);	xs_pHe.push_back(1.97778);
  eV_pHe.push_back(1.47619);	xs_pHe.push_back(1.93333);
  eV_pHe.push_back(1.55556);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.63492);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.71429);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.79365);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.87302);	xs_pHe.push_back(1.91111);
  eV_pHe.push_back(1.95238);	xs_pHe.push_back(1.91111);

  xsc_pHe = resampleArray(eV_pHe, xs_pHe, dx);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPHe_Emin), &eV_pHe[0], sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPHe_Emin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuPHe_H), &dx, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuPHe_H");

#ifdef XC_COMPARE
  // Compares the cross-section evaluation between CPU and GPU
  //
  if (myid==0) {
    const int Nenergy = 1000;
    ch.testCross(Nenergy, cuElems, xsc_H, xsc_He);
  }
  MPI_Barrier(MPI_COMM_WORLD);
#endif
}

__device__
cuFP_t cudaGeometric(int Z)
{
  if (Z>0 and Z< numRadii) {
    return cudaRadii[Z] * 1.0e-3;
  } else {
    return 0.0;
  }
}
		 

__device__
cuFP_t cudaElasticInterp(cuFP_t E, dArray<cuFP_t> xsc, int Z,
			 cudaElasticType etype = electron,
			 bool pin = true)
{
  // Bohr cross section (pi*a_0^2) in nm
  //
  const cuFP_t b_cross = 0.00879735542978;

  cuFP_t Emin, H;
  bool logV = false;
  int N = xsc._s;

  if (Z==1) {
    if (etype == electron) {
      H    = cuH_H;
      Emin = cuH_Emin;
    } else {
      H    = cuPH_H;
      Emin = cuPH_Emin;
      E    = log10(E);
      logV = true;
    }
  }
  else if (Z==2)
    if (etype == electron) {
      H    = cuHe_H;
      Emin = cuHe_Emin;
    } else {
      H    = cuPHe_H;
      Emin = cuPHe_Emin;
      E    = log10(E);
      logV = true;
    }
  else {
    return 0.0;
  }

  cuFP_t Emax = Emin + N*H, val = 0.0;

  // Enforce return value to grid boundaries for off-grid ordinates.
  // Otherwise, values will be extrapolated.
  //
  if      (pin and E <= Emin) val = xsc._v[0];
  else if (pin and E >= Emax) val = xsc._v[N-1];
  else {

    int indx = 0;
    if (E >= Emax)      indx = xsc._s - 2;
    else if (E <= Emin) indx = 0;
#if cuREAL == 4
    else                indx = floorf( (E - Emin)/H );
#else
    else                indx = floor ( (E - Emin)/H );
#endif
    
    // Sanity
    if (indx<0) indx = 0;
    if (indx>xsc._s - 2) indx = xsc._s - 2;

    cuFP_t a = (E - Emin - H*(indx+0))/H;
    cuFP_t b = (Emin + H*(indx+1) - E)/H;
    
    val = a*xsc._v[indx] + b*xsc._v[indx+1];

    if ((logV and val>3.0) or val>80.0) {
      if (pin)
	printf("E=%e a=%e b=%e val=%e [pinned]\n", E, a, b, val);
      else
	printf("E=%e a=%e b=%e val=%e [extrap]\n", E, a, b, val);
    }
  }

  if (logV) val = pow(10.0, val);
  
  return b_cross * val;
}


// Global symbols for coordinate transformation
//
__device__ __constant__
cuFP_t ionEminGrid, ionEmaxGrid, ionDeltaEGrid;

__device__ __constant__
int ionEgridNumber, ionRadRecombNumber;

// The grid energy ranges are set in Ion.cc as Ion class static
// variables
//
void atomicData::cuda_initialize_textures()
{
  size_t ionSize = IonList.size();

  // Interpolation data array
  //
  cuF0array.resize(ionSize, 0);
  cuFFarray.resize(ionSize, 0);
  cuRCarray.resize(ionSize, 0);
  cuCEarray.resize(ionSize, 0);
  cuCIarray.resize(ionSize, 0);
  cuPIarray.resize(ionSize   );

  // Texture object array
  //
  cuIonElem.resize(ionSize);

  // Total photo-ionization rate
  //
  std::vector<cuFP_t> phRate(ionSize, 0.0);

  size_t k = 0;

  for (auto v : IonList) {

    IonPtr I = v.second;
    cuIonElement& E = cuIonElem[k];

    E.IPval = 0.0;
    if (E.C<= E.Z) {
      lQ Q(I->Z, I->C);
      E.IPval = I->getIP(Q);
    }

    // The free-free array
    //
    if (E.C>1) {		// Must NOT BE neutral
      hipTextureDesc texDesc;

      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.readMode       = hipReadModeElementType;
      texDesc.filterMode     = hipFilterModePoint;
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
  
      // Temporary storage
      //
      std::vector<cuFP_t> h_buffer0(I->NfreeFreeGrid, 0.0);

      cuFP_t *d_Interp;

      cuda_safe_call(hipMalloc((void **)&d_Interp, I->NfreeFreeGrid*CHCUMK*sizeof(cuFP_t)),
		     __FILE__, __LINE__,
		     "Error allocating d_Interp1 for texture construction");
  
      std::vector<cuFP_t> h_buffer1(I->NfreeFreeGrid*CHCUMK, 0.0);

      double delC = 1.0/(CHCUMK-1);

      // Copy cross section values to buffer
      //
      for (int i = 0; i < I->NfreeFreeGrid; i++) {

	h_buffer0[i] = I->freeFreeGrid[i].back();
	
	// Unit normalized cumulative distribution
	//
	size_t tsize = I->freeFreeGrid[i].size();
	std::vector<double> temp(tsize);
	for (int j = 0; j < tsize; j++) {	
	  temp[j] = I->freeFreeGrid[i][j]/h_buffer0[i];
	}

	// End points
	//
	h_buffer1[i                              ] = I->kgrid[0];
	h_buffer1[i + (CHCUMK-1)*I->NfreeFreeGrid] = I->kgrid[tsize-1];

	// Remap to even grid
	//
	for (int j=1; j<CHCUMK-1; j++) {

	  double C = delC*j;

	  // Points to first element that is not < C
	  // but may be equal
	  std::vector<double>::iterator lb = 
	    std::lower_bound(temp.begin(), temp.end(), C);
    
	  // Assign upper end of range to the
	  // found element
	  //
	  std::vector<double>::iterator ub = lb;
	  //
	  // If is the first element, increment
	  // the upper boundary
	  //
	  if (lb == temp.begin()) { if (temp.size()>1) ub++; }
	  //
	  // Otherwise, decrement the lower boundary
	  //
	  else { lb--; }
    
	  // Compute the associated indices
	  //
	  size_t ii = lb - temp.begin();
	  size_t jj = ub - temp.begin();
	  double kk = I->kgrid[ii];
	  
	  // Linear interpolation
	  //
	  if (*ub > *lb) {
	    double d = *ub - *lb;
	    double a = (C - *lb) / d;
	    double b = (*ub - C) / d;

	    kk  = a * I->kgrid[ii] + b * I->kgrid[jj];
	  }

	  h_buffer1[i + j*I->NfreeFreeGrid] = kk;

	} // END: cumululative array loop

      } // END: energy loop

      // Copy 1-dim data to device
      //
      size_t tsize = I->NfreeFreeGrid*sizeof(cuFP_t);

      // Allocate CUDA array in device memory (a one-dimension 'channel')
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc<int2>();
#endif
      
      cuda_safe_call(hipMallocArray(&cuF0array[k], &channelDesc1, I->NfreeFreeGrid), __FILE__, __LINE__, "malloc cuArray");

      cuda_safe_call(hipMemcpyToArray(cuF0array[k], 0, 0, &h_buffer0[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

      // Specify 1-d texture
      //
      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuF0array[k];

      cuda_safe_call(hipCreateTextureObject(&E.ff_0, &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");

      // Copy data to device
      tsize = I->NfreeFreeGrid*CHCUMK*sizeof(cuFP_t);
      cuda_safe_call(hipMemcpy(d_Interp, &h_buffer1[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");
    
      // cuda 2d Array Descriptor
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<int2>();
#endif
      // cuda 2d Array
      //
      cuda_safe_call(hipMalloc3DArray(&cuFFarray[k], &channelDesc2, make_hipExtent(I->NfreeFreeGrid, CHCUMK, 1), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");
      
      // Array creation
      //
      hipMemcpy3DParms copyParams = {0};
  
      copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, I->NfreeFreeGrid*sizeof(cuFP_t), I->NfreeFreeGrid, CHCUMK);
      copyParams.dstArray = cuFFarray[k];
      copyParams.extent   = make_hipExtent(I->NfreeFreeGrid, CHCUMK, 1);
      copyParams.kind     = hipMemcpyDeviceToDevice;
      
      cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array  = cuFFarray[k];
    
      cuda_safe_call
	(hipCreateTextureObject(&E.ff_d, &resDesc, &texDesc, NULL),
	 __FILE__, __LINE__, "Failure in 2d texture creation");
      
      cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
    }

    // Radiative recombination texture (1-d)
    //
    if (E.C>1) {
      // Allocate CUDA array in device memory (a one-dimension 'channel')
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    
      // Size of interpolation array
      //
      size_t tsize = I->NradRecombGrid*sizeof(cuFP_t);

      hipTextureDesc texDesc;
      
      memset(&texDesc, 0, sizeof(hipTextureDesc));
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.readMode = hipReadModeElementType;
      texDesc.normalizedCoords = 0;
      
      thrust::host_vector<cuFP_t> tt(I->NradRecombGrid);
      
      cuda_safe_call(hipMallocArray(&cuRCarray[k], &channelDesc, I->NradRecombGrid), __FILE__, __LINE__, "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      for (size_t n = 0; n < I->NradRecombGrid; n++) tt[n] = I->radRecombGrid[n];
    
      cuda_safe_call(hipMemcpyToArray(cuRCarray[k], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");

      // Specify texture

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuRCarray[k];
      
      cuda_safe_call(hipCreateTextureObject(&E.rc_d, &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");
    }

    // The collisional excitation array
    //
    if (E.C <= E.Z and I->NcollideGrid>0) {

      E.ceEmin = I->collideEmin;
      E.ceEmax = I->collideEmax;
      E.ceDelE = I->delCollideE;
      E.NColl  = I->NcollideGrid;

      hipTextureDesc texDesc;
      
      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.readMode = hipReadModeElementType;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
  
      // Temporary storage
      //
      cuFP_t *d_Interp;
      cuda_safe_call(hipMalloc((void **)&d_Interp, I->NcollideGrid*2*sizeof(cuFP_t)),
		     __FILE__, __LINE__,
		     "Error allocating d_Interp for texture construction");
  
      std::vector<cuFP_t> h_buffer(I->NcollideGrid*2, 0.0);

      // Copy vectors to buffer
      //
      for (int i = 0; i < I->NcollideGrid; i++) {
	h_buffer[i                  ] = I->collideDataGrid[i].back().first;
	h_buffer[i + I->NcollideGrid] = I->collideDataGrid[i].back().second;
      }
      
      // Copy data to device
      cuda_safe_call(hipMemcpy(d_Interp, &h_buffer[0], I->NcollideGrid*2*sizeof(cuFP_t), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");
    
      // hipArray Descriptor
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
      // cuda Array
      //
      cuda_safe_call(hipMalloc3DArray(&cuCEarray[k], &channelDesc, make_hipExtent(I->NcollideGrid, 2, 1), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");
    
      // Array creation
      //
      hipMemcpy3DParms copyParams = {0};
      
      copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, I->NcollideGrid*sizeof(cuFP_t), I->NcollideGrid, 2);
      copyParams.dstArray = cuCEarray[k];
      copyParams.extent   = make_hipExtent(I->NcollideGrid, 2, 1);
      copyParams.kind     = hipMemcpyDeviceToDevice;
      
      cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");
      
      hipResourceDesc resDesc;
      
      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array  = cuCEarray[k];
      
      cuda_safe_call
	(hipCreateTextureObject(&E.ce_d, &resDesc, &texDesc, NULL),
	 __FILE__, __LINE__, "Failure in 2d texture creation");
      
      cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
    }

    // Collisional ionization texture (1-d)
    //
    if (E.C <= E.Z) {

      E.ciEmin = I->ionizeEmin;
      E.ciEmax = I->ionizeEmax;
      E.ciDelE = I->DeltaEGrid;
      E.NIonz  = I->NionizeGrid;

      // Allocate CUDA array in device memory (a one-dimension 'channel')
      //
#if cuREAL == 4
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
#endif
    
      // Size of interpolation array
      //
      size_t tsize = I->NionizeGrid*sizeof(cuFP_t);
      
      hipTextureDesc texDesc;

      memset(&texDesc, 0, sizeof(hipTextureDesc));
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModePoint;
      texDesc.readMode = hipReadModeElementType;
      texDesc.normalizedCoords = 0;
      
      thrust::host_vector<cuFP_t> tt(I->NionizeGrid);
      
      cuda_safe_call(hipMallocArray(&cuCIarray[k], &channelDesc, I->NionizeGrid), __FILE__, __LINE__, "malloc cuArray");

      // Copy to device memory some data located at address h_data
      // in host memory
      for (size_t n = 0; n < I->NionizeGrid; n++) tt[n] = I->ionizeDataGrid[n];
      
      cuda_safe_call(hipMemcpyToArray(cuCIarray[k], 0, 0, &tt[0], tsize, hipMemcpyHostToDevice), __FILE__, __LINE__, "copy texture to array");
      
      // Specify texture

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = cuCIarray[k];
      
      cuda_safe_call(hipCreateTextureObject(&E.ci_d, &resDesc, &texDesc, NULL), __FILE__, __LINE__, "create texture object");

      // Photoionization array
      //
      if (I->ib_type != Ion::none) {

	thrust::host_vector<cuFP_t> piCum(CHCUMK, 0.0);
	piCum[CHCUMK-1] = 1.0;
      
	double delC = 1.0/(CHCUMK-1);
      
	if (not I->IBinit) I->IBcreate();
      
	E.piTotl = I->IBtotl;

	// Copy cross section values to buffer
	//
	for (int j=1; j<CHCUMK-1; j++) {

	  // Location in cumulative cross section grid
	  //
	  double C = delC*j;

	  // Interpolate the cross section array
	  //
	
	  // Points to first element that is not < rn
	  // but may be equal
	  std::vector<double>::iterator lb = 
	    std::lower_bound(I->IBcum.begin(), I->IBcum.end(), C);
	  
	  // Assign upper end of range to the
	  // found element
	  //
	  std::vector<double>::iterator ub = lb;
	  //
	  // If is the first element, increment
	  // the upper boundary
	  //
	  if (lb == I->IBcum.begin()) { if (I->IBcum.size()>1) ub++; }
	  //
	  // Otherwise, decrement the lower boundary
	  //
	  else { lb--; }
	  
	  // Compute the associated indices
	  //
	  size_t ii = lb - I->IBcum.begin();
	  size_t jj = ub - I->IBcum.begin();
	  double nu = I->nugrid[ii];
	  
	  // Linear interpolation
	  //
	  if (*ub > *lb) {
	    double d = *ub - *lb;
	    double a = (C - *lb) / d;
	    double b = (*ub - C) / d;
	    nu  = a * I->nugrid[ii] + b * I->nugrid[jj];
	  }
	  
	  piCum[j] = (nu - 1.0)*I->ip;
	}
	
	// std::cout << "Allocating pi_0[" << k << "]" << std::endl;

	// Create storage on device
	cuPIarray[k] = piCum;

	// Assign pointer
	E.pi_0 = thrust::raw_pointer_cast(&cuPIarray[k][0]);

      } // END: cumululative array loop

    } // END: ions with electrons
    
    // Increment counter
    k++;	
    
  } // END: IonList

}




void atomicData::cuda_initialize_grid_constants()
{
  double Emin, Emax, delE;
  int NE, NR;

  for (auto v : IonList) {
    Emin = v.second->EminGrid;
    Emax = v.second->EmaxGrid;
    delE = v.second->DeltaEGrid;

    NE   = v.second->NfreeFreeGrid;

    if (v.first.second>1) {
      NR = v.second->NradRecombGrid;
      break;
    }
  }

  cuFP_t f;

  // Copy constants to device
  //
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEminGrid), &(f=Emin),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEminGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEmaxGrid), &(f=Emax),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEmaxGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionDeltaEGrid), &(f=delE),
				    sizeof(cuFP_t), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionDeltaEGrid");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionEgridNumber), &NE,
				    sizeof(int), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionEgridNumber");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(ionRadRecombNumber), &NR,
				    sizeof(int), size_t(0),
				    hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying ionRadRecombNumber");
}


__device__
void computeFreeFree
(cuFP_t E, cuFP_t rr, cuFP_t& ph, cuFP_t& xc,
 const dArray<cuIonElement> elems, int kindx)
{
  cuIonElement* elem = &elems._v[kindx];

  // value of h-bar * c in eV*nm
  //
  constexpr double hbc = 197.327;

  // Enforce minimum and maximum energies
  //
  if (E<ionEminGrid) E = ionEminGrid;
  if (E>ionEmaxGrid) E = ionEmaxGrid;

#if cuREAL == 4
  size_t indx = std::floor ( (E - ionEminGrid)/ionDeltaEGrid );
#else
  size_t indx = std::floor ( (E - ionEminGrid)/ionDeltaEGrid );
#endif
    
  if (indx >= ionEgridNumber - 1) indx = ionEgridNumber-2;

  double eA = ionEminGrid + ionDeltaEGrid*indx;
  double eB = ionEminGrid + ionDeltaEGrid*(indx+1);
  
  double A = (eB - E)/ionDeltaEGrid;
  double B = (E - eA)/ionDeltaEGrid;
  
  // Location in cumulative cross section grid
  //
  double rn = rr;
  double dC = 1.0/(CHCUMK-1);
  int lb    = rn/dC;
  cuFP_t k[4];

  // Interpolate the cross section array
  //
#if cuREAL == 4
  k[0]  = tex3D<float>(elem->ff_d, indx,   lb  , 0);
  k[1]  = tex3D<float>(elem->ff_d, indx+1, lb  , 0);
  k[2]  = tex3D<float>(elem->ff_d, indx,   lb+1, 0);
  k[3]  = tex3D<float>(elem->ff_d, indx+1, lb+1, 0);
#else
  k[0] = int2_as_double(tex3D<int2>(elem->ff_d, indx,   lb  , 0));
  k[1] = int2_as_double(tex3D<int2>(elem->ff_d, indx+1, lb  , 0));
  k[2] = int2_as_double(tex3D<int2>(elem->ff_d, indx,   lb+1, 0));
  k[3] = int2_as_double(tex3D<int2>(elem->ff_d, indx+1, lb+1, 0));
#endif
  
  // Linear interpolation
  //
  double a = (rn - dC*(lb+0)) / dC;
  double b = (dC*(lb+1) - rn) / dC;

  double K = A*(a*k[0] + b*k[2]) + B*(a*k[1] + b*k[3]);

  // Assign the photon energy
  //
  ph = pow(10, K) * hbc;

  // Use the integrated cross section from the differential grid
  //

  xc = 
#if cuREAL == 4
    A*tex1D<float>(elem->ff_0, indx  ) +
    B*tex1D<float>(elem->ff_0, indx+1) ;
#else
    A*int2_as_double(tex1D<int2>(elem->ff_0, indx  )) +
    B*int2_as_double(tex1D<int2>(elem->ff_0, indx+1)) ;
#endif
}


__global__
void testElasticE
(dArray<cuFP_t> energy,
 dArray<cuFP_t> xc,
 dArray<cuFP_t> xsc_H,
 dArray<cuFP_t> xsc_He,
 const dArray<cuIonElement> elems, int kindx)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {

    cuIonElement elem = elems._v[kindx];

    if (elem.Z==1 and elem.C==1) {
      xc._v[tid] = cudaElasticInterp(energy._v[tid], xsc_H, 1, cudaElasticType::electron);
    } else if (elem.Z==2 and elem.C==1) {
      xc._v[tid] = cudaElasticInterp(energy._v[tid], xsc_He, 2, cudaElasticType::electron);
    }
    else
      xc._v[tid] = 0.0;
  }

  __syncthreads();
}

__global__
void testFreeFree
(dArray<cuFP_t> energy,
 dArray<cuFP_t> randsl,
 dArray<cuFP_t> ph, dArray<cuFP_t> xc,
 const dArray<cuIonElement> elems, int kindx)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {
    computeFreeFree(energy._v[tid], randsl._v[tid], 
		    ph._v[tid], xc._v[tid], elems, kindx);
  }

  __syncthreads();
}


__device__
void computeColExcite
(cuFP_t E, cuFP_t& ph, cuFP_t& xc,
 const dArray<cuIonElement> elems, int k)
{
  cuIonElement* elem = &elems._v[k];

  if (E < elem->ceEmin or E > elem->ceEmax) {

    xc = 0.0;
    ph = 0.0;

  } else {

    // Interpolate the values
    //
#if cuREAL == 4
    int indx = std::floor ( (E - elem->ceEmin)/elem->ceDelE );
#else
    int indx = std::floor ( (E - elem->ceEmin)/elem->ceDelE );
#endif
    // Sanity check
    //
    if (indx > elem->NColl-2) indx = elem->NColl - 2;
    if (indx < 0)            indx = 0;
    
    double eA   = elem->ceEmin + elem->ceDelE*indx;
    double eB   = elem->ceEmin + elem->ceDelE*(indx+1);
    
    double A = (eB - E)/elem->ceDelE;
    double B = (E - eA)/elem->ceDelE;
    
#if cuREAL == 4
    xc = 
      A*tex3D<float>(elem->ce_d, indx,   0, 0) +
      B*tex3D<float>(elem->ce_d, indx+1, 0, 0) ;
    ph = 
      A*tex3D<float>(elem->ce_d, indx,   1, 0) +
      B*tex3D<float>(elem->ce_d, indx+1, 1, 0) ;
#else
    xc = 
      A*int2_as_double(tex3D<int2>(elem->ce_d, indx  , 0, 0)) +
      B*int2_as_double(tex3D<int2>(elem->ce_d, indx+1, 0, 0)) ;
    ph= 
      A*int2_as_double(tex3D<int2>(elem->ce_d, indx  , 1, 0)) +
      B*int2_as_double(tex3D<int2>(elem->ce_d, indx+1, 1, 0)) ;
#endif
  }
  // DONE
}

__global__ void testColExcite
(dArray<cuFP_t> energy,
 dArray<cuFP_t> ph, dArray<cuFP_t> xc,
 const dArray<cuIonElement> elems, int kindx)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {
    computeColExcite(energy._v[tid], ph._v[tid], xc._v[tid], elems, kindx);
  }

  __syncthreads();
}

__device__
void computeColIonize
(cuFP_t E, cuFP_t& xc, const dArray<cuIonElement> elems, int kindx)
{
  cuIonElement* elem = &elems._v[kindx];

  if (E < elem->ciEmin or E > elem->ciEmax) {

    xc = 0.0;

  } else {

    // Interpolate the values
    //
#if cuREAL == 4
    int indx = std::floor ( (E - elem->ciEmin)/elem->ciDelE );
#else
    int indx = std::floor ( (E - elem->ciEmin)/elem->ciDelE );
#endif
    // Sanity check
    //
    if (indx > elem->NIonz-2) indx = elem->NIonz - 2;
    if (indx < 0)             indx = 0;
    
    double eA   = elem->ciEmin + elem->ciDelE*indx;
    double eB   = elem->ciEmin + elem->ciDelE*(indx+1);
    
    double A = (eB - E)/elem->ciDelE;
    double B = (E - eA)/elem->ciDelE;
    
#if cuREAL == 4
    xc = 
      A*tex1D<float>(elem->ci_d, indx  ) +
      B*tex1D<float>(elem->ci_d, indx+1) ;
#else
    xc = 
      A*int2_as_double(tex1D<int2>(elem->ci_d, indx  )) +
      B*int2_as_double(tex1D<int2>(elem->ci_d, indx+1)) ;
#endif
  }
}


__device__
void computePhotoIonize
(cuFP_t rr, cuFP_t& ph, cuFP_t& xc,
 const dArray<cuIonElement> elems, int kindx)
{
  cuIonElement* elem = &elems._v[kindx];

  constexpr cuFP_t dC = 1.0/CHCUMK;
  int indx  = rr/dC;
  if (indx > CHCUMK-2) indx = CHCUMK - 2;

  // Linear interpolation
  //
  double a = (rr - dC*(indx+0)) / dC;
  double b = (dC*(indx+1) - rr) / dC;

  ph = a*elem->pi_0[indx+0] + b*elem->pi_0[indx+1];
  xc = elem->piTotl;
}


__global__ void testColIonize
(dArray<cuFP_t> energy, dArray<cuFP_t> ph, dArray<cuFP_t> xc,
 const dArray<cuIonElement> elems, int kindx)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {
    computeColIonize(energy._v[tid], xc._v[tid], elems, kindx);
    ph._v[tid] = elems._v[kindx].IPval;
  }

  __syncthreads();
}

__device__
void computeRadRecomb
(cuFP_t E, cuFP_t& xc, const dArray<cuIonElement> elems, int kindx)
{
  cuIonElement* elem = &elems._v[kindx];

  if (E < ionEminGrid or E > ionEmaxGrid) {

    xc = 0.0;

  } else {

    // Interpolate the values
    //
#if cuREAL == 4
    int indx = std::floor ( (E - ionEminGrid)/ionDeltaEGrid );
#else
    int indx = std::floor ( (E - ionEminGrid)/ionDeltaEGrid );
#endif
    // Sanity check
    //
    if (indx > ionRadRecombNumber-2) indx = ionRadRecombNumber - 2;
    if (indx < 0)                    indx = 0;
    
    double eA   = ionEminGrid + ionDeltaEGrid*indx;
    double eB   = ionEminGrid + ionDeltaEGrid*(indx+1);
    
    double A = (eB - E)/ionDeltaEGrid;
    double B = (E - eA)/ionDeltaEGrid;
    
#if cuREAL == 4
    xc = 
      A*tex1D<float>(elem->rc_d, indx  ) +
      B*tex1D<float>(elem->rc_d, indx+1) ;
#else
    xc = 
      A*int2_as_double(tex1D<int2>(elem->rc_d, indx  )) +
      B*int2_as_double(tex1D<int2>(elem->rc_d, indx+1)) ;
#endif
  }
  // DONE
}

__global__
void testRadRecomb
(dArray<cuFP_t> energy, dArray<cuFP_t> xc,
 const dArray<cuIonElement> elems, int kindx)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of evals
  //
  const unsigned int N = energy._s;

  if (tid < N) {
    computeRadRecomb(energy._v[tid], xc._v[tid], elems, kindx);
  }

  __syncthreads();
}


// Defined in atomicData for standalone version
void atomicData::testCross(int Nenergy) {}

// Defined in atomicData for production version
void atomicData::testCross(int Nenergy,
			   thrust::device_vector<cuIonElement> & cuElems,
			   thrust::device_vector<cuFP_t> & xsc_H,
			   thrust::device_vector<cuFP_t> & xsc_He)
{
  // Timers
  //
  Timer serial, cuda;

  // Initial header
  //
  std::string separator(10+(14+3)*8, '-');
  std::cout << separator << std::endl
	    << " Cross-section comparison for " << Nenergy << " samples"
	    << std::endl << separator << std::endl;

  // Loop over ions and tabulate statistics
  //
  size_t k = 0;

  thrust::host_vector<cuFP_t> energy_h(Nenergy), randsl_h(Nenergy);

  for (auto v : IonList) {

    IonPtr I = v.second;
    cuIonElement& E = cuIonElem[k];

    // Make an energy grid
    //
    double dE = (I->EmaxGrid - I->EminGrid)/(Nenergy-1) * 0.999;
    for (int i=0; i<Nenergy; i++) {
      energy_h[i] = I->EminGrid + dE*i;
      randsl_h[i] = static_cast<cuFP_t>(rand())/RAND_MAX;
    }

    thrust::device_vector<cuFP_t> energy_d = energy_h;
    thrust::device_vector<cuFP_t> randsl_d = randsl_h;

    // Only free-free for non-neutral species
    //
    thrust::device_vector<cuFP_t> eFF_d(Nenergy), xFF_d(Nenergy);
    thrust::device_vector<cuFP_t> eCE_d(Nenergy), xCE_d(Nenergy);
    thrust::device_vector<cuFP_t> eCI_d(Nenergy), xCI_d(Nenergy);
    thrust::device_vector<cuFP_t> xRC_d(Nenergy), xEE_d(Nenergy);

    unsigned int gridSize  = Nenergy/BLOCK_SIZE;
    if (Nenergy > gridSize*BLOCK_SIZE) gridSize++;

    cuda.start();

    testElasticE<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), toKernel(xEE_d),
					   toKernel(xsc_H), toKernel(xsc_He),
					   toKernel(cuElems), k);
    if (E.C>1)
      testFreeFree<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), toKernel(randsl_d),
					     toKernel(eFF_d), toKernel(xFF_d),
					     toKernel(cuElems), k);
    if (E.C<=E.Z)
      testColExcite<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), 
					      toKernel(eCE_d), toKernel(xCE_d),
					      toKernel(cuElems), k);

    if (E.C<=E.Z)
      testColIonize<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), 
					      toKernel(eCI_d), toKernel(xCI_d),
					      toKernel(cuElems), k);
      
    if (E.C>1)
      testRadRecomb<<<gridSize, BLOCK_SIZE>>>(toKernel(energy_d), 
					      toKernel(xRC_d),
					      toKernel(cuElems), k);
      
    thrust::host_vector<cuFP_t> xEE_h = xEE_d;
    thrust::host_vector<cuFP_t> eFF_h = eFF_d;
    thrust::host_vector<cuFP_t> xFF_h = xFF_d;
    thrust::host_vector<cuFP_t> eCE_h = eCE_d;
    thrust::host_vector<cuFP_t> xCE_h = xCE_d;
    thrust::host_vector<cuFP_t> eCI_h = eCI_d;
    thrust::host_vector<cuFP_t> xCI_h = xCI_d;
    thrust::host_vector<cuFP_t> xRC_h = xRC_d;
    
    cuda.stop();
    
    std::vector<double> xEE_0(Nenergy, 0);
    std::vector<double> eFF_0(Nenergy, 0), xFF_0(Nenergy, 0);
    std::vector<double> eCE_0(Nenergy, 0), xCE_0(Nenergy, 0);
    std::vector<double> eCI_0(Nenergy, 0), xCI_0(Nenergy, 0), xRC_0(Nenergy, 0);
    
    serial.start();
    
    const bool debug = false;

    Elastic elastic;

    for (int i=0; i<Nenergy; i++) {
				// Neutral-electron
      auto retEE = 0.0;
      if (E.C==1) retEE = elastic(E.Z, energy_h[i]);
      if (retEE>0.0) {
	xEE_0[i] = (xEE_h[i] - retEE)/retEE;
	/*
	std::cout << std::setw(16) << energy_h[i]
		  << std::setw(16) << xEE_h[i]/b_cross
		  << std::setw(16) << retEE/b_cross
		  << std::setw(4)  << E.Z
		  << std::setw(4)  << E.C
		  << std::endl;
	*/
      }

				// Free-free
      auto retFF = I->freeFreeCrossTest(energy_h[i], randsl_h[i], 0);
      if (retFF.first>0.0)
	xFF_0[i]   = (xFF_h[i] - retFF.first )/retFF.first;
      if (retFF.second>0.0)
	eFF_0[i]   = (eFF_h[i] - retFF.second)/retFF.second;

      if (debug and retFF.first>0.0)
	std::cout << std::setw(12) << "Free free"
		  << std::setw( 4) << E.Z
		  << std::setw( 4) << E.C
		  << std::setw(14) << energy_h[i]
		  << std::setw(14) << xFF_h[i]
		  << std::setw(14) << retFF.first
		  << std::endl;

				// Collisional excitation
      auto retCE = I->collExciteCross(energy_h[i], 0).back();
      if (retCE.first>0.0) {
	xCE_0[i]   = (xCE_h[i] - retCE.first )/retCE.first;
	if (debug)
	  std::cout << std::setw(12) << "Excite"
		    << std::setw( 4) << E.Z
		    << std::setw( 4) << E.C
		    << std::setw(14) << energy_h[i]
		    << std::setw(14) << xCE_h[i]
		    << std::setw(14) << retCE.first
		    << std::endl;
      }
				// Collisional ionization

      auto retCI = I->directIonCross(energy_h[i], 0);
      if (retCI>0.0) {
	xCI_0[i]   = (xCI_h[i] - retCI)/retCI;
	if (debug)
	  std::cout << std::setw(12) << "Ionize"
		    << std::setw( 4) << E.Z
		    << std::setw( 4) << E.C
		    << std::setw(14) << energy_h[i]
		    << std::setw(14) << xCI_h[i]
		    << std::setw(14) << retCI
		    << std::endl;
      }

				// Radiative recombination

      auto retRC = I->radRecombCross(energy_h[i], 0).back();
      if (retRC>0.0) {
	xRC_0[i]   = (xRC_h[i] - retRC)/retRC;
	if (debug)
	  std::cout << std::setw(12) << "Rad recomb"
		    << std::setw( 4) << E.Z
		    << std::setw( 4) << E.C
		    << std::setw(14) << energy_h[i]
		    << std::setw(14) << xRC_h[i]
		    << std::setw(14) << retRC
		    << std::endl;
      }

    }

    serial.stop();

    std::sort(xEE_0.begin(), xEE_0.end());
    std::sort(xFF_0.begin(), xFF_0.end());
    std::sort(eFF_0.begin(), eFF_0.end());
    std::sort(xCE_0.begin(), xCE_0.end());
    std::sort(eCE_0.begin(), eCE_0.end());
    std::sort(xCI_0.begin(), xCI_0.end());
    std::sort(eCI_0.begin(), eCI_0.end());
    std::sort(xRC_0.begin(), xRC_0.end());
    
    std::vector<double> quantiles = {0.01, 0.05, 0.1, 0.2, 0.5, 0.8, 0.9, 0.95, 0.99};

    std::cout << "Ion (" << I->Z << ", " << I->C << ")" << std::endl;

    std::cout << std::setw(10) << "Quantile"
	      << " | " << std::setw(14) << "ne xc"
	      << " | " << std::setw(14) << "ff xc"
	      << " | " << std::setw(14) << "ff ph"
	      << " | " << std::setw(14) << "CE xc"
	      << " | " << std::setw(14) << "CE ph"
	      << " | " << std::setw(14) << "CI_xc"
	      << " | " << std::setw(14) << "CI_ph"
	      << " | " << std::setw(14) << "RC_xc"
	      << std::endl << std::setfill('-')
	      <<          std::setw(10) << '-'
	      << " + " << std::setw(14) << '-'
	      << " + " << std::setw(14) << '-'
	      << " + " << std::setw(14) << '-'
	      << " + " << std::setw(14) << '-'
	      << " + " << std::setw(14) << '-'
	      << " + " << std::setw(14) << '-'
	      << " + " << std::setw(14) << '-'
	      << " + " << std::setw(14) << '-'
	      << std::endl << std::setfill(' ');

    for (auto v : quantiles) {
      int indx = std::min<int>(std::floor(v*Nenergy+0.5), Nenergy-1);
      double FF_xc = 0.0, FF_ph = 0.0, CE_xc = 0.0, CE_ph = 0.0;
      double CI_ph = 0.0, CI_xc = 0.0, RC_xc = 0.0, EE_xc = 0.0;
      
      EE_xc = xEE_0[indx];

      if (E.C>1) {
	FF_xc = xFF_0[indx];
	FF_ph = eFF_0[indx];
	RC_xc = xRC_0[indx];
      }

      if (E.C<=E.Z) {
	CE_xc = xCE_0[indx];
	CE_ph = eCE_0[indx];
	CI_xc = xCI_0[indx];
	CI_ph = eCI_0[indx];
      }

      std::cout << std::setw(10) << v
		<< " | " << std::setw(14) << EE_xc
		<< " | " << std::setw(14) << FF_xc
		<< " | " << std::setw(14) << FF_ph
		<< " | " << std::setw(14) << CE_xc
		<< " | " << std::setw(14) << CE_ph
		<< " | " << std::setw(14) << CI_xc
		<< " | " << std::setw(14) << CI_ph
		<< " | " << std::setw(14) << RC_xc
		<< std::endl;
    }

    k++;

  } // END: Ion list

  std::cout << separator << std::endl
	    << "Serial time: " << serial() << std::endl
	    << "Cuda time  : " << cuda()   << std::endl
	    << separator << std::endl;
}


enum cudaInterTypes { 
  nothing    = 0,
  neut_neut  = 1,
  neut_elec  = 2,
  neut_prot  = 3,
  ion_ion    = 4,
  ion_elec   = 5,
  free_free  = 6,
  col_excite = 7,
  col_ionize = 8,
  recombine  = 9,
  elec_elec  = 10
};

// This is only used for debugging
//
__constant__ char cudaInterNames[11][12] = { 
  "nothing",
  "neut_neut",
  "neut_elec",
  "neut_prot",
  "ion_ion",
  "ion_elec",
  "free_free",
  "col_excite",
  "col_ionize",
  "recombine",
  "elec_elec"
};

static std::string interLabels[] =
  {
    "Any type",			// 0
    "Neutral-neutral",		// 1
    "Neutral-electron",		// 2
    "Neutral-proton",		// 3
    "Ion-electron",		// 4
    "Ion-ion",			// 5
    "Free-free",		// 6
    "Collisional",		// 7
    "Ionization",		// 8
    "Recombination",		// 9
    "Electron-electron"		// 10
  };

// use_cons value
//
__constant__ int    cuSp0, cuCons, cuElec, cuEcon;

const int maxAtomicNumber = 15;
__constant__ cuFP_t cuda_atomic_weights[maxAtomicNumber], cuFloorEV;
__constant__ cuFP_t cuVunit, cuMunit, cuTunit, cuLunit, cuEunit;
__constant__ cuFP_t cuLogL, cuCrossfac, cuMinMass, cuEV;
__constant__ bool   cuNewRecombAlg, cuNoCool, cuRecombIP;
__constant__ bool   cuSpreadDef;

const int coulSelNumT = 2000;
__constant__ cuFP_t coulSelA[coulSelNumT];
__constant__ cuFP_t coulSelTau_i, coulSelTau_m, coulSelTau_f, coulSelTau_z, coulSelDel;

__global__
void testConstantsIon(int idev)
{
  printf("** -----------------------------------------\n");
  printf("** Ion constants [%d]\n", idev                 );
  printf("** -----------------------------------------\n");
  printf("** Spec posn  = %d\n",     cuSp0               );
  printf("** Cons posn  = %d\n",     cuCons              );
  printf("** Elec posn  = %d\n",     cuElec              );
  printf("** Econ posn  = %d\n",     cuEcon              );
  printf("** Lunit      = %13.6e\n", cuLunit             );
  printf("** Tunit      = %13.6e\n", cuTunit             );
  printf("** Vunit      = %13.6e\n", cuVunit             );
  printf("** Munit      = %13.6e\n", cuMunit             );
  printf("** Eunit      = %13.6e\n", cuEunit             );
  printf("** Egrid(min) = %13.6e\n", ionEminGrid         );
  printf("** Egrid(max) = %13.6e\n", ionEmaxGrid         );
  printf("** Egrid(del) = %13.6e\n", ionDeltaEGrid       );
  printf("** Egrid(num) = %d\n",     ionEgridNumber      );
  printf("** Rgrid(num) = %d\n",     ionRadRecombNumber  );
  printf("** log Lambda = %13.6e\n", cuLogL              );
  printf("** cross fac  = %13.6e\n", cuCrossfac          );

  if (cuRecombIP) 
    printf("** Rcmb IP    = true\n"                      );
  else
    printf("** Rcmb IP    = false\n"                     );
  if (cuNoCool) 
    printf("** No cool    = true\n"                      );
  else
    printf("** No cool    = false\n"                     );
  if (cuSpreadDef) 
    printf("** Spread def = true\n"                      );
  else
    printf("** Spread def = false\n"                     );
  printf("** -----------------------------------------\n");
}

// Coulombic algorithm initialization
//
double cuCA_f(double x)
{
  return 1.0/tanh(x) - 1.0/x;
}

double cuCA_deriv(double x)
{
  if (x>100.0) return 1.0/(x*x);
  double z = sinh(x);
  return 1.0/(x*x) - 1.0/(z*z);
}

double cuCA_func(cuFP_t tau, cuFP_t x)
{
  const double tol = 1.0e-12;
  const int itmax  = 1000;
  double fac       = exp(-2.0*tau), corr;
  
  for (int i=0; i<itmax; i++) {
    corr  = (cuCA_f(x) - fac)/cuCA_deriv(x);
    x    -= corr;
    if (fabs(corr)<tol) break;
  }
  
  return x;
}

// Initialization of counter array for debugging
//
#ifdef XC_DEEP9
__global__ void setCountersToZero()
{
  for (int T=0; T<11; T++) {
    w_countr[T] = 0;
    w_weight[T] = 0.0;
  }
}
#endif

// Link static parameters from CollideIon.cc (decide how to link these
// later)
//
static double FloorEv      = 0.05;
static bool   newRecombAlg = false;

static int Sp0 = 0, Sp0base = 0;

void CollideIon::cuda_initialize()
{
  static bool done = false;

  if (done) return;
  done = true;

  if (c0->cudaDevice>=0) {
    hipSetDevice(c0->cudaDevice);
  } else {
    std::cerr << "ERROR: c0->cudaDevice not set but CUDA requested"
	      << std::endl;
    MPI_Finalize();
    exit(33);
  }

  // Cache this: deviceProp is a CollideIon data member
  //
  hipGetDeviceProperties(&deviceProp, c0->cudaDevice);

  thrust::host_vector<cuIonElement> elems;

  int minSp = std::numeric_limits<int>::max();

  for (auto s : SpList) {
    speciesKey k = s.first;
    int Z = k.first;
    int C = k.second;

    // Scan
    bool found = false;

    for (auto & E : ad.cuIonElem) {
      if (E.Z == Z and E.C == C) {
	E.I   = s.second;
	minSp = std::min<int>(minSp, s.second);
	elems.push_back(E);
	found = true;
	break;
      }
    }

    if (not found) {
      std::cout << "CollideIon::cuda_initialize: [Z, C] = ["
		<< Z << ", " << C << "] not found" << std::endl;
    }
  }

  // This will be the base position of species fractions
  //
  for (auto & E : elems) E.I -= minSp;
  Sp0base  = minSp;

  int spc0val = minSp;
  Sp0 = minSp;

  if (use_cons>=0) minSp = std::min<int>(minSp, use_cons);
  if (use_elec>=0) minSp = std::min<int>(minSp, use_elec);

  spc0val -= minSp;
  Sp0      = spc0val;

  int consval = use_cons;
  int elecval = use_elec;
  int elccons = -1;

  if (use_cons) consval = use_cons - minSp;
  if (use_elec) elecval = use_elec - minSp;
  if (elc_cons) elccons = elecval + 4;

  cuElems = elems;

  // Coulombic velocity selection
  //
  cuFP_t tau_i = 0.0001, tau_m = 1.0e-8, tau_f = 4.0, tau_z = 40.0;
  std::vector<cuFP_t> hA(coulSelNumT);

  cuFP_t del = (log(tau_f) - log(tau_i))/(coulSelNumT-1);
  cuFP_t A   = 0.5/tau_i, T;
  for (int i=0; i<coulSelNumT; i++) {
    T = tau_i*exp(del*i);
    hA[i] = A = cuCA_func(T, A);
  }

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuSp0), &spc0val, sizeof(int)), 
		 __FILE__, __LINE__, "Error copying cuSp0");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuCons), &consval, sizeof(int)), 
		 __FILE__, __LINE__, "Error copying cuCons");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuElec), &elecval, sizeof(int)), 
		 __FILE__, __LINE__, "Error copying cuElec");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuEcon), &elccons, sizeof(int)), 
		 __FILE__, __LINE__, "Error copying cuEcon");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(coulSelTau_i), &tau_i, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying coulSelTau_i");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(coulSelTau_m), &tau_m, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying coulSelTau_m");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(coulSelTau_f), &tau_f, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying coulSelTau_f");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(coulSelTau_z), &tau_z, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying coulSelTau_z");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(coulSelDel), &del, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying coulSelDel");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(coulSelA), &hA[0], coulSelNumT*sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying coulSelA");

  cuda_atomic_weights_init();

  // For debugging only
  //
  if (myid==0)
    testConstantsIon<<<1, 1>>>(c0->cudaDevice);

#if HAVE_LIBCUDA==1
    if (use_cuda and aType == Trace)
      cuda_part_stats_print();
    else
#endif

#ifdef XC_DEEP9
  setCountersToZero<<<1, 1>>>();
#endif

  // For collision stats
  //
  cuda_part_stats_initialize();
}


void CollideIon::cuda_atomic_weights_init()
{
  cudaElasticInit();

  std::vector<cuFP_t> weights(maxAtomicNumber);

  weights[0]  = 0.000548579909; // Mass of electron
  weights[1]  = 1.0079;	       // Hydrogen
  weights[2]  = 4.0026;	       // Helium
  weights[3]  = 6.941;	       // Lithum
  weights[4]  = 9.0122;	       // Beryllium
  weights[5]  = 10.811;	       // Boron
  weights[6]  = 12.011;	       // Carbon
  weights[7]  = 14.007;	       // Nitrogen
  weights[8]  = 15.999;	       // Oxygen
  weights[9]  = 18.998;	       // Florine
  weights[10] = 20.180;	       // Neon
  weights[11] = 22.990;	       // Sodium
  weights[12] = 24.305;	       // Magnesium
  weights[13] = 26.982;	       // Aluminium
  weights[14] = 28.085;	       // Silicon

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuda_atomic_weights), &weights[0], sizeof(cuFP_t)*maxAtomicNumber), 
		 __FILE__, __LINE__, "Error copying cuda_atomic_weights");

  cuFP_t v = TreeDSMC::Vunit;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuVunit), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuVunit");
  v = TreeDSMC::Lunit;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuLunit), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuLunit");
  v = TreeDSMC::Munit;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuMunit), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuMunit");
  v = TreeDSMC::Tunit;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuTunit), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuTunit");
  v = TreeDSMC::Eunit;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuEunit), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuEunit");
  v = eV;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuEV), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuEV");
  v = logL;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuLogL), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuLogL");
  v = crossfac;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuCrossfac), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuCrossfac");
  v = FloorEv;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuFloorEV), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuFloorEV");
  v = esu;
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuEsu), &v, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuEsu");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuSpreadDef), &SpreadDef, sizeof(bool)), 
		 __FILE__, __LINE__, "Error copying cuSpreadDef");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuNewRecombAlg), &newRecombAlg, sizeof(bool)), 
		 __FILE__, __LINE__, "Error copying cuNewRecombAlg");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuNoCool), &NOCOOL, sizeof(bool)), 
		 __FILE__, __LINE__, "Error copying cuNoCool");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuRecombIP), &Recomb_IP, sizeof(bool)), 
		 __FILE__, __LINE__, "Error copying cuRecombIP");

  cuFP_t minMass = 1.0e-12;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuMinMass), &minMass, sizeof(cuFP_t)), 
		 __FILE__, __LINE__, "Error copying cuMinMass");
}  


// CURAND initialization
//
__global__ void initCurand(dArray<hiprandState> state,
			   int offset, int count, unsigned long long seed)
{
  for (int c = blockIdx.x * blockDim.x + threadIdx.x; 
       c < count; 
       c += blockDim.x * gridDim.x) {

    unsigned long long sq = c + offset;
    unsigned long long of = 0ll;

    hiprand_init(seed, sq, of, &state._v[c+offset]);
  }
}

// This is a port of the cell initialization for the scalar version
//
__global__ void cellInitKernel(dArray<cudaParticle> in,    // Particles (all active)
			       dArray<cuFP_t> Ivel2,       // Mean-squared ion velocity (per cell)
			       dArray<cuFP_t> Evel2,       // Mean-squared electron velocity (per cell)
			       dArray<cuFP_t> PiProb,      // Relative electron fraction for BN algorithm (per cell)
			       dArray<cuFP_t> ABrate,      // Plasma rate for BN algorithm (per cell)
			       dArray<cuFP_t> volC,        // Cell's volume
			       dArray<cuFP_t> tauC,        // Cell's time step
			       dArray<int>    cellI,       // Index to beginning of bodies for this cell
			       dArray<int>    cellN,	   // Number of bodies per cell
			       const
			       dArray<cuIonElement> elems) // Species array
{
  const cuFP_t dfac = cuMunit/amu / (cuLunit*cuLunit*cuLunit);

  if (cuEcon >= DATTRIB_CUDA) {
    printf("cellInit: econs OAB, econs=%d/%d\n", cuEcon, DATTRIB_CUDA);
  }

  if (cuElec+3>=DATTRIB_CUDA) {
    printf("cellInit: epos OAB, epos+3=%d/%d\n", cuElec+3, DATTRIB_CUDA);
  }

  for (int c = blockIdx.x * blockDim.x + threadIdx.x; 
       c < cellI._s; 
       c += blockDim.x * gridDim.x) {

    // Number of particles in _this_ cell
    //
    int nbods = cellN._v[c];

    // NB: cellI is the offset into the body list for _this_ cell

    cuFP_t massP = 0.0, numbP = 0.0, massE = 0.0;
    cuFP_t evel2 = 0.0, ivel2 = 0.0, numQ2 = 0.0;
    cuFP_t meanM = 0.0, densQ = 0.0, densE = 0.0;

    cuFP_t crm1[3], crm2[3];
    for (int k=0; k<3; k++) crm1[k] = crm2[k] = 0.0;

    for (size_t i=0; i<nbods; i++) {
	
      // Sanity check
      //
      if (i + cellI._v[c] >= in._s) {
	printf("Cell init: Wanted %lu/%lu\n", i + cellI._v[c], in._s);
      }
      
      // The particle
      cudaParticle* p = &in._v[i + cellI._v[c]];
      
      // Mass per cell
      massP += p->mass;
      
      // Mass-weighted trace fraction
      // [Iterate through all Trace ionization states]
      cuFP_t ee = 0.0;

      for (int k=0; k<elems._s; k++) {
	cuIonElement* E = &elems._v[k];
	
	cuFP_t ff       = p->datr[E->I + cuSp0];
	cuFP_t ww       = ff/cuda_atomic_weights[E->Z];
	cuFP_t qq       = E->C - 1;
	
	// Mean number
	numbP += p->mass * ww;
	
	// Electron fraction
	ee += ww * qq;
	
	// Charge
	densE += p->mass * ww * qq;
	
	// Charge squared
	numQ2 += p->mass * ww * qq*qq;
	if (E->C>1) densQ += p->mass * ww;
      }
	
      cuFP_t eVel2 = 0.0, iVel2 = 0.0;
      for (int l=0; l<3; l++) {
	cuFP_t ve  = p->datr[cuElec+l];
#ifdef SANITY_DEBUG
	if (::isnan(ve)) {
	  printf("Weird electron\n");
	}
#endif
	eVel2 += ve*ve;
	cuFP_t vi  = p->vel[l];
	iVel2 += vi*vi;

	crm1[l] += p->mass * vi;
	crm2[l] += p->mass * vi*vi;
      }
	
      evel2 += p->mass * ee * eVel2;
      ivel2 += p->mass * iVel2;
      massE += p->mass * ee;
    }
  
    if (numbP>0.0) meanM       = massP/numbP;
    if (massP>0.0) Ivel2._v[c] = ivel2/massP;
    if (massP>0.0) Evel2._v[c] = evel2/massP;
    if (densQ>0.0) numQ2      /= densQ;
      
    cuFP_t ddfac = dfac/volC._v[c];

#ifdef XC_DEEP7
    cuFP_t tmp1 = densQ, tmp2 = densE;
#endif
    densQ *= ddfac;
    densE *= ddfac;

    // Compute per channel Coulombic probabilities
    //
    // Ion probabilities
    //
    cuFP_t muii = meanM/2.0;
    cuFP_t muee = cuda_atomic_weights[0]/2.0;
    cuFP_t muie = cuda_atomic_weights[0] * meanM/(cuda_atomic_weights[0] + meanM);
    
#ifdef XC_DEEP7
    printf("coulombic: MUii=%e MUee=%e MUie=%e denQ=%e denE=%e numQ=%e Ivel2=%e Evel2=%e volC=%e masC=%e nbod=%d numQ=%e numE=%e\n", muii, muee, muie, densQ, densE, numQ2, Ivel2._v[c], Evel2._v[c], volC._v[c], massP, nbods, tmp1, tmp2);
#endif
    
    // Ion-Ion
    PiProb._v[c*4 + 0] =
      densQ +
      densE * pow(2.0*Ivel2._v[c], 1.5) * muii*muii /
      (pow(Ivel2._v[c] + Evel2._v[c], 1.5) * muie*muie * numQ2);
    //                                               ^
    //                                               |
    // The density is weighted by q^2 for each species
    
    // Ion-Electron
    PiProb._v[c*4 + 1] =
      densQ * pow(Ivel2._v[c] + Evel2._v[c], 1.5) * muie*muie * numQ2 /
      (pow(2.0*Ivel2._v[c], 1.5) * muii*muii) +
      densE ;
    
    // Electron-Ion
    PiProb._v[c*4 + 2] =
      densQ +
      densE * pow(Ivel2._v[c] + Evel2._v[c], 1.5) * muie*muie /
      (pow(2.0*Evel2._v[c], 1.5) * muee*muee * numQ2);
    
    // Electron-Electron
    PiProb._v[c*4 + 3] =
      densQ * pow(2.0*Evel2._v[c], 1.5) * muee*muee * numQ2 /
      (pow(Ivel2._v[c] + Evel2._v[c], 1.5) * muie*muie) +
      densE;
      
#ifdef XC_DEEP10
      printf("coul2: PP1=%e PP2=%e PP3=%e PP4=%e\n", PiProb._v[c*4+0], PiProb._v[c*4+1], PiProb._v[c*4+2], PiProb._v[c*4+3]);
#endif

    // Rate coefficients
    ABrate._v[c*4 + 0] = 2.0*M_PI * PiProb._v[c*4 + 0] * cuLogL * pow(numQ2*numQ2, 2.0);
      
    ABrate._v[c*4 + 1] = 2.0*M_PI * PiProb._v[c*4 + 1] * cuLogL * pow(numQ2, 2.0);
      
    ABrate._v[c*4 + 2] = 2.0*M_PI * PiProb._v[c*4 + 2] * cuLogL * pow(numQ2, 2.0);
      
    ABrate._v[c*4 + 3] = 2.0*M_PI * PiProb._v[c*4 + 3] * cuLogL ;

#ifdef XC_DEEP10
      printf("coul2: AB1=%e AB2=%e AB3=%e AB4=%e\n", ABrate._v[c*4+0], ABrate._v[c*4+1], ABrate._v[c*4+2], ABrate._v[c*4+3]);
#endif

  } // END: cell

}


// To ferry pair particle info . . . 
//
struct cuEnergyInfo
{
  cuFP_t Eta1, Eta2, Mu1, Mu2;
  cuFP_t kEi, kEe1, kEe2, kE1s, kE2s;
  cuFP_t vel, eVel1, eVel2, sVel1, sVel2;
  cuFP_t iE1, iE2, Sum1, Sum2;
};

// STL container pretty-print for std::array
//
__device__
void printEI(cuFP_t xc, cuEnergyInfo& E)
{
  printf("[xc=%e Eta1=%f Eta2=%f Mu1=%f Mu2=%f kEi=%e kEe1=%e kEe2=%e iE1=%e iE2=%e]\n",
	 xc, E.Eta1, E.Eta2, E.Mu1, E.Mu2, E.kEi, E.kEe1, E.kEe2, E.iE1, E.iE2);
}


// Computes the cross section for a pair of particles
//
__device__
void setupCrossSection(dArray<cudaParticle>   in,      // Particle array
		       dArray<cuIonElement>   elems,   // Info for all trace species
		       int                    C,       // Cell index
		       int                    I1,      // Index of Particle 1
		       int                    I2,      // Index of Particle 2
		       hiprandState*           state,   // Random number generator
		       cuEnergyInfo*          Einfo    // Return computed particle info
		       )
{
  const int Nsp = elems._s;

  // Sanity checks
  //
  if (I1 >= in._s) {
    printf("cross section: i1 wanted %d/%lu\n", I1, in._s);
  }

  if (I2 >= in._s) {
    printf("cross section: i2 wanted %d/%lu\n", I2, in._s);
  }

  // Pointer to particle structure for convenience
  //
  cudaParticle* p1 = &in._v[I1];
  cudaParticle* p2 = &in._v[I2];
	
  // Superparticle stats
  //
  cuFP_t Eta1=0.0, Eta2=0.0, Mu1=0.0, Mu2=0.0, Sum1=0.0, Sum2=0.0;
	
  for (int k=0; k<Nsp; k++) {

    cuIonElement* E = &elems._v[k];
	  
    // Number fraction of ions
    //
    cuFP_t one = p1->datr[E->I+cuSp0] / cuda_atomic_weights[E->Z];
    cuFP_t two = p2->datr[E->I+cuSp0] / cuda_atomic_weights[E->Z];
	  
    // Electron number fraction
    //
    Eta1 += one * (E->C - 1);
    Eta2 += two * (E->C - 1);
	  
    Sum1 += one;
    Sum2 += two;
  }

  // The number of electrons per particle
  //
  Eta1 /= Sum1;
  Eta2 /= Sum2;

  // The molecular weight
  //
  Mu1 = 1.0/Sum1;
  Mu2 = 1.0/Sum2;
	
  // Velocity and KE quantities
  //
  cuFP_t vel   = 0.0;
  cuFP_t eVel0 = 0.0, eVel1 = 0.0, eVel2 = 0.0;
  cuFP_t sVel1 = 0.0, sVel2 = 0.0;
  cuFP_t eKE1  = 0.0, eKE2  = 0.0;
  
  for (int k=0; k<3; k++) {
    cuFP_t del = p1->vel[k] - p2->vel[k];
    vel += del*del;
    
    cuFP_t rvel0 = p1->datr[cuElec+k] - p2->datr[cuElec+k];
    cuFP_t rvel1 = p1->datr[cuElec+k] - p2->vel[k];
    cuFP_t rvel2 = p2->datr[cuElec+k] - p1->vel[k];
    
    eVel0 += rvel0*rvel0;
    eVel1 += rvel1*rvel1;
    eVel2 += rvel2*rvel2;
	  
    rvel1 = p1->datr[cuElec+k] - p1->vel[k];
    rvel2 = p2->datr[cuElec+k] - p2->vel[k];
	  
    sVel1 += rvel1*rvel1;
    sVel2 += rvel2*rvel2;
	  
    eKE1 += p1->datr[cuElec+k] * p1->datr[cuElec+k];
    eKE2 += p2->datr[cuElec+k] * p2->datr[cuElec+k];
  }
	
  // Energy available in the center of mass of the atomic collision
  //
  vel   = (sqrt(vel) + 1.0e-32) * cuVunit;
  //                   ^
  //                   |
  // Prevent divide by zero in unusual circumstances

  eVel0 = sqrt(eVel0) * cuVunit / vel;
  eVel1 = sqrt(eVel1) * cuVunit / vel;
  eVel2 = sqrt(eVel2) * cuVunit / vel;
  sVel1 = sqrt(sVel1) * cuVunit / vel;
  sVel2 = sqrt(sVel2) * cuVunit / vel;
	
  cuFP_t  m1  = Mu1 * amu;
  cuFP_t  m2  = Mu2 * amu;
  cuFP_t  me  = cuda_atomic_weights[0] * amu;

  cuFP_t mu0  = m1 * m2 / (m1 + m2);
  cuFP_t mu1  = m1 * me / (m1 + me);
  cuFP_t mu2  = m2 * me / (m2 + me);
  
  // Available COM energy

  cuFP_t kEi  = 0.5  * mu0 * vel * vel / eV;
  cuFP_t kEe1 = 0.5  * mu1 * eVel2*eVel2 * vel*vel / eV;
  cuFP_t kEe2 = 0.5  * mu2 * eVel1*eVel1 * vel*vel / eV;
  cuFP_t kE1s = 0.5  * mu1 * sVel1*sVel1 * vel*vel / eV;
  cuFP_t kE2s = 0.5  * mu2 * sVel2*sVel2 * vel*vel / eV;
	
  // Assign energy info for return
  //
  if (Einfo != 0x0) {
    Einfo->Eta1  = Eta1;
    Einfo->Eta2  = Eta2;

    Einfo->Sum1  = Sum1;
    Einfo->Sum2  = Sum2;

    Einfo->Mu1   = Mu1;
    Einfo->Mu2   = Mu2;

    Einfo->vel   = vel/cuVunit;
    Einfo->eVel1 = eVel1;
    Einfo->eVel2 = eVel2;

    Einfo->sVel1 = sVel1;
    Einfo->sVel2 = sVel2;

    Einfo->kEi   = kEi;
    Einfo->kEe1  = kEe1;
    Einfo->kEe2  = kEe2;
    Einfo->kE1s  = kE1s;
    Einfo->kE2s  = kE2s;

    cuFP_t kfac  = 0.5 * cuda_atomic_weights[0] * cuVunit*cuVunit*amu/eV;
    Einfo->iE1   = eKE1 * kfac;
    Einfo->iE2   = eKE2 * kfac;

#ifdef XC_DEEP13
    printf("ETEST: eVel1=%e eVel2=%e ke1=%e ke2=%e\n", eVel1, eVel2, kEe1, kEe2);
#endif
  }
}

// Computes the cross section for a pair of particles
//
__device__
cuFP_t singleCrossSection(dArray<cudaParticle>   in,      // Particle array
			  dArray<cuIonElement>   elems,   // Info for all trace species
			  cuFP_t*                delph,   // Inelastic energy change
			  dArray<cuFP_t>         xsc_H,	  // Cross section arrays
			  dArray<cuFP_t>         xsc_He,  // ..
			  dArray<cuFP_t>         xsc_pH,  // ..
			  dArray<cuFP_t>         xsc_pHe, // ..
			  int                    C,       // Cell index
			  int                    I1,      // Index of Particle 1
			  int                    I2,      // Index of Particle 2
			  int                    T,	  // Interaction type
			  cuSpeciesDef*          J1,      // Particle state info
			  cuSpeciesDef*          J2,	  // ..
			  hiprandState*           state,   // Random number generator
			  cuEnergyInfo*          Einfo    // Return computed particle info
			  )
{
  // For convenience in checking species
  //
  const cuSpeciesKey cuProton    {1, 2};
  const cuSpeciesKey cuElectron  {0xffff, 0xffff};
  //                              ^       ^
  //                              |       |
  // 2^16-1 is max ushort---------+-------+
  // as defined in NTC.H

  // Zero return energy by default
  //
  *delph = 0.0;

  // Species info
  //
  auto Z1 = J1->sp.first;
  auto C1 = J1->sp.second;
  auto P1 = C1 - 1;

  auto Z2 = J2->sp.first;
  auto C2 = J2->sp.second;
  auto P2 = C2 - 1;

  //-------------------------------
  // *** Both particles neutral
  //-------------------------------
  if (T == neut_neut) {

    // Geometric cross sections based on atomic radius
    //
    double rad = cudaGeometric(Z1) + cudaGeometric(Z2);
    cuFP_t crs = M_PI*rad*rad * cuCrossfac;
	      
    if (crs>0.0) {
#ifdef XC_DEEP1
      printf("xsc: xnn=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
      printf("xsc: (Z, P)=(%d, %d) xnn=%e\n", Z1, P1, crs);
#endif
      // Double counting
      if (Z1 == Z2) crs *= 0.5;
    }

    return crs;
  }

  // --------------------------------------
  // *** Neutral atom-proton scattering
  // --------------------------------------

  if (T == neut_prot) {
	    
    cuFP_t crs = 0;
	    
    // Particle 2 is proton
    //
    if (J2->sp == cuProton) {
	
      // Particle 1 is neutral hydrogen
      if (Z1==1 and P1==0) {
	crs = cudaElasticInterp(Einfo->kEi, xsc_pH, 1, proton) * cuCrossfac;
#ifdef XC_DEEP12
	printf("H TEST: xnn=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
      }
      
      // Particle 1 is neutral helium
      if (Z1==2 and P1==0) {
	crs = cudaElasticInterp(Einfo->kEi, xsc_pHe, 2, proton) * cuCrossfac;
#ifdef XC_DEEP12
	printf("He TEST: xnn=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
      }	

      if (crs>0.0) {
#ifdef XC_DEEP1
	printf("xsc: xnp=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: kEi=%e (Z, P)=(%d, %d) xnp=%e\n", Einfo->kEi, Z1, C1, crs);
#endif
      }
    }
	    
    // Particle 1 is proton
    //
    if (J1->sp == cuProton) {
	
      // Particle 2 is neutral hydrogen
      if (Z2==1 and P2==0)
	crs = cudaElasticInterp(Einfo->kEi, xsc_pH, 1, proton) * cuCrossfac;

      // Particle 2 is neutral helium
      if (Z2==2 and P2==0)
	crs = cudaElasticInterp(Einfo->kEi, xsc_pHe, 2, proton) * cuCrossfac;

      if (crs>0.0) {
#ifdef XC_DEEP1
	printf("xsc: xnp=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: kEi=%e (Z, P)=(%d, %d) xnp=%e\n", Einfo->kEi, Z2, C2, crs);
#endif
      }
    }

    return crs;
  }
  // END: neut_prot

  // --------------------------------------
  // *** Neutral atom-electron scattering
  // --------------------------------------
  
  if (T == neut_elec) {

    cuFP_t crs = 0;

    // Particle 1 ION, Particle 2 ELECTRON
    //
    if (Z1<=2 and P1==0 and J2->sp==cuElectron) {
    
      // Hydrogen
      //
      if (Z1==1)
	crs = cudaElasticInterp(Einfo->kEe1, xsc_H,  1, electron) * Einfo->eVel2 * cuCrossfac;
      
      // Helium
      //
      if (Z1==2)
	crs = cudaElasticInterp(Einfo->kEe1, xsc_He, 2, electron) * Einfo->eVel2 * cuCrossfac;
      
      if (crs>0.0) {
#ifdef XC_DEEP1
	printf("xsc: xne=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e xne=%e [He 1]\n",
	       Einfo->kEe1, Z1, C1, Einfo->eVel2, Einfo->Eta2, crs);
#endif
      }
    }
    
	  
    // Particle 2 ION, Particle 1 ELECTRON
    //
    if (Z2<=2 and P2==0 and J1->sp==cuElectron) {

      // Hydrogen
      //
      if (Z2==1)
	crs = cudaElasticInterp(Einfo->kEe2, xsc_H, 1, electron) * Einfo->eVel1 * cuCrossfac;
      
      // Helium
      //
      if (Z2==2)
	crs = cudaElasticInterp(Einfo->kEe2, xsc_He, 2, electron) * Einfo->eVel1 * cuCrossfac;
	    
      if (crs>0.0) {
#ifdef XC_DEEP1
	printf("xsc: xne=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e xne=%e [He 2]\n",
	       Einfo->kEe2, Z2, C2, Einfo->eVel1, Einfo->Eta1, crs);
#endif
      }
    }

    return crs;
  }

	  
  //-------------------------------
  // *** Free-free
  //-------------------------------
  if (T == free_free) {
  
    cuFP_t crs = 0, rn;

    // Particle 1 ION, Particle 2 ELECTRON
    //
    if (C1>1 and J2->sp==cuElectron) {
      cuFP_t ke = Einfo->kEe1 > cuFloorEV ? Einfo->kEe1 : cuFloorEV, ff, ph;
      cuFP_t rn;
#if cuREAL == 4
      rn = hiprand_uniform(state);
#else
      rn = hiprand_uniform_double(state);
#endif
      computeFreeFree(ke, rn, ph, ff, elems, J1->k);
	    
      crs  = Einfo->eVel2 * ff;
	    
      if (crs>0.0) {
	*delph = ph;
	      
#ifdef XC_DEEP1
	printf("xsc: xf=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e xf=%e dE=%e\n",
	       Einfo->kEe1, Z1, C1, Einfo->eVel2, Einfo->Eta2, ff, ph);
#endif
      }
    }
	    
    // Particle 2 ION, Particle 1 ELECTRON
    if (C2>1 and J1->sp==cuElectron) {

      cuFP_t ke = Einfo->kEe2 > cuFloorEV ? Einfo->kEe2 : cuFloorEV;
      
#if cuREAL == 4
      rn = hiprand_uniform(state);
#else
      rn = hiprand_uniform_double(state);
#endif
      cuFP_t ff, ph;
      computeFreeFree(ke, rn, ph, ff, elems, J2->k);
	    
      crs = Einfo->eVel1 * ff;
	    
      if (crs>0.0) {
	*delph = ph;
#ifdef XC_DEEP1
	printf("xsc: xf=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e xf=%e dE=%e\n",
	       Einfo->kEe2, Z1, C2, Einfo->eVel1, Einfo->Eta1, ff, ph);
#endif
      }
    }

    return crs;
  }
  // END: free-free 
	  
  //-------------------------------
  // *** Collisional excitation
  //-------------------------------

  if (T == col_excite) {
    
    cuFP_t crs = 0;

    // Particle 1 nucleus has BOUND ELECTRON, Particle 2 has FREE ELECTRON
    //
    //  +--- Charge of the current subspecies
    //  |
    //  |         +--- Partner is an electron
    //  |         |
    //  V         V
    if (P1<Z1 and J2->sp==cuElectron) {
      cuFP_t ke = Einfo->kEe1 > cuFloorEV ? Einfo->kEe1 : cuFloorEV, ph, xc;

      computeColExcite(ke, ph, xc, elems, J1->k);
	    
      crs = Einfo->eVel2 * xc;
	    
      if (crs > 0.0) {
	*delph = ph;
#ifdef XC_DEEP1
	printf("xsc: xc=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e xc=%e dE=%e\n",
	       ke, Z1, C1, Einfo->eVel2, Einfo->Eta2, xc, ph);
#endif
      }
    }
	  
	  
    // Particle 2 nucleus has BOUND ELECTRON, Particle 1 has FREE ELECTRON
    //
    //  +--- Charge of the current subspecies
    //  |
    //  |         +--- Partner is an electron
    //  |         |
    //  V         V
    if (P2<Z2 and J1->sp==cuElectron) {
	    
      cuFP_t ke = Einfo->kEe2 > cuFloorEV ? Einfo->kEe2 : cuFloorEV, ph, xc;

      computeColExcite(ke, ph, xc, elems, J2->k);
	    
      crs = Einfo->eVel1 * xc;
	    
      if (crs > 0.0) {
	*delph = ph;
#ifdef XC_DEEP1
	printf("xsc: xc=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e xc=%e dE=%e\n",
	       ke, Z2, C2, Einfo->eVel2, Einfo->Eta2, xc, ph);
#endif
      }
    }

    return crs;
  }
  // END: colexcite

  //-------------------------------
  // *** Ionization cross section
  //-------------------------------
  
  if (T == col_ionize) {

    cuFP_t crs = 0;

    // Particle 1 nucleus has BOUND ELECTRON, Particle 2 has FREE ELECTRON
    //
    //  +--- Charge of the current subspecies, must have a bound electron
    //  |
    //  |         +--- Partner is an electron
    //  |         |
    //  V         V
    if (P1<Z1 and J2->sp==cuElectron) {
      
      cuFP_t ke = Einfo->kEe1 > cuFloorEV ? Einfo->kEe1 : cuFloorEV, xc;

      computeColIonize(ke, xc, elems, J1->k);
	    
      crs = Einfo->eVel2 * xc;
      
      if (crs > 0.0) {
	*delph = elems._v[J1->k].IPval;
#ifdef XC_DEEP1
	printf("xsc: [ie] io=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: [ie] kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e io=%e dE=%e\n",
	       Einfo->kEe1, Z1, C1, Einfo->eVel2, Einfo->Eta2, xc, 0.0);
#endif
      }
    }
	  
    // Particle 2 nucleus has BOUND ELECTRON, Particle 1 has FREE ELECTRON
    //
    //  +--- Charge of the current subspecies, must have a bound electron
    //  |
    //  |         +--- Partner is an electron
    //  |         |
    //  V         V
    if (P2<Z2 and J1->sp==cuElectron) {
	    
      cuFP_t ke = Einfo->kEe2 > cuFloorEV ? Einfo->kEe2 : cuFloorEV, xc;
      computeColIonize(ke, xc, elems, J2->k);
	    
      crs = Einfo->eVel1 * xc;
      
      if (crs > 0.0) {
	*delph = elems._v[J2->k].IPval;
#ifdef XC_DEEP1
	printf("xsc: [ei] io=%e cv=%e\n", crs, crs*Einfo->vel);
#endif
#ifdef XC_DEEP4
	printf("xsc: [ei] kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e io=%e dE=%e\n",
	       Einfo->kEe2, Z2, C2, Einfo->eVel1, Einfo->Eta1, xc, 0.0);
#endif
      }
    }
      
    return crs;
  }
	  
  //-------------------------------
  // *** Radiative recombination
  //-------------------------------
	  
  if (T == recombine) {

    cuFP_t crs = 0;

    // The "new" algorithm uses the electron energy of the ion's
    // electron rather than the standard particle partner.
    //
    if (cuNewRecombAlg) {

      // Particle 1 is ION, Particle 2 has ELECTRON
      //
      //  +--- Ion charge, must not be neutral
      //  |
      //  |        +--- Partner is an electron
      //  |        |
      //  v        v
      if (P1>0 and J2->sp==cuElectron) {
	      
	cuFP_t ke = Einfo->kE1s > cuFloorEV ? Einfo->kE1s : cuFloorEV, xc;
	computeRadRecomb(ke, xc, elems, J1->k);
	  
	crs = Einfo->sVel1 * xc;
      }
	    
      // Particle 2 is ION, Particle 1 has ELECTRON
      //
      //  +--- Ion charge, must not be neutral
      //  |
      //  |        +--- Partner is an electron
      //  |        |
      //  v        v
      if (P2>0 and J1->sp==cuElectron) {
	
	cuFP_t ke = Einfo->kE2s > cuFloorEV ? Einfo->kE2s : cuFloorEV, xc;
	computeRadRecomb(ke, xc, elems, J2->k);
	
	crs = Einfo->sVel2 * xc;
      }

    }
    // END: new recomb algorithm
    else {
      // Particle 1 is ION, Particle 2 has ELECTRON
      //
      //  +--- Charge of the current subspecies, not neutral
      //  |
      //  |        +--- Partner is an electron
      //  |        |
      //  V        V
      if (P1>0 and J2->sp==cuElectron) {
	
	cuFP_t ke = Einfo->kEe1 > cuFloorEV ? Einfo->kEe1 : cuFloorEV, xc;
	computeRadRecomb(ke, xc, elems, J1->k);
	      
	crs = Einfo->eVel2 * xc;

	if (crs > 0.0) {
#ifdef XC_DEEP1
	  printf("xsc: [ie] rc=%e cv=%e eta1=%e eta2=%e\n", crs, crs*Einfo->vel, Einfo->Eta1, Einfo->Eta2);
#endif
#ifdef XC_DEEP4
	  printf("xsc: [ie] kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e rc=%e dE=%e\n",
		 Einfo->kEe1, Z1, C1, Einfo->eVel2, Einfo->Eta2, xc, 0.0);
#endif
	}
      }
	    
      // Particle 2 is ION, Particle 1 has ELECTRON
      //
      //  +--- Charge of the current subspecies, not neutral
      //  |
      //  |        +--- Partner is an electron
      //  |        |
      //  V        V
      if (P2>0 and J1->sp==cuElectron) {
	      
	cuFP_t ke = Einfo->kEe2 > cuFloorEV ? Einfo->kEe2 : cuFloorEV, xc;
	computeRadRecomb(ke, xc, elems, J2->k);
	      
	crs = Einfo->eVel1 * xc;
	      
	if (crs > 0.0) {
#ifdef XC_DEEP1
	  printf("xsc: [ei] rc=%e cv=%e eta1=%e eta2=%e\n", crs, crs*Einfo->vel, Einfo->Eta1, Einfo->Eta2);
#endif
#ifdef XC_DEEP4
	  printf("xsc: [ei] kEe=%e (Z, P)=(%d, %d) gVel=%e eta=%e rc=%e dE=%e\n",
		 Einfo->kEe2, Z2, C2, Einfo->eVel1, Einfo->Eta1, xc, 0.0);
#endif
	}
      }
    }
    // END: original recomb algorithm
    
    return crs;
  }
  // END: recombination

  return 0.0;
}


// Return random 3d unit vector
//
__device__
void cudaUnitVector(cuFP_t *ret, hiprandState* state)
{
  enum UV {trig, gaus};		// Method choice
  static const UV uv(gaus);	// Pick the method
  
  if (uv == trig) {
#if cuREAL == 4
    cuFP_t cos_th = 1.0 - 2.0*hiprand_uniform(state);
    cuFP_t phi    = 2.0*M_PI*hiprand_uniform(state);
#else
    cuFP_t cos_th = 1.0 - 2.0*hiprand_uniform_double(state);
    cuFP_t phi    = 2.0*M_PI*hiprand_uniform_double(state);
#endif

    cuFP_t sin_th = sqrt(fabs(1.0 - cos_th*cos_th));

    ret[0] = sin_th*cos(phi);
    ret[1] = sin_th*sin(phi);
    ret[2] = cos_th;
  } else {
    cuFP_t nrm = 0.0;
    for (int i=0; i<3; i++) {
#if cuREAL == 4
      cuFP_t rn = hiprand_normal(state);
#else
      cuFP_t rn = hiprand_normal_double(state);
#endif
      ret[i] = rn;
      nrm += ret[i]*ret[i];
    }
    nrm = sqrt(nrm);

    if (nrm>0.0) {
      for (int i=0; i<3; i++) ret[i] /= nrm;
    }
  } 
}



__device__
cuFP_t cuCA_eval(cuFP_t tau)
{
  if      (tau >= coulSelTau_z) return 0.0;
  else if (tau >= coulSelTau_f) return 3.0*exp(-2.0*tau);
  else if (tau <= coulSelTau_m) return 1.0/(2.0*coulSelTau_m);
  else if (tau <= coulSelTau_i) return 1.0/(2.0*tau);
  else {
    cuFP_t lo  = log(coulSelTau_i);
    cuFP_t hi  = log(coulSelTau_f);
    cuFP_t lv  = log(tau), lo_t, hi_t;
    int indx   = floor( (lv - lo)/coulSelDel );

    if (indx<0) indx = 0;
    if (indx>coulSelNumT-2) indx = coulSelNumT - 2;

    lo_t = log(coulSelTau_i) + coulSelDel*indx;
    hi_t = log(coulSelTau_i) + coulSelDel*(indx+1);

    cuFP_t A = (hi_t - lv)/coulSelDel;
    cuFP_t B = (lv - lo_t)/coulSelDel;
    
    return A*coulSelA[indx-1] + B*coulSelA[indx];
  }
}

__global__ void photoIonizeKernel(dArray<cudaParticle> in,    // Particle array
				  dArray<cuFP_t>       dT,    // Time steps
				  dArray<int>          cellI, // Particle offset for each cell
				  dArray<int>          cellN, // Number of bodes for each cell
				  dArray<hiprandState>  randS, // Cuda random number objects
				  const
				  dArray<cuIonElement> elems  // Species map
				  )
{
  const int Nsp = elems._s;

  for (int cid = blockIdx.x * blockDim.x + threadIdx.x; 
       cid < cellI._s; 
       cid += blockDim.x * gridDim.x) {

    hiprandState* state = &randS._v[cid];

    int n0     = cellI._v[cid];
    int nbods  = cellN._v[cid];
    
    for (size_t i=0; i<nbods; i++) {
      
      int n = n0 + i;

      // Photoionize all subspecies
      //
      for (int s=0; s<Nsp; s++) {
	const cuIonElement& elem = elems._v[s];
	
	int Z = elem.Z;
	int C = elem.C;
	int I = elem.I;
      
	if (C<=Z) {
	  cuFP_t rn, Ep, Pr;
	  // Select random variate and pick a new photon for each body
	  //
#if cuREAL == 4
	  rn = hiprand_uniform(state);
#else
	  rn = hiprand_uniform_double(state);
#endif
	  computePhotoIonize(rn, Ep, Pr, elems, s);
	
	  // Compute the probability and get the residual electron energy
	  //
	  Pr *= cuTunit * dT._v[n];
	  cuFP_t ww  = in._v[n].datr[I+cuSp0] * Pr;
	
	  if (Pr >= 1.0) {	// Limiting case
	    ww = in._v[n].datr[I+cuSp0];
	    in._v[n].datr[I  +cuSp0]  = 0.0;
	    in._v[n].datr[I+1+cuSp0] += ww;
	  } else {		// Normal case
	    in._v[n].datr[I  +cuSp0] -= ww;
	    in._v[n].datr[I+1+cuSp0] += ww;
	  }
	
	}
	// End: bound electron block
      }
      // End: species loop
    }
    // End: particles per cell
  }
  // End: cell loop
}


// Compute the kinetic energy for all particles in a single cell
//
__device__
cuFP_t cellEnergy(int cid,		      // Cell index
		  dArray<cudaParticle> in,    // Particle array
		  dArray<int>          cellI, // Particle offset for each cell
		  dArray<int>          cellN, // Number of bodes for each cell
		  const
		  dArray<cuIonElement> elems  // Species map
		  )
{
  const int Nsp = elems._s;

  int n0     = cellI._v[cid];
  int nbods  = cellN._v[cid];
  cuFP_t sum = 0.0;
    
  for (size_t i=0; i<nbods; i++) {
      
    int n = n0 + i;
    cudaParticle* p = &in._v[n];
    cuFP_t eta = 0.0;

    for (int s=0; s<Nsp; s++) {
      const cuIonElement& elem = elems._v[s];
	
      int Z = elem.Z;
      int C = elem.C;
      int I = elem.I;
      
      cuFP_t cc = p->datr[I+cuSp0] / cuda_atomic_weights[Z];
      eta += cc * (C - 1);
    }

    // Velocity and KE quantities
    //
    cuFP_t vi2 = 0.0, ve2 = 0.0;
    for (int k=0; k<3; k++) {
      vi2 += p->vel[k] * p->vel[k];
      ve2 += p->datr[cuElec+k] * p->datr[cuElec+k];
    }
    
    sum += 0.5*p->mass*(vi2 + eta*cuda_atomic_weights[0]*ve2);
    if (cuCons>=0.0) sum -= p->datr[cuCons];
    if (cuEcon>=0.0) sum -= p->datr[cuEcon];
      
  } // End: particles per cell

  return sum;
}

__global__
void totalCellEnergy(dArray<cudaParticle> in,    // Particle array
		     dArray<int>          cellI, // Particle offset for each cell
		     dArray<int>          cellN, // Number of bodes for each cell
		     dArray<cuFP_t>       cellE, // Per cell energy
		     const
		     dArray<cuIonElement> elems  // Species map
		     )
{
  // Loop through all cells
  //
  for (int cid = blockIdx.x * blockDim.x + threadIdx.x; 
       cid < cellI._s; 
       cid += blockDim.x * gridDim.x) {

    cellE._v[cid] = cellEnergy(cid, in, cellI, cellN, elems);

  }
  // END: cell loop
}


//! Select tau given random number U in [0,1)
__device__
cuFP_t cuCA_get(cuFP_t tau, cuFP_t U)
{
  cuFP_t A = cuCA_eval(tau);
  if (U<1.0e-14)
    return -1.0;
  else if (A<1.0e-10)
    return 2.0*U - 1.0;
  else if (A>40.0)
    return 1.0 + log(U)/A;
  else
    return log(exp(-A) + 2.0*U*sinh(A))/A;
}

// Return 3d Colombic scattering vector
//
__device__
void cudaCoulombVector(cuFP_t *rel, cuFP_t Tau, hiprandState *state)
{
				// Normalize
  cuFP_t rel2 = 0.0;
  for (int i=0; i<3; i++) rel2 += rel[i]*rel[i];
  cuFP_t vfac = sqrt(rel2);
  if (vfac>0.0) for (int i=0; i<3; i++) rel[i] /= vfac;

				// Random generation
#if cuREAL == 4
  cuFP_t rn  = hiprand_uniform_double(state);
  cuFP_t phi = 2.0*M_PI*hiprand_uniform(state);
#else
  cuFP_t rn  = hiprand_uniform_double(state);
  cuFP_t phi = 2.0*M_PI*hiprand_uniform_double(state);
#endif


  cuFP_t tau = 100.0;
  if (Tau < tau) tau = Tau;

  cuFP_t cosx   = cuCA_get(tau, rn);
#ifdef SANITY_DEBUG
  if (::isnan(cosx)) {
    printf("Crazy cosx for Tau=%f tau=%f rn=%f\n", Tau, tau, rn);
  }
#endif
  cuFP_t sinx   = sqrt(fabs(1.0 - cosx*cosx));
  cuFP_t cosp   = cos(phi);
  cuFP_t sinp   = sin(phi);
  cuFP_t g_perp = sqrt(rel[1]*rel[1] + rel[2]*rel[2]);

				// Compute randomly-oriented
				// perpendicular vector
  cuFP_t h[3];
  if (g_perp>0.0) {
    h[0] = g_perp * cosp;
    h[1] = -(rel[1]*rel[0]*cosp + rel[2]*sinp)/g_perp;
    h[2] = -(rel[2]*rel[0]*cosp - rel[1]*sinp)/g_perp;
  } else {
    h[0] = 0.0;
    h[1] = cosp;
    h[2] = sinp;
  }
  
  for (int i=0; i<3; i++) rel[i] = rel[i]*cosx - h[i]*sinx;
}

__device__
void cudaDeferredEnergy
(
 cuFP_t E,
 cuFP_t m1,   cuFP_t m2,
 cuFP_t a,    cuFP_t b,
 cuFP_t *E1,  cuFP_t *E2
)
{
#ifdef XC_DEEP14
  printf("deferE=%e\n", E);
#endif

  if (cuCons>=0) {

    if (cuEcon<0) {
      E1[0] += 0.5*E;
      E2[0] += 0.5*E;
    } else {
      if (m1 < 1.0) {
	E1[1] += a*E/(a + b);
	E2[0] += b*E/(a + b);
      }
      else if (m2 < 1.0) {
	E1[0] += a*E/(a + b);
	E2[1] += b*E/(a + b);
      }
      else {
	E1[0]  += a*E/(a + b);
	E2[0]  += b*E/(a + b);
      }
    }
  }
}

__device__
void cudaScatterTrace
(cuFP_t m1,     cuFP_t m2,
 cuFP_t eta1, cuFP_t eta2,
 cuFP_t W1,   cuFP_t W2,
 cuFP_t *E1,  cuFP_t *E2,
 cuFP_t *v1,  cuFP_t *v2, cuFP_t delE,  
 hiprandState *state
 )
{
  // BEGIN: Energy conservation
  {

#ifdef XC_DEEP3
    // KE debug check
    //
    cuFP_t KEi = 0.0;
    {
      cuFP_t k1 = 0.0, k2 = 0.0;
      for (int k=0; k<3; k++) {
	k1 += v1[k]*v1[k];
	k2 += v2[k]*v2[k];
      }
      KEi = 0.5*W1*m1*k1 + 0.5*W2*m2*k2;
    }
#endif

    // Total effective mass in the collision (atomic mass units)
    //
    cuFP_t mt = m1 + m2;

    // Reduced mass (atomic mass units)
    //
    cuFP_t mu = m1 * m2 / mt;

    // Set COM frame
    //
    cuFP_t vcom[3], vrel[3];
    cuFP_t vi = 0.0, vfac = 1.0;
    
    for (size_t k=0; k<3; k++) {
      vcom[k] = (m1*v1[k] + m2*v2[k])/mt;
      vrel[k] = v1[k] - v2[k];
      vi += vrel[k] * vrel[k];
    }
				// Energy in COM
    cuFP_t kE = 0.5*W2*mu*vi;
				// Energy reduced by loss
    cuFP_t totE = kE - delE;

#ifdef XC_DEEP3
    cuFP_t fixE = 0.0;
#endif

    // KE is positive
    //
    if (kE>0.0) {
      // More loss energy requested than available?
      //
      if (totE < 0.0) {
	// Add to energy bucket for these particles
	//
	cudaDeferredEnergy(-totE, m1, m2, W1, W2, E1, E2);
#ifdef XC_DEEP3
	fixE = -totE;
	printf("deferE[1]=%e kE=%e dE=%e\n", totE, kE, delE);
#endif
	totE = 0.0;
      }
      // Update the outgoing energy in COM
      vfac = sqrt(totE/kE);
    }
    // KE is zero (limiting case)
    //
    else {
      if (delE>0.0) {
	// Defer all energy loss
	//
	cudaDeferredEnergy(delE, m1, m2, W1, W2, E1, E2);
#ifdef XC_DEEP3
	fixE = delE;
	printf("deferE[0]=%e\n", delE);
#endif
	delE = 0.0;
      } else {
	// Apply delE to COM
	//
	vi = -2.0*delE/(W1*mu);
      }
    }

    // Assign interaction energy variables
    //
    cudaUnitVector(vrel, state);
  
    vi = sqrt(vi);
    for (auto & v : vrel) v *= vi;
    //                         ^
    //                         |
    // Velocity in center of mass, computed from v1, v2 and adjusted
    // according to the inelastic energy loss
    //

    // BEGIN: energy conservation algorithm

    cuFP_t vrat = 1.0, q = W2/W1, cq = 1.0 - W2/W1;

    if (cq > 0.0 and q < 1.0) {

      cuFP_t uu[3];
      cuFP_t v1i2 = 0.0, b1f2 = 0.0, qT = 0.0;
      cuFP_t udif = 0.0, vcm2 = 0.0;

      for (size_t i=0; i<3; i++) {
	uu[i] = vcom[i] + m2/mt*vrel[i]*vfac;
	vcm2 += vcom[i] * vcom[i];
	v1i2 += v1[i] * v1[i];
	b1f2 += uu[i] * uu[i];
	qT   += v1[i] * uu[i];
	udif += (v1[i] - uu[i]) * (v1[i] - uu[i]);
      }
      
      if (v1i2 > 0.0 and b1f2 > 0.0) qT *= q/v1i2;
      
      cuFP_t csign = 1.0;
      if (qT<0.0) csign = -1.0;

      vrat = 
	( -qT + csign*sqrt(qT*qT + cq*(q*b1f2/v1i2 + 1.0)) )/cq;
    }

    // Assign new velocities
    //
    for (int i=0; i<3; i++) {
      cuFP_t v0 = vcom[i] + m2/mt*vrel[i]*vfac;
    
      v1[i] = cq*v1[i]*vrat + q*v0;
      v2[i] = vcom[i] - m1/mt*vrel[i]*vfac;
    }

#ifdef XC_DEEP3
    // KE debug check
    //
    {
      cuFP_t k1 = 0.0, k2 = 0.0;
      for (int k=0; k<3; k++) {
	k1 += v1[k]*v1[k];
	k2 += v2[k]*v2[k];
      }
      cuFP_t KEf = 0.5*W1*m1*k1 + 0.5*W2*m2*k2;
      cuFP_t KEd = KEi - KEf - delE + fixE;
      cuFP_t KEm = 0.5*(KEi + KEf);
      if (fabs(KEd)/KEm > EDEL_TOL) {
	printf("**ERROR deltaE: R=%e KEi=%e KEf=%e dKE=%e kE=%e delE=%e fixE=%e\n", KEd/KEm, KEi, KEf, KEd, kE, delE, fixE);
      }
      else if (false) {
	printf("OK deltaE: R=%e KEi=%e KEf=%e dKE=%e kE=%e delE=%e fixE=%e\n", KEd/KEm, KEi, KEf, KEd, kE, delE, fixE);
      }
    }
#endif

  } // END: Energy conservation algorithm
    
} // END: cudaScatterTrace


// Uses full molecular weight for scattering that explicitly conserves
// center-of-mass energy
//
__device__
void cudaScatterTraceExplicit
(cuFP_t m1,     cuFP_t m2,
 cuFP_t eta1, cuFP_t eta2,
 cuFP_t W1,   cuFP_t W2,
 cuFP_t *E1,  cuFP_t *E2,
 cuFP_t *v1,  cuFP_t *v2, cuFP_t delE,  
 hiprandState *state
 )
{
  cuFP_t M1 = W1*m1;
  cuFP_t M2 = W2*m2;

#ifdef XC_DEEP3
  // KE debug check
  //
  cuFP_t KEi = 0.0;
  {
    cuFP_t k1 = 0.0, k2 = 0.0;
    for (int k=0; k<3; k++) {
      k1 += v1[k]*v1[k];
      k2 += v2[k]*v2[k];
    }
    KEi = 0.5*M1*k1 + 0.5*M2*k2;
  }
#endif

  // Total effective mass in the collision (atomic mass units)
  //
  cuFP_t mt = M1 + M2;

  // Reduced mass (atomic mass units)
  //
  cuFP_t mu = M1 * M2 / mt;

  // Set COM frame
  //
  cuFP_t vcom[3], vrel[3];
  cuFP_t vi = 0.0, vfac = 1.0;
    
  for (size_t k=0; k<3; k++) {
    vcom[k] = (M1*v1[k] + M2*v2[k])/mt;
    vrel[k] = v1[k] - v2[k];
    vi += vrel[k] * vrel[k];
  }
				// Energy in COM
  cuFP_t kE = 0.5*mu*vi;
				// Energy reduced by loss
  cuFP_t totE = kE - delE;

#ifdef XC_DEEP3
  cuFP_t fixE = 0.0;
#endif

  // KE is positive
  //
  if (kE>0.0) {
    // More loss energy requested than available?
    //
    if (totE < 0.0) {
      // Add to energy bucket for these particles
      //
      cudaDeferredEnergy(-totE, m1, m2, W1, W2, E1, E2);
#ifdef XC_DEEP3
      fixE = -totE;
      printf("deferE[1]=%e kE=%e dE=%e\n", totE, kE, delE);
#endif
      totE = 0.0;
    }
    // Update the outgoing energy in COM
    vfac = sqrt(totE/kE);
  }
  // KE is zero (limiting case)
  //
  else {
    if (delE>0.0) {
      // Defer all energy loss
      //
      cudaDeferredEnergy(delE, m1, m2, W1, W2, E1, E2);
#ifdef XC_DEEP3
      fixE = delE;
      printf("deferE[0]=%e\n", delE);
#endif
      delE = 0.0;
    } else {
      // Apply delE to COM
      //
      vi = -2.0*delE/mu;
    }
  }

  // Assign interaction energy variables
  //
  cudaUnitVector(vrel, state);
  
  vi = sqrt(vi);
  for (auto & v : vrel) v *= vi;
  //                         ^
  //                         |
  // Velocity in center of mass, computed from v1, v2 and adjusted
  // according to the inelastic energy loss
  //

  // Assign new velocities
  //
  for (int i=0; i<3; i++) {
    v1[i] = vcom[i] + M2/mt*vrel[i]*vfac;
    v2[i] = vcom[i] - M1/mt*vrel[i]*vfac;
  }

#ifdef XC_DEEP3
  // KE debug check
  //
  {
    cuFP_t k1 = 0.0, k2 = 0.0;
    for (int k=0; k<3; k++) {
      k1 += v1[k]*v1[k];
      k2 += v2[k]*v2[k];
    }
    cuFP_t KEf = 0.5*M1*k1 + 0.5*M2*k2;
    cuFP_t KEd = KEi - KEf - delE + fixE;
    cuFP_t KEm = 0.5*(KEi + KEf);
    if (fabs(KEd)/KEm > EDEL_TOL) {
      printf("**ERROR deltaE: R=%e KEi=%e KEf=%e dKE=%e kE=%e delE=%e fixE=%e\n", KEd/KEm, KEi, KEf, KEd, kE, delE, fixE);
    }
    else if (false) {
      printf("OK deltaE: R=%e KEi=%e KEf=%e dKE=%e kE=%e delE=%e fixE=%e\n", KEd/KEm, KEi, KEf, KEd, kE, delE, fixE);
    }
  }
#endif

}
// END: cudaScatterTraceExplicit


__device__
void computeCoulombicScatter(dArray<cudaParticle>   in,
			     dArray<cuFP_t>         coul4,
			     dArray<int>            cellI,
			     dArray<int>            cellN,
			     dArray<cuFP_t>         PiProb,
			     dArray<cuFP_t>         ABrate,
			     const
			     dArray<cuIonElement>   elems,
			     dArray<cuFP_t>         spTau,
			     hiprandState*           state,
			     int                    C
			     )
{
  int nbods = cellN._v[C];

  // Can't have a collision with one body in the cell!
  //
  if (nbods<2) return;

  const int Nsp = elems._s;

  cuFP_t V1[3], V2[3];

  // Assign storage to pointer for swapping
  //
  cuFP_t *v1 = &V1[0], *v2 = &V2[0];

  int npair = nbods/2;
  bool odd = false;
  if ( (nbods/2)*2 != nbods) {
    odd = true;
    npair++;
  }

  // Time step in physical units
  //
  cuFP_t dT = spTau._v[C] * cuTunit;

  // Initial particle index
  //
  int n0 = cellI._v[C];


  cuFP_t na[4], nab[4];
  for (int l=0; l<4; l++) na[l] = nab[l] = 0.0;

  // Compute weights from all pairs
  //
  for (int n=0; n<npair; n++) {

    int i1 = n0 + n*2 + 0;
    int i2 = n0 + n*2 + 1;

    if (n==npair-1 and odd) {
      i1 = n0 + nbods - 3;
      i2 = n0 + nbods - 1;
    }

    cudaParticle* p1 = &in._v[i1];
    cudaParticle* p2 = &in._v[i2];

    cuFP_t Eta1 = 0.0, Eta2 = 0.0, Sum1 = 0.0, Sum2 = 0.0;
    cuFP_t Frc1 = 0.0, Frc2 = 0.0;

    for (int k=0; k<Nsp; k++) {
      cuIonElement& E = elems._v[k];
	  
      // Number fraction of ions
      cuFP_t one = p1->datr[E.I+cuSp0] / cuda_atomic_weights[E.Z];
      cuFP_t two = p2->datr[E.I+cuSp0] / cuda_atomic_weights[E.Z];
	  
      // Charged fraction
      if (E.C>1) {
	Frc1 += p1->datr[E.I+cuSp0];
	Frc2 += p2->datr[E.I+cuSp0];
      }

      // Electron number fraction
      Eta1 += one * (E.C - 1);
      Eta2 += two * (E.C - 1);
	  
      Sum1 += one;
      Sum2 += two;
    }

    // The number of electrons per particle
    Eta1 /= Sum1;
    Eta2 /= Sum2;

    // The molecular weight
    cuFP_t Mu1 = 1.0/Sum1;
    cuFP_t Mu2 = 1.0/Sum2;
	
    // Proportional to number of true particles in each superparticle
    //
    cuFP_t W1 = p1->mass/Mu1, ww1;
    cuFP_t W2 = p2->mass/Mu2, ww2;

    for (int l=0; l<4; l++) {

      if (l==0) {
	ww1 = W1 * Frc1;
	ww2 = W2 * Frc2;

	na[0] += ww1;
	if ((ww1>ww2 ? ww1 : ww2) > 0.0)
	  nab[0] += ww1*ww2/(ww1>ww2 ? ww1 : ww2);

      } else if (l==1) {
	ww1 = W1 * Frc1;
	ww2 = W2 * Eta2;

	na[1] += ww1;
	if ((ww1>ww2 ? ww1 : ww2) > 0.0)
	  nab[1] += ww1*ww2/(ww1>ww2 ? ww1 : ww2);

      } else if (l==2) {
	ww1 = W1 * Eta1;
	ww2 = W2 * Frc2;

	na[2] += ww1;
	if ((ww1>ww2 ? ww1 : ww2) > 0.0)
	  nab[2] += ww1*ww2/(ww1>ww2 ? ww1 : ww2);

      } else {
	ww1 = W1 * Eta1;
	ww2 = W2 * Eta2;

	na[3] += ww1;
	if ((ww1>ww2 ? ww1 : ww2) > 0.0)
	  nab[3] += ww1*ww2/(ww1>ww2 ? ww1 : ww2);
      }
    }
  }


  // Now, compute interactions for all pairs
  //
  for (int n=0; n<npair; n++) {

    int i1 = n0 + n*2 + 0;
    int i2 = n0 + n*2 + 1;

    if (n==npair-1 and odd) {
      i1 = n0 + nbods - 3;
      i2 = n0 + nbods - 1;
    }

    cudaParticle* p1 = &in._v[i1];
    cudaParticle* p2 = &in._v[i2];

    // Particle quantities
    //
    cuFP_t Eta1 = 0.0, Eta2 = 0.0, Sum1 = 0.0, Sum2 = 0.0;
    cuFP_t Frc1 = 0.0, Frc2 = 0.0;

    for (int k=0; k<Nsp; k++) {
      cuIonElement& E = elems._v[k];
	  
      // Number fraction of ions
      cuFP_t one = p1->datr[E.I+cuSp0] / cuda_atomic_weights[E.Z];
      cuFP_t two = p2->datr[E.I+cuSp0] / cuda_atomic_weights[E.Z];
	  
      // Charged fraction
      if (E.C>1) {
	Frc1 += p1->datr[E.I+cuSp0];
	Frc2 += p2->datr[E.I+cuSp0];
      }

      // Electron number fraction
      Eta1 += one * (E.C - 1);
      Eta2 += two * (E.C - 1);
	  
      Sum1 += one;
      Sum2 += two;
    }

    // The number of electrons per particle
    Eta1 /= Sum1;
    Eta2 /= Sum2;

    // The molecular weight
    cuFP_t Mu1 = 1.0/Sum1;
    cuFP_t Mu2 = 1.0/Sum2;
	
    // Proportional to number of true particles in each superparticle
    //
    cuFP_t W1 = p1->mass/Mu1;
    cuFP_t W2 = p2->mass/Mu2;

#ifdef XC_DEEP3
    // KE debug check
    //
    cuFP_t KE1i = 0.0, KE2i = 0.0;
    {
      cuFP_t k1i = 0.0, k1e = 0.0, k2i = 0.0, k2e = 0.0;
      for (int k=0; k<3; k++) {
	k1i += p1->vel[k]*p1->vel[k];
	k2i += p2->vel[k]*p2->vel[k];
	k1e += p1->datr[cuElec+k]*p1->datr[cuElec+k];
	k2e += p2->datr[cuElec+k]*p2->datr[cuElec+k];
      }
      KE1i = 0.5*W1*Mu1*k1i + 0.5*W1*Eta1*cuda_atomic_weights[0]*k1e;
      KE2i = 0.5*W2*Mu2*k2i + 0.5*W2*Eta2*cuda_atomic_weights[0]*k2e;
    }
#endif

    for (int l=0; l<4; l++) {

      cuFP_t ww1, ww2, KE = 0.0;
      cuFP_t m1 = Mu1, m2 = Mu2;

      // Neutrality rejection
      //
      if (l==0 or l==1) {
	if (hiprand_uniform(state) > Frc1) continue;
      }

      if (l==2) {
	if (hiprand_uniform(state) > Frc2) continue;
      }

      if (l==0) {

	ww1 = Frc1 * W1;
	ww2 = Frc2 * W2;
	if (ww1 <= 0.0 or ww2 <= 0.0) continue;
	if (ww1 > ww2) {
	  if ( hiprand_uniform(state) > ww2/ww1 ) continue;
	} else {
	  if ( hiprand_uniform(state) > ww1/ww2 ) continue;
	}

	for (int k=0; k<3; k++) {
				// Particle 1 is an ion
	  v1[k]  = p1->vel[k];
				// Particle 2 is an ion
	  v2[k]  = p2->vel[k];

	  KE += (v1[k] - v2[k]) * (v1[k] - v2[k]);
	}

      }	else if (l==1) {

	ww1 = Frc1 * W1;
	ww2 = Frc2 * W2;
	if (ww1 <= 0.0 or ww2 <= 0.0) continue;

#if cuREAL == 4
	cuFP_t rn = hiprand_uniform(state);
#else
	cuFP_t rn = hiprand_uniform_double(state);
#endif
	if (ww1 > ww2) {
	  if ( rn > ww2/ww1 ) continue;
	} else {
	  if ( rn > ww1/ww2 ) continue;
	}

	for (int k=0; k<3; k++) {
				// Particle 1 is the ion
	  v1[k]  = p1->vel[k];
				// Particle 2 is the electron
	  v2[k]  = p2->datr[cuElec+k];

	  KE += (v1[k] - v2[k]) * (v1[k] - v2[k]);
	}

	m2 = Eta2 * cuda_atomic_weights[0];
	
      } else if (l==2) {

	ww1 = Eta1 * W1;
	ww2 = Frc2 * W2;
	if (ww1 <= 0.0 or ww2 <= 0.0) continue;
#if cuREAL == 4
	cuFP_t rn = hiprand_uniform(state);
#else
	cuFP_t rn = hiprand_uniform_double(state);
#endif
	if (ww1 > ww2) {
	  if ( rn > ww2/ww1 ) continue;
	} else {
	  if ( rn > ww1/ww2 ) continue;
	}

	for (int k=0; k<3; k++) {
				// Particle 2 is the ion
	  v2[k]  = p2->vel[k];
				// Particle 1 is the electron
	  v1[k]  = p1->datr[cuElec+k];

	  KE += (v1[k] - v2[k]) * (v1[k] - v2[k]);
	}

	m1 = Eta1 * cuda_atomic_weights[0];

      } else {

	ww1 = Eta1 * W1;
	ww2 = Eta2 * W2;
	if (ww1 <= 0.0 or ww2 <= 0.0) continue;
#if cuREAL == 4
	cuFP_t rn = hiprand_uniform(state);
#else
	cuFP_t rn = hiprand_uniform_double(state);
#endif
	if (ww1 > ww2) {
	  if ( rn > ww2/ww1 ) continue;
	} else {
	  if ( rn > ww1/ww2 ) continue;
	}

	for (int k=0; k<3; k++) {
				// Both particles are electrons
	  v1[k]  = p1->datr[cuElec+k];
	  v2[k]  = p2->datr[cuElec+k];

	  KE += (v1[k] - v2[k]) * (v1[k] - v2[k]);
	}

	m1 = Eta1 * cuda_atomic_weights[0];
	m2 = Eta2 * cuda_atomic_weights[0];
      }

      double Q1 = Eta1;
      double Q2 = Eta2;
	
      if (m1 < cuMinMass) m1 = cuMinMass;
      if (m2 < cuMinMass) m2 = cuMinMass;

      m1 *= amu;
      m2 *= amu;

      // Total effective mass in the collision
      //
      cuFP_t mt = m1 + m2;

      // Reduced mass (atomic mass units)
      //
      cuFP_t mu = m1 * m2 / mt;
      
      // KE
      //
      KE *= 0.5 * mu * cuVunit * cuVunit;

      // Coulombic rate (physical units)
      //
      cuFP_t pVel = sqrt(2.0*KE/mu);
      cuFP_t KE2  = 2.0*KE;
      if (KE2/eV < cuFloorEV) KE2 = cuFloorEV * eV;
      cuFP_t afac = cuEsu*cuEsu*Q1*Q2/KE2;
      cuFP_t tau  = ABrate._v[C*4 + l]*afac*afac*pVel * dT;

#ifdef XC_DEEP11
	printf("coul5: l=%d pVel=%e afac=%e dt=%e tau=%e mu=%e\n",
	       l, pVel, afac, dT, tau, mu/amu);
#endif

      coul4._v[C*4+l] = tau;

      // Set COM frame
      //
      cuFP_t vcom[3], vrel[3];
      cuFP_t vi = 0.0;
	
      for (size_t k=0; k<3; k++) {
	vcom[k] = (m1*v1[k] + m2*v2[k])/mt;
	vrel[k] = v1[k] - v2[k];
	vi += vrel[k] * vrel[k];
      }
      // Energy in COM
      //
      cuFP_t mW = W1 > W2 ? W2 : W1;
      cuFP_t kE = 0.5 * mW * mu * vi;

      // KE is positive?
      //
      if (kE>0.0) {
	// Assign interaction energy variables
	//
	// printf("nab[%d]=%e na[%d]=%e rat=%f\n", l, nab[l], l, na[l], nab[l]/na[l]);
	cudaCoulombVector(vrel, tau*nab[l]/na[l], state);
	
	// Sanity check
	//
	cuFP_t ncheck = 0.0;
	for (int k=0; k<3; k++) ncheck += vrel[k]*vrel[k];
	if ( fabs(ncheck-1.0) > 1.0e-14) {
	  printf("Norm check error=%e\n", ncheck - 1.0);
	}

	vi   = sqrt(vi);
	for (size_t k=0; k<3; k++) vrel[k] *= vi;
	//                                    ^
	//                                    |
	// Velocity in center of mass, computed from v1, v2 and adjusted
	// according to the inelastic energy loss
	//

	for (size_t k=0; k<3; k++) {
	  v1[k] = vcom[k] + m2/mt*vrel[k];
	  v2[k] = vcom[k] - m1/mt*vrel[k];
	}
    
	if (l==0) {
	  for (int k=0; k<3; k++) {
				// Particle 1 is an ion
	    p1->vel[k] = v1[k];
				// Particle 2 is an ion
	    p2->vel[k] = v2[k];
	  }
	} else if (l==1) {
	  for (int k=0; k<3; k++) {
				// Particle 1 is the ion
	    p1->vel[k] = v1[k];
				// Particle 2 is the electron
	    p2->datr[cuElec+k] = v2[k];
	  }
	} else if (l==2) {
	  for (int k=0; k<3; k++) {
				// Particle 2 is the ion
	    p2->vel[k] = v2[k];
				// Particle 1 is the electron
	    p1->datr[cuElec+k] = v1[k];
	  }
	} else {
	  for (int k=0; k<3; k++) {
				// Both particles are electrons
	    p1->datr[cuElec+k] = v1[k];
	    p2->datr[cuElec+k] = v2[k];
	  }
	} // l==3

      } // kE>0.0

    } // l loop

#ifdef XC_DEEP3
    // KE debug check
    //
    cuFP_t KE1f = 0.0, KE2f = 0.0;
    {
      cuFP_t k1i = 0.0, k1e = 0.0, k2i = 0.0, k2e = 0.0;
      for (int k=0; k<3; k++) {
	k1i += p1->vel[k]*p1->vel[k];
	k2i += p2->vel[k]*p2->vel[k];
	k1e += p1->datr[cuElec+k]*p1->datr[cuElec+k];
	k2e += p2->datr[cuElec+k]*p2->datr[cuElec+k];
      }
      KE1f = 0.5*W1*Mu1*k1i + 0.5*W1*Eta1*cuda_atomic_weights[0]*k1e;
      KE2f = 0.5*W2*Mu2*k2i + 0.5*W2*Eta2*cuda_atomic_weights[0]*k2e;
    }

    cuFP_t KEi  = KE1i + KE2i;
    cuFP_t KEf  = KE1f + KE2f;
    cuFP_t delE = KEi  - KEf;
    if (fabs(delE)/KEi > EDEL_TOL) {
      printf("**ERROR in Coulombic: delE/KEi=%e [%8d, %8d]\n", delE/KEi, i1, i2);
    }
#endif

  } // body loop

} // END: computeCoulombicScatter


__device__
void computeEta(cuFP_t*                F,
		cuFP_t*                Eta,
		const
		dArray<cuIonElement>   elems
		)
{
  int    Nsp = elems._s;
  cuFP_t Sum = 0.0;
  *Eta = 0.0;

  for (int k=0; k<Nsp; k++) {

    cuIonElement* E = &elems._v[k];
	  
    // Number fraction of ions
    //
    cuFP_t fac = F[E->I] / cuda_atomic_weights[E->Z];
	  
    // Electron number fraction
    //
    *Eta += fac * (E->C - 1);
    Sum  += fac;
  }

  // The number of electrons per particle
  //
  *Eta /= Sum;
}

template <class T>
__device__
int xc_lower_bound (int first, int last, int fN, dArray<float> cum,
		    const T& val)
{
  int count = last - first, step, cur;
  while (count>0)
  {
    cur  = first;
    step = count/2;
    cur += step;
    if (cum._v[fN+cur]<val) {
      first  = cur + 1;
      count -= step + 1;
    }
    else count = step;
  }
  return first;
}


// Compute particle interactions for each cell given the computed
// cross sections
//
__global__ void partInteractions(dArray<cudaParticle>   in,
				 dArray<hiprandState>    randS,
				 dArray<cuFP_t>         coul4,
				 dArray<cuFP_t>         nSel,
				 dArray<int>            cellI,
				 dArray<int>            cellN,
				 dArray<cuFP_t>         deltE,
				 dArray<cuFP_t>         volC,
				 dArray<cuFP_t>         Ivel2,
				 dArray<cuFP_t>         Evel2,
				 dArray<cuFP_t>         xsc_H,
				 dArray<cuFP_t>         xsc_He,
				 dArray<cuFP_t>         xsc_pH,
				 dArray<cuFP_t>         xsc_pHe,
				 dArray<cuFP_t>         PiProb,
				 dArray<cuFP_t>         ABrate,
				 const
				 dArray<cuIonElement>   elems,
				 dArray<cuFP_t>         spTau,
				 dArray<cuInteract>     iK,
				 dArray<cuFP_t>         F1,
				 dArray<cuFP_t>         F2,
				 dArray<int>            prs,
				 dArray<float>          cum
				 )
{
  const int Nsp   = elems._s;
  const int numxc = iK._s;
  
  // For convenience in checking species
  //
  const cuSpeciesKey cuProton    {1, 2};
  const cuSpeciesKey cuElectron  {0xffff, 0xffff};
  //                              ^       ^
  //                              |       |
  // 2^16-1 is max ushort---------+-------+
  // as defined in NTC.H

  // Energy return info
  //
  cuEnergyInfo EI;
  // cuFP_t Mue = cuda_atomic_weights[0];
  
  // Cell loop with grid stride
  //
  for (int cid = blockIdx.x * blockDim.x + threadIdx.x; cid < cellI._s; cid += blockDim.x * gridDim.x) {

#ifdef XC_DEEP3
    cuFP_t begE = cellEnergy(cid, in, cellI, cellN, elems);
#endif
    hiprandState* state = &randS._v[cid];
    
    int n0     = cellI._v[cid];
    int nbods  = cellN._v[cid];
    cuFP_t vol = volC ._v[cid];
    
    // For selection diagnostics
    //
    nSel._v[cid] = 0.0;

    // For species state indexing
    //
    int fP = cid*Nsp;

    // For per cell indexing
    //
    int fN = cid*PLIST_LEN;

    // Compute Coulombic (plasma) interactions
    //
    computeCoulombicScatter(in, coul4, cellI, cellN, PiProb, ABrate, elems, spTau, state, cid);
    
    // For storing pair info
    //
    union {
      int   I2;
      short SS[2];
    } conv_;


    cuFP_t E1[2] = {0.0, 0.0}, E2[2] = {0.0, 0.0};

    // Compute total cross sections for interactions in this cell
    //
    cuFP_t mtotal = 0.0;
    
    enum AccumType {ion_ion, ion_electron, electron_electron};
    
    AccumType type = AccumType::ion_ion;

    double IEadjust = 0.0;

    // Interaction-type loop
    //
    for (int k=0; k<numxc; k++) {

      int           T = thrust::get<0>(iK._v[k]);
      cuSpeciesDef J1 = thrust::get<1>(iK._v[k]);
      cuSpeciesDef J2 = thrust::get<2>(iK._v[k]);

#ifdef XC_NOCHANGE
      if (T==8 or T==9) continue;
#endif
      int count = 0;

      // Pair-wise cross-section loop
      //
      for (int i=0; i<nbods; i++) {
      
	cuFP_t mass = in._v[n0+i].mass;
	mtotal += mass;
      
	for (int j=i+1; j<nbods; j++) {
	
	  setupCrossSection(in, elems, cid, n0+i, n0+j, state, &EI);

	  cuFP_t ph, xc;
	  
	  // p1 on p2
	  //
	  xc = singleCrossSection(in, elems, &ph, xsc_H, xsc_He, xsc_pH, xsc_pHe,
				  cid, n0+i, n0+j, T, &J1, &J2, state, &EI) * EI.vel;

	  if (xc>0.0) {
	    conv_.SS[0] = i;
	    conv_.SS[1] = j;
	    prs._v[fN+count] = conv_.I2;
	    cum._v[fN+count] = xc;
	    count++;
	    if (false and T==7) {
	      cuFP_t val = xc/EI.vel;
	      printf("col_excite [%d]: pair [%d, %d]/%d xc=%e e1=%e e2=%e\n", cid, i, j, nbods, val, EI.kEe1, EI.kEe2);
	    }
	  }

	  // p2 on p1
	  //
	  if (J1.sp != J2.sp) {
	    xc = singleCrossSection(in, elems, &ph, xsc_H, xsc_He, xsc_pH, xsc_pHe,
				    cid, n0+j, n0+i, T, &J1, &J2, state, &EI) * EI.vel;

	    if (xc>0.0) {
	      conv_.SS[0] = j;
	      conv_.SS[1] = i;
	      prs._v[fN+count] = conv_.I2;
	      cum._v[fN+count] = xc;
	      count++;
	      if (false and T==7) {
		cuFP_t val = xc/EI.vel;
		printf("col_excite [%d]: pair [%d, %d]/%d xc=%e e1=%e e2=%e\n", cid, j, i, nbods, val, EI.kEe1, EI.kEe2);
	      }
	    }
	  }
	
	  if (count >= PLIST_LEN) break;

	} // END: inner body loop

	if (count >= PLIST_LEN) break;

      } // END: outer body loop


      if (count==0) continue;

      // Compute cumulative cross section
      //
      for (int c=1; c<count; c++) cum._v[fN+c] += cum._v[fN+c-1];

      // Interaction probability
      //
      cuFP_t Prob  = mtotal/vol * cuMunit/amu *
	spTau._v[cid] * cum._v[fN+count-1] * 1e-14 / (cuLunit*cuLunit);

      // Number of interaction candidate pairs
      //
      cuFP_t nsel = Prob * (nbods-1);
      
      if (J1.sp == J2.sp) nsel *= 0.5;
#if cuREAL == 4
      int    npairs = ceilf(nsel);
#else
      int    npairs = ceil(nsel);
#endif
    
      nSel._v[cid] += nsel;

      if (false and T==7) printf("col_excite [%d]: npairs=%d nsel=%e\n", cid, npairs, nsel);

#ifdef XC_DEEP12
      printf("NPAIR=%8d NSEL=%13.6e T=%d\n", npairs, nsel, T);
#endif

      // Compute interactions for all pairs
      //
      for (int r=0; r<npairs; r++) {
	
	//  Only use fractional part on final candidate
	//
	cuFP_t frc = nsel - r;

	if (frc < 1.0) {
#if cuREAL == 4
	  cuFP_t R0 = hiprand_uniform(state);
#else
	  cuFP_t R0 = hiprand_uniform_double(state);
#endif
#ifdef XC_DEEP12
	  printf("FRC=%13.6e R=%13.6e T=%2d [%s] \n", frc, R0, T, cudaInterNames[T]);
#endif
	  if (frc < R0) break;
	}

	// Pick interaction pair
	//
#if cuREAL == 4
	cuFP_t R0 = hiprand_uniform(state);
#else
	cuFP_t R0 = hiprand_uniform_double(state);
#endif
	cuFP_t RC = R0*cum._v[fN+count-1];
	int cc    = xc_lower_bound(0, count-1, fN, cum, RC);
	if (cc==0)
	  conv_.I2  = prs._v[fN];
	else
	  conv_.I2  = prs._v[fN+cc-1];

	// Sanity check
	//
	if (cc >= count) printf("**ERROR cc=%d is bigger than count=%d (%d, %d)\n", cc, count, conv_.SS[0], conv_.SS[1]);

	int    n1 = conv_.SS[0] + n0;
	int    n2 = conv_.SS[1] + n0;

#ifdef SANITY_DEBUG
	if (n1==n2) {
	  printf("Crazy error! n1[%d]=n2[%d] nbods=%d\n", n1, n2, nbods);
      }
#endif
	setupCrossSection(in, elems, cid, n1, n2, state, &EI);

	cuFP_t dE=0, ph;
	cuFP_t curXC =
	  singleCrossSection(in, elems, &ph, 
			     xsc_H, xsc_He, xsc_pH, xsc_pHe,
			     cid, n1, n2, T, &J1, &J2, state, &EI);
      
#ifdef XC_DEEP5
	printf("ctest: T=%12s cross=%e selcM=%e Vel=%e Tau=%e ph=%e\n", cudaInterNames[T], curXC, nsel, EI.vel, spTau._v[cid], ph);
	if (false and T==7) {		// col_excite
	  if (cc==0) {
	    printf("col_excite [%d]: cc=%d [%e < %e] xc=%e (%d/%d) [%d, %d] e1=%e e2=%e\n", cid, cc,
		   RC, cum._v[fN], curXC, r, npairs, n1-n0, n2-n0, EI.kEe1, EI.kEe2);
	  } else {
	    printf("col_excite [%d]: cc=%d [%e < %e < %e] xc=%e (%d/%d) [%d, %d] e1=%e e2=%e\n", cid, cc,
		   cum._v[fN+cc-1], RC, cum._v[fN+cc], curXC, r, npairs, n1-n0, n2-n0, EI.kEe1, EI.kEe2);
	  }
	}
#endif
	if (curXC <= 0.0) continue;

	
	cudaParticle* p1 = &in._v[n1];
	cudaParticle* p2 = &in._v[n2];
      
	// Electron and molecular weight
	//
	{
	  cuFP_t sum1 = 0.0, sum2 = 0.0;
	  for (int k=0; k<Nsp; k++) {
	    cuIonElement& E = elems._v[k];
	    
	    F1._v[fP+k] = p1->datr[E.I+cuSp0];
	    F2._v[fP+k] = p2->datr[E.I+cuSp0];
	    sum1 += F1._v[fP+k];
	    sum2 += F2._v[fP+k];
	  }
	  for (int k=0; k<Nsp; k++) {
	    F1._v[fP+k] /= sum1;
	    F2._v[fP+k] /= sum2;
	  }
	}
	  
	cuFP_t w1  = p1->mass/EI.Mu1;
	cuFP_t w2  = p2->mass/EI.Mu2;
	cuFP_t W1  = w1;
	cuFP_t W2  = w2;
	
	// For electron energy conservation during ionization level change
	//
	cuFP_t elecAdj[2] = {0.0, 0.0};
	
	unsigned short Z1 = J1.sp.first;
	unsigned short Z2 = J2.sp.first;
	
	unsigned short C1 = J1.sp.second;
	unsigned short C2 = J2.sp.second;
	
	if (J1.sp==cuElectron) W1 *= EI.Eta1;
	if (J2.sp==cuElectron) W2 *= EI.Eta2;

	cuFP_t GG = cuMunit/amu;
	cuFP_t N0 = GG;
	  
	if (W1>W2) N0 *= W2;
	else       N0 *= W1;

	if (J2.sp == cuElectron and
	    J1.sp != cuElectron) {
	  Prob = p1->datr[J1.I+cuSp0] * EI.Eta2;
	  type = AccumType::ion_electron;
	}
	else if (J1.sp == cuElectron and
		 J2.sp != cuElectron) {
	  Prob = p2->datr[J2.I+cuSp0] * EI.Eta1;
	  type = AccumType::ion_electron;
	}
	else if (J1.sp != cuElectron and
		 J2.sp != cuElectron) {
	  Prob = p1->datr[J1.I+cuSp0] * p2->datr[J2.I+cuSp0];
	  type = AccumType::ion_ion;
	}
	else if (J1.sp == cuElectron and
		 J2.sp == cuElectron) {
	  printf("CRAZY pair: two electrons\n");
	  type = AccumType::electron_electron;
	}
	
	//-----------------------------
	// Parse each interaction type
	//-----------------------------
	  
	if (T == neut_neut) {
#ifdef XC_DEEP2
	  printf("testT: nnDE=%e W=%e Z1=%d Z2=%d\n", 0.0, Prob, Z1, Z2);
#endif
#ifdef XC_DEEP9
	  atomicAdd(&w_countr[T], 1ull);
	  atomicAdd(&w_weight[T], Prob);
#endif
	}
	  
	if (T == neut_elec) {
#ifdef XC_DEEP2
	  if (J1.sp != cuElectron)
	    printf("testT: neDE=%e W=%e Z1=%d\n", 0.0, Prob, Z1);
	  else
	    printf("testT: neDE=%e W=%e Z2=%d\n", 0.0, Prob, Z2);
#endif
#ifdef XC_DEEP9
	  atomicAdd(&w_countr[T], 1ull);
	  atomicAdd(&w_weight[T], Prob);
#endif
	}
	  
	if (T == neut_prot) {
#ifdef XC_DEEP2
	  if (Z1==1 and C1==2)
	    printf("testT: npDE=%e W=%e Z2=%d C2=%d\n", 0.0, Prob, Z2, C2);
	  else
	    printf("testT: npDE=%e W=%e Z1=%d C1=%d\n", 0.0, Prob, Z1, C1);
#endif
#ifdef XC_DEEP9
	    atomicAdd(&w_countr[T], 1ull);
	    atomicAdd(&w_weight[T], Prob);
#endif
	}
	  
	if (T == free_free) {
	    
	  dE = ph * Prob;
	    
	  // Sanity
#ifdef SANITY_DEBUG
	  if (::isnan(dE)) {
	    printf("Crazy dE value in free-free: XE=%e P=%e\n", dE, Prob);
	    dE = 0.0;
	  }
#endif	  
#ifdef XC_DEEP2
	  if (J2.sp == cuElectron)
	    printf("testT: ffDE=%e W=%e Z=%d C=%d\n", dE, Prob, Z1, C1);
	  else
	    printf("testT: ffDE=%e W=%e Z=%d C=%d\n", dE, Prob, Z2, C2);
#endif
#ifdef XC_DEEP9
	  atomicAdd(&w_countr[T], 1ull);
	  atomicAdd(&w_weight[T], Prob);
#endif
	}
	  
	if (T == col_excite) {
	    
	  dE = ph * Prob;
	    
	  // Sanity
#ifdef SANITY_DEBUG
	  if (::isnan(dE)) {
	    printf("Crazy dE value in col excite: XE=%e P=%e\n", ph, Prob);
	    dE = 0.0;
	  }
#endif
#ifdef XC_DEEP2
	  if (J2.sp == cuElectron)
	    printf("testT: ceDE=%e W=%e Z=%d C=%d\n", dE, Prob, Z1, C1);
	  else
	    printf("testT: ceDE=%e W=%e Z=%d C=%d\n", dE, Prob, Z2, C2);
#endif
#ifdef XC_DEEP9
	    atomicAdd(&w_countr[T], 1ull);
	    atomicAdd(&w_weight[T], Prob);
#endif
	} // END: col_excite
	  
	if (T == col_ionize) {
	    
	  // Ion is p1, electron is p2
	  //
	  if (J2.sp == cuElectron) {
	      
	    cuFP_t WW = Prob;
#ifdef SANITY_DEBUG
	    if (J1.I>Nsp-2) {
	      printf("Crazy ionize I1=%d\n", J1.I);
	    }
#endif
	    int pos = fP + J1.I;
	      
	    if (WW < F1._v[pos]) {
	      F1._v[pos  ] -= WW;
	      F1._v[pos+1] += WW;
	    } else {
	      WW = F1._v[pos];
	      F1._v[pos  ]  = 0.0;
	      F1._v[pos+1] += WW;
	    }
	      
#ifdef XC_DEEP9
	    atomicAdd(&w_countr[T], 1ull);
	    atomicAdd(&w_weight[T], WW);
#endif
	    Prob = WW;
	      
	    dE = ph * Prob;
	      
	    // Sanity
#ifdef SANITY_DEBUG
	    if (::isnan(dE)) {
	      printf("Crazy dE value in col ionize: XE=%e P=%e\n", ph, Prob);
	      dE = 0.0;
	    }
#endif	    
	    // The kinetic energy of the ionized electron is lost
	    // from the COM KE
	    //
	    cuFP_t wEta;
	    computeEta(&F1._v[fP], &wEta, elems);
	    wEta = wEta - EI.Eta1;

	    cuFP_t Echg = EI.iE1 * wEta;
#ifdef XC_DEEP0
	    printf("Ionize[1]: W=%e eV=%e,%e sys=%e\n", wEta, EI.iE1, EI.iE2, Echg*N0*eV/cuEunit);
#endif
	    elecAdj[0] += Echg;
	      
	    // Energy for ionized electron comes from COM
	    //
	    dE += Echg;
	      
	    // Sanity
#ifdef SANITY_DEBUG
	    if (::isnan(dE)) {
	      printf("Crazy dE value in col ionize: XE=%e P=%e E1=%e E2=%e\n", ph, Prob, EI.iE1*WW, EI.IE2*WW);
	      dE = 0.0;
	    }
#endif	    
	      
#ifdef XC_DEEP2
	    printf("testT: ciDE=%e W=%e Z=%d C=%d\n", dE, Prob, Z1, C1);
#endif
	  }
	  // END: ion-electron
	    
	  // Ion is p2, electron is p1
	  //
	  else if (J1.sp == cuElectron) {
	      
	    cuFP_t WW = Prob;
	      
#ifdef SANITY_DEBUG
	    if (J2.I > Nsp-2) {
	      printf("Crazy ionize I2=%d\n", J2.I);
	    }
#endif
	    int pos = fP + J2.I;
	      
	    if (WW < F2._v[pos]) {
	      F2._v[pos  ] -= WW;
	      F2._v[pos+1] += WW;
	    } else {
	      WW = F2._v[pos];
	      F2._v[pos  ]  = 0.0;
	      F2._v[pos+1] += WW;
	    }
	      
#ifdef XC_DEEP9
	    atomicAdd(&w_countr[T], 1ull);
	    atomicAdd(&w_weight[T], WW);
#endif
	    Prob = WW;
	      
	    dE = ph * Prob;
	      
	    // Sanity
#ifdef SANITY_DEBUG
	    if (::isnan(dE)) {
	      printf("Crazy dE value in col ionize: XE=%e P=%e\n", XE, Prob);
	      dE = 0.0;
	    }
#endif
	      
	    // The kinetic energy of the ionized electron is lost
	    // from the COM KE
	    //
	    cuFP_t wEta;
	    computeEta(&F2._v[fP], &wEta, elems);
	    wEta = wEta - EI.Eta2;
	    
	    cuFP_t Echg = EI.iE2 * wEta;
#ifdef XC_DEEP0
	    printf("Ionize[2]: W=%e eV=%e,%e sys=%e\n", wEta, EI.iE2, EI.iE1, Echg*N0*eV/cuEunit);
#endif
	    elecAdj[0] += Echg;
	      
	    // Energy for ionized electron comes from COM
	    //
	    dE += Echg;
	      
	    // Sanity
#ifdef SANITY_DEBUG
	    if (::isnan(dE)) {
	      printf("Crazy dE value in col ionize: XE=%e P=%e E1=%e E2=%e\n", XE, Prob, EI.iE1*WW, EI.iE2*WW);
	      dE = 0.0;
	    }
#endif	    
	      
#ifdef XC_DEEP2
	    printf("testT: ciDE=%e W=%e Z=%d C=%d\n", dE, Prob, Z2, C2);
#endif
	  }
	  // END: electron-ion
	    
	  // Sanity check
	  //
	  else {
	    printf("**ERROR: col_ionize without a valid state [no e]: p1=[%d, %d, %d] p2=[%d, %d, %d]\n", Z1, C1, J1.I, Z2, C2, J2.I);
	  }
	  
	} // END: ionize
	  
	if (T == recombine) {
	    
	  if (Prob > 1.0) {
	    printf("In recombine: crazy prob not possible: Prob=%e\n", Prob);
	  }
	    
	  if (J2.sp == cuElectron) {		// Ion is p1
	      
	    cuFP_t WW = Prob;
	      
	    int pos = fP + J1.I;
	      
#ifdef SANITY_DEBUG
	    if (C1<=1 or Z2!=0) {
	      
	      printf("Crazy recombine [p1] (%d %d %d) (%d %d %d) (%d %d) T=%d N=%d\n",
		     Z1, C1, J1.I, Z2, C2, J2.I, T, numxc)
		}
	    
	    if (WW < 0.0 or WW > 1.0)
	      printf("Crazy W: Z1=%d C1=%d I1=%d Z2=%d C2=%d I2=%d: ww=%e f1=%e P0=%e\n",
		     Z1, C1, J1.I, 
		     Z2, C2, J2.I,
		     WW, F1._v[pos], Prob);
#endif
	    if (WW < F1._v[pos]) {
	      F1._v[pos  ] -= WW;
	      F1._v[pos-1] += WW;
	    } else {
	      WW = F1._v[pos];
	      F1._v[pos  ]  = 0.0;
	      F1._v[pos-1] += WW;
	    }
	      
#ifdef XC_DEEP9
	    atomicAdd(&w_countr[T], 1ull);
	    atomicAdd(&w_weight[T], WW);
#endif
	    Prob = WW;
	      
	    // Electron KE lost in recombination is radiated
	    //
	    cuFP_t wEta;
	    computeEta(&F1._v[fP], &wEta, elems);
	    wEta = EI.Eta1 - wEta;
	    
	    cuFP_t Edel = (EI.iE2 - EI.iE1) * wEta;
	    cuFP_t Echg = EI.iE1 * wEta;
	      
#ifdef XC_DEEP0
	    printf("Recombine[1]: W=%e E=%e eV=%e\n", wEta, EI.iE1, Echg);
#endif
	      
	    dE += Edel;
	    elecAdj[1] += Echg;
	      
	    // KE Echg2 + IP is radiated.  Echg2 is lost from the COM
	    // but Echg1 is used as a proxy to conserve internal energy
	    //
	      
	    // Sanity
#ifdef SANITY_DEBUG
	    if (::isnan(dE)) {
	      printf("Crazy dE value in recomb: P=%e\n", Prob);
	      dE = 0.0;
	    }
#endif	    
	      
#ifdef XC_DEEP2
	    printf("testT: rcDE=%e W=%e Z=%d C=%d\n", ph, Prob, Z1, C1);
#endif
	  } // END: ion-electron
	  else if (J1.sp == cuElectron) { // Ion is p2
	      
	    cuFP_t WW = Prob;
	      
	    int pos = fP + J2.I;
	      
#ifdef SANITY_DEBUG
	    if (C2<=1 or Z1!=0) {
	      printf("Crazy recombine [p2] (%d %d %d) (%d %d %d) (%d %d) T=%d N=%d\n",
		     Z1, C1, J1.I, Z2, C2, J2.I, T, numxc);
	    }
	      
	    if (WW < 0.0 or WW > 1.0)
	      printf("Crazy W: Z1=%d C1=%d I1=%d Z2=%d C2=%d I2=%d: ww=%e f2=%e P0=%e cf=%e\n",
		     Z1, C1, J1.I, 
		     Z2, C2, J2.I,
		     WW, F2._v[pos], Prob);
#endif	    
	    if (WW < F2._v[pos]) {
	      F2._v[pos  ] -= WW;
	      F2._v[pos-1] += WW;
	    } else {
	      WW = F2._v[pos];
	      F2._v[pos  ]  = 0.0;
	      F2._v[pos-1] += WW;
	    }
	      
#ifdef XC_DEEP9
	    atomicAdd(&w_countr[T], 1ull);
	    atomicAdd(&w_weight[T], WW);
#endif
	    Prob = WW;
	      
	    // Electron KE lost in recombination is radiated
	    //
	    cuFP_t wEta;
	    computeEta(&F2._v[fP], &wEta, elems);
	    wEta = EI.Eta2 - wEta;

	    cuFP_t Edel = (EI.iE1 - EI.iE2) * wEta;
	    cuFP_t Echg = EI.iE2 * wEta;
	      
	    // Echg2 is lost from the electron pool by the algorithm
	    //
	    dE += Edel;
	    elecAdj[1] += Echg;
	      
#ifdef XC_DEEP0
	    printf("Recombine[2]: W=%e E=%e ke=%e eV=%e sys=%e\n", wEta, EI.iE2, Echg*N0*eV/cuEunit);
#endif
	    // Echg1 + IP is radiated.  Echg1 is lost from the COM but
	    // Echg2 is used as a proxy to conserve internal energy
	    //
	      
	    // Sanity
#ifdef SANITY_DEBUG
	    if (::isnan(dE)) {
	      printf("Crazy dE value in recomb: P=%e\n", Prob);
	      dE = 0.0;
	    }
#endif	    
	      
#ifdef XC_DEEP2
	    printf("testT: rcDE=%e W=%e Z=%d C=%d\n", ph, Prob, Z2, C2);
#endif
	  } // END: electron-ion
	  else {
	    printf("**ERROR: recombine without a valid state [no e]: (%d %d %d) (%d %d %d) T=%d N=%d\n",
		   Z1, C1, J1.I, Z2, C2, J2.I, T, numxc);
	  }
	  // END: unexpected
	  
	} // END: recomb
	

#ifdef XC_DEEP6
	printf("ctest: E=%e\n", EE);
#endif
	// Cumulate ionization and recombation energy adjustment
	//
	IEadjust += (elecAdj[1] - elecAdj[0]) * N0;
	  
	// Recompute electron fraction from possibly new weights
	//
	{
	  cuFP_t sum1 = 0.0, sum2 = 0.0;
	  for (int k=0; k<Nsp; k++) {
	    
	    if (F1._v[fP+k]<0.0)
	      printf("**ERROR: F1[%d]=%e\n", k, F1._v[fP+k]);
	    
	    if (F2._v[fP+k]<0.0)
	      printf("**ERROR: F2[%d]=%e\n", k, F2._v[fP+k]);
	      
	    sum1 += F1._v[fP+k];
	    sum2 += F2._v[fP+k];
	  }
	    
	  // Sanity check
	  //
	  if (fabs(sum1 - 1.0) > 1.0e-10) {
	    printf("**ERROR: sum1=%e\n", sum1-1.0);
	    for (int k=0; k<Nsp; k++) {
	      cuIonElement& E = elems._v[k];
	      printf("**[%d]=%13.6e  %13.6e\n", k, F1._v[fP+k], p1->datr[E.I+cuSp0]);
	    }
	  }
	    
	  if (fabs(sum2 - 1.0) > 1.0e-10) {
	    printf("**ERROR: sum2=%e\n", sum2-1.0);
	    for (int k=0; k<Nsp; k++) {
	      cuIonElement& E = elems._v[k];
	      printf("**[%d]=%13.6e  %13.6e\n", k, F2._v[fP+k], p2->datr[E.I+cuSp0]);
	    }
	  }
	  
	  // Deep sanity check
	  //
	  if (false) {
	    for (int k=0; k<Nsp; k++) {
	      cuIonElement& E = elems._v[k];
	      cuFP_t dif = fabs(F1._v[fP+k] - p1->datr[E.I+cuSp0]);
	      if (dif > 0.2) {
		printf("**WARNING: dF1[%d]=%13.6e  new=%13.6e  old=%13.6e\n", k, dif, F1._v[fP+k], p1->datr[E.I+cuSp0]);
	      }
	      dif = fabs(F2._v[fP+k] - p2->datr[E.I+cuSp0]);
	      if (dif > 0.2) {
		printf("**WARNING: dF2[%d]=%13.6e  new=%13.6e  old=%13.6e\n", k, dif, F2._v[fP+k], p2->datr[E.I+cuSp0]);
	      }
	    }
	  }
	  
	  cuFP_t Sum1 = 0.0, Sum2 = 0.0, Eta1 = 0.0, Eta2 = 0.0;
	  cuFP_t Test1 = 0.0, Test2 = 0.0;
	  for (int k=0; k<Nsp; k++) {
	    cuIonElement& E = elems._v[k];
	    
	    p1->datr[E.I+cuSp0] = F1._v[fP+k]/sum1;
	    p2->datr[E.I+cuSp0] = F2._v[fP+k]/sum2;
	    
	    // Number fraction of ions
	    cuFP_t one = p1->datr[E.I+cuSp0] / cuda_atomic_weights[E.Z];
	    cuFP_t two = p2->datr[E.I+cuSp0] / cuda_atomic_weights[E.Z];
	    
	    // Electron number fraction
	    Eta1 += one * (E.C - 1);
	    Eta2 += two * (E.C - 1);
	      
	    Sum1 += one;
	    Sum2 += two;
	      
	    Test1 += p1->datr[E.I+cuSp0];
	    Test2 += p2->datr[E.I+cuSp0];
	  }
	    
	  // Deep sanity check
	  //
	  if (true) {
	    if (fabs(Test1 - 1.0) > 1.0e-12) {
	      printf("**WARNING: dF1=%13.6e\n", Test1 - 1.0);
	    }
	    if (fabs(Test2 - 1.0) > 1.0e-12) {
	      printf("**WARNING: dF2=%13.6e\n", Test2 - 1.0);
	    }
	  }
	  
	  // Recompute the number of electrons per particle
	  //
	  EI.Eta1 = (Eta1 /= Sum1);
	  EI.Eta2 = (Eta2 /= Sum2);
	  
	  // Reassign the weights for standard Trace algorithm
	  //
	  if (J1.sp==cuElectron) W1 = w1 * EI.Eta1;
	  if (J2.sp==cuElectron) W2 = w2 * EI.Eta2;
	}

	if (J1.sp==cuElectron) {
	  printf("ERROR: particle 1 is electron\n");
	}

	// Convert energy change to system units
	//
	dE *= N0*cuEV/cuEunit;

	if (cuNoCool) dE = 0.0;

	// The ions have the molecular weight in an interaction. The
	// elctrons have the true electron weight, assigned below.
	//
	cuFP_t m1  = EI.Mu1;
	cuFP_t m2  = EI.Mu2;

	cuFP_t v1[3], v2[3];
	
	// Particle 1 is always Ion
	//
	for (int k=0; k<3; k++) v1[k] = p1->vel[k];
	
	// Particle 2 is Ion
	//
	if (type == AccumType::ion_ion) {
	  for (int k=0; k<3; k++) v2[k] = p2->vel[k];
	}
	// Particle 2 is Electron
	//
	else {			
	  m2 = cuda_atomic_weights[0];
	  for (int k=0; k<3; k++) v2[k] = p2->datr[cuElec+k];
	}

	// Perform the scatter
	//
#ifdef EXPLICIT_ECOM
	cudaScatterTraceExplicit
	  (m1, m2, EI.Eta1, EI.Eta2, W1, W2,
	   &E1[0], &E2[0], &v1[0], &v2[0], dE, state);
#else
	cudaScatterTrace
	  (m1, m2, EI.Eta1, EI.Eta2, W1, W2,
	   &E1[0], &E2[0], &v1[0], &v2[0], dE, state);
#endif

	// Copy scattered velocities back to particle
	//
	for (int k=0; k<3; k++) {
	  p1->vel[k] = v1[k];
	  if (type == AccumType::ion_ion) p2->vel[k] = v2[k];
	  else                            p2->datr[cuElec+k] = v2[k];
	}

      }
      // END: pair loop
    }
    // END: interaction loop
    
    for (size_t i=0; i<nbods; i++) {
      cudaParticle* p = &in._v[n0+i];
      if (cuCons>=0) {
	p->datr[cuCons] += (E1[0] + E2[0])/nbods;
	if (cuEcon>=0)
	  p->datr[cuEcon] += (E1[1] + E2[1])/nbods;
	else
	  p->datr[cuCons] += (E1[1] + E2[1])/nbods;
      }
    }


    // Spread deferred (test)
    //
    if (cuSpreadDef) {
      cuFP_t totCons = 0.0, totEcon = 0.0;
      for (size_t i=0; i<nbods; i++) {
	cudaParticle* p = &in._v[n0+i];
	if (cuCons>=0) totCons += p->datr[cuCons];
	if (cuEcon>=0) totEcon += p->datr[cuEcon];
      }
      for (size_t i=0; i<nbods; i++) {
	cudaParticle* p = &in._v[n0+i];
	if (cuCons>=0) p->datr[cuCons] = totCons/nbods;
	if (cuEcon>=0) p->datr[cuEcon] = totEcon/nbods;
      }
    }

#ifdef XC_DEEP3
    deltE._v[cid] = cellEnergy(cid, in, cellI, cellN, elems) - begE;
#endif
  }
  // END: Grid stride loop for cells
}


// Allocate one generator per particle (overkill, could be tuned to
// save memory)
//
void CollideIon::cuda_random_init(int N)
{
  int offset = d_randS.size();	// Current number of generators
  
  if (offset > N*2) {		// Too many generators?
    
    d_randS.resize(N);
    
  }
  
  if (offset < N) {		// Need more generators?
    
    std::cout << "Node " << myid
	      << ": CUDA random: size was " << offset
	      << ", new size will be " << N
	      << std::endl;
    
    d_randS.resize(N);
    
    int count    = N - offset;
    int stride   = count/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    int gridSize = (count+BLOCK_SIZE*stride-1)/(BLOCK_SIZE*stride);
    
    unsigned long long seed = 11 + myid;
    
    initCurand<<<gridSize, BLOCK_SIZE>>>
      (toKernel(d_randS), offset, count, seed);
  }
}


// Compute collisions on the GPU for all cells
//
void * CollideIon::collide_thread_cuda(void * arg)
{
  // Return if no cuda device is assigned by the Component instance
  //
  if (c0->cudaDevice<0) return(NULL);

  // This will only be done once
  //
  cuda_initialize();
  

  // Species constants as defined on device
  //
  const cuSpeciesKey cuProton    {1, 2};
  const cuSpeciesKey cuElectron  {0xffff, 0xffff};
  //                              ^       ^
  //                              |       |
  // 2^16-1 is max ushort---------+-------+
  // as defined in NTC.H
  const cuSpeciesDef electronDef {cuElectron, 0, 0};

  // Get the thread id
  //
  int id = static_cast<int>(((thrd_pass_arguments*)arg)->id);
  
  thread_timing_beg(id);
  
  if (id==0) {
    std::ostringstream sout;
    sout << "Collide::collide: ENTERING cuda collide_thread, T=" << tnow;
    (*barrier)(sout.str(), __FILE__, __LINE__);
  }
  
  // Initialize cell loop diagnostics
  //
  pre_cell_loop(id);
  
  // Start execution timer
  //
  cellTime[id].start();
  
  // Number of cells to process
  //
  size_t Ncells = cellist[id].size();
  
  // Structures for cell boundaries and counts
  //
  thrust::host_vector<int>    cellI, cellN;
  thrust::host_vector<cuFP_t> h_volC, h_tauC;
  
  size_t Pcount = 0, Count = 0, Single = 0;
  typedef std::pair<long int, int> partIndex;
  std::vector<partIndex> bods;
  

  // DEEP DEBUG
  if (false) {
    unsigned elem = 0, celltot = 0, cellsum = 0;
    for (auto v : cellist) {
      elem++;
      celltot += v.size();
    }
    
    std::cout << "[" << myid << "] cells=" << celltot
	      << "/" << elem << std::endl;
    MPI_Reduce(&celltot, &cellsum, 1, MPI_UNSIGNED, MPI_SUM, 0, MPI_COMM_WORLD);
    if (myid==0) std::cout << "[sum] cells=" << cellsum << std::endl;
  }
  // END DEBUG
  
  // Loop over cells to get count of bodies and cells to process.
  // This will be transferred to the GPU device.
  //
  for (unsigned j=0; j<cellist[id].size(); j++ ) {
    
    // The current cell
    //
    pCell *c = cellist[id][j];
    
    // Skip cell if this time has already been computed
    //
    if (c->time >= tnow) {
      continue;
    }
    
    auto number = c->bods.size();
    if (number>1) {
      
      cellI.push_back(Pcount);	//<--Offset into body list for this cell
      cellN.push_back(number);	//<--Number of bodies in this cell
      
				//<--Volume of this cell
      h_volC.push_back(c->Volume());
      //<--Time step for this cell
      h_tauC.push_back(dtime / (1<<c->maxplev));
      
      Pcount += number;		//<--Augment offset for this cell
      
				//<--Create bodies list
      for (auto b : c->bods) bods.push_back(partIndex(b, j));
      
      Count++;
    } else {
      Single++;
    }
  }
  
  if (false) {
    std::cout << "TID=" << std::setw(2) << id
	      << " # cells="  << cellist[id].size()
	      << " # active=" << bods.size()
	      << " # multi =" << Count
	      << " # single=" << Single
	      << std::endl;
  }
  
  // Prepare for cudaParticle staging
  //
  if (c0->host_particles.capacity()<Pcount) c0->host_particles.reserve(Pcount);
  c0->host_particles.resize(Pcount);
  
  // Species map info
  //
  int minSp = std::numeric_limits<int>::max(), maxSp = 0;

  for (auto s : SpList) {
    minSp = std::min<int>(minSp, s.second);
    maxSp = std::max<int>(maxSp, s.second);
  }
  
  int minSp0 = minSp;

  // Augment species position counter
  if (use_elec>=0) {		// for electrons
    minSp = std::min<int>(minSp, use_elec);
    if (elc_cons) maxSp = std::max<int>(maxSp, use_elec+4);
    else          maxSp = std::max<int>(maxSp, use_elec+3);
  } else {
    throw GenericError("use_elec must be set to use CUDA Trace implementation",  __FILE__, __LINE__);
  }
  
  if (use_cons>=0) {
    minSp = std::min<int>(minSp, use_cons);
    maxSp = std::max<int>(maxSp, use_cons);
  }
  
  // Make maxSP +1 beyond the last species weight
  //
  maxSp++;
  
  thrust::host_vector<cuInteract> ilist;
  for (auto s : SpList) {
    speciesKey k = s.first;
    cuSpeciesKey cuk(k);

    int kpos = 0;
    for (kpos=0; kpos<ad.cuIonElem.size(); kpos++)
      if (ad.cuIonElem[kpos].I == s.second) break;

    cuSpeciesDef def1 {cuk, kpos, s.second-minSp0};

    for (auto ss : SpList) {
      speciesKey kk = ss.first;
      cuSpeciesKey cukk(kk);

      for (kpos=0; kpos<ad.cuIonElem.size(); kpos++)
	if (ad.cuIonElem[kpos].I == ss.second) break;

      cuSpeciesDef def2 {cukk, kpos, ss.second-minSp0};

      if (k.second==1) {
	if (kk.second==1 and k.first <= kk.first)
	  ilist.push_back({neut_neut, def1, def2, 0.0});
				    
	// H
	if (k.first==1 and kk.first==1 and kk.second==2)
	  ilist.push_back({neut_prot, def1, def2, 0.0});

	// He
	if (k.first==2 and kk.first==1 and kk.second==2)
	  ilist.push_back({neut_prot, def1, def2, 0.0});
      }
    }

    // The rest are electron interactions

    // Atom must be neutral
    if (k.second==1)
      ilist.push_back({neut_elec, def1, electronDef, 0.0});

    // Atom must be charged for FREE-FREE 
    if (k.second>1 and !(NoDelC & 0x8))
      ilist.push_back({free_free, def1, electronDef, 0.0});

    // Atom must have at least one electron for COLLISIONAL EXCITATION
    if (k.second <= k.first and !(NoDelC & 0x4))
      ilist.push_back({col_excite, def1, electronDef, 0.0});

    // Atom must have at least one electron for IONIZATION
    if (k.second <= k.first and !(NoDelC & 0x2))
      ilist.push_back({col_ionize, def1, electronDef, 0.0});

    // Atom must be charged for RECOMBINATION
    if (k.second>1 and !(NoDelC & 0x1))
      ilist.push_back({recombine, def1, electronDef, 0.0});
  }


  // For deep debugging, only print on first call
  //
  static bool firstime = true;
  //                        ^
  //                        |
  // False for production --+
  //
  if (firstime and myid==0) {
    std::cout << std::string(60, '-') << std::endl
	      << "Interactions" << std::endl
	      << "------------" << std::endl;
    for (auto v : ilist) {
      auto TT = thrust::get<0>(v);
      auto k1 = thrust::get<1>(v);
      auto k2 = thrust::get<2>(v);
      std::cout << std::left
		<< std::setw(20) << interLabels[TT]
		<< " [" << std::setw(6) << k1.sp.first
		<< ", " << std::setw(6) << k1.sp.second << "]"
		<< " [" << std::setw(6) << k2.sp.first
		<< ", " << std::setw(6) << k2.sp.second << "]"
		<< " k1=" << std::setw(3) << k1.k
		<< " I1=" << std::setw(3) << k1.I
		<< " k2=" << std::setw(3) << k2.k
		<< " I2=" << std::setw(3) << k2.I
		<< std::endl;
    }
    std::cout << std::string(60, '-') << std::endl;
    firstime = false;
  }

  // Electron position in cudaParticle datr for convenience
  //
  int ePos = use_elec - minSp;	
  
  // Copy particles to DEVICE
  //
  thrust::host_vector<cuFP_t> h_tauP(Pcount);
  thrust::host_vector<cuFP_t>::iterator pit = h_tauP.begin();
  thrust::host_vector<cudaParticle>::iterator hit = c0->host_particles.begin();
  
  int nOK = 0;
  
  for (auto b : bods) {
    PartPtr h = Particles()[b.first];
    nOK = ParticleHtoD(h, *(hit++), minSp, maxSp);
    if (nOK) break;
    *(pit++) = h_tauC[b.second];
  }
  
  // Try to exit smoothly if particles can't be copied to cudaParticle
  // structures
  //
  if (nOK) {
    if (myid==0) {
      std::cerr << "CollideIon::collide_thread_cuda: "
		<< "Increase CUDA particle attribute size in cudaParticle.cuH"
		<< std::endl;
    }
    MPI_Finalize();
    exit(34);
  }
  
  thrust::device_vector<cuFP_t>       d_tauP(h_tauP);
  thrust::device_vector<cudaParticle> d_part(c0->host_particles);
  
  // Copy cell boundaries and counts to DEVICE
  //
  thrust::device_vector<int>    d_cellI = cellI;
  thrust::device_vector<int>    d_cellN = cellN;
  
  // Grid size computation
  //
  int N        = cellI.size();	// Number of cells
  int stride   = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
  int gridSize = (N+BLOCK_SIZE*stride-1)/(BLOCK_SIZE*stride);
  
  // These do not need copying back
  thrust::device_vector<cuFP_t> d_Ivel2(N), d_Evel2(N);
  thrust::device_vector<cuFP_t> d_PiProb(N*4), d_ABrate(N*4);
  thrust::device_vector<cuFP_t> d_volC(h_volC), d_tauC(h_tauC);
  
  // Initialize per cell info
  //
  cellInitKernel<<<gridSize, BLOCK_SIZE>>>
    (toKernel(d_part),		// Particle array (input)
     toKernel(d_Ivel2),		// Mean squared ion velocity (output)
     toKernel(d_Evel2),		// Mean squared electron velocity (output)
     toKernel(d_PiProb),	// For BN algorithm (output)
     toKernel(d_ABrate),	// For BN algorithm (output)
     toKernel(d_volC),		// Cell volume (input)
     toKernel(d_tauC),		// Cell time step (input)
     toKernel(d_cellI),		// Cell index (input)
     toKernel(d_cellN),		// Cell body count (input)
     toKernel(cuElems));	// Ionization state info (input)
  
  
  // Update random number generators count
  //
  cuda_random_init(N);

  // These are all for device storage do not need copying back
  //
  unsigned int XCsize = ilist.size();
  
  thrust::device_vector<cuFP_t>     d_delph(N), d_Coul4(N*4), d_Nsel(N);

  int esz = cuElems.size();
  thrust::device_vector<cuFP_t>     d_F1(N*esz), d_F2(N*esz);
  thrust::device_vector<cuInteract> d_Init(ilist);

  // Work space for pair selection in kernel
  //
  thrust::device_vector<int>        d_pairs(N*PLIST_LEN);
  thrust::device_vector<float>      d_xccum(N*PLIST_LEN);
  thrust::device_vector<cuFP_t>     d_deltE(N);

#ifdef XC_DEEPT
  std::cout << "**TIME=" << tnow << std::endl;
#endif
  
#ifdef CELL_ECHK
  thrust::device_vector<cuFP_t>     d_Ebeg(N), d_Efin(N);

  totalCellEnergy<<<gridSize, BLOCK_SIZE>>>
  (toKernel(d_part),   toKernel(d_cellI),  toKernel(d_cellN),
   toKernel(d_Ebeg),   toKernel(cuElems));
#endif

  partInteractions<<<gridSize, BLOCK_SIZE>>>
    (toKernel(d_part),   toKernel(d_randS),  toKernel(d_Coul4),  toKernel(d_Nsel),
     toKernel(d_cellI),  toKernel(d_cellN),  toKernel(d_deltE),  toKernel(d_volC),   
     toKernel(d_Ivel2),  toKernel(d_Evel2),  toKernel(xsc_H),    
     toKernel(xsc_He),   toKernel(xsc_pH),   toKernel(xsc_pHe),  
     toKernel(d_PiProb), toKernel(d_ABrate), toKernel(cuElems),  
     toKernel(d_tauC),   toKernel(d_Init),
     toKernel(d_F1),     toKernel(d_F2),
     toKernel(d_pairs),  toKernel(d_xccum)
     );
  
#ifdef CELL_ECHK
  totalCellEnergy<<<gridSize, BLOCK_SIZE>>>
  (toKernel(d_part),   toKernel(d_cellI),  toKernel(d_cellN),
   toKernel(d_Efin),   toKernel(cuElems));

  // Copy diagnostics to host
  //
  thrust::host_vector<cuFP_t> h_Ebeg = d_Ebeg;
  thrust::host_vector<cuFP_t> h_Efin = d_Efin;
  thrust::host_vector<cuFP_t> h_dltE = d_deltE;
  
  int countBad = 0;
  cuFP_t sumEdif = 0.0, sumEtot = 0.0, maxEcel = 0.0;
  for (int n=0; n<N; n++) {
    sumEtot += h_Ebeg[n];
    sumEdif += h_Efin[n] - h_Ebeg[n];
    if (fabs(h_Efin[n]/h_Ebeg[n] - 1.0) > EDEL_TOL) countBad++;
    cuFP_t emean = 0.5*(h_Ebeg[n] + h_Efin[n]);
    cuFP_t reldE = fabs(h_dltE[n])/emean;
    maxEcel = std::max<cuFP_t>(maxEcel, reldE);
    if (reldE > EDEL_TOL) {
      std::cout << "[" << std::setw(4) << n << "] delta="
		<< std::setw(16) << h_dltE[n] << " (" << reldE << ")"
		<< std::endl;
    }
  }

  if (countBad)
    std::cout << "Total energy dif=" << sumEdif << " total=" << sumEtot
	      << " [" << sumEdif/sumEtot << "] max/cell=" << maxEcel
	      << " | bad: " << countBad << "/" << N
	      << std::endl;
#endif


#ifdef XC_DEEP9
  {
    hipMemcpyFromSymbol(&xc_counter[0], HIP_SYMBOL(w_countr), 11*sizeof(unsigned long long));
    hipMemcpyFromSymbol(&xc_weight[0],  HIP_SYMBOL(w_weight), 11*sizeof(cuFP_t));

    setCountersToZero<<<1, 1>>>();
  }
#endif
  
  // Photoionization
  //
  if (use_photoIB) {
    photoIonizeKernel<<<gridSize, BLOCK_SIZE>>>
      (toKernel(d_part),  toKernel(d_tauP), 
       toKernel(d_cellI), toKernel(d_cellN),
       toKernel(d_randS), toKernel(cuElems));
  }
  
  // Finally, copy back particles to host
  // 
  c0->host_particles = d_part;
  
  // Copy particles to HOST
  //
  unsigned velDiff = 0, velTotl = 0, spcDiff = 0, spcTotl = 0;
  hit = c0->host_particles.begin();

  for (auto p : c0->host_particles) {
    long int curr = p.indx;
    PartPtr h = Particles()[curr];
    
    if (false) {
      std::cout << " pos dev: (";
      for (int k=0; k<3; k++) std::cout << std::setw(18) << p.pos[k];
      std::cout << ")" << std::endl << "pos host: (";
      for (int k=0; k<3; k++) std::cout << std::setw(18) << h->pos[k];
      std::cout << ")" << std::endl << " vel dev: (";
      for (int k=0; k<3; k++) std::cout << std::setw(18) << p.vel[k];
      std::cout << ")" << std::endl << "vel host: (";
      for (int k=0; k<3; k++) std::cout << std::setw(18) << h->vel[k];
      std::cout << ")" << std::endl << " elc dev: (";
      for (int k=0; k<3; k++) std::cout << std::setw(18) << p.datr[k+ePos];
      std::cout << ")" << std::endl << "elc host: (";
      for (int k=0; k<3; k++) std::cout << std::setw(18) << h->dattrib[k+use_elec];
      std::cout << ")" << std::endl;
    }

    if (false) {
      bool diff = false;
      
      for (int k=0; k<3; k++) {
	if (fabs(p.vel[k] - h->vel[k]) >
	    1.0e-10*(fabs(p.vel[k]) + fabs(h->vel[k])) ) diff = true;
	if (fabs(p.datr[k+ePos] - h->dattrib[k+use_elec]) >
	    1.0e-10*(fabs(p.datr[k+ePos]) + fabs(h->dattrib[k+use_elec])) ) diff = true;
      }
      if (diff) velDiff++;
      velTotl++;
    }

    if (true) {
      bool diff = false;

      for (int s=0; s<SpList.size(); s++) {
	cuFP_t dif = p.datr[s + Sp0] - h->dattrib[s + Sp0base];
	if (fabs(dif) > 1.0e-8) diff = true;
      }
      if (diff) spcDiff++;
      spcTotl++;
    }
    
    ParticleDtoH(p, h, minSp, maxSp);
  }

  if (false) {
    std::cout << "[" << myid << "] diffs=" << velDiff
	      << "/" << velTotl << std::endl;
  }
  
  if (false and spcDiff) {
    std::cout << "[" << myid << "] diffs=" << spcDiff
	      << "/" << spcTotl << std::endl;
  }

  if (id==0) {
    std::ostringstream sout;
    sout << "Collide::collide: AFTER cell loop, T=" << tnow;
    (*barrier)(sout.str(), __FILE__, __LINE__);
  }
  
  cellSoFar[id] = cellTime[id].stop();
  
  // Diagnostics at end of cell loop
  //
  post_cell_loop(id);
  
  // Copy diagnostics to host
  //
  thrust::host_vector<cuFP_t> h_Coul4 = d_Coul4;
  thrust::host_vector<cuFP_t> h_Nsel  = d_Nsel;
  
  for (int n=0; n<N; n++) {
    for (int l=0; l<4; l++) tauD[id][l].push_back(h_Coul4[n*4+l]);
    selD[id].push_back(h_Nsel[n]);
  }
  
  thread_timing_end(id);
  
  return (NULL);
}

// -*- C++ -*-
