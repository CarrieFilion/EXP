#include <EmpOrth9thd.h>

#include <iostream>
#include <iomanip>
#include <map>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

void EmpCylSL::initialize_cuda
(std::vector<hipArray_t>& cuArray,
 thrust::host_vector<hipTextureObject_t>& tex
 )
{
  // Number of texture arrays
  //
  size_t ndim = (MMAX+1)*rank3;

  // Interpolation data array
  //
  cuArray.resize(ndim);

  // Create texture objects
  //
  tex.resize(ndim);
  thrust::fill(tex.begin(), tex.end(), 0);

  hipTextureDesc texDesc;

  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.addressMode[2] = hipAddressModeClamp;
  
  // Temporary storage
  //
  float *d_Interp;
  cuda_safe_call(hipMalloc((void **)&d_Interp, NUMX*NUMY*6*sizeof(float)),
		 __FILE__, __LINE__,o
		 "Error allocating d_Interp for texture construction");
  
  std::vector<float> h_buffer(NUMX*NUMY*6, 0.0);
  size_t k = 0;

  for (size_t mm=0; mm<=MMAX; mm++) {

    for (size_t n=0; n<rank3; n++) {

      // Copy table to flat array
      //
      for (int j=0; j<NUMY; j++) {
	for (int i=0; i<NUMX; i++) {
	  h_buffer[i+j*NUMX              ]   = potC   [mm][n][i][j];
	  h_buffer[i+j*NUMX + NUMX*NUMY  ]   = rforceC[mm][n][i][j];
	  h_buffer[i+j*NUMX + NUMX*NUMY*2]   = zforceC[mm][n][i][j];
	  if (mm) {
	    h_buffer[i+j*NUMX + NUMX*NUMY*3] = potS   [mm][n][i][j];
	    h_buffer[i+j*NUMX + NUMX*NUMY*4] = rforceS[mm][n][i][j];
	    h_buffer[i+j*NUMX + NUMX*NUMY*5] = zforceS[mm][n][i][j];
	  }
	}
      }
      
      // Copy data to device
      cuda_safe_call(hipMemcpy(d_Interp, &h_buffer[0], NUMX*NUMY*6*sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying texture table to device");

      // hipArray Descriptor
      //
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

      // cuda Array
      //
      cuda_safe_call(hipMalloc3DArray(&cuArray[k], &channelDesc, make_hipExtent(NUMX, NUMY, 6), 0), __FILE__, __LINE__, "Error allocating cuArray for 3d texture");

      // Array creation
      //
      hipMemcpy3DParms copyParams = {0};
      
      copyParams.srcPtr   = make_hipPitchedPtr(d_Interp, NUMX*sizeof(float), NUMX, NUMY);
      copyParams.dstArray = cuArray[k];
      copyParams.extent   = make_hipExtent(NUMX, NUMY, 6);
      copyParams.kind     = hipMemcpyDeviceToDevice;

      cuda_safe_call(hipMemcpy3D(&copyParams), __FILE__, __LINE__, "Error in copying 3d pitched array");

      hipResourceDesc resDesc;

      memset(&resDesc, 0, sizeof(hipResourceDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array  = cuArray[k];

      cuda_safe_call
	(hipCreateTextureObject(&tex[k], &resDesc, &texDesc, NULL),
	 __FILE__, __LINE__, "Failure in 2d texture creation");
      
      k++;

    } // radial order loop

  } // harmonic subspace loop


  assert(k == ndim);

  /*
  if (false) {
    printf("**HOST** Texture compare\n");
    {
      for (int l : {0, 1, 2}) {
	for (int j=0; j<10; j++) {
	  for (int i : {3980, 3990, 3995, 3999}) 
	    printf("%5d %5d %5d %13.7e\n", l, j, i, table[l].ef[j+1][i]);
	}
      }
    }
  }
  */

  cuda_safe_call(hipFree(d_Interp), __FILE__, __LINE__, "Failure freeing device memory");
}


cudaMappingConstants EmpCylSL::getCudaMappingConstants()
{
  cudaMappingConstants ret;

  ret.rscale = ASCALE;
  ret.hscale = HSCALE;
  ret.xmin   = XMIN;
  ret.xmax   = XMAX;
  ret.ymin   = YMIN;
  ret.ymax   = YMAX;
  ret.numr   = 0;
  ret.numx   = NUMX;
  ret.numy   = NUMY;
  ret.dxi    = dX;
  ret.dyi    = dY;
  ret.cmap   = (CMAP ? 1 : 0);

  return ret;
}
