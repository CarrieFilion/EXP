#include "hip/hip_runtime.h"
#include <Component.H>
#include "expand.h"

#include <boost/make_shared.hpp>

unsigned Component::cudaStreamData::totalInstances=0;

Component::cudaStreamData::cudaStreamData()
{
  // Not sure why this breaks thrust, but it does . . .
  /*
  cuda_safe_call(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking),
		 __FILE__, __LINE__,
		 "Component::cudaStreamData: error creating stream");
  */
  // Need blocking until thrust bug in binary search is fixed
  cuda_safe_call(hipStreamCreate(&stream),
		 __FILE__, __LINE__,
		 "Component::cudaStreamData: error creating stream");
  instance = totalInstances++;
}

Component::cudaStreamData::~cudaStreamData()
{
  cuda_safe_call(hipStreamDestroy(stream), __FILE__, __LINE__,
		 "Component::cudaStreamData: error destroying stream");
  totalInstances--;
}

void Component::cuda_initialize()
{
  // Initialize streams
  //
  cuRingData.resize(cuStreams);
  cuRing = boost::make_shared<cuRingType>(cuRingData);
}

struct LevelFunctor
{
  __host__ __device__
  unsigned operator()(const cudaParticle &p) const
  {
    return p.level;
  }
};


template<typename Iterator, typename Pointer>
__global__
void getLower(Iterator first, Iterator last, int val, Pointer result)
{
  auto it  = thrust::lower_bound(thrust::cuda::par, first, last, val);
  *result  = thrust::distance(first, it);
}

template<typename Iterator, typename Pointer>
__global__
void getUpper(Iterator first, Iterator last, int val, Pointer result)
{
  auto it  = thrust::upper_bound(thrust::cuda::par, first, last, val);
  *result  = thrust::distance(first, it);
}

std::pair<unsigned int, unsigned int>
Component::CudaSortByLevel(Component::cuRingType cr, int minlev, int maxlev)
{
  std::pair<unsigned, unsigned> ret;

  try {
    auto exec = thrust::cuda::par.on(cr->stream);
    
    thrust::device_vector<cudaParticle>::iterator
      pbeg = cr->cuda_particles.begin(),
      pend = cr->cuda_particles.end();
    
    thrust::sort(exec, pbeg, pend, LessCudaLev());
    
    // Convert from cudaParticle to a flat vector to prevent copying
    // the whole particle structure in getBound.  Perhaps this
    // temporary should be part of the data storage structure?
    thrust::device_vector<unsigned> lev(cr->cuda_particles.size());

    thrust::transform(exec, pbeg, pend, lev.begin(), cuPartToLevel());

    // Perform in the sort on the int vector of levels on the GPU
    thrust::device_vector<unsigned> retV(1);

    // Use this single thread call to maintain synchronization with cr->stream
    //
    getLower<<<1, 1, 0, cr->stream>>>(lev.begin(), lev.end(), minlev, &retV[0]);
				// Wait for completion before memcpy
    hipStreamSynchronize(cr->stream); ret.first = retV[0];
				// If maxlev==multistep: upper bound
				// is at end, so skip explicit computation
    if (maxlev < multistep) {
      getUpper<<<1, 1, 0, cr->stream>>>(lev.begin(), lev.end(), maxlev, &retV[0]);
      hipStreamSynchronize(cr->stream); ret.second = retV[0];
    } else {
      ret.second = thrust::distance(pbeg, pend);
    }
    
  }
  catch(std::bad_alloc &e) {
    std::cerr << "Ran out of memory while sorting" << std::endl;
    exit(-1);
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  return ret;
}

void Component::CudaSortBySequence(Component::cuRingType cr)
{
  thrust::device_vector<cudaParticle>::iterator
    pbeg = cr->cuda_particles.begin(),
    pend = cr->cuda_particles.end();

  thrust::sort(thrust::cuda::par.on(cr->stream), pbeg, pend, LessCudaSeq());
}

void Component::ParticlesToCuda()
{
  auto npart = Particles().size();
  
  if (host_particles.capacity()<npart) host_particles.reserve(npart);
  host_particles.resize(npart);

  hostPartItr hit = host_particles.begin();
  for (auto pit : Particles()) {
    ParticleHtoD(pit.second, *(hit++));
  }
}

void Component::HostToDev(Component::cuRingType cr)
{
  auto npart = thrust::distance(cr->first, cr->last);
  
  if (npart) {		  // Don't bother trying to copy zero particles

    if (cr->cuda_particles.capacity()<npart) cr->cuda_particles.reserve(npart);
    cr->cuda_particles.resize(npart);
  
    hipMemcpyAsync(thrust::raw_pointer_cast(&cr->cuda_particles[0]),
		    thrust::raw_pointer_cast(&(*cr->first)),
		    npart*sizeof(cudaParticle),
		    hipMemcpyHostToDevice, cr->stream);
  }
}

void Component::DevToHost(Component::cuRingType cr)
{
  auto npart = thrust::distance(cr->first, cr->last);
  
  if (npart) {		  // Don't bother trying to copy zero particles

    hipMemcpyAsync(thrust::raw_pointer_cast(&(*cr->first)),
		    thrust::raw_pointer_cast(&cr->cuda_particles[0]),
		    npart*sizeof(cudaParticle),
		    hipMemcpyDeviceToHost, cr->stream);
  }
}


void Component::CudaToParticles()
{
  for (auto v : host_particles) ParticleDtoH(v, particles[v.indx]);
}

struct cudaZeroAcc : public thrust::unary_function<cudaParticle, cudaParticle>
{
  __host__ __device__
  cudaParticle operator()(cudaParticle& p)
  {
    for (size_t k=0; k<3; k++) p.acc[k] = 0.0;
    p.pot = p.potext = 0.0;
    return p;
  }
};

void Component::ZeroPotAccel(int minlev)
{
  #pragma message "Please ignore the 'statement unreachable' warning here"
  
  return;			// Don't need this now, since this
				// duplicates zeroing performed on
				// host
  // Copy particles to host vector
  //
  ParticlesToCuda();

  // Loop over bunches
  //
  size_t psize  = host_particles.size();

  Component::hostPartItr begin = host_particles.begin();
  Component::hostPartItr first = begin;
  Component::hostPartItr last  = begin;
  Component::hostPartItr end   = host_particles.end();

  if (psize <= bunchSize) last = end;
  else std::advance(last, bunchSize);

  Component::cuRingType cr = *cuRing.get();

  while (thrust::distance(first, last)) {
    
    cr->first = first;
    cr->last  = last;

    // Copy bunch to device
    //
    HostToDev(cr);

    if (multistep) {
      std::pair<unsigned int, unsigned int>
	ret = CudaSortByLevel(cr, minlev, multistep);
      
      thrust::transform(thrust::cuda::par.on(cr->stream),
			cr->cuda_particles.begin()+ret.first, cr->cuda_particles.end(),
			cr->cuda_particles.begin()+ret.first, cudaZeroAcc());
    } else {
      thrust::transform(thrust::cuda::par.on(cr->stream),
			cr->cuda_particles.begin(), cr->cuda_particles.end(),
			cr->cuda_particles.begin(), cudaZeroAcc());
    }

    // Copy device to host
    //
    DevToHost(cr);

    // Advance iterators
    //
    first = last;
    size_t nadv = thrust::distance(first, end);
    if (nadv <= bunchSize) last = end;
    else thrust::advance(last, bunchSize);

    // Advance stream iterators
    //
    cr++;
  }

  CudaToParticles();
}
