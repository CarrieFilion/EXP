#include "hip/hip_runtime.h"
#include <Component.H>
#include "expand.h"
#include "cudaParticle.cuH"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#include <boost/make_shared.hpp>

unsigned Component::cudaStreamData::totalInstances=0;

Component::cudaStreamData::cudaStreamData()
{
  // Not sure why this breaks thrust, but it does . . .
  /*
  cuda_safe_call(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking),
		 __FILE__, __LINE__,
		 "Component::cudaStreamData: error creating stream");
  */

  // Need blocking until thrust bug in binary search is fixed
  cuda_safe_call(hipStreamCreate(&stream),
		 __FILE__, __LINE__,
		 "Component::cudaStreamData: error creating stream");
  instance = totalInstances++;
}

Component::cudaStreamData::~cudaStreamData()
{
  cuda_safe_call(hipStreamDestroy(stream), __FILE__, __LINE__,
		 "Component::cudaStreamData: error destroying stream");
  totalInstances--;
}

void Component::cuda_initialize()
{
  cuStream = boost::make_shared<cudaStreamData>();
}

struct LevelFunctor
{
  int _t;

  LevelFunctor(int t=0) : _t(t) {}

  __host__ __device__
  int operator()(const cudaParticle &p) const
  {
    return p.lev[_t];
  }
};


Component::I2vec Component::CudaSortLevelChanges(Component::cuSharedStream cr)
{
  // The plan: for the current active level search above and below for
  // particles for correction to coefficient matrix
  //
  // 1. Sort all particles by current level
  // 2. Get indices to range for each level L
  // 3. Within each level L, compute the ranges for changes,
  //    delta L = [-L, multistep-L]
  // 4. For each (L, delta L), compute the coefficient changes and
  //    apply to the appropriate coefficient matrices

  I2vec ret(multistep+1);
  for (auto & v : ret) v.resize(multistep+1);

  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    thrust::device_vector<cudaParticle>::iterator
      pbeg = cuStream->cuda_particles.begin(),
      pend = cuStream->cuda_particles.end();
    
    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      thrust::sort(pbeg, pend, LessCudaLev2());
    } else {
      thrust::sort(exec, pbeg, pend, LessCudaLev2());
    }
    
    pbeg = cuStream->cuda_particles.begin();
    pend = cuStream->cuda_particles.end();

    cudaParticle trg;

    for (int target=0; target<=multistep; target++) {

      trg.lev[0] = target;

      for (int del=0; del<=multistep; del++) {

	if (del==target) {
	  ret[target][del] = {0, 0};
	  continue;
	}
	
	trg.lev[1] = del;

	thrust::device_vector<cudaParticle>::iterator lo, hi;

	if (thrust_binary_search_workaround) {
	  hipStreamSynchronize(cuStream->stream);
	  lo  = thrust::lower_bound(pbeg, pend, trg, LessCudaLev2());
	} else {
	  lo = thrust::lower_bound(exec, pbeg, pend, trg, LessCudaLev2());
	}
	
	hipStreamSynchronize(cuStream->stream);

	if (thrust_binary_search_workaround) {
	  hi = thrust::upper_bound(pbeg, pend, trg, LessCudaLev2());
	} else {
	  hi = thrust::upper_bound(exec, pbeg, pend, trg, LessCudaLev2());
	}

	hipStreamSynchronize(cuStream->stream);

	ret[target][del] = {thrust::distance(pbeg, lo), 
			    thrust::distance(pbeg, hi)};
      }
    }

  }
  catch(std::bad_alloc &e) {
    std::cerr << "Ran out of memory while sorting" << std::endl;
    exit(-1);
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  if (false) {
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
    std::cout << "--- " << name << " [" << myid << "]" << std::endl;
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
    for (int m1=0; m1<=multistep; m1++) {
      for (int m2=0; m2<=multistep; m2++) {
	std::ostringstream sout;
	sout << "(" << ret[m1][m2].first << ", " << ret[m1][m2].second << ")";
	std::cout << std::setw(15) << sout.str();
      }
      std::cout << std::endl;
    }
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
  }

  return ret;
}


void Component::CudaSortByLevel()
{
  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    thrust::device_vector<cudaParticle>::iterator
      pbeg = cuStream->cuda_particles.begin(),
      pend = cuStream->cuda_particles.end();
    
    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      thrust::sort(pbeg, pend, LessCudaLev());
    } else {
      thrust::sort(exec, pbeg, pend, LessCudaLev());
    }
    
    // Convert from cudaParticle to a flat vector to prevent copying
    // the whole particle structure in getBound.  Perhaps this
    // temporary should be part of the data storage structure?
    //
    cuStream->levList.resize(cuStream->cuda_particles.size());

    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      thrust::transform(pbeg, pend, cuStream->levList.begin(), cuPartToLevel());
    } else {
      thrust::transform(exec, pbeg, pend, cuStream->levList.begin(), cuPartToLevel());
    }
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }

}

std::pair<unsigned int, unsigned int>
Component::CudaGetLevelRange(int minlev, int maxlev)
{
  std::pair<unsigned, unsigned> ret;

  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);

    // Get unsigned from input
    //
    unsigned int minl = static_cast<unsigned>(minlev);
    unsigned int maxl = static_cast<unsigned>(maxlev);

    thrust::device_vector<int>::iterator lbeg = cuStream->levList.begin();
    thrust::device_vector<int>::iterator lend = cuStream->levList.end();
    thrust::device_vector<int>::iterator lo, hi;

    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      lo  = thrust::lower_bound(lbeg, lend, minl);
    } else {
      lo = thrust::lower_bound(exec, lbeg, lend, minl);
    }
	
    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      hi = thrust::upper_bound(lbeg, lend, maxl);
    } else {
      hi = thrust::upper_bound(exec, lbeg, lend, maxl);
    }

    ret.first  = thrust::distance(lbeg, lo);
    ret.second = thrust::distance(lbeg, hi);
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  return ret;
}

void Component::CudaSortBySequence(Component::cuSharedStream cr)
{
  thrust::device_vector<cudaParticle>::iterator
    pbeg = cr->cuda_particles.begin(),
    pend = cr->cuda_particles.end();

  thrust::sort(thrust::cuda::par.on(cr->stream), pbeg, pend, LessCudaSeq());
}

void Component::ParticlesToCuda(PartMap::iterator beg, PartMap::iterator fin)
{
  if (step_timing and use_cuda) comp->timer_cuda.start();

  auto npart = std::distance(beg, fin);
  
  if (host_particles.capacity()<npart) host_particles.reserve(npart);
  host_particles.resize(npart);

  hostPartItr hit = host_particles.begin();
  for (auto pit=beg; pit!=fin; pit++) {
    ParticleHtoD(pit->second, *(hit++));
  }

  if (step_timing and use_cuda) comp->timer_cuda.stop();
}

void Component::HostToDev(Component::cuSharedStream cr)
{
  auto npart = thrust::distance(cr->first, cr->last);
  
  if (npart) {		  // Don't bother trying to copy zero particles

    if (cr->cuda_particles.capacity()<npart) cr->cuda_particles.reserve(npart);
    cr->cuda_particles.resize(npart);
  
    hipMemcpyAsync(thrust::raw_pointer_cast(&cr->cuda_particles[0]),
		    thrust::raw_pointer_cast(&(*cr->first)),
		    npart*sizeof(cudaParticle),
		    hipMemcpyHostToDevice, cr->stream);
  }
}

void Component::DevToHost(Component::cuSharedStream cr)
{
  auto npart = thrust::distance(cr->first, cr->last);
  
  if (npart) {		  // Don't bother trying to copy zero particles

    hipMemcpyAsync(thrust::raw_pointer_cast(&(*cr->first)),
		    thrust::raw_pointer_cast(&cr->cuda_particles[0]),
		    npart*sizeof(cudaParticle),
		    hipMemcpyDeviceToHost, cr->stream);
  }
}


void Component::CudaToParticles(hostPartItr beg, hostPartItr end)
{
  if (step_timing and use_cuda) comp->timer_cuda.start();

  // DEBUG PRINTING
  const int imax = 0;
  int icnt = 0;

  for (hostPartItr v=beg; v!=end; v++) {
    cudaParticle & p = *v;
    if (icnt < imax) {
      std::cout << "[" << icnt++ << ", " << myid << "] " << p << std::endl;
    }
    ParticleDtoH(p, particles[p.indx]);
  }

  if (step_timing and use_cuda) comp->timer_cuda.stop();
}

struct cudaZeroAcc : public thrust::unary_function<cudaParticle, cudaParticle>
{
  __host__ __device__
  cudaParticle operator()(cudaParticle& p)
  {
    for (size_t k=0; k<3; k++) p.acc[k] = 0.0;
    p.pot = p.potext = 0.0;
    return p;
  }
};

void Component::ZeroPotAccel(int minlev)
{
  size_t psize  = particles.size();
  
  if (multistep) {
    std::pair<unsigned int, unsigned int>
      ret = CudaGetLevelRange(minlev, multistep);
    
    thrust::transform(// thrust::cuda::par.on(cuStream->stream),
		      thrust::cuda::par,
		      cuStream->cuda_particles.begin()+ret.first, cuStream->cuda_particles.end(),
		      cuStream->cuda_particles.begin()+ret.first, cudaZeroAcc());
  } else {
    thrust::transform(// thrust::cuda::par.on(cuStream->stream),
		      thrust::cuda::par,
		      cuStream->cuda_particles.begin(), cuStream->cuda_particles.end(),
		      cuStream->cuda_particles.begin(), cudaZeroAcc());
  }
  
}

struct getMass : public thrust::unary_function<cudaParticle, cuFP_t>

{
 __host__ __device__
 cuFP_t operator()(const cudaParticle& p) const
  {
    return p.mass;
  }
};

struct getPos : public thrust::unary_function<cudaParticle, cuFP_t>
{
  int _t;
  getPos(int t) : _t(t) {}

 __host__ __device__
 cuFP_t operator()(const cudaParticle& p) const
  {
    return p.mass * p.pos[_t];
  }
};

struct getVel : public thrust::unary_function<cudaParticle, cuFP_t>
{
  int _t;
  getVel(int t) : _t(t) {}

 __host__ __device__
 cuFP_t operator()(const cudaParticle& p) const
  {
    return p.mass * p.vel[_t];
  }
};

struct getAcc : public thrust::unary_function<cudaParticle, cuFP_t>
{
  int _t;
  getAcc(int t) : _t(t) {}

 __host__ __device__
 cuFP_t operator()(const cudaParticle& p) const
  {
    return p.mass * p.acc[_t];
  }
};

void Component::fix_positions_cuda(unsigned mlevel)
{
				// Zero center
  for (int i=0; i<3; i++) center[i] = 0.0;

  				// Zero variables
  mtot = 0.0;
  for (int k=0; k<dim; k++) com[k] = cov[k] = coa[k] = 0.0;

				// Zero multistep counters at and
				// above this level
  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    for (unsigned mm=mlevel; mm<=multistep; mm++) {

      cudaParticle trg;
      trg.lev[0] = mm;
      
      thrust::device_vector<cudaParticle>::iterator
	pbeg = cuStream->cuda_particles.begin(),
	pend = cuStream->cuda_particles.end();
    
      thrust::device_vector<cudaParticle>::iterator lo, hi;

      hipStreamSynchronize(cuStream->stream);

      if (thrust_binary_search_workaround) {
	hipStreamSynchronize(cuStream->stream);
	lo  = thrust::lower_bound(pbeg, pend, trg, LessCudaLev());
      } else {
	lo = thrust::lower_bound(exec, pbeg, pend, trg, LessCudaLev());
      }
      
      hipStreamSynchronize(cuStream->stream);

      if (thrust_binary_search_workaround) {
	hi = thrust::upper_bound(pbeg, pend, trg, LessCudaLev());
      } else {
	hi = thrust::upper_bound(exec, pbeg, pend, trg, LessCudaLev());
      }
      
      com_mas[mm] = thrust::transform_reduce(lo, hi, getMass(), 0.0, thrust::plus<cuFP_t>());
      for (unsigned k=0; k<3; k++)  {
	com_lev[3*mm+k] = thrust::transform_reduce(lo, hi, getPos(k), 0.0, thrust::plus<cuFP_t>());
	cov_lev[3*mm+k] = thrust::transform_reduce(lo, hi, getVel(k), 0.0, thrust::plus<cuFP_t>());
	coa_lev[3*mm+k] = thrust::transform_reduce(lo, hi, getAcc(k), 0.0, thrust::plus<cuFP_t>());
      }
    }
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  std::vector<double> com1(3, 0.0), cov1(3, 0.0), coa1(3, 0.0);
  double              mtot1 = 0.0;

  for (unsigned mm=0; mm<=multistep; mm++) {
    for (int k=0; k<3; k++) {
      com1[k] += com_lev[3*mm + k];
      cov1[k] += cov_lev[3*mm + k];
      coa1[k] += coa_lev[3*mm + k];
    }
    mtot1 += com_mas[mm];
  }

  MPI_Allreduce(&mtot1, &mtot, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&com1[0], com, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&cov1[0], cov, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&coa1[0], coa, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
    
  if (VERBOSE>5) {
				// Check for NaN
    bool com_nan = false, cov_nan = false, coa_nan = false;
    for (int k=0; k<3; k++)
      if (std::isnan(com[k])) com_nan = true;
    for (int k=0; k<3; k++)
      if (std::isnan(cov[k])) cov_nan = true;
    for (int k=0; k<3; k++)
      if (std::isnan(coa[k])) coa_nan = true;
    if (com_nan && myid==0)
      cerr << "Component [" << name << "] com has a NaN" << endl;
    if (cov_nan && myid==0)
      cerr << "Component [" << name << "] cov has a NaN" << endl;
    if (coa_nan && myid==0)
      cerr << "Component [" << name << "] coa has a NaN" << endl;
  }
				// Compute component center of mass and
				// center of velocity, and center of accel

  if (mtot > 0.0) {
    for (int k=0; k<dim; k++) com[k]  /= mtot;
    for (int k=0; k<dim; k++) cov[k]  /= mtot;
    for (int k=0; k<dim; k++) coa[k]  /= mtot;
  }

  if (com_system and not consp) {
    for (int k=0; k<dim; k++) com0[k] = com[k];
    for (int k=0; k<dim; k++) cov0[k] = cov[k];
  }

  if (com_system) {	   // Use local center of accel for com update
    for (int k=0; k<dim; k++) acc0[k]  = coa[k];
  } else {			// No mass, no acceleration?
    for (int k=0; k<dim; k++) acc0[k]  = 0.0;
  }

  if ((EJ & Orient::CENTER) && !EJdryrun) {
    Vector ctr = orient->currentCenter();
    bool ok    = true;
    for (int i=0; i<3; i++) {
      if (std::isnan(ctr[i+1])) ok = false;
    } 
    if (ok) {
      for (int i=0; i<3; i++) center[i] += ctr[i+1];
    } else if (myid==0) {
      cout << "Orient: center failure, T=" << tnow 
	   << ", adjustment skipped" << endl;
    }
  }
}

struct testCountLevel :  public thrust::unary_function<cudaParticle, int>
{
  int _l;

  testCountLevel(int l) : _l(l) {}

  __host__ __device__
  int operator()(const cudaParticle& p) const
  {
    if (p.lev[0] == _l) return 1;
    return 0;
  }
};


void Component::print_level_lists_cuda(double T)
{
				// Retrieve counts per level
  std::vector<int> cntr(multistep+1);
  for (int m=0; m<=multistep; m++) {
    cntr[m] = thrust::transform_reduce(cuStream->cuda_particles.begin(),
				       cuStream->cuda_particles.end(),
				       testCountLevel(m),
				       0, thrust::plus<int>());
  }

  if (myid==0) {
				// Sum reduce to root
    MPI_Reduce(MPI_IN_PLACE, &cntr[0], multistep+1, MPI_INT, MPI_SUM,
	       0, MPI_COMM_WORLD);

    int tot=0;
    for (int n=0; n<=multistep; n++) tot += cntr[n];

    if (tot) {

      std::ostringstream ofil;
      ofil << runtag << ".levels";
      std::ofstream out(ofil.str().c_str(), ios::app);

      int sum=0;
      out << setw(60) << setfill('-') << '-' << endl;
      std::ostringstream sout;
      sout << "--- Component <" << name 
	   << ", " << id  << ">, T=" << T;
      out << std::setw(60) << std::left << sout.str().c_str() << std::endl;
      out << std::setw(60) << '-' << std::endl << std::setfill(' ');
      out << std::setw(3)  << "L" 
	  << std::setw(10) << "Number" 
	  << std::setw(10) << "dN/dL" 
	  << std::setw(10) << "N(<=L)"
	  << std::endl
	  << std::setw(60) << std::setfill('-') << '-'
	  << std::endl << std::setfill(' ');
      for (int n=0; n<=multistep; n++) {
	sum += cntr[n];
	out << std::setw(3)  << n 
	    << std::setw(10) << cntr[n] << std::setprecision(3) << std::fixed
	    << std::setw(10) << static_cast<double>(cntr[n])/tot
	    << std::setw(10) << static_cast<double>(sum)    /tot;
	out << std::endl;
      }
      out << std::endl << std::setw(3) << "T" << std::setw(10) << tot
	  << std::endl << std::endl << std::right;
    } else {
      std::cout << "print_level_lists [" << name 
		<< ", T=" << tnow << "]: tot=" << tot << std::endl;
    }

  } else {
				// Sum reduce counts to root
    MPI_Reduce(&cntr[0], 0, multistep+1, MPI_INT, MPI_SUM,
	       0, MPI_COMM_WORLD);
  }

}

// -*- C++ -*-
