#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "expand.h"
#include "Component.H"
#include "cudaParticle.cuH"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>

#include <boost/make_shared.hpp>

unsigned Component::cudaStreamData::totalInstances=0;

using PII=std::pair<int, int>;

struct testCountLevel :  public thrust::unary_function<cudaParticle, int>
{
  int _l;

  __host__ __device__
  testCountLevel(int l) : _l(l) {}

  __host__ __device__
  int operator()(const cudaParticle& p) const
  {
    if (p.lev[0] == _l) return 1;
    return 0;
  }
};

struct testCountLevel2 :  public thrust::unary_function<int, int>
{
  int _l;

__host__ __device__
  testCountLevel2(int l) : _l(l) {}

  __host__ __device__
  int operator()(const int p) const
  {
    if (p == _l) return 1;
    return 0;
  }
};

Component::cudaStreamData::cudaStreamData()
{
  // Not sure why this breaks thrust, but it does . . .
  /*
  cuda_safe_call(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking),
		 __FILE__, __LINE__,
		 "Component::cudaStreamData: error creating stream");
  */

  // Need blocking until thrust bug in binary search is fixed
  cuda_safe_call(hipStreamCreate(&stream),
		 __FILE__, __LINE__,
		 "Component::cudaStreamData: error creating stream");
  instance = totalInstances++;
}

Component::cudaStreamData::~cudaStreamData()
{
  cuda_safe_call(hipStreamDestroy(stream), __FILE__, __LINE__,
		 "Component::cudaStreamData: error destroying stream");
  totalInstances--;
}

void Component::cuda_initialize()
{
  cuStream = boost::make_shared<cudaStreamData>();
}


// Comparison operator for [from, to] pairs
struct pairLess
{
  __host__ __device__
  bool operator()(const thrust::pair<int, int>& lhs,
		  const thrust::pair<int, int>& rhs) const
  {
    return lhs.first < rhs.first or (lhs.first == rhs.first and lhs.second < rhs.second);
  }
};


Component::I2vec Component::CudaSortLevelChanges()
{
  // The plan: for the current active level search above and below for
  // particles for correction to coefficient matrix
  //
  // 1. Sort all particles by current level
  // 2. Get indices to range for each level L
  // 3. Within each level L, compute the ranges for changes,
  //    delta L = [-L, multistep-L]
  // 4. For each (L, delta L), compute the coefficient changes and
  //    apply to the appropriate coefficient matrices

  I2vec ret(multistep+1);
  for (auto & v : ret) v.resize(multistep+1);

  // Particle number
  //
  auto N = cuStream->cuda_particles.size();

  // Resize pair list and pair index
  //
  cuStream->levPair.resize(N);
  cuStream->indx2.  resize(N);

  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    thrust::device_vector<cudaParticle>::iterator
      pbeg = cuStream->cuda_particles.begin(),
      pend = cuStream->cuda_particles.end();
    
    // This gets a vector of pairs [current index, desired index],
    // leaving the order of particle structures unchanged
    //
    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      thrust::transform(pbeg, pend, cuStream->levPair.begin(), cuPartToChange());
    } else {
      thrust::transform(exec, pbeg, pend, cuStream->levPair.begin(), cuPartToChange());
    }
    
    // Make the initial sequential index
    //
    thrust::sequence(cuStream->indx2.begin(), cuStream->indx2.end(), 0, 1);
  
    // Sort the keys and make the index
    //
    thrust::sort_by_key(cuStream->levPair.begin(), cuStream->levPair.end(),
			cuStream->indx2.begin());

    // This will be [from, to] pair for determining the change matrix
    //
    thrust::pair<int, int> tr2;

    for (int target=0; target<=multistep; target++) {

      // From level is 'target'
      //
      tr2.first = target;

      for (int del=0; del<=multistep; del++) {

	if (del==target) {
	  ret[target][del] = {0, 0};
	  continue;
	}
	
	// To level is 'del'
	//
	tr2.second = del;

	thrust::device_vector<thrust::pair<int, int>>::iterator
	  lbeg = cuStream->levPair.begin(), lo;

	thrust::device_vector<thrust::pair<int, int>>::iterator
	  lend = cuStream->levPair.end(),   hi;

	// Determine upper and lower indices into indx2 for the [from,
	// to] pair
	//
	if (thrust_binary_search_workaround) {
	  hipStreamSynchronize(cuStream->stream);
	  lo  = thrust::lower_bound(lbeg, lend, tr2, pairLess());
	} else {
	  lo = thrust::lower_bound(exec, lbeg, lend, tr2, pairLess());
	}
	
	hipStreamSynchronize(cuStream->stream);

	if (thrust_binary_search_workaround) {
	  hi = thrust::upper_bound(lbeg, lend, tr2);
	} else {
	  hi = thrust::upper_bound(exec, lbeg, lend, tr2);
	}

	hipStreamSynchronize(cuStream->stream);

	ret[target][del] = {thrust::distance(lbeg, lo), 
			    thrust::distance(lbeg, hi)};
      }
    }

  }
  catch(std::bad_alloc &e) {
    std::cerr << "Ran out of memory while sorting" << std::endl;
    exit(-1);
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  // Debugging output for level changes
  //
  if (false) {
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
    std::cout << "--- " << name << " [" << myid << "]" << std::endl;
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
    for (int m1=0; m1<=multistep; m1++) {
      for (int m2=0; m2<=multistep; m2++) {
	std::cout << std::setw(15) << ret[m1][m2].second - ret[m1][m2].first;
      }
      std::cout << std::endl;
    }
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
  }

  return ret;
}


void Component::CudaSortByLevel()
{
  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    // Convert from cudaParticle to a flat vector of levels.  The
    // order of the particle structures will remain fixed
    //
    cuStream->levList.resize(cuStream->cuda_particles.size());

    thrust::device_vector<cudaParticle>::iterator
      pbeg = cuStream->cuda_particles.begin(),
      pend = cuStream->cuda_particles.end();

    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      thrust::transform(pbeg, pend, cuStream->levList.begin(), cuPartToLevel());
    } else {
      thrust::transform(exec, pbeg, pend, cuStream->levList.begin(), cuPartToLevel());
    }

    // Make room for an index
    //
    cuStream->indx1.resize(cuStream->cuda_particles.size());

    // Make the initial sequential index
    //
    thrust::sequence(cuStream->indx1.begin(), cuStream->indx1.end(), 0, 1);
  
    // First sort the keys and indices by the keys.  This gives a
    // indirect index back to the particles and a sorted levList for
    // determining the partition of the indirect index into levels
    //
    thrust::sort_by_key(cuStream->levList.begin(), cuStream->levList.end(),
			cuStream->indx1.begin());
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
}



std::pair<unsigned int, unsigned int>
Component::CudaGetLevelRange(int minlev, int maxlev)
{
  std::pair<unsigned, unsigned> ret;

  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);

    // Get unsigned from input
    //
    unsigned int minl = static_cast<unsigned>(minlev);
    unsigned int maxl = static_cast<unsigned>(maxlev);

    thrust::device_vector<int>::iterator lbeg = cuStream->levList.begin();
    thrust::device_vector<int>::iterator lend = cuStream->levList.end();
    thrust::device_vector<int>::iterator lo, hi;

    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      lo = thrust::lower_bound(lbeg, lend, minl);
    } else {
      lo = thrust::lower_bound(exec, lbeg, lend, minl);
    }
	
    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      hi = thrust::upper_bound(lbeg, lend, maxl);
    } else {
      hi = thrust::upper_bound(exec, lbeg, lend, maxl);
    }

    ret.first  = thrust::distance(lbeg, lo);
    ret.second = thrust::distance(lbeg, hi);

    if (false) {
      thrust::host_vector<int> testH(cuStream->levList);
      for (int n=0; n<10; n++) std::cout << " " << testH[n];
      std::cout << std::endl;

      std::cout << "Number of zeros="
		<< thrust::transform_reduce(cuStream->cuda_particles.begin(),
					    cuStream->cuda_particles.end(),
					    testCountLevel(0),
					    0, thrust::plus<int>())
		<< ", "
		<< thrust::transform_reduce(cuStream->levList.begin(),
					    cuStream->levList.end(),
					    testCountLevel2(0),
					    0, thrust::plus<int>())
		<< " lower=" << ret.first << " upper=" << ret.second
		<< std::endl;
    }
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  return ret;
}

void Component::ParticlesToCuda(PartMap::iterator beg, PartMap::iterator fin)
{
  if (step_timing and use_cuda) comp->timer_cuda.start();

  auto npart = std::distance(beg, fin);
  
  // Allocate particle memory and iterators
  //
  if (host_particles.capacity()<npart) host_particles.reserve(npart);
  host_particles.resize(npart);

  cuStream->first = host_particles.begin();
  cuStream->last  = host_particles.end();

  // Translate the EXP particle to Cuda particle structures
  //
  hostPartItr hit = host_particles.begin();
  for (auto pit=beg; pit!=fin; pit++) {
    ParticleHtoD(pit->second, *(hit++));
  }

  if (step_timing and use_cuda) comp->timer_cuda.stop();
}

void Component::HostToDev(Component::cuSharedStream cr)
{
  auto npart = thrust::distance(cr->first, cr->last);
  
  if (npart) {		  // Don't bother trying to copy zero particles

    // Resize the device array, if necessary
    //
    if (cr->cuda_particles.capacity()<npart) cr->cuda_particles.reserve(npart);
    cr->cuda_particles.resize(npart);
  
    // Copy the cuda particle structures to the device
    //
    hipMemcpyAsync(thrust::raw_pointer_cast(&cr->cuda_particles[0]),
		    thrust::raw_pointer_cast(&(*cr->first)),
		    npart*sizeof(cudaParticle),
		    hipMemcpyHostToDevice, cr->stream);
  }

  // Make the level index after a particle copy to device
  //
  CudaSortByLevel();
}

void Component::DevToHost(Component::cuSharedStream cr)
{
  auto npart = thrust::distance(cr->first, cr->last);
  
  if (npart) {		  // Don't bother trying to copy zero particles

    hipMemcpyAsync(thrust::raw_pointer_cast(&(*cr->first)),
		    thrust::raw_pointer_cast(&cr->cuda_particles[0]),
		    npart*sizeof(cudaParticle),
		    hipMemcpyDeviceToHost, cr->stream);
  }
}


void Component::CudaToParticles(hostPartItr beg, hostPartItr end)
{
  if (step_timing and use_cuda) comp->timer_cuda.start();

  // DEBUG PRINTING (enable by setting imax>0)
  //
  const int imax = 0;
  int icnt = 0;

  // Translate the Cuda particle to the EXP particle structures
  //
  for (hostPartItr v=beg; v!=end; v++) {
    cudaParticle & p = *v;
    if (icnt < imax) {
      std::cout << "[" << icnt++ << ", " << myid << "] " << p << std::endl;
    }
    ParticleDtoH(p, particles[p.indx]);
  }

  if (step_timing and use_cuda) comp->timer_cuda.stop();
}

// No longer used because we need to deal with indirection
//
struct cudaZeroAcc : public thrust::unary_function<cudaParticle, cudaParticle>
{
  __host__ __device__
  cudaParticle operator()(cudaParticle& p)
  {
    for (size_t k=0; k<3; k++) p.acc[k] = 0.0;
    p.pot = p.potext = 0.0;
    return p;
  }
};

__global__ void
zeroPotAccKernel(dArray<cudaParticle> P, dArray<int> I, int stride, PII lohi)
{
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {

      cudaParticle & p = P._v[I._v[npart]];
      
      for (int k=0; k<3; k++) p.acc[k] = 0.0;
      p.pot = p.potext = 0.0;

    } // Particle index block
    
  } // END: stride loop
}


void Component::ZeroPotAccel(int minlev)
{
  size_t psize  = particles.size();
  
  std::pair<unsigned int, unsigned int> lohi, cur;

  if (multistep)
    lohi = CudaGetLevelRange(minlev, multistep);
  else
    lohi = {0, cuStream->cuda_particles.size()};
    
  unsigned int Ntotal = lohi.second - lohi.first;
  unsigned int Npacks = Ntotal/bunchSize + 1;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cudaDevice);

  // Loop over bunches
  //
  for (int n=0; n<Npacks; n++) {

    // Current bunch
    //
    cur. first = lohi.first + bunchSize*n;
    cur.second = lohi.first + bunchSize*(n+1);
    cur.second = std::min<unsigned int>(cur.second, lohi.second);
    
    if (cur.second <= cur.first) break;
    
    // Compute grid
    //
    unsigned int N         = cur.second - cur.first;
    unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    unsigned int gridSize  = N/BLOCK_SIZE/stride;
      
    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;
      

    // Pack the com values into a matrix, one particle per row
    // 
    zeroPotAccKernel<<<gridSize, BLOCK_SIZE, 0, cuStream->stream>>>
      (toKernel(cuStream->cuda_particles), toKernel(cuStream->indx1),
       stride, cur);
  }
  
}


__global__ void comKernel
(dArray<cudaParticle> P, dArray<int> I, dArray<cuFP_t> com,
 int stride, PII lohi)
{
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;
    int npart = i + lohi.first;

    if (npart < lohi.second) {

      cudaParticle & p = P._v[I._v[npart]];
    
      com._v[i*10+0] = p.mass;
      for (int k=0; k<3; k++) {
	com._v[i*10+1+k] = p.pos[k];
	com._v[i*10+4+k] = p.vel[k];
	com._v[i*10+7+k] = p.acc[k];
      }
    }
  }
}


// Convert linear index to row index for column reduction
//
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {

  T Ncols; // --- Number of columns
  
  __host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}
  
  __host__ __device__ T operator()(T i) { return i / Ncols; }
};

void Component::fix_positions_cuda(unsigned mlevel)
{
  const int maxBunch = 40000;

				// Zero center
  for (int i=0; i<3; i++) center[i] = 0.0;

  				// Zero variables
  mtot = 0.0;
  for (int k=0; k<dim; k++) com[k] = cov[k] = coa[k] = 0.0;

				// Zero multistep counters at and
				// above this level
  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    for (int mm=mlevel; mm<=multistep; mm++) {

      hipStreamSynchronize(cuStream->stream);

      thrust::device_vector<int>::iterator
	lbeg = cuStream->levList.begin(), lo,
	lend = cuStream->levList.end(),   hi;

      if (thrust_binary_search_workaround) {
	hipStreamSynchronize(cuStream->stream);
	lo  = thrust::lower_bound(lbeg, lend, mm);
      } else {
	lo = thrust::lower_bound(exec, lbeg, lend, mm);
      }
      
      hipStreamSynchronize(cuStream->stream);

      if (thrust_binary_search_workaround) {
	hi = thrust::upper_bound(lbeg, lend, mm);
      } else {
	hi = thrust::upper_bound(exec, lbeg, lend, mm);
      }
      
      // Sort particles and get coefficient size
      //
      PII lohi = {thrust::distance(lbeg, lo), thrust::distance(lbeg, hi)};
      PII cur;
  
      unsigned int Ntotal = thrust::distance(lo, hi);
      unsigned int Npacks = Ntotal/maxBunch + 1;

      com_mas[mm] = 0.0;
      for (unsigned k=0; k<3; k++)  {
	com_lev[3*mm+k] = 0.0;
	cov_lev[3*mm+k] = 0.0;
	coa_lev[3*mm+k] = 0.0;
      }

      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, cudaDevice);

      // Loop over bunches
      //
      for (int n=0; n<Npacks; n++) {

	// Current bunch
	//
	cur. first = lohi.first + maxBunch*n;
	cur.second = lohi.first + maxBunch*(n+1);
	cur.second = std::min<unsigned int>(cur.second, lohi.second);
	
	if (cur.second <= cur.first) break;
    
	// Compute grid
	//
	unsigned int N         = cur.second - cur.first;
	unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
	unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
	if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

	// Resize storage as needed
	//
	const int Ncols = 10;	// mass, pos, vel, acc
	thrust::device_vector<cuFP_t> ret(N*Ncols);
	
	// Allocate space for row sums and indices
	//
	thrust::device_vector<cuFP_t> d_col_sums   (Ncols);
	thrust::device_vector<int>    d_col_indices(Ncols);


	// Pack the com values into a matrix, one particle per row
	// 
	comKernel<<<gridSize, BLOCK_SIZE, 0, cuStream->stream>>>
	  (toKernel(cuStream->cuda_particles), toKernel(cuStream->indx1),
	   toKernel(ret), stride, cur);

	// Perform sum over columns by summing values with equal column indices
	//
	thrust::reduce_by_key
	  (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(N)),
	   thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(N)) + (N*Ncols),
	   thrust::make_permutation_iterator
	   (ret.begin(), thrust::make_transform_iterator(thrust::make_counting_iterator(0),(thrust::placeholders::_1 % N) * Ncols + thrust::placeholders::_1 / N)),
	   d_col_indices.begin(),
	   d_col_sums.begin(),
	   thrust::equal_to<int>(),
	   thrust::plus<cuFP_t>());

	// Sum the partial results
	//
	com_mas[mm] += d_col_sums[0];
	for (unsigned k=0; k<3; k++)  {
	  com_lev[3*mm+k] += d_col_sums[1+k];
	  cov_lev[3*mm+k] += d_col_sums[4+k];
	  coa_lev[3*mm+k] += d_col_sums[7+k];
	}
      }
    }
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  std::vector<double> com1(3, 0.0), cov1(3, 0.0), coa1(3, 0.0);
  double              mtot1 = 0.0;

  for (unsigned mm=0; mm<=multistep; mm++) {
    for (int k=0; k<3; k++) {
      com1[k] += com_lev[3*mm + k];
      cov1[k] += cov_lev[3*mm + k];
      coa1[k] += coa_lev[3*mm + k];
    }
    mtot1 += com_mas[mm];
  }

  MPI_Allreduce(&mtot1, &mtot, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&com1[0], com, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&cov1[0], cov, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&coa1[0], coa, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
    
  if (VERBOSE>5) {
				// Check for NaN
    bool com_nan = false, cov_nan = false, coa_nan = false;
    for (int k=0; k<3; k++)
      if (std::isnan(com[k])) com_nan = true;
    for (int k=0; k<3; k++)
      if (std::isnan(cov[k])) cov_nan = true;
    for (int k=0; k<3; k++)
      if (std::isnan(coa[k])) coa_nan = true;
    if (com_nan && myid==0)
      cerr << "Component [" << name << "] com has a NaN" << endl;
    if (cov_nan && myid==0)
      cerr << "Component [" << name << "] cov has a NaN" << endl;
    if (coa_nan && myid==0)
      cerr << "Component [" << name << "] coa has a NaN" << endl;
  }
				// Compute component center of mass and
				// center of velocity, and center of accel

  if (mtot > 0.0) {
    for (int k=0; k<dim; k++) com[k]  /= mtot;
    for (int k=0; k<dim; k++) cov[k]  /= mtot;
    for (int k=0; k<dim; k++) coa[k]  /= mtot;
  }

  if (com_system and not consp) {
    for (int k=0; k<dim; k++) com0[k] = com[k];
    for (int k=0; k<dim; k++) cov0[k] = cov[k];
  }

  if (com_system) {	   // Use local center of accel for com update
    for (int k=0; k<dim; k++) acc0[k]  = coa[k];
  } else {			// No mass, no acceleration?
    for (int k=0; k<dim; k++) acc0[k]  = 0.0;
  }

  if ((EJ & Orient::CENTER) && !EJdryrun) {
    Vector ctr = orient->currentCenter();
    bool ok    = true;
    for (int i=0; i<3; i++) {
      if (std::isnan(ctr[i+1])) ok = false;
    } 
    if (ok) {
      for (int i=0; i<3; i++) center[i] += ctr[i+1];
    } else if (myid==0) {
      cout << "Orient: center failure, T=" << tnow 
	   << ", adjustment skipped" << endl;
    }
  }
}


void Component::print_level_lists_cuda(double T)
{
				// Retrieve counts per level
  std::vector<int> cntr(multistep+1);
  for (int m=0; m<=multistep; m++) {
    cntr[m] = thrust::transform_reduce(cuStream->cuda_particles.begin(),
				       cuStream->cuda_particles.end(),
				       testCountLevel(m),
				       0, thrust::plus<int>());
  }

  if (myid==0) {
				// Sum reduce to root
    MPI_Reduce(MPI_IN_PLACE, &cntr[0], multistep+1, MPI_INT, MPI_SUM,
	       0, MPI_COMM_WORLD);

    int tot=0;
    for (int m=0; m<=multistep; m++) tot += cntr[m];

    if (tot) {

      std::ostringstream ofil;
      ofil << runtag << ".levels";
      std::ofstream out(ofil.str().c_str(), ios::app);

      int sum=0;
      out << setw(60) << setfill('-') << '-' << endl;
      std::ostringstream sout;
      sout << "--- Component <" << name 
	   << ", " << id  << ">, T=" << T;
      out << std::setw(60) << std::left << sout.str().c_str() << std::endl;
      out << std::setw(60) << '-' << std::endl << std::setfill(' ');
      out << std::setw(3)  << "L" 
	  << std::setw(10) << "Number" 
	  << std::setw(10) << "dN/dL" 
	  << std::setw(10) << "N(<=L)"
	  << std::endl
	  << std::setw(60) << std::setfill('-') << '-'
	  << std::endl << std::setfill(' ');
      for (int n=0; n<=multistep; n++) {
	sum += cntr[n];
	out << std::setw(3)  << n 
	    << std::setw(10) << cntr[n] << std::setprecision(3) << std::fixed
	    << std::setw(10) << static_cast<double>(cntr[n])/tot
	    << std::setw(10) << static_cast<double>(sum)    /tot;
	out << std::endl;
      }
      out << std::endl << std::setw(3) << "T" << std::setw(10) << tot
	  << std::endl << std::endl << std::right;
    } else {
      std::cout << "print_level_lists_cuda [" << name 
		<< ", T=" << tnow << "]: tot=" << tot << std::endl;
    }

  } else {
				// Sum reduce counts to root
    MPI_Reduce(&cntr[0], 0, multistep+1, MPI_INT, MPI_SUM,
	       0, MPI_COMM_WORLD);
  }

}

// No cuda code here but only used after CudaToParticles() call for
// testing
void Component::MakeLevlist()
{
  levlist.resize(multistep+1);
  for (auto & v : levlist) v.clear();
  for (auto & v : particles) levlist[v.second->level].push_back(v.first);
}
