#include "hip/hip_runtime.h"
#include <Component.H>
#include <SphericalBasis.H>

#include <cudaReduce.cuH>

__host__ __device__
int Ilm(int l, int m)
{
  if (l==0) return 0;
  return l*(l+1)/2 + m;
}

__host__ __device__
int Ilmn(int l, int m, char cs, int n, int nmax)
{
  int ret = 0;

  if (l==0) ret = n;
  else if (m==0) ret = l*l*nmax + n;
  else ret = (l*l + 2*m - 1 + (cs=='s' ? 1 : 0))*nmax + n;

  if (ret >= (l+1)*(l+1)*nmax) {
    printf("Ilmn oab: %4d %4d %4d [%4d : %4d : %4d]\n", l, m, n, ret, (l+1)*(l+1)*nmax, nmax);
  }

  return ret;
}

__host__ __device__
void legendre_v(int lmax, float x, float* p)
{
  float fact, somx2, pll, pl1, pl2;
  int m, l;

  p[0] = pll = 1.0f;
  if (lmax > 0) {
    somx2 = sqrt( (1.0f - x)*(1.0f + x) );
    fact = 1.0f;
    for (m=1; m<=lmax; m++) {
      pll *= -fact*somx2;
      p[Ilm(m, m)] = pll;
      fact += 2.0f;
    }
  }

  for (m=0; m<lmax; m++) {
    pl2 = p[Ilm(m, m)];
    p[Ilm(m+1, m)] = pl1 = x*(2*m+1)*pl2;
    for (l=m+2; l<=lmax; l++) {
      p[Ilm(l, m)] = pll = (x*(2*l-1)*pl1-(l+m-1)*pl2)/(l-m);
      pl2 = pl1;
      pl1 = pll;
    }
  }
}

__host__ __device__
void legendre_v2(int lmax, float x, float* p, float* dp)
{
  float fact, somx2, pll, pl1, pl2;
  int m, l;

  p[0] = pll = 1.0f;
  if (lmax > 0) {
    somx2 = sqrt( (1.0 - x)*(1.0 + x) );
    fact = 1.0;
    for (m=1; m<=lmax; m++) {
      pll *= -fact*somx2;
      p[Ilm(m, m)] = pll;
      fact += 2.0;
    }
  }

  for (m=0; m<lmax; m++) {
    pl2 = p[Ilm(m, m)];
    p[Ilm(m+1, m)] = pl1 = x*(2*m+1)*pl2;
    for (l=m+2; l<=lmax; l++) {
      p[Ilm(l, m)] = pll = (x*(2*l-1)*pl1-(l+m-1)*pl2)/(l-m);
      pl2 = pl1;
      pl1 = pll;
    }
  }

  if (1.0-fabs(x) < MINEPS) {
    if (x>0) x =   1.0 - MINEPS;
    else     x = -(1.0 - MINEPS);
  }

  somx2 = 1.0/(x*x - 1.0);
  dp[0] = 0.0;
  for (l=1; l<=lmax; l++) {
    for (m=0; m<l; m++)
      dp[Ilm(l, m)] = somx2*(x*l*p[Ilm(l, m)] - (l+m)*p[Ilm(l-1, m)]);
    dp[Ilm(l, l)] = somx2*x*l*p[Ilm(l, l)];
  }
}

__global__
void testConstants()
{
  printf("** Rscale = %f\n", cuRscale);
  printf("** Xmin   = %f\n", cuXmin);
  printf("** Xmax   = %f\n", cuXmax);
  printf("** Dxi    = %f\n", cuDxi);
  printf("** Numr   = %d\n", cuNumr);
  printf("** Cmap   = %d\n", cuCmap);
}

__device__
float cu_r_to_xi(float r)
{
  float ret;

  if (cuCmap==1) {
    ret = (r/cuRscale-1.0)/(r/cuRscale+1.0);
  } else if (cuCmap==2) {
    ret = log(r);
  } else {
    ret = r;
  }    

  return ret;
}
    
__device__
float cu_xi_to_r(float xi)
{
  float ret;

  if (cuCmap==1) {
    ret = (1.0+xi)/(1.0 - xi) * cuRscale;
  } else if (cuCmap==2) {
    ret = exp(xi);
  } else {
    ret = xi;
  }

  return ret;
}

__device__
float cu_d_xi_to_r(float xi)
{
  float ret;

  if (cuCmap==1) {
    ret = 0.5*(1.0-xi)*(1.0-xi)/cuRscale;
  } else if (cuCmap==2) {
    ret = exp(-xi);
  } else {
    ret = 1.0;
  }

  return ret;
}

void SphericalBasis::initialize_mapping_constants()
{
  // Copy constants to device
  //
  
  cudaMappingConstants f = getCudaMappingConstants();

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuRscale), &f.rscale, sizeof(float), size_t(0), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying cuRscale");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuXmin), &f.xmin, sizeof(float), size_t(0), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying cuXmin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuXmax), &f.xmax, sizeof(float), size_t(0), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying cuXmax");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuDxi), &f.dxi, sizeof(float), size_t(0), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying cuDxi");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuNumr), &f.numr, sizeof(int), size_t(0), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying cuNumr");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cuCmap), &f.cmap, sizeof(int), size_t(0), hipMemcpyHostToDevice), __FILE__, __LINE__, "Error copying cuCmap");
}

__global__
void testTexture(dArray<hipTextureObject_t> tex, int nmax)
{
  printf("**DEVICE Texture compare\n");
  for (int l : {0, 1, 2}) {
    for (int j=0; j<10; j++) {
      int k = 1 + l*nmax;
      for (int i : {3980, 3990, 3995, 3999}) 
	printf("%5d %5d %5d %13.7e\n", l, j, i, tex1D<float>(tex._v[k+j], i));
    }
  }
}

__global__ void coordKernel
(dArray<cudaParticle> in, dArray<float> mass, dArray<float> Afac,
 dArray<float> phi, dArray<float> Plm, dArray<int> Indx, 
 unsigned int Lmax, unsigned int stride, PII lohi, float rmax)
{
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;
  const int psiz  = (Lmax+1)*(Lmax+2)/2;

  /*
    vector<double> ctr;
    if (mix) mix->getCenter(ctr);
  */
  float ctr[3] {0.0f, 0.0f, 0.0f};

  for (int n=0; n<stride; n++) {
    int i = tid*stride + n;
    int npart = i + lohi.first;

    if (npart < lohi.second) {

      if (npart>=in._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

      cudaParticle p = in._v[npart];
    
      float xx = p.pos[0] - ctr[0];
      float yy = p.pos[1] - ctr[1];
      float zz = p.pos[2] - ctr[2];
      
      float r2 = (xx*xx + yy*yy + zz*zz);
      float r = sqrt(r2) + FSMALL;
      
      if (i>=mass._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

      mass._v[i] = -1.0;
      
      if (r<rmax) {
	
	mass._v[i] = p.mass;
	
	float costh = zz/r;
	phi._v[i] = atan2(yy,xx);
	
	if (i>=phi._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

	float *plm = &Plm._v[psiz*i];
	legendre_v(Lmax, costh, plm);

	if (psiz*(i+1)>Plm._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

	float x  = cu_r_to_xi(r);
	float xi = (x - cuXmin)/cuDxi;
	int indx = floor(xi);
	
	if (indx<0) indx = 0;
	if (indx>cuNumr-2) indx = cuNumr - 2;
	  
	Afac._v[i] = float(indx+1) - xi;
	if (Afac._v[i]<0.0 or Afac._v[i]>1.0)
	  printf("off grid: x=%f\n", xi);
	Indx._v[i] = indx;

	if (i>=Afac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=Indx._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
      }
    }
  }
}


__global__ void coefKernel
(dArray<float> coef, dArray<hipTextureObject_t> tex,
 dArray<float> Mass, dArray<float> Afac, dArray<float> Phi,
 dArray<float> Plm, dArray<int> Indx,  int stride, 
 int l, int m, unsigned Lmax, unsigned int nmax, PII lohi)
{
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;
  const int psiz  = (Lmax+1)*(Lmax+2)/2;
  const unsigned int N = lohi.second - lohi.first;

  float fac0 = 4.0*M_PI;

  for (int istr=0; istr<stride; istr++) {

    int i = tid*stride + istr;

    if (i<N) {

      float mass = Mass._v[i];

      if (i>=Mass._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

      if (mass>0.0) {

	if (i>=Phi._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

	float phi  = Phi._v[i];
	float cosp = cos(phi*m);
	float sinp = sin(phi*m);
	
	if (psiz*(i+1)>Plm._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	
	float *plm = &Plm._v[psiz*i];
	
	// Do the interpolation
	//
	float a = Afac._v[i];
	float b = 1.0 - a;
	int ind = Indx._v[i];
	
	if (i>=Afac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=Indx._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

	for (int n=0; n<nmax; n++) {

	  float p0 =
	    a*tex1D<float>(tex._v[0], ind  ) +
	    b*tex1D<float>(tex._v[0], ind+1) ;

	  int k = 1 + l*nmax + n;

	  if (k>=tex._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

	  float v = (
		     a*tex1D<float>(tex._v[k], ind  ) +
		     b*tex1D<float>(tex._v[k], ind+1)
		     ) * p0 * plm[Ilm(l, m)] * Mass._v[i] * fac0;
	  
	  
	  coef._v[(2*n+0)*N + i] = v * cosp;
	  coef._v[(2*n+1)*N + i] = v * sinp;

	  if ((2*n+0)*N+i>=coef._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	  if ((2*n+1)*N+i>=coef._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	}
      }
    }
  }
}

__global__ void
forceKernel(dArray<cudaParticle> in, dArray<float> coef,
	    dArray<hipTextureObject_t> tex, dArray<float> L1, dArray<float> L2,
	    int stride, unsigned Lmax, unsigned int nmax, PII lohi, float rmax)
{
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;
  const int psiz  = (Lmax+1)*(Lmax+2)/2;

  /*
    vector<double> ctr;
    if (mix) mix->getCenter(ctr);
  */
  float ctr[3] {0.0f, 0.0f, 0.0f};

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {
      
      if (npart>=in._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);

      cudaParticle p = in._v[npart];
      
      float xx = p.pos[0] - ctr[0];
      float yy = p.pos[1] - ctr[1];
      float zz = p.pos[2] - ctr[2];
      
      float r2 = (xx*xx + yy*yy + zz*zz);
      float r  = sqrt(r2) + FSMALL;
      
      float costh = zz/r;
      float phi   = atan2(yy, xx);
      float RR    = xx*xx + yy*yy;
      
      float *plm1 = &L1._v[psiz*tid];
      float *plm2 = &L2._v[psiz*tid];
      legendre_v2(Lmax, costh, plm1, plm2);

      int ioff = 0;
      // float rs = 0.0;
      float r0;

      if (r>rmax) {
	ioff = 1;
	r0   = r;
	r    = rmax;
	// rs = r/cuRscale;
      }

      float  x = cu_r_to_xi(r);
      float xi = (x - cuXmin)/cuDxi;
      float dx = cu_d_xi_to_r(x)/cuDxi;
      int  ind = floor(xi);
      
      if (ind<1) ind = 1;
      if (ind>cuNumr-2) ind = cuNumr - 2;
      
      float a = (float)(ind+1) - xi;
      if (a<0.0 or a>1.0) printf("forceKernel: off grid: x=%f\n", xi);
      float b = 1.0 - a;
      
      // Do the interpolation for the prefactor potential
      //
      float pm1 = tex1D<float>(tex._v[0], ind-1);
      float p00 = tex1D<float>(tex._v[0], ind  );
      float pp1 = tex1D<float>(tex._v[0], ind+1);

      // For force accumulation
      //
      float potl = 0.0;
      float potr = 0.0;
      float pott = 0.0;
      float potp = 0.0;

      // l loop
      //
      for (int l=0; l<Lmax; l++) {

	float fac1 = (2.0*l + 1.0)/(4.0*M_PI);

	// m loop
	//
	for (int m=0; m<=l; m++) {

	  int pindx = Ilm(l, m);

	  float Plm1 = plm1[pindx];
	  float Plm2 = plm2[pindx];
      
	  if (std::isnan(Plm1)) 
	    {
	      printf("Force isnan for Plm(%d, %d) ioff=%d\n", l, m, ioff);
	    }

	  if (std::isnan(Plm2)) 
	    {
	      printf("Force isnan for Plm2(%d, %d) ioff=%d\n", l, m, ioff);
	    }
	  

	  float pp_c = 0.0;
	  float dp_c = 0.0;
	  float pp_s = 0.0;
	  float dp_s = 0.0;
	  
	  int indxC = Ilmn(l, m, 'c', 0, nmax);
	  int indxS = Ilmn(l, m, 's', 0, nmax);

	  float cosp = cos(phi*m);
	  float sinp = sin(phi*m);

	  for (size_t n=0; n<nmax; n++) {
	
	    int k = 1 + l*nmax + n;
	
	    if (k>=tex._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	
	    float um1 = tex1D<float>(tex._v[k], ind-1);
	    float u00 = tex1D<float>(tex._v[k], ind  );
	    float up1 = tex1D<float>(tex._v[k], ind+1);
	    
	    float v = (a*u00 + b*up1)*(a*p00 + b*pp1);
	    
	    float dv =
	      dx * ( (b - 0.5)*um1*pm1 - 2.0*b*u00*p00 + (b + 0.5)*up1*pp1 );
	    
	    pp_c +=  v * coef._v[indxC+n];
	    dp_c += dv * coef._v[indxC+n];
	    if (m>0) {
	      pp_s +=  v * coef._v[indxS+n];
	      dp_s += dv * coef._v[indxS+n];
	    }

	  } // END: n loop
	  
	  pp_c *= -1.0;
	  dp_c *= -1.0;

	  if (m==0) {

	    if (ioff) {
	      pp_c *= pow(rmax/r0, (float)(l+1));
	      dp_c  = -pp_c/r0 * (float)(l+1);
	      if (std::isnan(pp_c)) 
		{
		  printf("Force nan: l=%d, r=%f\n", l, r);
		}
	    }
	    
	    potl += fac1 * pp_c * Plm1;
	    potr += fac1 * dp_c * Plm1;
	    pott += fac1 * pp_c * Plm2;
	    potp += 0.0;
	    
	  } else {

	    float cosm = cos(phi*m);
	    float sinm = sin(phi*m);
	    
	    if (ioff) {
	      float facp  = pow(rmax/r0,(double)(l+1));
	      float facdp = -facp/r0 * (l+1);
	      pp_c *= facp;
	      pp_s *= facp;
	      dp_c = pp_c * facdp;
	      dp_s = pp_s * facdp;

	      if (std::isnan(pp_c)) 
		{
		  printf("Force nan: c l=%d, m=%d, r=%f\n", l, m, r);
		}

	      if (std::isnan(dp_s)) 
		{
		  printf("Force nan: s l=%d, m=%d, r=%f\n", l, m, r);
		}

	      if (std::isnan(dp_c)) 
		{
		  printf("Force nan: dc l=%d, m=%d, r=%f\n", l, m, r);
		}

	      if (std::isnan(pp_s)) 
		{
		  printf("Force nan: ds l=%d, m=%d, r=%f\n", l, m, r);
		}
	    }

	    // Factorials
	    //
	    float numf = 1.0, denf = 1.0;
	    for (int i=1; i<=l-m; i++) numf *= i;
	    for (int i=1; i<=l+m; i++) denf *= i;
	    
	    float fac2 = 2.0 * numf/denf * fac1;
	    
	    potl += fac2 * Plm1 * ( pp_c*cosm + pp_s*sinm);
	    potr += fac2 * Plm1 * ( dp_c*cosm + dp_s*sinm);
	    pott += fac2 * Plm2 * ( pp_c*cosm + pp_s*sinm);
	    potp += fac2 * Plm1 * (-pp_c*sinm + pp_s*cosm)*m;
	  }

	} // END: m loop

      } // END: l loop

      in._v[npart].acc[0] = -(potr*xx/r - pott*xx*zz/(r*r*r));
      in._v[npart].acc[1] = -(potr*yy/r - pott*yy*zz/(r*r*r));
      in._v[npart].acc[2] = -(potr*zz/r - pott*RR/(r*r*r));
      if (RR > FSMALL) 
	{
	  in._v[npart].acc[0] +=  potp*yy/RR;
	  in._v[npart].acc[1] += -potp*xx/RR;
	}
      in._v[npart].pot    = potl;

      // Sanity check
      bool bad = false;
      for (int k=0; k<3; k++) {
	if (std::isnan(in._v[npart].acc[k])) bad = true;
      }

      if (bad) 
	{
	  printf("Force nan value: [%d] x=%f xi=%f dxi=%f a=%f i=%d o=%d\n",
		 in._v[npart].indx, x, xi, dx, a, ind, ioff);
	  if (ioff==0) 
	    {
	      printf("Force nan value, no ioff: [%d] x=%f xi=%f dxi=%f a=%f i=%d\n",
		     in._v[npart].indx, x, xi, dx, a, ind);
	    }
	  
	  /*
	  printf("Force nan value: [%d] xx=%f yy=%f zz=%f r=%f R=%f\n",
		 in._v[npart].indx, xx, yy, zz, r, RR, ioff);
	  */
	}
      

    } // Particle index block

  } // END: stride loop

}


void SphericalBasis::determine_coefficients_cuda(const Matrix& expcoef)
{
  std::cout << std::scientific;

  // Sort particles and get coefficient size
  //
  PII lohi = cC->CudaSortByLevel(mlevel, multistep);

  unsigned int N         = lohi.second - lohi.first;
  unsigned int stride    = 1;
  unsigned int gridSize  = N/BLOCK_SIZE/stride;

  /*
  if (gridSize>128) {
    stride = N/BLOCK_SIZE/128 + 1;
    gridSize = N/BLOCK_SIZE/stride;
  }
  */

  if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

  // unsigned int Nthread = gridSize*BLOCK_SIZE;

  std::cout << "**" << std::endl
	    << "** N      = " << N          << std::endl
	    << "** Stride = " << stride     << std::endl
	    << "** Block  = " << BLOCK_SIZE << std::endl
	    << "** Grid   = " << gridSize   << std::endl
	    << "**" << std::endl;

  // Create space for coefficient reduction
  //
  thrust::device_vector<float> dN_coef(2*nmax*N);
  thrust::device_vector<float> dc_coef(2*nmax*gridSize);
  thrust::device_vector<float> df_coef(2*nmax);

  // Texture objects
  //
  thrust::device_vector<hipTextureObject_t> t_d = tex;

  // Space for Legendre coefficients 
  //
  thrust::device_vector<float> plm_d((Lmax+1)*(Lmax+2)/2*N);
  thrust::device_vector<float> r_d(N), m_d(N), a_d(N), p_d(N);
  thrust::device_vector<int>   i_d(N);

  // Shared memory size for the reduction
  //
  int sMemSize = BLOCK_SIZE * sizeof(float);

  // For debugging
  //
  if (false) {
    testConstants<<<1, 1>>>();
    
    static bool firstime = true;
    testTexture<<<1, 1>>>(toKernel(t_d), nmax);
    firstime == false;
  }

  std::vector<float> coefs((Lmax+1)*(Lmax+1)*nmax);

  thrust::counting_iterator<int> index_begin(0);
  thrust::counting_iterator<int> index_end(gridSize*2*nmax);

  // Do the work
  //
				// Compute the coordinate
				// transformation
				// 
  coordKernel<<<gridSize, BLOCK_SIZE>>>
    (toKernel(cC->cuda_particles),
     toKernel(m_d), toKernel(a_d), toKernel(p_d), toKernel(plm_d),
     toKernel(i_d), Lmax, stride, lohi, rmax);

				// Compute the coefficient
				// contribution for each order
  for (int l=0; l<=Lmax; l++) {
    for (int m=0; m<=l; m++) {
      coefKernel<<<gridSize, BLOCK_SIZE>>>
	(toKernel(dN_coef), toKernel(t_d), toKernel(m_d),
	 toKernel(a_d), toKernel(p_d), toKernel(plm_d), toKernel(i_d),
	 stride, l, m, Lmax, nmax, lohi);

				// Begin the reduction per grid block
      int osize = nmax*2;	// 
      reduceSum<float, BLOCK_SIZE><<<gridSize, BLOCK_SIZE, sMemSize>>>
	(toKernel(dc_coef), toKernel(dN_coef), osize, N);
      
				// Finish the reduction for this order
				// in parallel
      thrust::reduce_by_key
	(
	 thrust::make_transform_iterator(index_begin, key_functor(gridSize)),
	 thrust::make_transform_iterator(index_end,   key_functor(gridSize)),
	 dc_coef.begin(), thrust::make_discard_iterator(), df_coef.begin()
	 );

      thrust::host_vector<float> ret = df_coef;
      for (size_t j=0; j<nmax; j++) {
	coefs[Ilmn(l, m, 'c', j, nmax)] = ret[2*j];
	if (m>0) coefs[Ilmn(l, m, 's', j, nmax)] = ret[2*j+1];
      }
    }
  }

  // DEBUG
  //
  if (false) {
    std::cout << "L=M=0 coefficients" << std::endl;
    for (size_t n=0; n<nmax; n++) {
      std::cout << std::setw(4)  << n
		<< std::setw(16) << coefs[Ilmn(0, 0, 'c', n, nmax)]
		<< std::setw(16) << expcoef[0][n+1]
		<< std::endl;
    }

    std::cout << "L=1, M=0 coefficients" << std::endl;
    for (size_t n=0; n<nmax; n++) {
      std::cout << std::setw(4)  << n
		<< std::setw(16) << coefs[Ilmn(1, 0, 'c', n, nmax)]
		<< std::setw(16) << expcoef[1][n+1]
		<< std::endl;
    }

    std::cout << "L=1, M=1c coefficients" << std::endl;
    for (size_t n=0; n<nmax; n++) {
      std::cout << std::setw(4)  << n
		<< std::setw(16) << coefs[Ilmn(1, 1, 'c', n, nmax)]
		<< std::setw(16) << expcoef[2][n+1]
		<< std::endl;
    }

    std::cout << "L=1, M=1s coefficients" << std::endl;
    for (size_t n=0; n<nmax; n++) {
      std::cout << std::setw(4)  << n
		<< std::setw(16) << coefs[Ilmn(1, 1, 's', n, nmax)]
		<< std::setw(16) << expcoef[3][n+1]
		<< std::endl;
    }
    
    std::cout << "L=2, M=0 coefficients" << std::endl;
    for (size_t n=0; n<nmax; n++) {
      std::cout << std::setw(4)  << n
		<< std::setw(16) << coefs[Ilmn(2, 0, 'c', n, nmax)]
		<< std::setw(16) << expcoef[4][n+1]
		<< std::endl;
    }

    std::cout << "L=2, M=1c coefficients" << std::endl;
    for (size_t n=0; n<nmax; n++) {
      std::cout << std::setw(4)  << n
		<< std::setw(16) << coefs[Ilmn(2, 1, 'c', n, nmax)]
		<< std::setw(16) << expcoef[5][n+1]
		<< std::endl;
    }

    std::cout << "L=2, M=1s coefficients" << std::endl;
    for (size_t n=0; n<nmax; n++) {
      std::cout << std::setw(4)  << n
		<< std::setw(16) << coefs[Ilmn(2, 1, 's', n, nmax)]
		<< std::setw(16) << expcoef[6][n+1]
		<< std::endl;
    }
  }

  //
  // TEST comparison of coefficients for debugging
  //
  if (false) {

    struct Element
    {
      double d;
      float  f;
      
      int  l;
      int  m;
      int  n;
      
      char cs;
    }
    elem;

    std::map<double, Element> compare;

    std::ofstream out("test.dat");

    //		l loop
    for (int l=0, loffset=0; l<=Lmax; loffset+=(2*l+1), l++) {
      //		m loop
      for (int m=0, moffset=0; m<=l; m++) {
	
	if (m==0) {
	  for (int n=1; n<=nmax; n++) {
	    elem.l = l;
	    elem.m = m;
	    elem.n = n;
	    elem.cs = 'c';
	    elem.d = expcoef[loffset+moffset][n];
	    elem.f = coefs[Ilmn(l, m, 'c', n-1, nmax)];
	    
	    double test = fabs(elem.d - elem.f);
	    if (fabs(elem.d)>1.0e-4) test /= fabs(elem.d);
	    
	    compare[test] = elem;
	    
	    out << std::setw( 5) << l
		<< std::setw( 5) << m
		<< std::setw( 5) << n
		<< std::setw( 5) << 'c'
		<< std::setw( 5) << Ilmn(l, m, 'c', n-1, nmax)
		<< std::setw(14) << elem.d
		<< std::setw(14) << elem.f
		<< std::endl;
	  }
	  
	  moffset++;
	}
	else {
	  for (int n=1; n<=nmax; n++) {
	    elem.l = l;
	    elem.m = m;
	    elem.n = n;
	    elem.cs = 'c';
	    elem.d = expcoef[loffset+moffset][n];
	    elem.f = coefs[Ilmn(l, m, 'c', n-1, nmax)];

	    out << std::setw( 5) << l
		<< std::setw( 5) << m
		<< std::setw( 5) << n
		<< std::setw( 5) << 'c'
		<< std::setw( 5) << Ilmn(l, m, 'c', n-1, nmax)
		<< std::setw(14) << elem.d
		<< std::setw(14) << elem.f
		<< std::endl;

	    double test = fabs(elem.d - elem.f);
	    if (fabs(elem.d)>1.0e-4) test /= fabs(elem.d);

	    compare[test] = elem;
	  }
	  for (int n=1; n<=nmax; n++) {
	    elem.l = l;
	    elem.m = m;
	    elem.n = n;
	    elem.cs = 's';
	    elem.d = expcoef[loffset+moffset+1][n];
	    elem.f = coefs[Ilmn(l, m, 's', n-1, nmax)];

	    out << std::setw( 5) << l
		<< std::setw( 5) << m
		<< std::setw( 5) << n
		<< std::setw( 5) << 's'
		<< std::setw( 5) << Ilmn(l, m, 's', n-1, nmax)
		<< std::setw(14) << elem.d
		<< std::setw(14) << elem.f
		<< std::endl;
	    
	    double test = fabs(elem.d - elem.f);
	    if (fabs(elem.d)>1.0e-4) test /= fabs(elem.d);
	    
	    compare[test] = elem;
	  }
	  moffset+=2;
	}
      }
    }
    
    std::map<double, Element>::iterator best = compare.begin();
    std::map<double, Element>::iterator midl = best;
    std::advance(midl, compare.size()/2);
    std::map<double, Element>::reverse_iterator last = compare.rbegin();
    
    std::cout << "Best case: ["
	      << std::setw( 2) << best->second.l << ", "
	      << std::setw( 2) << best->second.m << ", "
	      << std::setw( 2) << best->second.n << ", "
	      << std::setw( 2) << best->second.cs << "] = "
	      << std::setw(15) << best->second.d
	      << std::setw(15) << best->second.f
	      << std::setw(15) << fabs(best->second.d - best->second.f)
	      << std::endl;
  
    std::cout << "Mid case:  ["
	      << std::setw( 2) << midl->second.l << ", "
	      << std::setw( 2) << midl->second.m << ", "
	      << std::setw( 2) << midl->second.n << ", "
	      << std::setw( 2) << midl->second.cs << "] = "
	      << std::setw(15) << midl->second.d
	      << std::setw(15) << midl->second.f
	      << std::setw(15) << fabs(midl->second.d - midl->second.f)
	      << std::endl;
    
    std::cout << "Last case: ["
	      << std::setw( 2) << last->second.l << ", "
	      << std::setw( 2) << last->second.m << ", "
	      << std::setw( 2) << last->second.n << ", "
	      << std::setw( 2) << last->second.cs << "] = "
	      << std::setw(15) << last->second.d
	      << std::setw(15) << last->second.f
	      << std::setw(15) << fabs(last->second.d - last->second.f)
	      << std::endl;
  }
}


void SphericalBasis::determine_acceleration_cuda()
{
  std::cout << std::scientific;

  // Sort particles and do all particles at or above mlevel
  //
  PII lohi = cC->CudaSortByLevel(mlevel, multistep);

  unsigned int N         = lohi.second - lohi.first;
  unsigned int stride    = 1;
  unsigned int gridSize  = N/BLOCK_SIZE/stride;

  /*
  if (gridSize>128) {
    stride = N/BLOCK_SIZE/128 + 1;
    gridSize = N/BLOCK_SIZE/stride;
  }
  */

  if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

  unsigned int Nthread = gridSize*BLOCK_SIZE;

  std::cout << "**" << std::endl
	    << "** N      = " << N          << std::endl
	    << "** Stride = " << stride     << std::endl
	    << "** Block  = " << BLOCK_SIZE << std::endl
	    << "** Grid   = " << gridSize   << std::endl
	    << "**" << std::endl;

  // Texture objects
  //
  thrust::device_vector<hipTextureObject_t> t_d = tex;

  // Space for Legendre coefficients 
  //
  thrust::device_vector<float> plm1_d((Lmax+1)*(Lmax+2)/2*Nthread);
  thrust::device_vector<float> plm2_d((Lmax+1)*(Lmax+2)/2*Nthread);

  // Shared memory size for the reduction
  //
  int sMemSize = BLOCK_SIZE * sizeof(float);

  // Do the work
  //
  forceKernel<<<gridSize, BLOCK_SIZE, sMemSize>>>
    (toKernel(cC->cuda_particles), toKernel(dev_coefs), toKernel(t_d),
     toKernel(plm1_d), toKernel(plm2_d), stride, Lmax, nmax, lohi, rmax);
}

void SphericalBasis::HtoD_coefs(const Matrix& expcoef)
{
  host_coefs.resize((Lmax+1)*(Lmax+1)*nmax);

  // l loop
  //
  for (int l=0, loffset=0; l<=Lmax; loffset+=(2*l+1), l++) {
    // m loop
    //
    for (int m=0, moffset=0; m<=l; m++) {
	
      // n loop
      //
      for (int n=1; n<=nmax; n++) {
	host_coefs[Ilmn(l, m, 'c', n-1, nmax)] = expcoef[loffset+moffset][n];
	if (m>0) host_coefs[Ilmn(l, m, 's', n-1, nmax)] = expcoef[loffset+moffset+1][n];
      }

      if (m>0) moffset += 2;
      else     moffset += 1;
    }
  }

  dev_coefs = host_coefs;
}


void SphericalBasis::DtoH_coefs(Matrix& expcoef)
{
  host_coefs = dev_coefs;

  // l loop
  //
  for (int l=0, loffset=0; l<=Lmax; loffset+=(2*l+1), l++) {

    // m loop
    //
    for (int m=0, moffset=0; m<=l; m++) {
	
      // n loop
      //
      for (int n=1; n<=nmax; n++) {
	expcoef[loffset+moffset][n] = host_coefs[Ilmn(l, m, 'c', n-1, nmax)];
	if (m>0) expcoef[loffset+moffset+1][n] = host_coefs[Ilmn(l, m, 's', n-1, nmax)];
      }

      if (m>0) moffset += 2;
      else     moffset += 1;
    }
  }
}

void SphericalBasis::destroy_cuda()
{
  // std::cout << "texture object array size = " << tex.size() << std::endl;
  for (size_t i=0; i<tex.size(); i++) {
    std::ostringstream sout;
    sout << "trying to free TextureObject [" << i << "]";
    cuda_safe_call(hipDestroyTextureObject(tex[i]),
		   __FILE__, __LINE__, sout.str());
  }

  // std::cout << "cuInterpArray size = " << cuInterpArray.size() << std::endl;
  for (size_t i=0; i<cuInterpArray.size(); i++) {
    std::ostringstream sout;
    sout << "trying to free cuArray [" << i << "]";
    cuda_safe_call(hipFreeArray(cuInterpArray[i]),
		     __FILE__, __LINE__, sout.str());
  }
    
  // std::cout << "cuda memory freed" << std::endl;
}

void SphericalBasis::host_dev_force_compare()
{
  // Copy from device
  cC->host_particles = cC->cuda_particles;
  
  std::streamsize ss = std::cout.precision();
  std::cout.precision(4);

  std::cout << std::string(16+14*7, '-') << std::endl
	    << std::setw(8)  << "Index"  << std::setw(8)  << "Level"
	    << std::setw(14) << "ax [d]" << std::setw(14) << "ay [d]"
	    << std::setw(14) << "az [d]" << std::setw(14) << "ax [h]"
	    << std::setw(14) << "ay [h]" << std::setw(14) << "az [h]"
	    << std::setw(14) << "|Del a|/|a|"  << std::endl;

  // Compare first and last 5 from the device list
  //
  for (size_t i=0; i<5; i++) 
    {
      auto indx = cC->host_particles[i].indx;
      auto levl = cC->host_particles[i].level;
      
      std::cout << std::setw(8) << indx	<< std::setw(8) << levl;

      for (int k=0; k<3; k++)
	std::cout << std::setw(14) << cC->host_particles[i].acc[k];

      for (int k=0; k<3; k++)
	std::cout << std::setw(14) << cC->Particles()[indx].acc[k];

      double diff = 0.0, norm = 0.0;
      for (int k=0; k<3; k++) {
	double b  = cC->host_particles[i].acc[k];
	double a  = cC->Particles()[indx].acc[k];
	diff += (a - b)*(a - b);
	norm += a*a;
      }
      std::cout << std::setw(14) << sqrt(diff/norm) << std::endl;
    }
  
  for (size_t j=0; j<5; j++) 
    {
      size_t i = cC->host_particles.size() - 6 + j;

      auto indx = cC->host_particles[i].indx;
      auto levl = cC->host_particles[i].level;

      std::cout << std::setw(8) << indx	<< std::setw(8) << levl;
      
      for (int k=0; k<3; k++)
	std::cout << std::setw(14) << cC->host_particles[i].acc[k];

      for (int k=0; k<3; k++)
	std::cout << std::setw(14) << cC->Particles()[indx].acc[k];

      double diff = 0.0, norm = 0.0;
      for (int k=0; k<3; k++) {
	double b  = cC->host_particles[i].acc[k];
	double a  = cC->Particles()[indx].acc[k];
	diff += (a - b)*(a - b);
	norm += a*a;
      }
      std::cout << std::setw(14) << sqrt(diff/norm) << std::endl;
    }

  std::cout << std::string(16+14*7, '-') << std::endl;
  std::cout.precision(ss);
}

    
