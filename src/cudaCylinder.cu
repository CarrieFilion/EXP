#include "hip/hip_runtime.h"
#include <Component.H>
#include <Cylinder.H>
#include <cudaReduce.cuH>

// Define for debugging
//
// #define OFF_GRID_ALERT
// #define BOUNDS_CHECK
// #define VERBOSE

// Global symbols for coordinate transformation
//
__device__ __constant__
cuFP_t cylRscale, cylHscale, cylXmin, cylXmax, cylYmin, cylYmax, cylDxi, cylDyi, cylCen[3], cylBody[9], cylOrig[9];

__device__ __constant__
int   cylNumx, cylNumy, cylCmap;

__device__ __constant__
bool  cylOrient;

__host__ __device__
int Imn(int m, char cs, int n, int nmax)
{
  int ret = 0;

  if (m==0) ret = n;
  else ret = (2*m - 1 + (cs=='s' ? 1 : 0))*nmax + n;

#ifdef BOUNDS_CHECK
  // Verbose sanity check
  if (ret >= (2*m+1)*nmax) {
    printf("Imn oab: %4d %4d %4d [%4d : %4d ]\n", m, n, ret, (2*m+1)*nmax, nmax);
  }
#endif
  return ret;
}

__global__
void testConstantsCyl()
{
  printf("** Rscale = %f\n", cylRscale);
  printf("** Hscale = %f\n", cylHscale);
  printf("** Xmin   = %f\n", cylXmin);
  printf("** Xmax   = %f\n", cylXmax);
  printf("** Ymin   = %f\n", cylYmin);
  printf("** Ymax   = %f\n", cylYmax);
  printf("** Dxi    = %f\n", cylDxi);
  printf("** Dyi    = %f\n", cylDyi);
  printf("** Numx   = %d\n", cylNumx);
  printf("** Numy   = %d\n", cylNumy);
  printf("** Cmap   = %d\n", cylCmap);
}

				// R coordinate transformation
__device__
cuFP_t cu_r_to_xi_cyl(cuFP_t r)
{
  cuFP_t ret;

  if (cylCmap==1) {
    ret = (r/cylRscale - 1.0)/(r/cylRscale + 1.0);
  } else {
    ret = r;
  }    

  return ret;
}
    
__device__
cuFP_t cu_xi_to_r_cyl(cuFP_t xi)
{
  cuFP_t ret;

  if (cylCmap==1) {
    ret = (1.0 + xi)/(1.0 - xi) * cylRscale;
  } else {
    ret = xi;
  }

  return ret;
}

__device__
cuFP_t cu_d_xi_to_r_cyl(cuFP_t xi)
{
  cuFP_t ret;

  if (cylCmap==1) {
    ret = 0.5*(1.0 - xi)*(1.0 - xi) / cylRscale;
  } else {
    ret = 1.0;
  }

  return ret;
}

				// Z coordinate transformation
__device__
cuFP_t cu_z_to_y_cyl(cuFP_t z)
{ return z/(fabs(z)+FLT_MIN)*asinh(fabs(z/cylHscale)); }

__device__
cuFP_t cu_y_to_z_cyl(cuFP_t y)
{ return cylHscale*sinh(y); }

__device__
cuFP_t cu_d_y_to_z_cyl(cuFP_t y)
{ return cylHscale*cosh(y); }


void Cylinder::initialize_mapping_constants()
{
  // Copy constants to device
  //
  
  cudaMappingConstants f = getCudaMappingConstants();

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylRscale), &f.rscale, sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylRscale");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylHscale), &f.hscale, sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylHscale");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylXmin),   &f.xmin,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylXmin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylXmax),   &f.xmax,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylXmax");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylDxi),    &f.dxi,    sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylDxi");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylNumx),   &f.numx,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylNumx");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylYmin),   &f.ymin,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylYmin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylYmax),   &f.ymax,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylYmax");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylDyi),    &f.dyi,    sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylDxi");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylNumy),   &f.numy,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylNumy");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCmap),   &f.cmap,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCmap");

}


__global__ void coordKernelCyl
(dArray<cudaParticle> in, dArray<cuFP_t> mass, dArray<cuFP_t> phi,
 dArray<cuFP_t> Xfac, dArray<cuFP_t> Yfac,
 dArray<int> IndX, dArray<int> IndY,
 unsigned int stride, PII lohi, cuFP_t rmax)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i = tid*stride + n;	// Particle counter
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {

#ifdef BOUNDS_CHECK
      if (npart>=in._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle p = in._v[npart];
    
      cuFP_t xx=0.0, yy=0.0, zz=0.0;

      if (cylOrient) {
	for (int k=0; k<3; k++) xx += cylBody[0+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) yy += cylBody[3+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) zz += cylBody[6+k]*(p.pos[k] - cylCen[k]);
      } else {
	xx = p.pos[0] - cylCen[0];
	yy = p.pos[1] - cylCen[1];
	zz = p.pos[2] - cylCen[2];
      }
      
      cuFP_t R2 = xx*xx + yy*yy;
      cuFP_t r2 = R2 + zz*zz;
      cuFP_t R  = sqrt(R2);
      cuFP_t r  = sqrt(r2);
#ifdef BOUNDS_CHECK
      if (i>=mass._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      mass._v[i] = -1.0;
      
      if (r<=rmax) {
	
	mass._v[i] = p.mass;
	
	phi._v[i] = atan2(yy, xx);

#ifdef BOUNDS_CHECK
	if (i>=phi._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	// Interpolation indices
	//
	cuFP_t X  = (cu_r_to_xi_cyl(R) - cylXmin)/cylDxi;
	cuFP_t Y  = (cu_z_to_y_cyl(zz) - cylYmin)/cylDyi;

	int indX = floor(X);
	int indY = floor(Y);
	
	if (indX<0) indX = 0;
	if (indX>cylNumx-2) indX = cylNumx - 2;
	
	if (indY<0) indY = 0;
	if (indY>cylNumy-2) indY = cylNumy - 2;
	
	Xfac._v[i] = cuFP_t(indX+1) - X;
	IndX._v[i] = indX;

	Yfac._v[i] = cuFP_t(indY+1) - Y;
	IndY._v[i] = indY;

#ifdef OFF_GRID_ALERT
	if (Xfac._v[i]<-0.5 or Xfac._v[i]>1.5) printf("X off grid: x=%f\n", X);
	if (Yfac._v[i]<-0.5 or Yfac._v[i]>1.5) printf("Y off grid: y=%f\n", Y);
#endif
#ifdef BOUNDS_CHECK
	if (i>=Xfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=IndX._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=Yfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=IndY._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      }
    }
  }
}


__global__ void coefKernelCyl
(dArray<cuFP_t> coef, dArray<hipTextureObject_t> tex,
 dArray<cuFP_t> Mass, dArray<cuFP_t> Phi,
 dArray<cuFP_t> Xfac, dArray<cuFP_t> Yfac,
 dArray<int> indX, dArray<int> indY,
 int stride, int m, unsigned int nmax, PII lohi)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of particles to be evaluated
  //
  const unsigned int N = lohi.second - lohi.first;

  const cuFP_t norm = -4.0*M_PI;	// Biorthogonality factor

  for (int istr=0; istr<stride; istr++) {

    int i = tid*stride + istr;	// Particle counter

    if (i<N) {			// Allow for grid padding

      cuFP_t mass = Mass._v[i];
      
#ifdef BOUNDS_CHECK
      if (i>=Mass._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif      
      if (mass>0.0) {
#ifdef BOUNDS_CHECK
	if (i>=Phi._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	cuFP_t phi  = Phi._v[i];
	cuFP_t cosp = cos(phi*m);
	cuFP_t sinp = sin(phi*m);
	
	// Do the interpolation
	//
	cuFP_t delx0 = Xfac._v[i];
	cuFP_t dely0 = Yfac._v[i];
	cuFP_t delx1 = 1.0 - delx0;
	cuFP_t dely1 = 1.0 - dely0;

#ifdef BOUNDS_CHECK
	if (i>=Xfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=Yfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	cuFP_t c00 = delx0*dely0;
	cuFP_t c10 = delx1*dely0;
	cuFP_t c01 = delx0*dely1;
	cuFP_t c11 = delx1*dely1;

	int   indx = indX._v[i];
	int   indy = indY._v[i];

#ifdef BOUNDS_CHECK
	if (i>=indX._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=indY._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	for (int n=0; n<nmax; n++) {

	  // Texture maps are packed in slices
	  // ---------------------------------
	  // potC, rforceC, zforceC, potS, rforceS, zforceS
	  // 0     1        2        3     4        5

	  int k = m*nmax + n;

#if cuREAL == 4
	  cuFP_t d00  = tex3D<float>(tex._v[k], indx,   indy  , 0);
	  cuFP_t d10  = tex3D<float>(tex._v[k], indx+1, indy  , 0);
	  cuFP_t d01  = tex3D<float>(tex._v[k], indx,   indy+1, 0);
	  cuFP_t d11  = tex3D<float>(tex._v[k], indx+1, indy+1, 0);

#else
	  cuFP_t d00  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy  , 0));
	  cuFP_t d10  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy  , 0));
	  cuFP_t d01  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy+1, 0));
	  cuFP_t d11  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy+1, 0));
#endif

#ifdef BOUNDS_CHECK
	  if (k>=tex._s)            printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	  if ((2*n+0)*N+i>=coef._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	  coef._v[(2*n+0)*N + i] = (c00*d00 + c10*d10 + c01*d01 + c11*d11) * cosp * norm * mass;

	  if (m>0) {
	    // potS tables are offset from potC tables by +3
	    //
#if cuREAL == 4
	    d00  = tex3D<float>(tex._v[k], indx,   indy  , 3);
	    d10  = tex3D<float>(tex._v[k], indx+1, indy  , 3);
	    d01  = tex3D<float>(tex._v[k], indx,   indy+1, 3);
	    d11  = tex3D<float>(tex._v[k], indx+1, indy+1, 3);
#else
	    d00  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy  , 3));
	    d10  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy  , 3));
	    d01  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy+1, 3));
	    d11  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy+1, 3));
#endif

	    coef._v[(2*n+1)*N + i] = (c00*d00 + c10*d10 + c01*d01 + c11*d11) * sinp * norm * mass;

#ifdef BOUNDS_CHECK
	    if ((2*n+1)*N+i>=coef._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	  } // m>0
	  else {
	    coef._v[(2*n+1)*N + i] = 0.0;
	  }

	} // norder loop

      } else {
	// No contribution from off-grid particles
	for (int n=0; n<nmax; n++) {
	  coef._v[(2*n+0)*N + i] = 0.0;
	  if (m) coef._v[(2*n+1)*N + i] = 0.0;
	}

      } // mass value check

    } // particle index check

  } // stride loop
}

__global__ void
forceKernelCyl(dArray<cudaParticle> in, dArray<cuFP_t> coef,
	       dArray<hipTextureObject_t> tex,
	       int stride, unsigned int mmax, unsigned int nmax, PII lohi,
	       cuFP_t rmax, cuFP_t cylmass, bool external)
{
  // Thread ID
  //
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;

  // Maximum radius squared
  //
  const cuFP_t rmax2 = rmax*rmax;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {
      
#ifdef BOUNDS_CHECK
      if (npart>=in._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle p = in._v[npart];
      
      cuFP_t acc[3] = {0.0, 0.0, 0.0};
      cuFP_t xx=0.0, yy=0.0, zz=0.0;

      if (cylOrient) {
	for (int k=0; k<3; k++) xx += cylBody[0+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) yy += cylBody[3+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) zz += cylBody[6+k]*(p.pos[k] - cylCen[k]);
      } else {
	xx = p.pos[0] - cylCen[0];
	yy = p.pos[1] - cylCen[1];
	zz = p.pos[2] - cylCen[2];
      }

      cuFP_t phi = atan2(yy, xx);
      cuFP_t R2  = xx*xx + yy*yy;
      cuFP_t  R  = sqrt(R2) + FSMALL;
      
      const cuFP_t ratmin = 0.75;
      const cuFP_t maxerf = 3.0;
      const cuFP_t midpt  = ratmin + 0.5*(1.0 - ratmin);
      const cuFP_t rsmth  = 0.5*(1.0 - ratmin)/maxerf;

      cuFP_t ratio = sqrt( (R2 + zz*zz)/rmax2 );
      cuFP_t mfactor = 1.0, frac = 1.0, cfrac = 0.0;

      if (ratio >= 1.0) {
	cfrac      = 1.0 - mfactor;
      } else if (ratio > ratmin) {
	frac  = 0.5*(1.0 - erf( (ratio - midpt)/rsmth )) * mfactor;
	cfrac = 1.0 - frac;
      } else {
	frac  = mfactor;
      }

      cuFP_t fr = 0.0;
      cuFP_t fz = 0.0;
      cuFP_t fp = 0.0;
      cuFP_t pp = 0.0;
      
      if (ratio < 1.0) {

	cuFP_t X  = (cu_r_to_xi_cyl(R) - cylXmin)/cylDxi;
	cuFP_t Y  = (cu_z_to_y_cyl(zz) - cylYmin)/cylDyi;

	int indX = floor(X);
	int indY = floor(Y);
	
	if (indX < 0) indX = 0;
	if (indY < 0) indY = 0;
	if (indX >= cylNumx) indX = cylNumx - 1;
	if (indY >= cylNumy) indY = cylNumy - 1;

	cuFP_t delx0 = cuFP_t(indX+1) - X;
	cuFP_t dely0 = cuFP_t(indY+1) - Y;

#ifdef OFF_GRID_ALERT
	if (delx0<0.0 or delx0>1.0) printf("X off grid: x=%f\n", delx0);
	if (dely0<0.0 or dely0>1.0) printf("Y off grid: y=%f\n", dely0);
#endif

	cuFP_t delx1 = 1.0 - delx0;
	cuFP_t dely1 = 1.0 - dely0;
      
	cuFP_t c00 = delx0*dely0;
	cuFP_t c10 = delx1*dely0;
	cuFP_t c01 = delx0*dely1;
	cuFP_t c11 = delx1*dely1;

	cuFP_t cos1 = cos(phi);
	cuFP_t sin1 = sin(phi);

	cuFP_t ccos = 1.0;
	cuFP_t ssin = 0.0;

	for (int mm=0; mm<=mmax; mm++) {

	  for (int n=0; n<nmax; n++) {
      
	    cuFP_t fac0 = coef._v[Imn(mm, 'c', n, nmax)];
	    cuFP_t fac1 = fac0 * ccos;
	    cuFP_t fac2 = fac0 * ssin;
      
	    // Texture table index
	    //
	    int k = mm*nmax + n;

	    pp += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 0) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 0) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 0) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 0) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 0)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 0)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 0)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 0)) * c11 
#endif
	       );
	    
	    fr += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 1) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 1) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 1) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 1) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 1)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 1)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 1)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 1)) * c11 
#endif
	       );
      
	    fz += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 2) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 2) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 2) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 2) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 2)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 2)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 2)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 2)) * c11 
#endif
	       );
	    
	    fp += fac2 * mm *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 0) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 0) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 0) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 0) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 0)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 0)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 0)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 0)) * c11 
#endif
	       );
      
      
	    if (mm) {
	
	      cuFP_t fac0 =  coef._v[Imn(mm, 's', n, nmax)];
	      cuFP_t fac1 =  fac0 * ssin;
	      cuFP_t fac2 = -fac0 * ccos;

	      pp += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 3) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 3) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 3) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 3) * c11
#else		 
 		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 3)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 3)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 3)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 3)) * c11 
#endif
		 );
	      
	      fr += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 4) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 4) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 4) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 4) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 4)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 4)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 4)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 4)) * c11 
#endif
		 );
	      
	      fz += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 5) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 5) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 5) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 5) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 5)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 5)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 5)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 5)) * c11 
#endif
		 );
	      
	      fp += fac2 * mm *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 3) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 3) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 3) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 3) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 3)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 3)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 3)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 3)) * c11 
#endif
		 );
	      
	    }
	  }
	  
	  // Trig recursion to squeeze avoid internal FP fct call
	  //
	  cuFP_t cosM = ccos;
	  cuFP_t sinM = ssin;

	  ccos = cosM * cos1 - sinM * sin1;
	  ssin = sinM * cos1 + cosM * sin1;
	}

	acc[0] += ( fr*xx/R - fp*yy/R2 ) * frac;
	acc[1] += ( fr*yy/R + fp*xx/R2 ) * frac;
	acc[2] += fz * frac;
      }

      if (ratio > ratmin) {

	cuFP_t r3 = R2 + zz*zz;
	pp = -cylmass/sqrt(r3);	// -M/r
	fr = pp/r3;		// -M/r^3

	acc[0] += xx*fr * cfrac;
	acc[1] += yy*fr * cfrac;
	acc[2] += zz*fr * cfrac;
      }

      if (cylOrient) {
	for (int j=0; j<3; j++) {
	  for (int k=0; k<3; k++) in._v[npart].acc[j] += cylOrig[3*j+k]*acc[k];
	}
      } else {
	for (int j=0; j<3; j++) in._v[npart].acc[j] += acc[j];
      }

      if (external)
	in._v[npart].potext += pp;
      else
	in._v[npart].pot    += pp;

    } // Particle index block

  } // END: stride loop

}



template<typename T>
class LessAbs : public std::binary_function<bool, T, T>
{
public:
  T operator()( const T &a, const T &b ) const
  {
    return (fabs(a) < fabs(b));
  }
};

static bool initialize_cuda_cyl = true;

void Cylinder::determine_coefficients_cuda()
{
  if (initialize_cuda_cyl) {
    initialize_cuda();
    initialize_mapping_constants();
    initialize_cuda_cyl = false;
    // Only copy texture memory once
    t_d = tex;
  }

  std::cout << std::scientific;

  int deviceCount = 0;
  cuda_safe_call(hipGetDeviceCount(&deviceCount),
		 __FILE__, __LINE__, "could not get device count");

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceCount-1);

  // Sort particles and get coefficient size
  //
  PII lohi = cC->CudaSortByLevel(mlevel, mlevel);

  // Zero out coefficients
  //
  host_coefs.resize((2*mmax+1)*ncylorder);

  // Compute grid
  //
  unsigned int N         = lohi.second - lohi.first;
  unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
  unsigned int gridSize  = N/BLOCK_SIZE/stride;

  if (N == 0) {
    use[0] = 0.0;
    cylmass0[0] = 0.0;
    return;
  }

  if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

  std::vector<cuFP_t> ctr;
  for (auto v : cC->getCenter(Component::Local | Component::Centered)) ctr.push_back(v);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCen), &ctr[0], sizeof(cuFP_t)*3,
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCen");

  bool orient = (cC->EJ & Orient::AXIS) && !cC->EJdryrun;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrient), &orient,   sizeof(bool),  size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylOrient");

  if (orient) {
    std::vector<cuFP_t> trans(9);
    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++) trans[i*3+j] = cC->orient->transformBody()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylBody), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylBody");
  }

#ifdef VERBOSE
  std::cout << std::endl << "**" << std::endl
	    << "** N      = " << N           << std::endl
	    << "** I low  = " << lohi.first  << std::endl
	    << "** I high = " << lohi.second << std::endl
	    << "** Stride = " << stride      << std::endl
	    << "** Block  = " << BLOCK_SIZE  << std::endl
	    << "** Grid   = " << gridSize    << std::endl
	    << "** Xcen   = " << ctr[0]     << std::endl
	    << "** Ycen   = " << ctr[1]     << std::endl
	    << "** Zcen   = " << ctr[2]     << std::endl
	    << "**" << std::endl;
#endif

  // Create space for coefficient reduction
  //
  dN_coef.resize(2*ncylorder*N);
  dc_coef.resize(2*ncylorder*gridSize);
  df_coef.resize(2*ncylorder);

  // Space for coordinate arrays
  //
  m_d.resize(N);
  X_d.resize(N);
  Y_d.resize(N);
  p_d.resize(N);
  iX_d.resize(N);
  iY_d.resize(N);

  // Shared memory size for the reduction
  //
  int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);

  // For debugging (set to false to disable)
  //
  static bool firstime = false;

  if (firstime) {
    testConstantsCyl<<<1, 1>>>();
    hipDeviceSynchronize();
    firstime = false;
  }

  thrust::counting_iterator<int> index_begin(0);
  thrust::counting_iterator<int> index_end(gridSize*2*ncylorder);

  // Maximum radius on grid
  //
  cuFP_t rmax = rcylmax * acyl * M_SQRT1_2;

  // Do the work
  //
				// Compute the coordinate
				// transformation
				// 
  coordKernelCyl<<<gridSize, BLOCK_SIZE>>>
    (toKernel(cC->cuda_particles), toKernel(m_d), toKernel(p_d),
     toKernel(X_d), toKernel(Y_d), toKernel(iX_d), toKernel(iY_d),
     stride, lohi, rmax);

				// Compute the coefficient
				// contribution for each order
  int osize = ncylorder*2;	// 
  for (int m=0; m<=mmax; m++) {
    coefKernelCyl<<<gridSize, BLOCK_SIZE>>>
      (toKernel(dN_coef), toKernel(t_d), toKernel(m_d), toKernel(p_d),
       toKernel(X_d), toKernel(Y_d), toKernel(iX_d), toKernel(iY_d),
       stride, m, ncylorder, lohi);
    
				// Begin the reduction per grid block
				//
    reduceSum<cuFP_t, BLOCK_SIZE><<<gridSize, BLOCK_SIZE, sMemSize>>>
      (toKernel(dc_coef), toKernel(dN_coef), osize, N);
      
				// Finish the reduction for this order
				// in parallel
    thrust::reduce_by_key
      (
       thrust::make_transform_iterator(index_begin, key_functor(gridSize)),
       thrust::make_transform_iterator(index_end,   key_functor(gridSize)),
       dc_coef.begin(), thrust::make_discard_iterator(), df_coef.begin()
       );
    
    thrust::host_vector<cuFP_t> ret = df_coef;
    for (size_t j=0; j<ncylorder; j++) {
      host_coefs[Imn(m, 'c', j, ncylorder)] = ret[2*j];
      if (m>0) host_coefs[Imn(m, 's', j, ncylorder)] = ret[2*j+1];
    }
  }

  // DEBUG
  //
  if (true) {
    std::cout << std::string(2*4+4*20, '-') << std::endl
	      << "---- Cylindrical "      << std::endl
	      << std::string(2*4+4*20, '-') << std::endl;
    std::cout << "M=0 coefficients" << std::endl
	      << std::setprecision(10);
    

    std::cout << std::setw(4)  << "n"
	      << std::setw(4)  << "i"
	      << std::setw(20) << "GPU"
	      << std::setw(20) << "CPU"
	      << std::setw(20) << "diff"
	      << std::setw(20) << "rel diff"
	      << std::endl;

    int i = Imn(0, 'c', 0, ncylorder);
    auto cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(0, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(0, n, 'c');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }

    std::cout << "M=1c coefficients" << std::endl;

    i = Imn(1, 'c', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(1, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(1, n, 'c');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }

    std::cout << "M=1s coefficients" << std::endl;

    i = Imn(1, 's', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(1, 's', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(1, n, 's');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }

    std::cout << "M=2c coefficients" << std::endl;

    i = Imn(2, 'c', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(2, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(2, n, 'c');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }
    
    std::cout << "M=2s coefficients" << std::endl;

    i = Imn(2, 's', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(2, 's', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(2, n, 's');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }

    std::cout << std::string(2*4+4*20, '-') << std::endl;
  }


  //
  // TEST comparison of coefficients for debugging
  //
  if (true) {

    struct Element
    {
      double d;
      double f;
      
      int  m;
      int  n;
      
      char cs;
    }
    elem;

    std::multimap<double, Element> compare;

    std::ofstream out("test_cyl.dat");

    // m loop
    for (int m=0; m<=mmax; m++) {
	
      if (m==0) {
	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 'c';
	  elem.d = ortho->get_coef(m, n, 'c');
	  elem.f = host_coefs[Imn(m, 'c', n, ncylorder)];
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);
	  
	  compare.insert(std::make_pair(test, elem));;
	    
	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 'c'
	      << std::setw( 5) << Imn(m, 'c', n, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	}

      } else {
	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 'c';
	  elem.d = ortho->get_coef(m, n, 'c');
	  elem.f = host_coefs[Imn(m, 'c', n, ncylorder)];

	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 'c'
	      << std::setw( 5) << Imn(m, 'c', n, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);

	  compare.insert(std::make_pair(test, elem));;
	}

	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 's';
	  elem.d = ortho->get_coef(m, n, 's');
	  elem.f = host_coefs[Imn(m, 's', n, ncylorder)];

	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 's'
	      << std::setw( 5) << Imn(m, 's', n-1, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);
	  
	  compare.insert(std::make_pair(test, elem));;
	}
      }
    }
    
    std::map<double, Element>::iterator best = compare.begin();
    std::map<double, Element>::iterator midl = best;
    std::advance(midl, compare.size()/2);
    std::map<double, Element>::reverse_iterator last = compare.rbegin();
    
    std::cout << std::string(3*2 + 3*20 + 20, '-') << std::endl
	      << "---- Cylinder coefficients" << std::endl
	      << std::string(3*2 + 3*20 + 20, '-') << std::endl;

    std::cout << "Best case: ["
	      << std::setw( 2) << best->second.m << ", "
	      << std::setw( 2) << best->second.n << ", "
	      << std::setw( 2) << best->second.cs << "] = "
	      << std::setw(20) << best->second.d
	      << std::setw(20) << best->second.f
	      << std::setw(20) << fabs(best->second.d - best->second.f)
	      << std::endl;
  
    std::cout << "Mid case:  ["
	      << std::setw( 2) << midl->second.m << ", "
	      << std::setw( 2) << midl->second.n << ", "
	      << std::setw( 2) << midl->second.cs << "] = "
	      << std::setw(20) << midl->second.d
	      << std::setw(20) << midl->second.f
	      << std::setw(20) << fabs(midl->second.d - midl->second.f)
	      << std::endl;
    
    std::cout << "Last case: ["
	      << std::setw( 2) << last->second.m << ", "
	      << std::setw( 2) << last->second.n << ", "
	      << std::setw( 2) << last->second.cs << "] = "
	      << std::setw(20) << last->second.d
	      << std::setw(20) << last->second.f
	      << std::setw(20) << fabs(last->second.d - last->second.f)
	      << std::endl;
  }

  // Compute number and total mass of particles used in coefficient
  // determination
  //
  thrust::sort(m_d.begin(), m_d.end());

  auto m_it   = thrust::upper_bound(m_d.begin(), m_d.end(), 0.0);
  use[0]      = thrust::distance(m_it, m_d.end());
  cylmass0[0] = thrust::reduce  (m_it, m_d.end());
}


void Cylinder::determine_acceleration_cuda()
{
  if (initialize_cuda_cyl) {
    initialize_cuda();
    initialize_mapping_constants();
    initialize_cuda_cyl = false;
    // Only copy texture memory once
    t_d = tex;
  }

  std::cout << std::scientific;

  int deviceCount = 0;
  cuda_safe_call(hipGetDeviceCount(&deviceCount),
		 __FILE__, __LINE__, "could not get device count");

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceCount-1);

  // Sort particles and get coefficient size
  //
  PII lohi = cC->CudaSortByLevel(mlevel, multistep);

  // Compute grid
  //
  unsigned int N         = lohi.second - lohi.first;
  unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
  unsigned int gridSize  = N/BLOCK_SIZE/stride;

  if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

  std::vector<cuFP_t> ctr;
  for (auto v : cC->getCenter(Component::Local | Component::Centered)) ctr.push_back(v);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCen), &ctr[0], sizeof(cuFP_t)*3,
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCen");

  bool orient = (cC->EJ & Orient::AXIS) && !cC->EJdryrun;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrient), &orient,   sizeof(bool),  size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylOrient");

  if (orient) {
    std::vector<cuFP_t> trans(9);
    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++)
	trans[i*3+j] = cC->orient->transformBody()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylBody), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylBody");

    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++)
	trans[i*3+j] = cC->orient->transformOrig()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrig), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylOrig");
  }

#ifdef VERBOSE
  std::cout << std::endl << "**" << std::endl
	    << "** N      = " << N          << std::endl
	    << "** Stride = " << stride     << std::endl
	    << "** Block  = " << BLOCK_SIZE << std::endl
	    << "** Grid   = " << gridSize   << std::endl
	    << "** Xcen   = " << ctr[0]     << std::endl
	    << "** Ycen   = " << ctr[1]     << std::endl
	    << "** Zcen   = " << ctr[2]     << std::endl
	    << "**" << std::endl;
#endif

  // Shared memory size for the reduction
  //
  int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);

  // Maximum radius on grid
  //
  cuFP_t rmax = rcylmax * acyl;

  // Do the work
  //
  forceKernelCyl<<<gridSize, BLOCK_SIZE, sMemSize>>>
    (toKernel(cC->cuda_particles), toKernel(dev_coefs), toKernel(t_d),
     stride, mmax, ncylorder, lohi, rmax, cylmass, use_external);
}

void Cylinder::HtoD_coefs()
{
  // Check size
  host_coefs.resize((2*mmax+1)*ncylorder);

  // Copy from EmpCylSL
  
  // m loop
  //
  for (int m=0; m<=mmax; m++) {
    
    // n loop
    //
    for (int n=0; n<ncylorder; n++) {
      host_coefs[Imn(m, 'c', n, ncylorder)] = ortho->get_coef(m, n, 'c');
      if (m>0) host_coefs[Imn(m, 's', n, ncylorder)] = ortho->get_coef(m, n, 's');
    }
  }

  // Copy to device
  dev_coefs = host_coefs;
}


void Cylinder::DtoH_coefs(int M)
{
  // Copy from host device to EmpCylSL

  // m loop
  //
  for (int m=0; m<=mmax; m++) {
    
    // n loop
    //
    for (int n=0; n<ncylorder; n++) {
      ortho->set_coef(M, m, n, 'c') = host_coefs[Imn(m, 'c', n, ncylorder)];
      if (m>0) ortho->set_coef(M, m, n, 's') = host_coefs[Imn(m, 's', n, ncylorder)];
    }
  }
}

void Cylinder::destroy_cuda()
{
  // std::cout << "texture object array size = " << tex.size() << std::endl;
  for (size_t i=0; i<tex.size(); i++) {
    std::ostringstream sout;
    sout << "trying to free TextureObject [" << i << "]";
    cuda_safe_call(hipDestroyTextureObject(tex[i]),
		   __FILE__, __LINE__, sout.str());
  }

  // std::cout << "cuInterpArray size = " << cuInterpArray.size() << std::endl;
  for (size_t i=0; i<cuInterpArray.size(); i++) {
    std::ostringstream sout;
    sout << "trying to free cuPitch [" << i << "]";
    cuda_safe_call(hipFree(cuInterpArray[i]),
		     __FILE__, __LINE__, sout.str());
  }
    
  std::cout << "cuda memory freed" << std::endl;
}

void Cylinder::host_dev_force_compare()
{
  // Copy from device
  cC->host_particles = cC->cuda_particles;
  
  std::streamsize ss = std::cout.precision();
  std::cout.precision(10);

  std::cout << std::string(16+20*8, '-') << std::endl
	    << std::setw(8)  << "Index"  << std::setw(8)  << "Level"
	    << std::setw(20) << "ax [d]" << std::setw(20) << "ay [d]"
	    << std::setw(20) << "az [d]" << std::setw(20) << "ax [h]"
	    << std::setw(20) << "ay [h]" << std::setw(20) << "az [h]"
	    << std::setw(20) << "|Del a|/|a|"
	    << std::setw(20) << "|a|"    << std::endl;
  
  // Compare first and last 5 from the device list
  //
  for (size_t i=0; i<5; i++) 
    {
      auto indx = cC->host_particles[i].indx;
      auto levl = cC->host_particles[i].level;
      
      std::cout << std::setw(8) << indx	<< std::setw(8) << levl;

      for (int k=0; k<3; k++)
	std::cout << std::setw(20) << cC->host_particles[i].acc[k];

      for (int k=0; k<3; k++)
	std::cout << std::setw(20) << cC->Particles()[indx].acc[k];

      double diff = 0.0, norm = 0.0;
      for (int k=0; k<3; k++) {
	double b  = cC->host_particles[i].acc[k];
	double a  = cC->Particles()[indx].acc[k];
	diff += (a - b)*(a - b);
	norm += a*a;
      }
      std::cout << std::setw(20) << sqrt(diff/norm)
		<< std::setw(20) << sqrt(norm) << std::endl;
    }
  
  for (size_t j=0; j<5; j++) 
    {
      size_t i = cC->host_particles.size() - 5 + j;

      auto indx = cC->host_particles[i].indx;
      auto levl = cC->host_particles[i].level;

      std::cout << std::setw(8) << indx	<< std::setw(8) << levl;
      
      for (int k=0; k<3; k++)
	std::cout << std::setw(20) << cC->host_particles[i].acc[k];

      for (int k=0; k<3; k++)
	std::cout << std::setw(20) << cC->Particles()[indx].acc[k];

      double diff = 0.0, norm = 0.0;
      for (int k=0; k<3; k++) {
	double b  = cC->host_particles[i].acc[k];
	double a  = cC->Particles()[indx].acc[k];
	diff += (a - b)*(a - b);
	norm += a*a;
      }
      std::cout << std::setw(20) << sqrt(diff/norm)
		<< std::setw(20) << sqrt(norm) << std::endl;
    }

  std::cout << std::string(16+20*8, '-') << std::endl;
  std::cout.precision(ss);
}
