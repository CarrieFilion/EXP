#include "hip/hip_runtime.h"
#include <Component.H>
#include <Cylinder.H>
#include <cudaReduce.cuH>

// Define for debugging
//
// #define OFF_GRID_ALERT
// #define BOUNDS_CHECK
// #define VERBOSE

// Global symbols for coordinate transformation
//
__device__ __constant__
cuFP_t cylRscale, cylHscale, cylXmin, cylXmax, cylYmin, cylYmax, cylDxi, cylDyi, cylCen[3], cylBody[9], cylOrig[9];

__device__ __constant__
int   cylNumx, cylNumy, cylCmap;

__device__ __constant__
bool  cylOrient;

__host__ __device__
int Imn(int m, char cs, int n, int nmax)
{
  int ret = 0;

  if (m==0) ret = n;
  else ret = (2*m - 1 + (cs=='s' ? 1 : 0))*nmax + n;

#ifdef BOUNDS_CHECK
  // Verbose sanity check
  if (ret >= (2*m+1)*nmax) {
    printf("Imn oab: %4d %4d %4d [%4d : %4d ]\n", m, n, ret, (2*m+1)*nmax, nmax);
  }
#endif
  return ret;
}

__global__
void testConstantsCyl()
{
  printf("** Rscale = %f\n", cylRscale);
  printf("** Hscale = %f\n", cylHscale);
  printf("** Xmin   = %f\n", cylXmin);
  printf("** Xmax   = %f\n", cylXmax);
  printf("** Ymin   = %f\n", cylYmin);
  printf("** Ymax   = %f\n", cylYmax);
  printf("** Dxi    = %f\n", cylDxi);
  printf("** Dyi    = %f\n", cylDyi);
  printf("** Numx   = %d\n", cylNumx);
  printf("** Numy   = %d\n", cylNumy);
  printf("** Cmap   = %d\n", cylCmap);
}

				// R coordinate transformation
__device__
cuFP_t cu_r_to_xi_cyl(cuFP_t r)
{
  cuFP_t ret;

  if (cylCmap==1) {
    ret = (r/cylRscale - 1.0)/(r/cylRscale + 1.0);
  } else {
    ret = r;
  }    

  return ret;
}
    
__device__
cuFP_t cu_xi_to_r_cyl(cuFP_t xi)
{
  cuFP_t ret;

  if (cylCmap==1) {
    ret = (1.0 + xi)/(1.0 - xi) * cylRscale;
  } else {
    ret = xi;
  }

  return ret;
}

__device__
cuFP_t cu_d_xi_to_r_cyl(cuFP_t xi)
{
  cuFP_t ret;

  if (cylCmap==1) {
    ret = 0.5*(1.0 - xi)*(1.0 - xi) / cylRscale;
  } else {
    ret = 1.0;
  }

  return ret;
}

				// Z coordinate transformation
__device__
cuFP_t cu_z_to_y_cyl(cuFP_t z)
{ return z/(fabs(z)+FLT_MIN)*asinh(fabs(z/cylHscale)); }

__device__
cuFP_t cu_y_to_z_cyl(cuFP_t y)
{ return cylHscale*sinh(y); }

__device__
cuFP_t cu_d_y_to_z_cyl(cuFP_t y)
{ return cylHscale*cosh(y); }


void Cylinder::initialize_mapping_constants()
{
  // Copy constants to device
  //
  
  cudaMappingConstants f = getCudaMappingConstants();

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylRscale), &f.rscale, sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylRscale");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylHscale), &f.hscale, sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylHscale");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylXmin),   &f.xmin,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylXmin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylXmax),   &f.xmax,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylXmax");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylDxi),    &f.dxi,    sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylDxi");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylNumx),   &f.numx,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylNumx");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylYmin),   &f.ymin,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylYmin");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylYmax),   &f.ymax,   sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylYmax");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylDyi),    &f.dyi,    sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylDxi");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylNumy),   &f.numy,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylNumy");

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCmap),   &f.cmap,   sizeof(int),   size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCmap");

}


__global__ void coordKernelCyl
(dArray<cudaParticle> in, dArray<cuFP_t> mass, dArray<cuFP_t> phi,
 dArray<cuFP_t> Xfac, dArray<cuFP_t> Yfac,
 dArray<int> IndX, dArray<int> IndY,
 unsigned int stride, PII lohi, cuFP_t rmax)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i = tid*stride + n;	// Particle counter
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {

#ifdef BOUNDS_CHECK
      if (npart>=in._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle p = in._v[npart];
    
      cuFP_t xx=0.0, yy=0.0, zz=0.0;

      if (cylOrient) {
	for (int k=0; k<3; k++) xx += cylBody[0+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) yy += cylBody[3+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) zz += cylBody[6+k]*(p.pos[k] - cylCen[k]);
      } else {
	xx = p.pos[0] - cylCen[0];
	yy = p.pos[1] - cylCen[1];
	zz = p.pos[2] - cylCen[2];
      }
      
      cuFP_t R2 = xx*xx + yy*yy;
      cuFP_t r2 = R2 + zz*zz;
      cuFP_t R  = sqrt(R2);
      cuFP_t r  = sqrt(r2);
#ifdef BOUNDS_CHECK
      if (i>=mass._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      mass._v[i] = -1.0;
      
      if (r<=rmax) {
	
	mass._v[i] = p.mass;
	
	phi._v[i] = atan2(yy, xx);

#ifdef BOUNDS_CHECK
	if (i>=phi._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	// Interpolation indices
	//
	cuFP_t X  = (cu_r_to_xi_cyl(R) - cylXmin)/cylDxi;
	cuFP_t Y  = (cu_z_to_y_cyl(zz) - cylYmin)/cylDyi;

	int indX = floor(X);
	int indY = floor(Y);
	
	if (indX<0) indX = 0;
	if (indX>cylNumx-2) indX = cylNumx - 2;
	
	if (indY<0) indY = 0;
	if (indY>cylNumy-2) indY = cylNumy - 2;
	
	Xfac._v[i] = cuFP_t(indX+1) - X;
	IndX._v[i] = indX;

	Yfac._v[i] = cuFP_t(indY+1) - Y;
	IndY._v[i] = indY;

#ifdef OFF_GRID_ALERT
	if (Xfac._v[i]<-0.5 or Xfac._v[i]>1.5) printf("X off grid: x=%f\n", X);
	if (Yfac._v[i]<-0.5 or Yfac._v[i]>1.5) printf("Y off grid: y=%f\n", Y);
#endif
#ifdef BOUNDS_CHECK
	if (i>=Xfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=IndX._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=Yfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=IndY._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      }
    }
  }
}


__global__ void coefKernelCyl
(dArray<cuFP_t> coef, dArray<hipTextureObject_t> tex,
 dArray<cuFP_t> Mass, dArray<cuFP_t> Phi,
 dArray<cuFP_t> Xfac, dArray<cuFP_t> Yfac,
 dArray<int> indX, dArray<int> indY,
 int stride, int m, unsigned int nmax, PII lohi)
{
  // Thread ID
  //
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Total number of particles to be evaluated
  //
  const unsigned int N = lohi.second - lohi.first;

  const cuFP_t norm = -4.0*M_PI;	// Biorthogonality factor

  for (int istr=0; istr<stride; istr++) {

    int i = tid*stride + istr;	// Particle counter

    if (i<N) {			// Allow for grid padding

      cuFP_t mass = Mass._v[i];
      
#ifdef BOUNDS_CHECK
      if (i>=Mass._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif      
      if (mass>0.0) {
#ifdef BOUNDS_CHECK
	if (i>=Phi._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	cuFP_t phi  = Phi._v[i];
	cuFP_t cosp = cos(phi*m);
	cuFP_t sinp = sin(phi*m);
	
	// Do the interpolation
	//
	cuFP_t delx0 = Xfac._v[i];
	cuFP_t dely0 = Yfac._v[i];
	cuFP_t delx1 = 1.0 - delx0;
	cuFP_t dely1 = 1.0 - dely0;

#ifdef BOUNDS_CHECK
	if (i>=Xfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=Yfac._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	cuFP_t c00 = delx0*dely0;
	cuFP_t c10 = delx1*dely0;
	cuFP_t c01 = delx0*dely1;
	cuFP_t c11 = delx1*dely1;

	int   indx = indX._v[i];
	int   indy = indY._v[i];

#ifdef BOUNDS_CHECK
	if (i>=indX._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	if (i>=indY._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	for (int n=0; n<nmax; n++) {

	  // Texture maps are packed in slices
	  // ---------------------------------
	  // potC, rforceC, zforceC, potS, rforceS, zforceS
	  // 0     1        2        3     4        5

	  int k = m*nmax + n;

#if cuREAL == 4
	  cuFP_t d00  = tex3D<float>(tex._v[k], indx,   indy  , 0);
	  cuFP_t d10  = tex3D<float>(tex._v[k], indx+1, indy  , 0);
	  cuFP_t d01  = tex3D<float>(tex._v[k], indx,   indy+1, 0);
	  cuFP_t d11  = tex3D<float>(tex._v[k], indx+1, indy+1, 0);

#else
	  cuFP_t d00  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy  , 0));
	  cuFP_t d10  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy  , 0));
	  cuFP_t d01  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy+1, 0));
	  cuFP_t d11  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy+1, 0));
#endif

#ifdef BOUNDS_CHECK
	  if (k>=tex._s)            printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
	  if ((2*n+0)*N+i>=coef._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	  coef._v[(2*n+0)*N + i] = (c00*d00 + c10*d10 + c01*d01 + c11*d11) * cosp * norm * mass;

	  if (m>0) {
	    // potS tables are offset from potC tables by +3
	    //
#if cuREAL == 4
	    d00  = tex3D<float>(tex._v[k], indx,   indy  , 3);
	    d10  = tex3D<float>(tex._v[k], indx+1, indy  , 3);
	    d01  = tex3D<float>(tex._v[k], indx,   indy+1, 3);
	    d11  = tex3D<float>(tex._v[k], indx+1, indy+1, 3);
#else
	    d00  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy  , 3));
	    d10  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy  , 3));
	    d01  = int2_as_double(tex3D<int2>(tex._v[k], indx,   indy+1, 3));
	    d11  = int2_as_double(tex3D<int2>(tex._v[k], indx+1, indy+1, 3));
#endif

	    coef._v[(2*n+1)*N + i] = (c00*d00 + c10*d10 + c01*d01 + c11*d11) * sinp * norm * mass;

#ifdef BOUNDS_CHECK
	    if ((2*n+1)*N+i>=coef._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
	  } // m>0
	  else {
	    coef._v[(2*n+1)*N + i] = 0.0;
	  }

	} // norder loop

      } else {
	// No contribution from off-grid particles
	for (int n=0; n<nmax; n++) {
	  coef._v[(2*n+0)*N + i] = 0.0;
	  if (m) coef._v[(2*n+1)*N + i] = 0.0;
	}

      } // mass value check

    } // particle index check

  } // stride loop
}

__global__ void
forceKernelCyl(dArray<cudaParticle> in, dArray<cuFP_t> coef,
	       dArray<hipTextureObject_t> tex,
	       int stride, unsigned int mmax, unsigned int nmax, PII lohi,
	       cuFP_t rmax, cuFP_t cylmass, bool external)
{
  // Thread ID
  //
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;

  // Maximum radius squared
  //
  const cuFP_t rmax2 = rmax*rmax;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {
      
#ifdef BOUNDS_CHECK
      if (npart>=in._s) printf("out of bounds: %s:%d\n", __FILE__, __LINE__);
#endif
      cudaParticle p = in._v[npart];
      
      cuFP_t acc[3] = {0.0, 0.0, 0.0};
      cuFP_t xx=0.0, yy=0.0, zz=0.0;

      if (cylOrient) {
	for (int k=0; k<3; k++) xx += cylBody[0+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) yy += cylBody[3+k]*(p.pos[k] - cylCen[k]);
	for (int k=0; k<3; k++) zz += cylBody[6+k]*(p.pos[k] - cylCen[k]);
      } else {
	xx = p.pos[0] - cylCen[0];
	yy = p.pos[1] - cylCen[1];
	zz = p.pos[2] - cylCen[2];
      }

      cuFP_t phi = atan2(yy, xx);
      cuFP_t R2  = xx*xx + yy*yy;
      cuFP_t  R  = sqrt(R2) + FSMALL;
      
      const cuFP_t ratmin = 0.75;
      const cuFP_t maxerf = 3.0;
      const cuFP_t midpt  = ratmin + 0.5*(1.0 - ratmin);
      const cuFP_t rsmth  = 0.5*(1.0 - ratmin)/maxerf;

      cuFP_t ratio = sqrt( (R2 + zz*zz)/rmax2 );
      cuFP_t mfactor = 1.0, frac = 1.0, cfrac = 0.0;

      if (ratio >= 1.0) {
	cfrac      = 1.0 - mfactor;
      } else if (ratio > ratmin) {
	frac  = 0.5*(1.0 - erf( (ratio - midpt)/rsmth )) * mfactor;
	cfrac = 1.0 - frac;
      } else {
	frac  = mfactor;
      }

      cuFP_t fr = 0.0;
      cuFP_t fz = 0.0;
      cuFP_t fp = 0.0;
      cuFP_t pp = 0.0;
      
      if (ratio < 1.0) {

	cuFP_t X  = (cu_r_to_xi_cyl(R) - cylXmin)/cylDxi;
	cuFP_t Y  = (cu_z_to_y_cyl(zz) - cylYmin)/cylDyi;

	int indX = floor(X);
	int indY = floor(Y);
	
	if (indX < 0) indX = 0;
	if (indY < 0) indY = 0;
	if (indX >= cylNumx) indX = cylNumx - 1;
	if (indY >= cylNumy) indY = cylNumy - 1;

	cuFP_t delx0 = cuFP_t(indX+1) - X;
	cuFP_t dely0 = cuFP_t(indY+1) - Y;

#ifdef OFF_GRID_ALERT
	if (delx0<0.0 or delx0>1.0) printf("X off grid: x=%f\n", delx0);
	if (dely0<0.0 or dely0>1.0) printf("Y off grid: y=%f\n", dely0);
#endif

	cuFP_t delx1 = 1.0 - delx0;
	cuFP_t dely1 = 1.0 - dely0;
      
	cuFP_t c00 = delx0*dely0;
	cuFP_t c10 = delx1*dely0;
	cuFP_t c01 = delx0*dely1;
	cuFP_t c11 = delx1*dely1;

	cuFP_t cos1 = cos(phi);
	cuFP_t sin1 = sin(phi);

	cuFP_t ccos = 1.0;
	cuFP_t ssin = 0.0;

	for (int mm=0; mm<=mmax; mm++) {

	  for (int n=0; n<nmax; n++) {
      
	    cuFP_t fac0 = coef._v[Imn(mm, 'c', n, nmax)];
	    cuFP_t fac1 = fac0 * ccos;
	    cuFP_t fac2 = fac0 * ssin;
      
	    // Texture table index
	    //
	    int k = mm*nmax + n;

	    pp += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 0) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 0) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 0) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 0) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 0)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 0)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 0)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 0)) * c11 
#endif
	       );
	    
	    fr += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 1) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 1) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 1) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 1) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 1)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 1)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 1)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 1)) * c11 
#endif
	       );
      
	    fz += fac1 *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 2) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 2) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 2) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 2) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 2)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 2)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 2)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 2)) * c11 
#endif
	       );
	    
	    fp += fac2 * mm *
	      (
#if cuREAL == 4
	       tex3D<float>(tex._v[k], indX,   indY  , 0) * c00 +
	       tex3D<float>(tex._v[k], indX+1, indY  , 0) * c10 +
	       tex3D<float>(tex._v[k], indX,   indY+1, 0) * c01 +
	       tex3D<float>(tex._v[k], indX+1, indY+1, 0) * c11 
#else
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 0)) * c00 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 0)) * c10 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 0)) * c01 +
	       int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 0)) * c11 
#endif
	       );
      
      
	    if (mm) {
	
	      cuFP_t fac0 =  coef._v[Imn(mm, 's', n, nmax)];
	      cuFP_t fac1 =  fac0 * ssin;
	      cuFP_t fac2 = -fac0 * ccos;

	      pp += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 3) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 3) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 3) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 3) * c11
#else		 
 		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 3)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 3)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 3)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 3)) * c11 
#endif
		 );
	      
	      fr += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 4) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 4) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 4) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 4) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 4)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 4)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 4)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 4)) * c11 
#endif
		 );
	      
	      fz += fac1 *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 5) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 5) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 5) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 5) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 5)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 5)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 5)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 5)) * c11 
#endif
		 );
	      
	      fp += fac2 * mm *
		(
#if cuREAL == 4
		 tex3D<float>(tex._v[k], indX,   indY  , 3) * c00 +
		 tex3D<float>(tex._v[k], indX+1, indY  , 3) * c10 +
		 tex3D<float>(tex._v[k], indX,   indY+1, 3) * c01 +
		 tex3D<float>(tex._v[k], indX+1, indY+1, 3) * c11 
#else
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY  , 3)) * c00 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY  , 3)) * c10 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX,   indY+1, 3)) * c01 +
		 int2_as_double(tex3D<int2>(tex._v[k], indX+1, indY+1, 3)) * c11 
#endif
		 );
	      
	    }
	  }
	  
	  // Trig recursion to squeeze avoid internal FP fct call
	  //
	  cuFP_t cosM = ccos;
	  cuFP_t sinM = ssin;

	  ccos = cosM * cos1 - sinM * sin1;
	  ssin = sinM * cos1 + cosM * sin1;
	}

	acc[0] += ( fr*xx/R - fp*yy/R2 ) * frac;
	acc[1] += ( fr*yy/R + fp*xx/R2 ) * frac;
	acc[2] += fz * frac;
      }

      if (ratio > ratmin) {

	cuFP_t r3 = R2 + zz*zz;
	pp = -cylmass/sqrt(r3);	// -M/r
	fr = pp/r3;		// -M/r^3

	acc[0] += xx*fr * cfrac;
	acc[1] += yy*fr * cfrac;
	acc[2] += zz*fr * cfrac;
      }

      if (cylOrient) {
	for (int j=0; j<3; j++) {
	  for (int k=0; k<3; k++) in._v[npart].acc[j] += cylOrig[3*j+k]*acc[k];
	}
      } else {
	for (int j=0; j<3; j++) in._v[npart].acc[j] += acc[j];
      }

      if (external)
	in._v[npart].potext += pp;
      else
	in._v[npart].pot    += pp;

    } // Particle index block

  } // END: stride loop

}



template<typename T>
class LessAbs : public std::binary_function<bool, T, T>
{
public:
  T operator()( const T &a, const T &b ) const
  {
    return (fabs(a) < fabs(b));
  }
};

static bool initialize_cuda_cyl = true;

void Cylinder::determine_coefficients_cuda()
{
  if (initialize_cuda_cyl) {
    initialize_cuda();
    initialize_mapping_constants();
    initialize_cuda_cyl = false;
    // Only copy texture memory once
    t_d = tex;
  }

  std::cout << std::scientific;

  int deviceCount = 0;
  cuda_safe_call(hipGetDeviceCount(&deviceCount),
		 __FILE__, __LINE__, "could not get device count");

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceCount-1);

  // This will stay fixed for the entire run
  //
  host_coefs.resize((2*mmax+1)*ncylorder);

  // Set component center and orientation
  //
  std::vector<cuFP_t> ctr;
  for (auto v : cC->getCenter(Component::Local | Component::Centered)) ctr.push_back(v);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCen), &ctr[0], sizeof(cuFP_t)*3,
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCen");

  bool orient = (cC->EJ & Orient::AXIS) && !cC->EJdryrun;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrient), &orient,   sizeof(bool),  size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylOrient");

  if (orient) {
    std::vector<cuFP_t> trans(9);
    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++) trans[i*3+j] = cC->orient->transformBody()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylBody), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylBody");
  }

  Component::cuRingType cr = *cC->cuRing.get();

  // For debugging (set to false to disable)
  //
  static bool firstime = false;

  if (firstime) {
    testConstantsCyl<<<1, 1, 0, cr->stream>>>();
    hipDeviceSynchronize();
    firstime = false;
  }
  
  // Zero counter and coefficients
  //
  unsigned Ntot = 0;
  cylmass0[0] = 0.0;
  thrust::fill(host_coefs.begin(), host_coefs.end(), 0.0);

  // Maximum radius on grid
  //
  cuFP_t rmax = rcylmax * acyl * M_SQRT1_2;


  // Loop over bunches
  //
  size_t psize  = cC->Particles().size();

  PartMap::iterator begin = cC->Particles().begin();
  PartMap::iterator first = begin;
  PartMap::iterator last  = begin;
  PartMap::iterator end   = cC->Particles().end();

  std::advance(last, cC->bunchSize);

  while (std::distance(first, last)) {
    
    // Copy bunch to device
    //
    cC->ParticlesToCuda(cr, first, last);

    // Sort particles and get coefficient size
    //
    PII lohi = cC->CudaSortByLevel(cr, mlevel, mlevel);

    // Compute grid
    //
    unsigned int N         = lohi.second - lohi.first;
    unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

#ifdef VERBOSE
    static debug_max_count = 10;
    static debug_cur_count = 0;
    if (debug_cur_count++ < debug_max_count) {
      std::cout << std::endl << "**" << std::endl
		<< "** N      = " << N           << std::endl
		<< "** I low  = " << lohi.first  << std::endl
		<< "** I high = " << lohi.second << std::endl
		<< "** Stride = " << stride      << std::endl
		<< "** Block  = " << BLOCK_SIZE  << std::endl
		<< "** Grid   = " << gridSize    << std::endl
		<< "** Xcen   = " << ctr[0]     << std::endl
		<< "** Ycen   = " << ctr[1]     << std::endl
		<< "** Zcen   = " << ctr[2]     << std::endl
		<< "**" << std::endl;
    }
#endif
  
    if (N) {

      // Reserve space for coefficient reduction
      //
      if (dN_coef.capacity() < 2*ncylorder*N)
	dN_coef.reserve(2*ncylorder*N);
      
      if (dc_coef.capacity() < 2*ncylorder*gridSize)
	dc_coef.reserve(2*ncylorder*gridSize);

      if (m_d .capacity() < N) m_d .reserve(N);
      if (X_d .capacity() < N) X_d .reserve(N);
      if (Y_d .capacity() < N) Y_d .reserve(N);
      if (p_d .capacity() < N) p_d .reserve(N);
      if (iX_d.capacity() < N) iX_d.reserve(N);
      if (iY_d.capacity() < N) iY_d.reserve(N);
      
      
      // Set space for current step
      //
      dN_coef.resize(2*ncylorder*N);
      dc_coef.resize(2*ncylorder*gridSize);
      df_coef.resize(2*ncylorder);	// Should stay fixed, no reserve
    

      // Space for coordinate arrays on the current step
      //
      m_d .resize(N);
      X_d .resize(N);
      Y_d .resize(N);
      p_d .resize(N);
      iX_d.resize(N);
      iY_d.resize(N);

      // Shared memory size for the reduction
      //
      int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);
    

      thrust::counting_iterator<int> index_begin(0);
      thrust::counting_iterator<int> index_end(gridSize*2*ncylorder);

      // Do the work
      //
				// Compute the coordinate
				// transformation
				// 
      coordKernelCyl<<<gridSize, BLOCK_SIZE, 0, cr->stream>>>
	(toKernel(cr->cuda_particles), toKernel(m_d), toKernel(p_d),
	 toKernel(X_d), toKernel(Y_d), toKernel(iX_d), toKernel(iY_d),
	 stride, lohi, rmax);
      
				// Compute the coefficient
				// contribution for each order
      int osize = ncylorder*2;	// 
      for (int m=0; m<=mmax; m++) {
	coefKernelCyl<<<gridSize, BLOCK_SIZE, 0, cr->stream>>>
	  (toKernel(dN_coef), toKernel(t_d), toKernel(m_d), toKernel(p_d),
	   toKernel(X_d), toKernel(Y_d), toKernel(iX_d), toKernel(iY_d),
	   stride, m, ncylorder, lohi);
      
				// Begin the reduction per grid block
				//
	reduceSum<cuFP_t, BLOCK_SIZE><<<gridSize, BLOCK_SIZE, sMemSize, cr->stream>>>
	  (toKernel(dc_coef), toKernel(dN_coef), osize, N);
      
				// Finish the reduction for this order
				// in parallel
	thrust::reduce_by_key
	  (
	   thrust::cuda::par.on(cr->stream),
	   thrust::make_transform_iterator(index_begin, key_functor(gridSize)),
	   thrust::make_transform_iterator(index_end,   key_functor(gridSize)),
	   dc_coef.begin(), thrust::make_discard_iterator(), df_coef.begin()
	   );
    
	thrust::host_vector<cuFP_t> ret = df_coef;
	for (size_t j=0; j<ncylorder; j++) {
	  host_coefs[Imn(m, 'c', j, ncylorder)] += ret[2*j];
	  if (m>0) host_coefs[Imn(m, 's', j, ncylorder)] += ret[2*j+1];
	}
      }
    
      // Compute number and total mass of particles used in coefficient
      // determination
      //
      thrust::sort(m_d.begin(), m_d.end());

      auto m_it    = thrust::upper_bound(thrust::cuda::par.on(cr->stream),
					 m_d.begin(), m_d.end(), 0.0);
      use[0]      += thrust::distance(m_it, m_d.end());
      cylmass0[0] += thrust::reduce  (thrust::cuda::par.on(cr->stream),
				      m_it, m_d.end());
      Ntot        += N;
    }

    // Advance iterators
    //
    first = last;
    size_t nadv = std::distance(first, end);
    if (nadv < cC->bunchSize) last = end;
    else std::advance(last, cC->bunchSize);
  }

  if (Ntot == 0) {
    return;
  }

  // DEBUG, only useful for CUDAtest branch
  //
  if (false) {
    std::cout << std::string(2*4+4*20, '-') << std::endl
	      << "---- Cylindrical "      << std::endl
	      << std::string(2*4+4*20, '-') << std::endl;
    std::cout << "M=0 coefficients" << std::endl
	      << std::setprecision(10);
    

    std::cout << std::setw(4)  << "n"
	      << std::setw(4)  << "i"
	      << std::setw(20) << "GPU"
	      << std::setw(20) << "CPU"
	      << std::setw(20) << "diff"
	      << std::setw(20) << "rel diff"
	      << std::endl;

    int i = Imn(0, 'c', 0, ncylorder);
    auto cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(0, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(0, n, 'c');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }

    std::cout << "M=1c coefficients" << std::endl;

    i = Imn(1, 'c', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(1, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(1, n, 'c');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }

    std::cout << "M=1s coefficients" << std::endl;

    i = Imn(1, 's', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(1, 's', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(1, n, 's');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }

    std::cout << "M=2c coefficients" << std::endl;

    i = Imn(2, 'c', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(2, 'c', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(2, n, 'c');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }
    
    std::cout << "M=2s coefficients" << std::endl;

    i = Imn(2, 's', 0, ncylorder);
    cmax = std::max_element(host_coefs.begin()+i, host_coefs.begin()+i+ncylorder, LessAbs<cuFP_t>());

    for (size_t n=0; n<ncylorder; n++) {
      int    i = Imn(2, 's', n, ncylorder);
      cuFP_t a = host_coefs[i];
      cuFP_t b = ortho->get_coef(2, n, 's');
      std::cout << std::setw(4)  << n
		<< std::setw(4)  << i
		<< std::setw(20) << a
		<< std::setw(20) << b
		<< std::setw(20) << a - b
		<< std::setw(20) << (a - b)/fabs(*cmax)
		<< std::endl;
    }

    std::cout << std::string(2*4+4*20, '-') << std::endl;
  }


  //
  // TEST comparison of coefficients for debugging
  //
  if (false) {

    struct Element
    {
      double d;
      double f;
      
      int  m;
      int  n;
      
      char cs;
    }
    elem;

    std::multimap<double, Element> compare;

    std::ofstream out("test_cyl.dat");

    // m loop
    for (int m=0; m<=mmax; m++) {
	
      if (m==0) {
	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 'c';
	  elem.d = ortho->get_coef(m, n, 'c');
	  elem.f = host_coefs[Imn(m, 'c', n, ncylorder)];
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);
	  
	  compare.insert(std::make_pair(test, elem));;
	    
	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 'c'
	      << std::setw( 5) << Imn(m, 'c', n, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	}

      } else {
	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 'c';
	  elem.d = ortho->get_coef(m, n, 'c');
	  elem.f = host_coefs[Imn(m, 'c', n, ncylorder)];

	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 'c'
	      << std::setw( 5) << Imn(m, 'c', n, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);

	  compare.insert(std::make_pair(test, elem));;
	}

	for (int n=0; n<ncylorder; n++) {
	  elem.m = m;
	  elem.n = n;
	  elem.cs = 's';
	  elem.d = ortho->get_coef(m, n, 's');
	  elem.f = host_coefs[Imn(m, 's', n, ncylorder)];

	  out << std::setw( 5) << m
	      << std::setw( 5) << n
	      << std::setw( 5) << 's'
	      << std::setw( 5) << Imn(m, 's', n-1, ncylorder)
	      << std::setw(14) << elem.d
	      << std::setw(14) << elem.f
	      << std::endl;
	  
	  double test = fabs(elem.d - elem.f);
	  if (fabs(elem.d)>1.0e-12) test /= fabs(elem.d);
	  
	  compare.insert(std::make_pair(test, elem));;
	}
      }
    }
    
    std::map<double, Element>::iterator best = compare.begin();
    std::map<double, Element>::iterator midl = best;
    std::advance(midl, compare.size()/2);
    std::map<double, Element>::reverse_iterator last = compare.rbegin();
    
    std::cout << std::string(3*2 + 3*20 + 20, '-') << std::endl
	      << "---- Cylinder coefficients" << std::endl
	      << std::string(3*2 + 3*20 + 20, '-') << std::endl;

    std::cout << "Best case: ["
	      << std::setw( 2) << best->second.m << ", "
	      << std::setw( 2) << best->second.n << ", "
	      << std::setw( 2) << best->second.cs << "] = "
	      << std::setw(20) << best->second.d
	      << std::setw(20) << best->second.f
	      << std::setw(20) << fabs(best->second.d - best->second.f)
	      << std::endl;
  
    std::cout << "Mid case:  ["
	      << std::setw( 2) << midl->second.m << ", "
	      << std::setw( 2) << midl->second.n << ", "
	      << std::setw( 2) << midl->second.cs << "] = "
	      << std::setw(20) << midl->second.d
	      << std::setw(20) << midl->second.f
	      << std::setw(20) << fabs(midl->second.d - midl->second.f)
	      << std::endl;
    
    std::cout << "Last case: ["
	      << std::setw( 2) << last->second.m << ", "
	      << std::setw( 2) << last->second.n << ", "
	      << std::setw( 2) << last->second.cs << "] = "
	      << std::setw(20) << last->second.d
	      << std::setw(20) << last->second.f
	      << std::setw(20) << fabs(last->second.d - last->second.f)
	      << std::endl;
  }

}


void Cylinder::determine_acceleration_cuda()
{
  if (initialize_cuda_cyl) {
    initialize_cuda();
    initialize_mapping_constants();
    initialize_cuda_cyl = false;
    // Only copy texture memory once
    t_d = tex;
  }

  std::cout << std::scientific;

  int deviceCount = 0;
  cuda_safe_call(hipGetDeviceCount(&deviceCount),
		 __FILE__, __LINE__, "could not get device count");

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceCount-1);

  Component::cuRingType cr = *cC->cuRing.get();

  // Assign component center and orientation
  //
  std::vector<cuFP_t> ctr;
  for (auto v : cC->getCenter(Component::Local | Component::Centered)) ctr.push_back(v);

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylCen), &ctr[0], sizeof(cuFP_t)*3,
				    size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylCen");

  bool orient = (cC->EJ & Orient::AXIS) && !cC->EJdryrun;

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrient), &orient,   sizeof(bool),  size_t(0), hipMemcpyHostToDevice),
		 __FILE__, __LINE__, "Error copying cylOrient");

  if (orient) {
    std::vector<cuFP_t> trans(9);
    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++)
	trans[i*3+j] = cC->orient->transformBody()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylBody), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylBody");

    for (int i=0; i<3; i++) 
      for (int j=0; j<3; j++)
	trans[i*3+j] = cC->orient->transformOrig()[i][j];
  
    cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(cylOrig), &trans[0], sizeof(cuFP_t), size_t(0), hipMemcpyHostToDevice),
		   __FILE__, __LINE__, "Error copying cylOrig");
  }

  // Loop over bunches
  //
  size_t psize  = cC->Particles().size();

  PartMap::iterator begin = cC->Particles().begin();
  PartMap::iterator first = begin;
  PartMap::iterator last  = begin;
  PartMap::iterator end   = cC->Particles().end();

  std::advance(last, cC->bunchSize);

  unsigned Ntot = 0;

  while (std::distance(first, last)) {

    // Copy bunch to device
    //
    cC->ParticlesToCuda(cr, first, last);

    // Sort particles and get coefficient size
    //
    PII lohi = cC->CudaSortByLevel(cr, mlevel, multistep);

    // Compute grid
    //
    unsigned int N         = lohi.second - lohi.first;
    unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
    Ntot += N;

    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

#ifdef VERBOSE
    static debug_max_count = 10;
    static debug_cur_count = 0;
    if (debug_cur_count++ < debug_max_count) {
      std::cout << std::endl << "**" << std::endl
		<< "** N      = " << N          << std::endl
		<< "** Stride = " << stride     << std::endl
		<< "** Block  = " << BLOCK_SIZE << std::endl
		<< "** Grid   = " << gridSize   << std::endl
		<< "** Xcen   = " << ctr[0]     << std::endl
		<< "** Ycen   = " << ctr[1]     << std::endl
		<< "** Zcen   = " << ctr[2]     << std::endl
		<< "**" << std::endl;
    }
#endif
    
    // Shared memory size for the reduction
    //
    int sMemSize = BLOCK_SIZE * sizeof(cuFP_t);
    
    // Maximum radius on grid
    //
    cuFP_t rmax = rcylmax * acyl;
    
    // Do the work
    //
    forceKernelCyl<<<gridSize, BLOCK_SIZE, sMemSize, cr->stream>>>
      (toKernel(cr->cuda_particles), toKernel(dev_coefs), toKernel(t_d),
       stride, mmax, ncylorder, lohi, rmax, cylmass, use_external);

    // Copy particles back to host.  These copies could be staged in
    // streams and returned asynchronously.
    //
    cC->CudaToParticles(cr);

    // Advance iterators
    //
    first = last;
    size_t nadv = std::distance(first, end);
    if (nadv < cC->bunchSize) last = end;
    else std::advance(last, cC->bunchSize);

    // Advance stream iterator
    //
    cr++;
  }

  // DEBUGGING TEST
  if (false) {
    std::cout << std::string(10+7*16, '-') << std::endl;
    std::cout << "---- Acceleration in Cylinder [T=" << tnow
	      << ", N=" << Ntot << "]" << std::endl;
    std::cout << std::string(10+7*16, '-') << std::endl;
    first = last = begin;
    std::advance(last, 5);
    std::copy(first, last, std::ostream_iterator<PMapType>(std::cout, "\n"));
    first = begin;
    last  = end;
    std::advance(first, psize-5);
    std::copy(first, last, std::ostream_iterator<PMapType>(std::cout, "\n"));
    std::cout << std::string(10+7*16, '-') << std::endl;
  }
}

void Cylinder::HtoD_coefs()
{
  // Check size
  host_coefs.resize((2*mmax+1)*ncylorder); // Should stay fixed, no reserve

  // Copy from EmpCylSL
  
  // m loop
  //
  for (int m=0; m<=mmax; m++) {
    
    // n loop
    //
    for (int n=0; n<ncylorder; n++) {
      host_coefs[Imn(m, 'c', n, ncylorder)] = ortho->get_coef(m, n, 'c');
      if (m>0) host_coefs[Imn(m, 's', n, ncylorder)] = ortho->get_coef(m, n, 's');
    }
  }

  // Copy to device
  dev_coefs = host_coefs;
}


void Cylinder::DtoH_coefs(int M)
{
  // Copy from host device to EmpCylSL

  // m loop
  //
  for (int m=0; m<=mmax; m++) {
    
    // n loop
    //
    for (int n=0; n<ncylorder; n++) {
      ortho->set_coef(M, m, n, 'c') = host_coefs[Imn(m, 'c', n, ncylorder)];
      if (m>0) ortho->set_coef(M, m, n, 's') = host_coefs[Imn(m, 's', n, ncylorder)];
    }
  }
}

void Cylinder::destroy_cuda()
{
  // std::cout << "texture object array size = " << tex.size() << std::endl;
  for (size_t i=0; i<tex.size(); i++) {
    std::ostringstream sout;
    sout << "trying to free TextureObject [" << i << "]";
    cuda_safe_call(hipDestroyTextureObject(tex[i]),
		   __FILE__, __LINE__, sout.str());
  }

  // std::cout << "cuInterpArray size = " << cuInterpArray.size() << std::endl;
  for (size_t i=0; i<cuInterpArray.size(); i++) {
    std::ostringstream sout;
    sout << "trying to free cuPitch [" << i << "]";
    cuda_safe_call(hipFree(cuInterpArray[i]),
		     __FILE__, __LINE__, sout.str());
  }
    
  std::cout << "cuda memory freed" << std::endl;
}

void Cylinder::host_dev_force_compare()
{
  // Copy from device
  Component::cuRingType cr = *cC->cuRing.get();
  cr->host_particles = cr->cuda_particles;
  
  std::streamsize ss = std::cout.precision();
  std::cout.precision(10);

  std::cout << std::string(16+20*8, '-') << std::endl
	    << std::setw(8)  << "Index"  << std::setw(8)  << "Level"
	    << std::setw(20) << "ax [d]" << std::setw(20) << "ay [d]"
	    << std::setw(20) << "az [d]" << std::setw(20) << "ax [h]"
	    << std::setw(20) << "ay [h]" << std::setw(20) << "az [h]"
	    << std::setw(20) << "|Del a|/|a|"
	    << std::setw(20) << "|a|"    << std::endl;
  
  // Compare first and last 5 from the device list
  //
  for (size_t i=0; i<5; i++) 
    {
      auto indx = cr->host_particles[i].indx;
      auto levl = cr->host_particles[i].level;
      
      std::cout << std::setw(8) << indx	<< std::setw(8) << levl;

      for (int k=0; k<3; k++)
	std::cout << std::setw(20) << cr->host_particles[i].acc[k];

      for (int k=0; k<3; k++)
	std::cout << std::setw(20) << cC->Particles()[indx].acc[k];

      double diff = 0.0, norm = 0.0;
      for (int k=0; k<3; k++) {
	double b  = cr->host_particles[i].acc[k];
	double a  = cC->Particles()[indx].acc[k];
	diff += (a - b)*(a - b);
	norm += a*a;
      }
      std::cout << std::setw(20) << sqrt(diff/norm)
		<< std::setw(20) << sqrt(norm) << std::endl;
    }
  
  for (size_t j=0; j<5; j++) 
    {
      size_t i = cr->host_particles.size() - 5 + j;

      auto indx = cr->host_particles[i].indx;
      auto levl = cr->host_particles[i].level;

      std::cout << std::setw(8) << indx	<< std::setw(8) << levl;
      
      for (int k=0; k<3; k++)
	std::cout << std::setw(20) << cr->host_particles[i].acc[k];

      for (int k=0; k<3; k++)
	std::cout << std::setw(20) << cC->Particles()[indx].acc[k];

      double diff = 0.0, norm = 0.0;
      for (int k=0; k<3; k++) {
	double b  = cr->host_particles[i].acc[k];
	double a  = cC->Particles()[indx].acc[k];
	diff += (a - b)*(a - b);
	norm += a*a;
      }
      std::cout << std::setw(20) << sqrt(diff/norm)
		<< std::setw(20) << sqrt(norm) << std::endl;
    }

  std::cout << std::string(16+20*8, '-') << std::endl;
  std::cout.precision(ss);
}
